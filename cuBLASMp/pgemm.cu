#include "hip/hip_runtime.h"
/*
 * Copyright 2023 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <assert.h>
#include <math.h>
#include <mpi.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <vector>

#ifdef USE_CAL_MPI
#include <cal_mpi.h>
#endif

#include <cublasmp.h>

#include "helpers.h"
#include "matrix_generator.hxx"

int main(int argc, char* argv[])
{
    using input_t = __half;
    using output_t = __half;
    using compute_t = float;
    const hipDataType cuda_input_type = HIP_R_16F;
    const hipDataType cuda_output_type = HIP_R_16F;
    const hipblasComputeType_t cublas_compute_type = HIPBLAS_COMPUTE_32F;

    Options opts = { .m = 10,
                     .n = 10,
                     .k = 10,
                     .mbA = 2,
                     .nbA = 2,
                     .mbB = 2,
                     .nbB = 2,
                     .mbC = 2,
                     .nbC = 2,
                     .ia = 3,
                     .ja = 3,
                     .ib = 3,
                     .jb = 1,
                     .ic = 1,
                     .jc = 1,
                     .p = 2,
                     .q = 1,
                     .grid_layout = 'c',
                     .verbose = false };

    opts.parse(argc, argv);
    opts.validate();
    opts.print();

    MPI_Init(nullptr, nullptr);

    const int64_t m = opts.m;
    const int64_t n = opts.n;
    const int64_t k = opts.k;
    const int64_t ia = opts.ia;
    const int64_t ja = opts.jb;
    const int64_t ib = opts.ib;
    const int64_t jb = opts.jb;
    const int64_t ic = opts.ic;
    const int64_t jc = opts.jc;
    const int64_t mbA = opts.mbA;
    const int64_t nbA = opts.nbA;
    const int64_t mbB = opts.mbB;
    const int64_t nbB = opts.nbB;
    const int64_t mbC = opts.mbC;
    const int64_t nbC = opts.nbC;

    const int nprow = opts.p;
    const int npcol = opts.q;

    int rank, nranks;
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    const int myprow = (opts.grid_layout == 'c' ? rank % nprow : rank / npcol);
    const int mypcol = (opts.grid_layout == 'c' ? rank / nprow : rank % npcol);

    const int local_device = getLocalDevice();
    CUDA_CHECK(hipSetDevice(local_device));
    CUDA_CHECK(hipFree(nullptr));

    cal_comm_t cal_comm = nullptr;
#ifdef USE_CAL_MPI
    CAL_CHECK(cal_comm_create_mpi(MPI_COMM_WORLD, rank, nranks, local_device, &cal_comm));
#else
    cal_comm_create_params_t params;
    params.allgather = allgather;
    params.req_test = request_test;
    params.req_free = request_free;
    params.data = (void*)(MPI_COMM_WORLD);
    params.rank = rank;
    params.nranks = nranks;
    params.local_device = local_device;
    CAL_CHECK(cal_comm_create(params, &cal_comm));
#endif

    hipStream_t stream = nullptr;
    CUDA_CHECK(hipStreamCreate(&stream));

    cublasMpHandle_t handle = nullptr;
    CUBLAS_CHECK(cublasMpCreate(&handle, stream));

    cublasMpGrid_t grid = nullptr;

    cublasMpMatrixDescriptor_t descA = nullptr;
    cublasMpMatrixDescriptor_t descB = nullptr;
    cublasMpMatrixDescriptor_t descC = nullptr;

    double* d_A = nullptr;
    double* d_B = nullptr;
    double* d_C = nullptr;

    double* d_work = nullptr;

    compute_t alpha = 1.0;
    compute_t beta = 1.0;

    size_t workspaceInBytesOnDevice = 0;
    size_t workspaceInBytesOnHost = 0;

    const int64_t global_m_a = (ia - 1) + m;
    const int64_t global_n_a = (ja - 1) + k;
    const int64_t global_m_b = (ib - 1) + k;
    const int64_t global_n_b = (jb - 1) + n;
    const int64_t global_m_c = (ic - 1) + m;
    const int64_t global_n_c = (jc - 1) + n;

    const int64_t llda = cublasMpNumroc(global_m_a, mbA, myprow, 0, nprow);
    const int64_t loc_n_a = cublasMpNumroc(global_n_a, nbA, mypcol, 0, npcol);

    const int64_t lldb = cublasMpNumroc(global_m_b, mbB, myprow, 0, nprow);
    const int64_t loc_n_b = cublasMpNumroc(global_n_b, nbB, mypcol, 0, npcol);

    const int64_t lldc = cublasMpNumroc(global_m_c, mbC, myprow, 0, nprow);
    const int64_t loc_n_c = cublasMpNumroc(global_n_c, nbC, mypcol, 0, npcol);

    std::vector<input_t> h_A(llda * loc_n_a, 0);
    std::vector<input_t> h_B(lldb * loc_n_b, 0);
    std::vector<output_t> h_C(lldc * loc_n_c, 0);

    generate_random_matrix(m, k, h_A.data(), mbA, nbA, ia, ja, llda, nprow, npcol, myprow, mypcol);
    generate_random_matrix(k, n, h_B.data(), mbB, nbB, ib, jb, lldb, nprow, npcol, myprow, mypcol);
    generate_random_matrix(m, n, h_C.data(), mbC, nbC, ic, jc, lldc, nprow, npcol, myprow, mypcol);

    CUDA_CHECK(hipMallocAsync(&d_A, llda * loc_n_a * sizeof(input_t), stream));
    CUDA_CHECK(hipMallocAsync(&d_B, lldb * loc_n_b * sizeof(input_t), stream));
    CUDA_CHECK(hipMallocAsync(&d_C, lldc * loc_n_c * sizeof(output_t), stream));

    CUDA_CHECK(hipMemcpyAsync(d_A, h_A.data(), llda * loc_n_a * sizeof(input_t), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, h_B.data(), lldb * loc_n_b * sizeof(input_t), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_C, h_C.data(), lldc * loc_n_c * sizeof(output_t), hipMemcpyHostToDevice, stream));

    CUBLAS_CHECK(cublasMpGridCreate(
        handle,
        nprow,
        npcol,
        opts.grid_layout == 'c' ? CUBLASMP_GRID_LAYOUT_COL_MAJOR : CUBLASMP_GRID_LAYOUT_ROW_MAJOR,
        cal_comm,
        &grid));

    CUBLAS_CHECK(cublasMpMatrixDescriptorCreate(
        handle, global_m_a, global_n_a, mbA, nbA, 0, 0, llda, cuda_input_type, grid, &descA));
    CUBLAS_CHECK(cublasMpMatrixDescriptorCreate(
        handle, global_m_b, global_n_b, mbB, nbB, 0, 0, lldb, cuda_input_type, grid, &descB));
    CUBLAS_CHECK(cublasMpMatrixDescriptorCreate(
        handle, global_m_c, global_n_c, mbC, nbC, 0, 0, lldc, cuda_output_type, grid, &descC));

    CUBLAS_CHECK(cublasMpGemm_bufferSize(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m,
        n,
        k,
        &alpha,
        d_A,
        ia,
        ja,
        descA,
        d_B,
        ib,
        jb,
        descB,
        &beta,
        d_C,
        ic,
        jc,
        descC,
        cublas_compute_type,
        &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    CUDA_CHECK(hipMallocAsync(&d_work, workspaceInBytesOnDevice, stream));

    std::vector<int8_t> h_work(workspaceInBytesOnHost);

    CAL_CHECK(cal_stream_sync(cal_comm, stream));
    CAL_CHECK(cal_comm_barrier(cal_comm, stream));

    const double begin = MPI_Wtime();

    CUBLAS_CHECK(cublasMpGemm(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        m,
        n,
        k,
        &alpha,
        d_A,
        ia,
        ja,
        descA,
        d_B,
        ib,
        jb,
        descB,
        &beta,
        d_C,
        ic,
        jc,
        descC,
        cublas_compute_type,
        d_work,
        workspaceInBytesOnDevice,
        h_work.data(),
        workspaceInBytesOnHost));

    CAL_CHECK(cal_stream_sync(cal_comm, stream));
    CAL_CHECK(cal_comm_barrier(cal_comm, stream));

    const double end = MPI_Wtime();

    printf("Duration: %lf GFlops: %lf\n", end - begin, (2 * m * n * k * 1e-9) / (end - begin));

    CUBLAS_CHECK(cublasMpMatrixDescriptorDestroy(handle, descA));
    CUBLAS_CHECK(cublasMpMatrixDescriptorDestroy(handle, descB));
    CUBLAS_CHECK(cublasMpMatrixDescriptorDestroy(handle, descC));

    CUBLAS_CHECK(cublasMpGridDestroy(handle, grid));

    CUBLAS_CHECK(cublasMpDestroy(handle));

    CUDA_CHECK(hipFreeAsync(d_A, stream));
    CUDA_CHECK(hipFreeAsync(d_B, stream));
    CUDA_CHECK(hipFreeAsync(d_C, stream));
    CUDA_CHECK(hipFreeAsync(d_work, stream));

    CAL_CHECK(cal_comm_barrier(cal_comm, stream));

    CAL_CHECK(cal_comm_destroy(cal_comm));

    CUDA_CHECK(hipStreamDestroy(stream));

    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Finalize();

    return 0;
};