/*
 * Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <math.h>
#include <mpi.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <vector>

#ifdef USE_CAL_MPI
#include <cal_mpi.h>
#endif

#include <cublasmp.h>

#include "helpers.h"
#include "matrix_generator.hxx"

int main(int argc, char* argv[])
{
    Options opts = { .m = 10,
                     .n = 10,
                     .k = 10,
                     .mbA = 2,
                     .nbA = 2,
                     .mbB = 2,
                     .nbB = 2,
                     .mbC = 2,
                     .nbC = 2,
                     .ia = 3,
                     .ja = 3,
                     .ib = 3,
                     .jb = 1,
                     .ic = 1,
                     .jc = 1,
                     .p = 2,
                     .q = 1,
                     .grid_layout = 'c',
                     .verbose = false };

    opts.parse(argc, argv);
    opts.validate();
    opts.print();

    MPI_Init(nullptr, nullptr);

    const int64_t m = opts.m;
    const int64_t n = opts.n;
    const int64_t ia = opts.ia;
    const int64_t ja = opts.ja;
    const int64_t ib = opts.ib;
    const int64_t jb = opts.jb;
    const int64_t mbA = opts.mbA;
    const int64_t nbA = opts.nbA;
    const int64_t mbB = opts.mbB;
    const int64_t nbB = opts.nbB;

    const int nprow = opts.p;
    const int npcol = opts.q;

    int rank, nranks;
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    const int myprow = (opts.grid_layout == 'c' ? rank % nprow : rank / npcol);
    const int mypcol = (opts.grid_layout == 'c' ? rank / nprow : rank % npcol);

    const int local_device = getLocalDevice();
    CUDA_CHECK(hipSetDevice(local_device));
    CUDA_CHECK(hipFree(nullptr));

    cal_comm_t cal_comm = nullptr;
#ifdef USE_CAL_MPI
    CAL_CHECK(cal_comm_create_mpi(MPI_COMM_WORLD, rank, nranks, local_device, &cal_comm));
#else
    cal_comm_create_params_t params;
    params.allgather = allgather;
    params.req_test = request_test;
    params.req_free = request_free;
    params.data = (void*)(MPI_COMM_WORLD);
    params.rank = rank;
    params.nranks = nranks;
    params.local_device = local_device;
    CAL_CHECK(cal_comm_create(params, &cal_comm));
#endif

    hipStream_t stream = nullptr;
    CUDA_CHECK(hipStreamCreate(&stream));

    cublasMpHandle_t handle = nullptr;
    CUBLASMP_CHECK(cublasMpCreate(&handle, stream));

    cublasMpGrid_t grid = nullptr;

    cublasMpMatrixDescriptor_t descA = nullptr;
    cublasMpMatrixDescriptor_t descB = nullptr;

    double* d_A = nullptr;
    double* d_B = nullptr;

    double* d_work = nullptr;

    double alpha = 1.0;

    size_t workspaceInBytesOnDevice = 0;
    size_t workspaceInBytesOnHost = 0;

    const int64_t global_m_a = (ia - 1) + m;
    const int64_t global_n_a = (ja - 1) + m;
    const int64_t global_m_b = (ib - 1) + m;
    const int64_t global_n_b = (jb - 1) + n;

    const int64_t llda = cublasMpNumroc(global_m_a, mbA, myprow, 0, nprow);
    const int64_t loc_n_a = cublasMpNumroc(global_n_a, nbA, mypcol, 0, npcol);

    const int64_t lldb = cublasMpNumroc(global_m_b, mbB, myprow, 0, nprow);
    const int64_t loc_n_b = cublasMpNumroc(global_n_b, nbB, mypcol, 0, npcol);

    std::vector<double> h_A(llda * loc_n_a, 0);
    std::vector<double> h_B(lldb * loc_n_b, 0);

    generate_diag_matrix(m, m, h_A.data(), mbA, nbA, ia, ja, llda, nprow, npcol, myprow, mypcol);
    generate_random_matrix(m, n, h_B.data(), mbB, nbB, ib, jb, lldb, nprow, npcol, myprow, mypcol);

    CUDA_CHECK(hipMallocAsync(&d_A, llda * loc_n_a * sizeof(double), stream));
    CUDA_CHECK(hipMallocAsync(&d_B, lldb * loc_n_b * sizeof(double), stream));

    CUDA_CHECK(hipMemcpyAsync(d_A, h_A.data(), llda * loc_n_a * sizeof(double), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, h_B.data(), lldb * loc_n_b * sizeof(double), hipMemcpyHostToDevice, stream));

    CUBLASMP_CHECK(cublasMpGridCreate(
        nprow,
        npcol,
        opts.grid_layout == 'c' ? CUBLASMP_GRID_LAYOUT_COL_MAJOR : CUBLASMP_GRID_LAYOUT_ROW_MAJOR,
        cal_comm,
        &grid));

    CUBLASMP_CHECK(
        cublasMpMatrixDescriptorCreate(global_m_a, global_n_a, mbA, nbA, 0, 0, llda, HIP_R_64F, grid, &descA));
    CUBLASMP_CHECK(
        cublasMpMatrixDescriptorCreate(global_m_b, global_n_b, mbB, nbB, 0, 0, lldb, HIP_R_64F, grid, &descB));

    CUBLASMP_CHECK(cublasMpTrsm_bufferSize(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N,
        HIPBLAS_DIAG_NON_UNIT,
        m,
        n,
        &alpha,
        d_A,
        ia,
        ja,
        descA,
        d_B,
        ib,
        jb,
        descB,
        HIPBLAS_COMPUTE_64F,
        &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    CUDA_CHECK(hipMallocAsync(&d_work, workspaceInBytesOnDevice, stream));

    std::vector<int8_t> h_work(workspaceInBytesOnHost);

    CAL_CHECK(cal_stream_sync(cal_comm, stream));
    CAL_CHECK(cal_comm_barrier(cal_comm, stream));

    const double begin = MPI_Wtime();

    CUBLASMP_CHECK(cublasMpTrsm(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N,
        HIPBLAS_DIAG_NON_UNIT,
        m,
        n,
        &alpha,
        d_A,
        ia,
        ja,
        descA,
        d_B,
        ib,
        jb,
        descB,
        HIPBLAS_COMPUTE_64F,
        d_work,
        workspaceInBytesOnDevice,
        h_work.data(),
        workspaceInBytesOnHost));

    CAL_CHECK(cal_stream_sync(cal_comm, stream));
    CAL_CHECK(cal_comm_barrier(cal_comm, stream));

    const double end = MPI_Wtime();

    if (rank == 0)
    {
        printf(
            "Duration: %lf GFlops: %lf\n",
            end - begin,
            ((((0.5 * m * (m - 1)) + ((0.5 * m * (m + 1)))) * n) * 1e-9) / (end - begin));
    }

    CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descA));
    CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descB));

    CUBLASMP_CHECK(cublasMpGridDestroy(grid));

    CUBLASMP_CHECK(cublasMpDestroy(handle));

    CUDA_CHECK(hipFreeAsync(d_A, stream));
    CUDA_CHECK(hipFreeAsync(d_B, stream));
    CUDA_CHECK(hipFreeAsync(d_work, stream));

    CAL_CHECK(cal_comm_barrier(cal_comm, stream));

    CAL_CHECK(cal_comm_destroy(cal_comm));

    CUDA_CHECK(hipStreamDestroy(stream));

    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Finalize();

    if (rank == 0)
    {
        printf("[SUCCEEDED]\n");
    }

    return 0;
};