#include "hip/hip_runtime.h"
/*
 * Copyright 2024 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <assert.h>
#include <hip/hip_fp8.h>
#include <math.h>
#include <mpi.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <vector>

#ifdef USE_CAL_MPI
#include <cal_mpi.h>
#endif

#include <cublasmp.h>
#include <nvshmem.h>

#include "helpers.h"
#include "matrix_generator.hxx"

int main(int argc, char* argv[])
{
    using input_t = __half;
    using output_t = __half;
    using compute_t = float;
    const hipDataType cuda_input_type = HIP_R_16F;
    const hipDataType cuda_output_type = HIP_R_16F;
    const hipblasComputeType_t cublas_compute_type = HIPBLAS_COMPUTE_32F;
    const hipblasOperation_t transA = HIPBLAS_OP_T;
    const hipblasOperation_t transB = HIPBLAS_OP_N;

    MPI_Init(nullptr, nullptr);

    int rank, nranks;
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    const int local_device = getLocalDevice();
    CUDA_CHECK(hipSetDevice(local_device));
    CUDA_CHECK(hipFree(nullptr));

    cal_comm_t cal_comm = nullptr;
#ifdef USE_CAL_MPI
    CAL_CHECK(cal_comm_create_mpi(MPI_COMM_WORLD, rank, nranks, local_device, &cal_comm));
#else
    cal_comm_create_params_t params;
    params.allgather = allgather;
    params.req_test = request_test;
    params.req_free = request_free;
    params.data = (void*)(MPI_COMM_WORLD);
    params.rank = rank;
    params.nranks = nranks;
    params.local_device = local_device;
    CAL_CHECK(cal_comm_create(params, &cal_comm));
#endif

    nvshmemx_init_attr_t attr;
    nvshmemx_uniqueid_t id;

    if (rank == 0)
    {
        NVSHMEM_CHECK(nvshmemx_get_uniqueid(&id));
    }

    MPI_CHECK(MPI_Bcast(&id, sizeof(nvshmemx_uniqueid_t), MPI_BYTE, 0, MPI_COMM_WORLD));

    nvshmemx_set_attr_uniqueid_args(rank, nranks, &id, &attr);
    NVSHMEM_CHECK(nvshmemx_hostlib_init_attr(NVSHMEMX_INIT_WITH_UNIQUEID, &attr));

    hipStream_t stream = nullptr;
    CUDA_CHECK(hipStreamCreate(&stream));

    cublasMpHandle_t handle = nullptr;
    CUBLASMP_CHECK(cublasMpCreate(&handle, stream));

    cublasMpMatrixDescriptor_t descA = nullptr;
    cublasMpMatrixDescriptor_t descB = nullptr;
    cublasMpMatrixDescriptor_t descC = nullptr;

    cublasMpMatmulDescriptor_t matmulDesc = nullptr;

    output_t* d_X1 = nullptr;
    output_t* d_X2 = nullptr;

    void* d_work = nullptr;

    compute_t alpha = 1.0;
    compute_t beta = 0.0;

    size_t workspaceInBytesOnDevice = 0;
    size_t workspaceInBytesOnHost = 0;

    cublasMpGrid_t grid_col_major = nullptr;
    cublasMpGrid_t grid_row_major = nullptr;

    CUBLASMP_CHECK(cublasMpGridCreate(nranks, 1, CUBLASMP_GRID_LAYOUT_COL_MAJOR, cal_comm, &grid_col_major));
    CUBLASMP_CHECK(cublasMpGridCreate(1, nranks, CUBLASMP_GRID_LAYOUT_ROW_MAJOR, cal_comm, &grid_row_major));

    // AG + Matmul
    {
        const int64_t m = 16 * nranks;
        const int64_t n = 16 * nranks;
        const int64_t k = 16;

        const int64_t loc_a_m = k;
        const int64_t loc_a_n = m / nranks;
        const int64_t loc_b_m = k;
        const int64_t loc_b_n = n / nranks;
        const int64_t loc_c_m = m / nranks;
        const int64_t loc_c_n = n / nranks;

        std::vector<input_t> h_X0(loc_a_m * loc_a_n, input_t(0));
        std::vector<input_t> h_W0(loc_b_m * loc_b_n, input_t(0));
        std::vector<output_t> h_X1(loc_c_m * loc_c_n * nranks, output_t(0));

        generate_random_matrix(k, m, h_X0.data(), loc_a_m, loc_a_n, 1, 1, loc_a_m, 1, nranks, 1, rank);
        generate_random_matrix(k, n, h_W0.data(), loc_b_m, loc_b_n, 1, 1, loc_b_m, 1, nranks, 1, rank);
        generate_random_matrix(m, n, h_X1.data(), loc_c_m, loc_c_n, 1, 1, loc_c_m, nranks, 1, rank, 1);

        input_t* d_X0 = nullptr;
        input_t* d_W0 = nullptr;

        CUDA_CHECK(hipMalloc((void**)&d_X0, loc_a_m * loc_a_n * sizeof(input_t)));
        CUDA_CHECK(hipMalloc((void**)&d_W0, loc_b_m * loc_b_n * sizeof(input_t)));
        CUDA_CHECK(hipMalloc((void**)&d_X1, loc_c_m * loc_c_n * nranks * sizeof(output_t)));

        CUDA_CHECK(
            hipMemcpyAsync(d_X0, h_X0.data(), loc_a_m * loc_a_n * sizeof(input_t), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(
            hipMemcpyAsync(d_W0, h_W0.data(), loc_b_m * loc_b_n * sizeof(input_t), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(
            d_X1, h_X1.data(), loc_c_m * loc_c_n * nranks * sizeof(output_t), hipMemcpyHostToDevice, stream));

        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            k, m, loc_a_m, loc_a_n, 0, 0, loc_a_m, cuda_input_type, grid_row_major, &descA));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            k, n, loc_b_m, loc_b_n, 0, 0, loc_b_m, cuda_input_type, grid_row_major, &descB));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            m, n, loc_c_m, loc_c_n, 0, 0, loc_c_m, cuda_output_type, grid_col_major, &descC));

        const cublasMpMatmulAlgoType_t algoType = CUBLASMP_MATMUL_ALGO_TYPE_SPLIT_P2P;

        CUBLASMP_CHECK(cublasMpMatmulDescriptorCreate(&matmulDesc, cublas_compute_type));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_TRANSA, &transA, sizeof(transA)));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_TRANSB, &transB, sizeof(transB)));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_ALGO_TYPE, &algoType, sizeof(algoType)));

        CUBLASMP_CHECK(cublasMpMatmul_bufferSize(
            handle,
            matmulDesc,
            m,
            n,
            k,
            &alpha,
            d_X0,
            1,
            1,
            descA,
            d_W0,
            1,
            1,
            descB,
            &beta,
            nullptr,
            1,
            1,
            descC,
            d_X1,
            1,
            1,
            descC,
            &workspaceInBytesOnDevice,
            &workspaceInBytesOnHost));

        d_work = nvshmem_malloc(workspaceInBytesOnDevice);

        std::vector<int8_t> h_work(workspaceInBytesOnHost);

        CAL_CHECK(cal_stream_sync(cal_comm, stream));
        CAL_CHECK(cal_comm_barrier(cal_comm, stream));

        const double begin = MPI_Wtime();

        CUBLASMP_CHECK(cublasMpMatmul(
            handle,
            matmulDesc,
            m,
            n,
            k,
            &alpha,
            d_X0,
            1,
            1,
            descA,
            d_W0,
            1,
            1,
            descB,
            &beta,
            nullptr,
            1,
            1,
            descC,
            d_X1,
            1,
            1,
            descC,
            d_work,
            workspaceInBytesOnDevice,
            h_work.data(),
            workspaceInBytesOnHost));

        CAL_CHECK(cal_stream_sync(cal_comm, stream));
        CAL_CHECK(cal_comm_barrier(cal_comm, stream));

        const double end = MPI_Wtime();

        if (rank == 0)
        {
            printf("AG + Matmul: %lf (s) %lf (GFlops)\n", end - begin, (2 * m * n * k * 1e-9) / (end - begin));
        }

        CUBLASMP_CHECK(cublasMpMatmulDescriptorDestroy(matmulDesc));

        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descA));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descB));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descC));

        CUDA_CHECK(hipFree(d_X0));
        CUDA_CHECK(hipFree(d_W0));
        nvshmem_free(d_work);
    }

    // Matmul + RS
    {
        const int64_t m = 16;
        const int64_t n = 16 * nranks;
        const int64_t k = 16 * nranks;

        const int64_t loc_a_m = k / nranks;
        const int64_t loc_a_n = m;
        const int64_t loc_b_m = k / nranks;
        const int64_t loc_b_n = n / nranks;
        const int64_t loc_c_m = m;
        const int64_t loc_c_n = n / nranks;

        std::vector<input_t> h_W1(loc_a_m * loc_a_n, input_t(0));
        std::vector<output_t> h_X2(loc_c_m * loc_c_n, output_t(0));

        generate_random_matrix(k, m, h_W1.data(), loc_a_m, loc_a_n, 1, 1, loc_a_m, nranks, 1, rank, 1);
        generate_random_matrix(m, n, h_X2.data(), loc_c_m, loc_c_n, 1, 1, loc_c_m, 1, nranks, 1, rank);

        input_t* d_W1 = nullptr;

        CUDA_CHECK(hipMalloc((void**)&d_W1, loc_a_m * loc_a_n * sizeof(input_t)));
        CUDA_CHECK(hipMalloc((void**)&d_X2, loc_c_m * loc_c_n * sizeof(output_t)));

        CUDA_CHECK(
            hipMemcpyAsync(d_W1, h_W1.data(), loc_a_m * loc_a_n * sizeof(input_t), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(
            hipMemcpyAsync(d_X2, h_X2.data(), loc_c_m * loc_c_n * sizeof(output_t), hipMemcpyHostToDevice, stream));

        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            k, m, loc_a_m, loc_a_n, 0, 0, loc_a_m, cuda_input_type, grid_col_major, &descA));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            k, n, loc_b_m, loc_b_n, 0, 0, loc_b_m, cuda_input_type, grid_col_major, &descB));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            m, n, loc_c_m, loc_c_n, 0, 0, loc_c_m, cuda_output_type, grid_row_major, &descC));

        const cublasMpMatmulAlgoType_t algoType = CUBLASMP_MATMUL_ALGO_TYPE_SPLIT_P2P;

        CUBLASMP_CHECK(cublasMpMatmulDescriptorCreate(&matmulDesc, cublas_compute_type));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_TRANSA, &transA, sizeof(transA)));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_TRANSB, &transB, sizeof(transB)));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_ALGO_TYPE, &algoType, sizeof(algoType)));

        CUBLASMP_CHECK(cublasMpMatmul_bufferSize(
            handle,
            matmulDesc,
            m,
            n,
            k,
            &alpha,
            d_W1,
            1,
            1,
            descB,
            d_X1,
            1,
            1,
            descA,
            &beta,
            nullptr,
            1,
            1,
            descC,
            d_X2,
            1,
            1,
            descC,
            &workspaceInBytesOnDevice,
            &workspaceInBytesOnHost));

        d_work = nvshmem_malloc(workspaceInBytesOnDevice);

        std::vector<int8_t> h_work(workspaceInBytesOnHost);

        CAL_CHECK(cal_stream_sync(cal_comm, stream));
        CAL_CHECK(cal_comm_barrier(cal_comm, stream));

        const double begin = MPI_Wtime();

        CUBLASMP_CHECK(cublasMpMatmul(
            handle,
            matmulDesc,
            m,
            n,
            k,
            &alpha,
            d_W1,
            1,
            1,
            descB,
            d_X1,
            1,
            1,
            descA,
            &beta,
            nullptr,
            1,
            1,
            descC,
            d_X2,
            1,
            1,
            descC,
            d_work,
            workspaceInBytesOnDevice,
            h_work.data(),
            workspaceInBytesOnHost));

        CAL_CHECK(cal_stream_sync(cal_comm, stream));
        CAL_CHECK(cal_comm_barrier(cal_comm, stream));

        const double end = MPI_Wtime();

        if (rank == 0)
        {
            printf("Matmul + RS: %lf (s) %lf (GFlops)\n", end - begin, (2 * m * n * k * 1e-9) / (end - begin));
        }

        CUBLASMP_CHECK(cublasMpMatmulDescriptorDestroy(matmulDesc));

        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descA));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descB));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descC));

        CUDA_CHECK(hipFree(d_X1));
        CUDA_CHECK(hipFree(d_W1));
        CUDA_CHECK(hipFree(d_X2));
        nvshmem_free(d_work);
    }

    CUBLASMP_CHECK(cublasMpGridDestroy(grid_col_major));
    CUBLASMP_CHECK(cublasMpGridDestroy(grid_row_major));

    CUBLASMP_CHECK(cublasMpDestroy(handle));

    nvshmemx_hostlib_finalize();

    CAL_CHECK(cal_comm_barrier(cal_comm, stream));

    CAL_CHECK(cal_comm_destroy(cal_comm));

    CUDA_CHECK(hipStreamDestroy(stream));

    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Finalize();

    if (rank == 0)
    {
        printf("[SUCCEEDED]\n");
    }

    return 0;
};