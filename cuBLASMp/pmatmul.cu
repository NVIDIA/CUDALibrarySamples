#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <hip/hip_fp8.h>
#include <math.h>
#include <mpi.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <vector>

#ifdef USE_CAL_MPI
#include <cal_mpi.h>
#endif

#include <cublasmp.h>
#include <nvshmem.h>

#include "helpers.h"
#include "matrix_generator.hxx"

int main(int argc, char* argv[])
{
    using input_t = __half;
    using output_t = __half;
    using compute_t = float;
    const hipDataType cuda_input_type = HIP_R_16F;
    const hipDataType cuda_output_type = HIP_R_16F;
    const hipblasComputeType_t cublas_compute_type = HIPBLAS_COMPUTE_32F;
    const hipblasOperation_t transA = HIPBLAS_OP_T;
    const hipblasOperation_t transB = HIPBLAS_OP_N;

    MPI_Init(nullptr, nullptr);

    int rank, nranks;
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    const int local_device = getLocalDevice();
    CUDA_CHECK(hipSetDevice(local_device));
    CUDA_CHECK(hipFree(nullptr));

    cal_comm_t cal_comm = nullptr;
#ifdef USE_CAL_MPI
    CAL_CHECK(cal_comm_create_mpi(MPI_COMM_WORLD, rank, nranks, local_device, &cal_comm));
#else
    cal_comm_create_params_t params;
    params.allgather = allgather;
    params.req_test = request_test;
    params.req_free = request_free;
    params.data = (void*)(MPI_COMM_WORLD);
    params.rank = rank;
    params.nranks = nranks;
    params.local_device = local_device;
    CAL_CHECK(cal_comm_create(params, &cal_comm));
#endif

    hipStream_t stream = nullptr;
    CUDA_CHECK(hipStreamCreate(&stream));

    cublasMpHandle_t handle = nullptr;
    CUBLASMP_CHECK(cublasMpCreate(&handle, stream));

    cublasMpMatrixDescriptor_t descA = nullptr;
    cublasMpMatrixDescriptor_t descB = nullptr;
    cublasMpMatrixDescriptor_t descC = nullptr;

    cublasMpMatmulDescriptor_t matmulDesc = nullptr;

    output_t* d_X1 = nullptr;
    output_t* d_X2 = nullptr;

    void* d_work = nullptr;

    compute_t alpha = 1.0;
    compute_t beta = 0.0;

    size_t workspaceInBytesOnDevice = 0;
    size_t workspaceInBytesOnHost = 0;

    cublasMpGrid_t grid_col_major = nullptr;
    cublasMpGrid_t grid_row_major = nullptr;

    CUBLASMP_CHECK(cublasMpGridCreate(nranks, 1, CUBLASMP_GRID_LAYOUT_COL_MAJOR, cal_comm, &grid_col_major));
    CUBLASMP_CHECK(cublasMpGridCreate(1, nranks, CUBLASMP_GRID_LAYOUT_ROW_MAJOR, cal_comm, &grid_row_major));

    const bool ta = (transA != HIPBLAS_OP_N);

    // AG + Matmul
    {
        const int64_t m = 64 * nranks;
        const int64_t n = 64 * nranks;
        const int64_t k = 64;

        const int64_t loc_a_m = ta ? k : m / nranks;
        const int64_t loc_a_n = ta ? m / nranks : k;
        const int64_t loc_b_m = k;
        const int64_t loc_b_n = n / nranks;
        const int64_t loc_c_m = m / nranks;
        const int64_t loc_c_n = n / nranks;

        std::vector<input_t> h_X0(loc_a_m * loc_a_n, input_t(0));
        std::vector<input_t> h_W0(loc_b_m * loc_b_n, input_t(0));
        std::vector<output_t> h_X1(loc_c_m * loc_c_n * nranks, output_t(0));

        generate_random_matrix(
            ta ? k : m,
            ta ? m : k,
            h_X0.data(),
            loc_a_m,
            loc_a_n,
            1,
            1,
            loc_a_m,
            ta ? 1 : nranks,
            ta ? nranks : 1,
            ta ? 1 : rank,
            ta ? rank : 1);
        generate_random_matrix(k, n, h_W0.data(), loc_b_m, loc_b_n, 1, 1, loc_b_m, 1, nranks, 1, rank);
        generate_random_matrix(m, n, h_X1.data(), loc_c_m, loc_c_n, 1, 1, loc_c_m, nranks, 1, rank, 1);

        input_t* d_X0 = nullptr;
        input_t* d_W0 = nullptr;

        CUDA_CHECK(hipMalloc((void**)&d_X0, loc_a_m * loc_a_n * sizeof(input_t)));
        CUDA_CHECK(hipMalloc((void**)&d_W0, loc_b_m * loc_b_n * sizeof(input_t)));
        CUDA_CHECK(hipMalloc((void**)&d_X1, loc_c_m * loc_c_n * nranks * sizeof(output_t)));

        CUDA_CHECK(
            hipMemcpyAsync(d_X0, h_X0.data(), loc_a_m * loc_a_n * sizeof(input_t), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(
            hipMemcpyAsync(d_W0, h_W0.data(), loc_b_m * loc_b_n * sizeof(input_t), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(
            d_X1, h_X1.data(), loc_c_m * loc_c_n * nranks * sizeof(output_t), hipMemcpyHostToDevice, stream));

        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            ta ? k : m,
            ta ? m : k,
            loc_a_m,
            loc_a_n,
            0,
            0,
            loc_a_m,
            cuda_input_type,
            ta ? grid_row_major : grid_col_major,
            &descA));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            k, n, loc_b_m, loc_b_n, 0, 0, loc_b_m, cuda_input_type, grid_row_major, &descB));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            m, n, loc_c_m, loc_c_n, 0, 0, loc_c_m, cuda_output_type, grid_col_major, &descC));

        const cublasMpMatmulAlgoType_t algoType = CUBLASMP_MATMUL_ALGO_TYPE_SPLIT_P2P;

        CUBLASMP_CHECK(cublasMpMatmulDescriptorCreate(&matmulDesc, cublas_compute_type));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_TRANSA, &transA, sizeof(transA)));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_TRANSB, &transB, sizeof(transB)));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_ALGO_TYPE, &algoType, sizeof(algoType)));

        CUBLASMP_CHECK(cublasMpMatmul_bufferSize(
            handle,
            matmulDesc,
            m,
            n,
            k,
            &alpha,
            d_X0,
            1,
            1,
            descA,
            d_W0,
            1,
            1,
            descB,
            &beta,
            nullptr,
            1,
            1,
            descC,
            d_X1,
            1,
            1,
            descC,
            &workspaceInBytesOnDevice,
            &workspaceInBytesOnHost));

        // NVSHMEM is initialized as part of cublasMpGridCreate.
        d_work = nvshmem_malloc(workspaceInBytesOnDevice);

        std::vector<int8_t> h_work(workspaceInBytesOnHost);

        CAL_CHECK(cal_stream_sync(cal_comm, stream));
        CAL_CHECK(cal_comm_barrier(cal_comm, stream));

        const double begin = MPI_Wtime();

        CUBLASMP_CHECK(cublasMpMatmul(
            handle,
            matmulDesc,
            m,
            n,
            k,
            &alpha,
            d_X0,
            1,
            1,
            descA,
            d_W0,
            1,
            1,
            descB,
            &beta,
            nullptr,
            1,
            1,
            descC,
            d_X1,
            1,
            1,
            descC,
            d_work,
            workspaceInBytesOnDevice,
            h_work.data(),
            workspaceInBytesOnHost));

        CAL_CHECK(cal_stream_sync(cal_comm, stream));
        CAL_CHECK(cal_comm_barrier(cal_comm, stream));

        const double end = MPI_Wtime();

        if (rank == 0)
        {
            printf("AG + Matmul: %lf (s) %lf (GFlops)\n", end - begin, (2 * m * n * k * 1e-9) / (end - begin));
        }

        CUBLASMP_CHECK(cublasMpMatmulDescriptorDestroy(matmulDesc));

        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descA));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descB));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descC));

        CUDA_CHECK(hipFree(d_X0));
        CUDA_CHECK(hipFree(d_W0));
        nvshmem_free(d_work);
    }

    // Matmul + RS
    {
        const int64_t m = 64;
        const int64_t n = 64 * nranks;
        const int64_t k = 64 * nranks;

        const int64_t loc_a_m = ta ? k / nranks : m;
        const int64_t loc_a_n = ta ? m : k / nranks;
        const int64_t loc_b_m = k / nranks;
        const int64_t loc_b_n = n / nranks;
        const int64_t loc_c_m = m;
        const int64_t loc_c_n = n / nranks;

        std::vector<input_t> h_W1(loc_a_m * loc_a_n, input_t(0));
        std::vector<output_t> h_X2(loc_c_m * loc_c_n, output_t(0));

        generate_random_matrix(
            ta ? k : m,
            ta ? m : k,
            h_W1.data(),
            loc_a_m,
            loc_a_n,
            1,
            1,
            loc_a_m,
            ta ? nranks : 1,
            ta ? 1 : nranks,
            ta ? rank : 1,
            ta ? 1 : rank);
        generate_random_matrix(m, n, h_X2.data(), loc_c_m, loc_c_n, 1, 1, loc_c_m, 1, nranks, 1, rank);

        input_t* d_W1 = nullptr;

        CUDA_CHECK(hipMalloc((void**)&d_W1, loc_a_m * loc_a_n * sizeof(input_t)));
        CUDA_CHECK(hipMalloc((void**)&d_X2, loc_c_m * loc_c_n * sizeof(output_t)));

        CUDA_CHECK(
            hipMemcpyAsync(d_W1, h_W1.data(), loc_a_m * loc_a_n * sizeof(input_t), hipMemcpyHostToDevice, stream));
        CUDA_CHECK(
            hipMemcpyAsync(d_X2, h_X2.data(), loc_c_m * loc_c_n * sizeof(output_t), hipMemcpyHostToDevice, stream));

        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            ta ? k : m,
            ta ? m : k,
            loc_a_m,
            loc_a_n,
            0,
            0,
            loc_a_m,
            cuda_input_type,
            ta ? grid_col_major : grid_row_major,
            &descA));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            k, n, loc_b_m, loc_b_n, 0, 0, loc_b_m, cuda_input_type, grid_col_major, &descB));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorCreate(
            m, n, loc_c_m, loc_c_n, 0, 0, loc_c_m, cuda_output_type, grid_row_major, &descC));

        const cublasMpMatmulAlgoType_t algoType = CUBLASMP_MATMUL_ALGO_TYPE_SPLIT_P2P;

        CUBLASMP_CHECK(cublasMpMatmulDescriptorCreate(&matmulDesc, cublas_compute_type));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_TRANSA, &transA, sizeof(transA)));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_TRANSB, &transB, sizeof(transB)));
        CUBLASMP_CHECK(cublasMpMatmulDescriptorAttributeSet(
            matmulDesc, CUBLASMP_MATMUL_DESCRIPTOR_ATTRIBUTE_ALGO_TYPE, &algoType, sizeof(algoType)));

        CUBLASMP_CHECK(cublasMpMatmul_bufferSize(
            handle,
            matmulDesc,
            m,
            n,
            k,
            &alpha,
            d_W1,
            1,
            1,
            descA,
            d_X1,
            1,
            1,
            descB,
            &beta,
            nullptr,
            1,
            1,
            descC,
            d_X2,
            1,
            1,
            descC,
            &workspaceInBytesOnDevice,
            &workspaceInBytesOnHost));

        d_work = nvshmem_malloc(workspaceInBytesOnDevice);

        std::vector<int8_t> h_work(workspaceInBytesOnHost);

        CAL_CHECK(cal_stream_sync(cal_comm, stream));
        CAL_CHECK(cal_comm_barrier(cal_comm, stream));

        const double begin = MPI_Wtime();

        CUBLASMP_CHECK(cublasMpMatmul(
            handle,
            matmulDesc,
            m,
            n,
            k,
            &alpha,
            d_W1,
            1,
            1,
            descA,
            d_X1,
            1,
            1,
            descB,
            &beta,
            nullptr,
            1,
            1,
            descC,
            d_X2,
            1,
            1,
            descC,
            d_work,
            workspaceInBytesOnDevice,
            h_work.data(),
            workspaceInBytesOnHost));

        CAL_CHECK(cal_stream_sync(cal_comm, stream));
        CAL_CHECK(cal_comm_barrier(cal_comm, stream));

        const double end = MPI_Wtime();

        if (rank == 0)
        {
            printf("Matmul + RS: %lf (s) %lf (GFlops)\n", end - begin, (2 * m * n * k * 1e-9) / (end - begin));
        }

        CUBLASMP_CHECK(cublasMpMatmulDescriptorDestroy(matmulDesc));

        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descA));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descB));
        CUBLASMP_CHECK(cublasMpMatrixDescriptorDestroy(descC));

        CUDA_CHECK(hipFree(d_X1));
        CUDA_CHECK(hipFree(d_W1));
        CUDA_CHECK(hipFree(d_X2));
        nvshmem_free(d_work);
    }

    CUBLASMP_CHECK(cublasMpGridDestroy(grid_col_major));
    CUBLASMP_CHECK(cublasMpGridDestroy(grid_row_major));

    CUBLASMP_CHECK(cublasMpDestroy(handle));

    CAL_CHECK(cal_comm_barrier(cal_comm, stream));

    CAL_CHECK(cal_comm_destroy(cal_comm));

    CUDA_CHECK(hipStreamDestroy(stream));

    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Finalize();

    if (rank == 0)
    {
        printf("[SUCCEEDED]\n");
    }

    return 0;
};