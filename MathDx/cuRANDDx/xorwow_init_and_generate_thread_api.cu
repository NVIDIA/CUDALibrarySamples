#include "hip/hip_runtime.h"
// Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include <iostream>
#include <vector>

#include <curanddx.hpp>
#include "common.hpp"

// This example demonstrates how to launch two kernels, one for initializing the states, one for generating normally distribute 
// random numbers with Box-Muller method, using XORWOW generator. 
// 
// cuRANDDx functions used in the example kernels:
// (1) state initialization using init() function, or the constructor of the RNG object
// (2) generate2() function as Box-Muller method is selected for normal distribution
//
// The generated RNs are the same as the results using cuRAND host API with HIPRAND_ORDERING_PSEUDO_LEGACY ordering
// The ordering requires that the result at offset n is from position  
// (n mod 4096) * 2^67 + (i/4096) in the original XORWOW sequence
// https://docs.nvidia.com/cuda/hiprand/host-api-overview.html#generator-types 

constexpr unsigned int subsequences = 4096;

template<class RNG>
__global__ void init_kernel(RNG* states, const unsigned long long seed, const typename RNG::offset_type offset) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    RNG rng;
    rng.init(seed, ((offset + tid) % subsequences), ((offset + tid) / subsequences)); // seed, subsequence, offset

    // Alternatively one can use the constructor directly
    //RNG rng(seed, ((offset + tid) % subsequences), ((offset + tid) / subsequences)); 
    
    states[tid] = rng;
}

template<class RNG, typename DataType>
__global__ void generate_kernel(RNG* states, float2* d_out, const size_t size, const DataType mean, const DataType stddev) {
    int       tid     = blockDim.x * blockIdx.x + threadIdx.x;
    const int threads = blockDim.x * gridDim.x;

    curanddx::normal<DataType, curanddx::box_muller> dist(mean, stddev);

    RNG rng = states[tid];

    for (auto idx = tid; idx < size / 2; idx += threads) {
        d_out[idx] = dist.generate2(rng);
    }

    // Each thread updates "states" in global memory so "states" can be used for later kernels
    states[tid] = rng;
}


template<unsigned int Arch>
int xorwow_init_and_generate_thread_api() {
    using RNG      = decltype(curanddx::Generator<curanddx::xorwow>() + curanddx::SM<Arch>() + curanddx::Thread());
    using DataType = float;

    // Allocate output memory
    DataType*    d_out;
    const size_t size = 50000;
    CUDA_CHECK_AND_EXIT(hipMalloc((void**)&d_out, size * sizeof(DataType)));

    const unsigned long long seed   = 1234ULL;
    const typename RNG::offset_type offset = 2ULL;

    const unsigned int block_dim = 256;
    const unsigned int grid_size = 16;

    const DataType mean = 0; 
    const DataType stddev = 2;

    // Allocate an array of states
    RNG* states;
    CUDA_CHECK_AND_EXIT(hipMalloc((void**)&states, block_dim * grid_size * sizeof(RNG)));

    // Invoke the init kernel first to set up the states
    init_kernel<RNG><<<grid_size, block_dim>>>(states, seed, offset);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());

    // Invoke the generate kernel to generate RNs and update the states
    generate_kernel<RNG, DataType><<<grid_size, block_dim>>>(states, (float2*)d_out, size, mean, stddev);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    std::vector<DataType> h_out(size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(h_out.data(), d_out, size * sizeof(DataType), hipMemcpyDeviceToHost));

    // cuRAND host API
    hiprandGenerator_t gen_curand;
    DataType*         d_ref;
    CUDA_CHECK_AND_EXIT(hipMalloc((void**)&d_ref, size * sizeof(DataType)));

    CURAND_CHECK_AND_EXIT(hiprandCreateGenerator(&gen_curand, HIPRAND_RNG_PSEUDO_XORWOW));
    CURAND_CHECK_AND_EXIT(hiprandSetPseudoRandomGeneratorSeed(gen_curand, seed));
    CURAND_CHECK_AND_EXIT(hiprandSetGeneratorOffset(gen_curand, offset));
    CURAND_CHECK_AND_EXIT(hiprandSetGeneratorOrdering(gen_curand, HIPRAND_ORDERING_PSEUDO_LEGACY));

    CURAND_CHECK_AND_EXIT(hiprandGenerateNormal(gen_curand, d_ref, size, mean, stddev));

    std::vector<DataType> h_ref(size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(h_ref.data(), d_ref, size * sizeof(DataType), hipMemcpyDeviceToHost));

    CURAND_CHECK_AND_EXIT(hiprandDestroyGenerator(gen_curand));
    CUDA_CHECK_AND_EXIT(hipFree(states));
    CUDA_CHECK_AND_EXIT(hipFree(d_out));
    CUDA_CHECK_AND_EXIT(hipFree(d_ref));

    // Compare Results between cuRAND host API and cuRANDDx
    if (h_out == h_ref) {
        std::cout << "SUCCESS: Same sequence is generated with cuRANDDx and cuRAND Host API using LEGACY ordering.\n";
        return 0;
    } else {
        unsigned int count {0};
        for (auto i = 0U; i < size; i++) {
            if (h_out[i] != h_ref[i] && count < 10) {
                std::cout << "array_curanddx[" << i << "] = " << h_out[i] << " array_curand[" << i << "] = " << h_ref[i]
                          << std::endl;
                count++;
            }
        }
        std::cout << "FAILED: Different sequence is generated with cuRANDDx and cuRAND Host API using LEGACY "
                     "ordering.\n";
        return 1;
    }
}

template<unsigned int Arch>
struct xorwow_init_and_generate_thread_api_functor {
    int operator()() { return xorwow_init_and_generate_thread_api<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<xorwow_init_and_generate_thread_api_functor>();
}
