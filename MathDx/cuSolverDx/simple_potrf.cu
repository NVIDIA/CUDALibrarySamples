#include "hip/hip_runtime.h"
// Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include <iostream>

#include <cusolverdx.hpp>
#include "common.hpp"

// This example shows how to perform Cholesky factorization for a Hermitian positive-definite matrix
// using cuSolverDx, and compare the result factors with cuSolver host API.

template<class Solver, typename DataType = typename Solver::a_data_type>
__global__ void potrf_kernel(DataType* A, typename Solver::status_type* info) {

    extern __shared__ unsigned char shared_mem[];
    DataType* As = reinterpret_cast<DataType*>(shared_mem);

    constexpr auto lda_smem = Solver::lda;
    constexpr auto lda_gmem = Solver::m_size;

    // Load data from global memory to registers
    common::io<Solver>::load(A, lda_gmem, As, lda_smem);

    Solver().execute(As, info);

    // store
    common::io<Solver>::store(As, lda_smem, A, lda_gmem);
}

template<int Arch>
int simple_potrf() {

    using namespace cusolverdx;
    using Solver = decltype(Size<32, 32>() + Precision<double>() + Type<type::complex>() + Function<potrf>() + Block() + LeadingDimension<33>() + SM<Arch>() + BlockDim<256>() +
                            FillMode<fill_mode::upper>());

#ifdef CUSOLVERDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using data_type      = typename example::a_data_type_t<Solver>;
    using cuda_data_type = typename example::a_cuda_data_type_t<Solver>;
#else
    using data_type      = typename Solver::a_data_type;
    using cuda_data_type = typename Solver::a_cuda_data_type;
#endif
    constexpr auto m = Solver::m_size;
    constexpr auto n = Solver::n_size;
    static_assert(m == n, "potrf is for Hermitian positive-definite matrix matrix only");
    constexpr auto lda_smem = Solver::lda;

    constexpr auto lda        = m;       // this is the leading dimension in global memory for A
    constexpr auto input_size = lda * n; // input global memory size for A

    std::cout << "Use compile-time leading dimension LDA for shared memory = " << lda_smem << std::endl;

    hipStream_t stream = nullptr;
    CUDA_CHECK_AND_EXIT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    std::vector<data_type> A(input_size);
    common::fillup_random_diagonal_dominant_matrix_col_major<data_type>(m, n, A.data(), lda, false, -2, 2); // input A is not symmetric

    // To get around cuSolver potrsBatched bug for CUDA <= 12.2, set the diagonal elements of input matrix A to be real
#if (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ <= 2)
    if constexpr (common::is_complex<data_type>()) {
        for (unsigned int row = 0; row < m; row++) {
            A[row + row * lda].y = 0;
        }
    }
#endif

    // Uncomment out the following lines to print out the matrix A
    // printf("A = \n");
    // common::print_matrix(m, n, A.data(), lda);
    // printf("=====\n");

    std::vector<data_type> L(input_size);
    int                    info   = 0;
    data_type*             d_A    = nullptr; /* device copy of A */
    int*                   d_info = nullptr; /* error info */

    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int)));

    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));

    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(potrf_kernel<Solver>), hipFuncAttributeMaxDynamicSharedMemorySize, Solver::shared_memory_size));
    //Invokes kernel
    potrf_kernel<Solver><<<1, Solver::block_dim, Solver::shared_memory_size, stream>>>(d_A, d_info);

    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(L.data(), d_A, sizeof(data_type) * A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK_AND_EXIT(hipStreamSynchronize(stream));

    printf("after cuSolverDx potrf kernel: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    // uncomment out the following lines to print out the result factors
    // printf("L = \n");
    // common::print_matrix(m, m, L.data(), lda);

    //=========================
    // cuSolver reference
    //=========================
    // Use dumb B as only factorization is performed
    std::vector<data_type> B;
    common::reference_cusolver_cholesky<data_type, cuda_data_type>(
        A, B, &info, m, 1, 1, (fill_mode_of_v<Solver> == fill_mode::lower));

    // check result
    const auto total_relative_error = common::check_error<data_type, data_type>(L.data(), A.data(), A.size());
    std::cout << "Solver: relative error of A between cuSolverDx and cuSolver results: " << total_relative_error << std::endl;

    /* free resources */
    CUDA_CHECK_AND_EXIT(hipFree(d_A));
    CUDA_CHECK_AND_EXIT(hipFree(d_info));
    CUDA_CHECK_AND_EXIT(hipDeviceReset());

    if (common::is_error_acceptable<data_type>(total_relative_error)) {
        std::cout << "Success compared with cuSolver API results" << std::endl;
        return 0;
    }
    std::cout << "Failure" << std::endl;
    return 1;
}

template<int Arch>
struct simple_potrf_functor {
    int operator()() { return simple_potrf<Arch>(); }
};


int main() { return common::run_example_with_sm<simple_potrf_functor>(); }
