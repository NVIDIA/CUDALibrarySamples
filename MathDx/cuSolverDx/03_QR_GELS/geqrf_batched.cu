#include "hip/hip_runtime.h"
// Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include <cusolverdx.hpp>

#include "../common/common.hpp"
#include "../common/cudart.hpp"
#include "../common/error_checking.hpp"
#include "../common/random.hpp"
#include "../common/example_sm_runner.hpp"
#include "../common/device_io.hpp"
#include "../common/print.hpp"
#include "../common/cusolver_reference_qr.hpp"

// This example demonstrates how to use cuSolverDx API to compute the QR factorization on a batched m x n matrix A.
// The results are compared with the reference values obtained with cuSolver host API.

template<class Solver, unsigned int BatchesPerBlock, typename DataType = typename Solver::a_data_type>
__global__ __launch_bounds__(Solver::max_threads_per_block) void kernel(DataType* A, const int lda_gmem, DataType* tau, const unsigned batches) {

    constexpr auto m = Solver::m_size;
    constexpr auto n = Solver::n_size;

    const auto     one_batch_size_a_gmem = (cusolverdx::arrangement_of_v_a<Solver> == cusolverdx::col_major) ? lda_gmem * n : m * lda_gmem;
    constexpr auto lda_smem              = Solver::lda;
    constexpr auto one_batch_size_a_smem = (cusolverdx::arrangement_of_v_a<Solver> == cusolverdx::col_major) ? lda_smem * n : m * lda_smem;

    extern __shared__ __align__(16) unsigned char shared_mem[];
    // Slice shared memory into pointers
    auto [As, tau_s] = cusolverdx::shared_memory::slice<DataType, DataType>(
        shared_mem,
        alignof(DataType), one_batch_size_a_smem * BatchesPerBlock,
        alignof(DataType)  // the size (number of elements) may be omitted for the last pointer
    );

    const auto batch_idx = blockIdx.x * BatchesPerBlock;
    if (batch_idx >= batches)
        return;
    auto Ag    = A + one_batch_size_a_gmem * batch_idx;
    auto tau_g = tau + min(m, n) * batch_idx;

    // Load data from global memory to shared memory
    common::io<Solver, BatchesPerBlock>::load_a(Ag, lda_gmem, As, lda_smem);

    Solver().execute(As, lda_smem, tau_s);

    // Store results back to global memory
    common::io<Solver, BatchesPerBlock>::store_a(As, lda_smem, Ag, lda_gmem);

    // store tau from shared memory to global memory
    int thread_id = threadIdx.x + Solver::block_dim.x * (threadIdx.y + Solver::block_dim.y * threadIdx.z);
    for (int i = thread_id; i < min(m, n) * BatchesPerBlock; i += Solver::max_threads_per_block) {
        tau_g[i] = tau_s[i];
    }
}

template<int Arch>
int geqrf_batched() {

    using namespace cusolverdx;
    using Base   = decltype(Size<16, 20>() + Precision<float>() + Type<type::real>() + Function<geqrf>() + Arrangement<arrangement::row_major>() + TransposeMode<non_trans>() + SM<Arch>() + Block() +
                          BlockDim<64>());
    using Solver = decltype(Base() + BatchesPerBlock<Base::suggested_batches_per_block>());

#ifdef CUSOLVERDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using data_type      = typename example::a_data_type_t<Solver>;
    using cuda_data_type = typename example::a_cuda_data_type_t<Solver>;
#else
    using data_type      = typename Solver::a_data_type;
    using cuda_data_type = typename Solver::a_cuda_data_type;
#endif
    constexpr unsigned bpb = Solver::batches_per_block;
    std::cout << "Using Suggested Batches per block = " << bpb << std::endl;
    std::cout << "Suggested BlockDim = " << Solver::suggested_block_dim.x << std::endl;
    std::cout << "BlockDim Used = " << Solver::block_dim.x << std::endl;

    constexpr auto m = Solver::m_size;
    constexpr auto n = Solver::n_size;

    constexpr bool is_col_maj_a = arrangement_of_v_a<Solver> == arrangement::col_major;

    const auto     lda          = is_col_maj_a ? m : n;
    constexpr auto input_size_a = m * n;

    const auto batches        = 2;
    const auto padded_batches = (batches + bpb - 1) / bpb * bpb;

    hipStream_t stream = nullptr;
    CUDA_CHECK_AND_EXIT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    std::vector<data_type> A(input_size_a * padded_batches);
    common::fillup_random_matrix<data_type>(is_col_maj_a, m, n, A.data(), lda, false, false, 2, 4, batches); // not symmetric and not diagonally dominant

    std::vector<data_type> L(input_size_a * padded_batches);
    std::vector<data_type> tau(min(m, n) * padded_batches, 0);
    std::vector<data_type> tau_ref(min(m, n) * padded_batches, 0);
    data_type*             d_A   = nullptr;
    data_type*             d_tau = nullptr;

    // Comment below to remove printing A matrix
    printf("A = \n");
    common::print_matrix<data_type, m, n, lda, is_col_maj_a>(A.data(), batches);

    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_tau), sizeof(data_type) * tau.size()));

    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));

    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel<Solver), bpb>, hipFuncAttributeMaxDynamicSharedMemorySize, Solver::shared_memory_size));

    //Invokes kernel
    kernel<Solver, bpb><<<padded_batches / bpb, Solver::block_dim, Solver::shared_memory_size, stream>>>(d_A, lda, d_tau, batches);
    CUDA_CHECK_AND_EXIT(hipGetLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(L.data(), d_A, sizeof(data_type) * A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(tau.data(), d_tau, sizeof(data_type) * tau.size(), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK_AND_EXIT(hipStreamSynchronize(stream));

    // Comment below to remove printing results after cuSolverDx execute
    printf("=====\n");
    printf(" after cuSolverDx\n");
    printf("L = \n");
    common::print_matrix<data_type, m, n, lda, is_col_maj_a>(L.data(), batches);


    /* free resources */
    CUDA_CHECK_AND_EXIT(hipFree(d_A));
    CUDA_CHECK_AND_EXIT(hipFree(d_tau));

    //=========================
    // cuSolver reference
    //=========================
    std::vector<data_type> dummy_b;
    common::reference_cusolver_qr<data_type, cuda_data_type, false>(A, dummy_b, tau_ref, m, n, 1, padded_batches, batches, is_col_maj_a);


    // check A
    const auto total_relative_error_a = common::check_error<data_type, data_type>(L.data(), A.data(), batches * input_size_a);
    std::cout << "GEQRF: relative error of A between cuSolverDx and cuSolver results: " << total_relative_error_a << std::endl;
    const auto total_relative_error_tau = common::check_error<data_type, data_type>(tau.data(), tau_ref.data(), batches * min(m, n));
    std::cout << "GEQRF: relative error of tau between cuSolverDx and cuSolver results: " << total_relative_error_tau << std::endl;

    // Comment below to remove printing results after cuSolver reference execute
    printf("Lref = \n");
    common::print_matrix<data_type, m, n, lda, is_col_maj_a>(A.data(), batches);
    printf("=====\n");


    if (!common::is_error_acceptable<data_type>(total_relative_error_a)) {
        std::cout << "Failure compared with cuSolver API results A" << std::endl;
        return 1;
    }
    if (!common::is_error_acceptable<data_type>(total_relative_error_tau)) {
        std::cout << "Failure compared with cuSolver API results TAU" << std::endl;
        //Print out tau for debugging. Do not delete
        for (int i = 0; i < min(m, n) * batches; ++i) {
            if (abs(tau[i] - tau_ref[i]) / abs(tau_ref[i]) > 1e-05) {
                printf("tau[%d] = %10.3f, tau_ref[%d] = %10.3f  differ \n", i, tau[i], i, tau_ref[i]);
            }
        }
        return 1;
    }

    std::cout << "Success compared with cuSolver API results, A and tau" << std::endl;
    return 0;
}

template<int Arch>
struct geqrf_batched_functor {
    int operator()() { return geqrf_batched<Arch>(); }
};


int main() { return common::run_example_with_sm<geqrf_batched_functor>(); }
