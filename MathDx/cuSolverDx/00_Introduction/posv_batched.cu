#include "hip/hip_runtime.h"
// Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include <cusolverdx.hpp>

#include "../common/common.hpp"
#include "../common/cudart.hpp"
#include "../common/error_checking.hpp"
#include "../common/random.hpp"
#include "../common/example_sm_runner.hpp"
#include "../common/device_io.hpp"
#include "../common/measure.hpp"
#include "../common/print.hpp"
#include "../common/cusolver_reference_cholesky.hpp"

// This example demonstrates how to use cuSolverDx API to solve a batched linear systems with multiple right hand side
// after performing Cholesky factorization of the batched symmetric, positive-definite matrix A.
// The results are compared with the reference values obtained with cuSolver host API.

template<class POSV, unsigned int BatchesPerBlock, class DataType = typename POSV::a_data_type>
__global__ __launch_bounds__(POSV::max_threads_per_block) void posv_kernel(DataType* A, const unsigned int lda_gmem, DataType* B, const unsigned int ldb_gmem, typename POSV::status_type* info, const unsigned int batches) {

    using namespace cusolverdx;
    constexpr auto m                     = POSV::m_size;
    constexpr auto nrhs                  = POSV::k_size;
    const auto     one_batch_size_a_gmem = lda_gmem * m;
    const auto     one_batch_size_b_gmem = (arrangement_of_v_b<POSV> == arrangement::col_major) ? ldb_gmem * nrhs : m * ldb_gmem;

    constexpr auto lda_smem              = POSV::lda;
    constexpr auto ldb_smem              = POSV::ldb;
    constexpr auto one_batch_size_a_smem = lda_smem * m;
    constexpr auto one_batch_size_b_smem = (arrangement_of_v_b<POSV> == arrangement::col_major) ? ldb_smem * nrhs : m * ldb_smem;

    extern __shared__ __align__(sizeof(DataType)) char shared_mem[];

    DataType* As = reinterpret_cast<DataType*>(shared_mem);
    DataType* Bs = As + one_batch_size_a_smem * BatchesPerBlock;

    const auto batch_idx = blockIdx.x * BatchesPerBlock;
    if (batch_idx >= batches)
        return;

    auto Ag = A + size_t(one_batch_size_a_gmem) * batch_idx;
    auto Bg = B + size_t(one_batch_size_b_gmem) * batch_idx;

    // Load data from global memory to shared memory
    common::io<POSV, BatchesPerBlock>::load_a(Ag, lda_gmem, As, lda_smem);
    common::io<POSV, BatchesPerBlock>::load_b(Bg, ldb_gmem, Bs, ldb_smem);

    POSV().execute(As, lda_smem, Bs, &info[batch_idx]);

    // Store results back to global memory
    common::io<POSV, BatchesPerBlock>::store_a(As, lda_smem, Ag, lda_gmem);
    common::io<POSV, BatchesPerBlock>::store_b(Bs, ldb_smem, Bg, ldb_gmem);
}

template<int Arch>
int simple_posv_batched() {

    using namespace cusolverdx;

    using POSV = decltype(Size<32 /* = m */, 32 /* = n */, 1 /* = k */>() + Precision<double>() + Type<type::complex>() + Function<function::posv>() + FillMode<lower>() +
                          Arrangement<col_major /* A, X, and B */>() + SM<Arch>() + Block());

    constexpr unsigned bpb = POSV::batches_per_block;
    std::cout << "Using Suggested Batches per block = " << bpb << std::endl;
    std::cout << "Using Suggested BlockDim = " << POSV::suggested_block_dim.x << std::endl;
    std::cout << "Using Specified BlockDim = " << POSV::block_dim.x << std::endl;

#ifdef CUSOLVERDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using data_type      = typename example::a_data_type_t<POSV>;
    using cuda_data_type = typename example::a_cuda_data_type_t<POSV>;
#else
    using data_type      = typename POSV::a_data_type;
    using cuda_data_type = typename POSV::a_cuda_data_type;
#endif
    constexpr auto m    = POSV::m_size;
    constexpr auto n    = POSV::n_size;
    constexpr auto nrhs = POSV::k_size;
    static_assert(m == n, "posv is for Hermitian positive-definite matrix matrix only");
    constexpr auto lda_smem = POSV::lda;
    constexpr auto ldb_smem = POSV::ldb;

    constexpr bool is_col_maj_a = arrangement_of_v_a<POSV> == arrangement::col_major;
    constexpr bool is_col_maj_b = arrangement_of_v_b<POSV> == arrangement::col_major;

    // no padding for global memory
    constexpr auto lda = m;
    constexpr auto ldb = is_col_maj_b ? m : nrhs;

    printf("Size m = %d, n = %d, nrhs = %d\n", m, n, nrhs);
    std::cout << "Using leading dimension LDA = " << lda_smem << ", LDB = " << ldb_smem << std::endl;

    const auto batches        = 2;
    const auto padded_batches = (batches + bpb - 1) / bpb * bpb;

    const auto one_batch_size_A = lda * n; // no padding for global memory
    const auto one_batch_size_B = m * nrhs;

    hipStream_t stream = nullptr;
    CUDA_CHECK_AND_EXIT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    std::vector<data_type> A(one_batch_size_A * padded_batches);
    std::vector<data_type> L(one_batch_size_A * padded_batches);

    common::fillup_random_diagonal_dominant_matrix<data_type>(arrangement_of_v_a<POSV> == col_major, m, n, A.data(), lda, false, 2, 4, batches);

    // To get around cuSolver potrsBatched bug for CUDA <=12.2, set the diagonal elements of input matrix A to be real
#if (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ <= 2)
    if constexpr (common::is_complex<data_type>()) {
        for (auto batch = 0; batch < batches; batch++) {
            for (unsigned int row = 0; row < m; row++) {
                A[row + row * lda + batch * one_batch_size_A].y = 0;
            }
        }
    }
#endif

    std::vector<data_type> B(one_batch_size_B * padded_batches);
    common::fillup_random_matrix<data_type>(arrangement_of_v_b<POSV> == col_major, m, nrhs, B.data(), ldb, false, false, -1, 1, batches);
    std::vector<data_type> X(one_batch_size_B * padded_batches);

    std::vector<int> info(padded_batches, 0);
    data_type*       d_A    = nullptr; /* device copy of A */
    data_type*       d_B    = nullptr; /* device copy of B */
    int*             d_info = nullptr; /* error info */


    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int) * padded_batches));

    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream));

    // Increase max dynamic shared memory for the kernel if needed.
    const auto sm_size = POSV::shared_memory_size;

    const auto kernel = posv_kernel<POSV, POSV::batches_per_block>;
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, sm_size));

    //Invokes kernel
    kernel<<<(batches + bpb - 1) / bpb, POSV::block_dim, sm_size, stream>>>(d_A, lda, d_B, ldb, d_info, batches);
    CUDA_CHECK_AND_EXIT(hipGetLastError());
    CUDA_CHECK_AND_EXIT(hipStreamSynchronize(stream));

    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(L.data(), d_A, sizeof(data_type) * A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(X.data(), d_B, sizeof(data_type) * B.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(info.data(), d_info, sizeof(int) * batches, hipMemcpyDeviceToHost, stream));

    CUDA_CHECK_AND_EXIT(hipStreamSynchronize(stream));
    if (std::accumulate(info.begin(), info.end(), 0) != 0) {
        std::cout << "non-zero d_info returned for at least one of the batches after cuSolverDx kernel \n";
        for (int j = 0; j < batches; j++) {
            if (info[j] != 0)
                std::cout << "info[" << j << "]=" << info[j] << std::endl;
        }
        return -1;
    }

    // Uncomment below to print the results after cuSolverDx execute
    //printf("after cuSolverDx execute\n");
    //printf("L = \n");
    //common::print_matrix<data_type, m, n, lda, is_col_maj_a>(L.data(), padded_batches);
    //printf("=====\n");
    //printf("X = \n");
    // common::print_matrix<data_type, n, nrhs, ldb, is_col_maj_b>(X.data(), padded_batches);
    // printf("=====\n");

    //=======================================================
    // cuSolver reference with potrfBatched and portsBatched
    //=======================================================
    common::reference_cusolver_cholesky<data_type, cuda_data_type, true>(A,
                                                                         B,
                                                                         info.data(),
                                                                         m,
                                                                         nrhs,
                                                                         padded_batches,
                                                                         (fill_mode_of_v<POSV> == fill_mode::lower),           /* is_lower? */
                                                                         is_col_maj_a,
                                                                         is_col_maj_b,
                                                                         batches);

    auto total_relative_error = common::check_error<data_type, data_type>(L.data(), A.data(), batches * one_batch_size_A);
    printf("BATCHED POSV: relative error of A between cuSolverDx and cuSolver results: = %e\n", total_relative_error);

    // Uncomment below to print the results after cuSolver reference execute
    // printf("after cuSolver API execute\n");
    // printf("A = \n");
    // common::print_matrix<data_type, m, n, lda, is_col_maj_a>(A.data(), padded_batches);
    // printf("B = \n");
    // common::print_matrix<data_type, n, nrhs, ldb, is_col_maj_b>(B.data(), padded_batches);
    // printf("=====\n");

    total_relative_error = common::check_error<data_type, data_type>(X.data(), B.data(), batches * one_batch_size_B);
    printf("BATCHED POSV: relative error of B between cuSolverDx and cuSolver results: = %e\n", total_relative_error);
    if (common::is_error_acceptable<data_type>(total_relative_error)) {
        std::cout << "Success compared to cuSolver potrSBatched Result " << std::endl;
    } else {
        std::cout << "Failure compared to cuSolver potrSBatched Result " << std::endl;
        return 1;
    }

    CUDA_CHECK_AND_EXIT(hipFree(d_A));
    CUDA_CHECK_AND_EXIT(hipFree(d_B));
    CUDA_CHECK_AND_EXIT(hipFree(d_info));

    CUDA_CHECK_AND_EXIT(hipDeviceReset());

    return 0;
}

template<int Arch>
struct simple_posv_batched_functor {
    int operator()() { return simple_posv_batched<Arch>(); }
};


int main() { return common::run_example_with_sm<simple_posv_batched_functor>(); }
