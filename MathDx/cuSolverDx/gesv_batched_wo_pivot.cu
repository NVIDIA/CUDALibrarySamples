#include "hip/hip_runtime.h"
// Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#include <iostream>

#include <cusolverdx.hpp>
#include "common.hpp"

// This example demonstrates how to use cuSolverDx API to solve a batched linear systems with multiple right hand side after performing LU
// factorization (without pivoting) of the batched general matrix A.  The results are compared with the reference values obtained with cuSolver host API.

template<class Solver, unsigned int BatchesPerBlock, typename DataType = typename Solver::a_data_type>
__global__ void kernel(DataType* A, unsigned int lda, DataType* B, unsigned int ldb, typename Solver::status_type* info, const unsigned int batches) {

    constexpr auto m    = Solver::m_size;
    constexpr auto n    = Solver::n_size;
    constexpr auto nrhs = Solver::nrhs;

    const auto lda_gmem              = cusolverdx::arrangement_of_v_a<Solver> == cusolverdx::col_major ? m : n;
    const auto ldb_gmem              = cusolverdx::arrangement_of_v_b<Solver> == cusolverdx::col_major ? n : nrhs;
    const auto one_batch_size_a_gmem = (cusolverdx::arrangement_of_v_b<Solver> == cusolverdx::col_major) ? lda * n: m * lda;
    const auto one_batch_size_b_gmem = (cusolverdx::arrangement_of_v_b<Solver> == cusolverdx::col_major) ? ldb * nrhs : n * ldb;

    constexpr auto lda_smem              = Solver::lda;
    constexpr auto ldb_smem              = Solver::ldb;
    constexpr auto one_batch_size_a_smem = (cusolverdx::arrangement_of_v_a<Solver> == cusolverdx::col_major) ? lda_smem * n : m * lda_smem;
    constexpr auto one_batch_size_b_smem = (cusolverdx::arrangement_of_v_b<Solver> == cusolverdx::col_major) ? ldb_smem * nrhs : n * ldb_smem;

    extern __shared__ unsigned char __align__(sizeof(DataType)) shared_mem[];

    DataType* As = reinterpret_cast<DataType*>(shared_mem);
    DataType* Bs = As + one_batch_size_a_smem * BatchesPerBlock;

    const auto batch_idx = blockIdx.x * BatchesPerBlock;
    if (batch_idx >= batches)
        return;

    auto Ag = A + size_t(one_batch_size_a_gmem) * batch_idx;
    auto Bg = B + size_t(one_batch_size_b_gmem) * batch_idx;

    // Load data from global memory to registers
    common::io<Solver, BatchesPerBlock>::load(Ag, lda_gmem, As, lda_smem);
    common::io<Solver, BatchesPerBlock>::load_rhs(Bg, ldb_gmem, Bs, ldb_smem);

    Solver().execute(As, lda_smem, Bs, ldb_smem, &info[batch_idx]);

    // store
    common::io<Solver, BatchesPerBlock>::store(As, lda_smem, Ag, lda_gmem);
    common::io<Solver, BatchesPerBlock>::store_rhs(Bs, ldb_smem, Bg, ldb_gmem);
}

template<int Arch>
int gesv_batched_wo_pivot() {

    using namespace cusolverdx;
    using Base   = decltype(Size<5, 5, 4>() + Precision<float>() + Type<type::real>() + Function<gesv_no_pivot>() + Arrangement<arrangement::col_major, col_major>() + LeadingDimension<5, 7>() +
                          SM<Arch>() + Block());
    using Solver = decltype(Base() + BatchesPerBlock<Base::suggested_batches_per_block>() + BlockDim<65, 1, 1>());

    constexpr unsigned bpb = Solver::batches_per_block;
    std::cout << "Using Suggested Batches per block = " << bpb << std::endl;
    std::cout << "Suggested BlockDim = " << Solver::suggested_block_dim.x << std::endl;
    std::cout << "BlockDim Used = " << Solver::block_dim.x << std::endl;

#ifdef CUSOLVERDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using data_type      = typename example::a_data_type_t<Solver>;
    using cuda_data_type = typename example::a_cuda_data_type_t<Solver>;
#else
    using data_type      = typename Solver::a_data_type;
    using cuda_data_type = typename Solver::a_cuda_data_type;
#endif
    constexpr auto m            = Solver::m_size;
    constexpr auto n            = Solver::n_size;
    constexpr auto nrhs         = Solver::nrhs;
    const auto     lda          = arrangement_of_v_a<Solver> == col_major ? m : n;
    const auto     ldb          = arrangement_of_v_b<Solver> == col_major ? n : nrhs;
    const auto     input_size_a = m * n;
    const auto     input_size_b = n * nrhs;

    const auto batches        = 14;
    const auto padded_batches = (batches + bpb - 1) / bpb * bpb;

    hipStream_t stream = nullptr;
    CUDA_CHECK_AND_EXIT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    std::vector<data_type> A(input_size_a * padded_batches);
    common::fillup_random_diagonal_dominant_matrix<data_type>(arrangement_of_v_a<Solver> == col_major, m, n, A.data(), lda, false, 2, 4, batches); // not symmetric
    std::vector<data_type> L(input_size_a * padded_batches);

    std::vector<data_type> B(input_size_b * padded_batches);
    common::fillup_random_matrix<data_type>(arrangement_of_v_b<Solver> == col_major, n, nrhs, B.data(), ldb, false, false, -1, 1, batches);
    std::vector<data_type> X(input_size_b * padded_batches);

    std::vector<int> info(padded_batches, 0);
    data_type*       d_A    = nullptr; /* device copy of A */
    data_type*       d_B    = nullptr; /* device copy of A */
    int*             d_info = nullptr; /* error info */

    // printf("A = \n");
    // common::print_matrix(m, n * batches, A.data(), lda);
    // const unsigned int b_dim_fast = (arrangement_of_v_b<Solver> == col_major) ? n : nrhs;
    // printf("B = \n");
    // common::print_matrix(b_dim_fast, B.size() / b_dim_fast, B.data(), ldb);

    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK_AND_EXIT(hipMalloc(reinterpret_cast<void**>(&d_info), sizeof(int) * info.size()));

    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice, stream));

    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel<Solver), bpb>, hipFuncAttributeMaxDynamicSharedMemorySize, Solver::shared_memory_size));

    //Invokes kernel
    kernel<Solver, bpb><<<padded_batches / bpb, Solver::block_dim, Solver::shared_memory_size, stream>>>(d_A, lda, d_B, ldb, d_info, batches);
    CUDA_CHECK_AND_EXIT(hipGetLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(L.data(), d_A, sizeof(data_type) * A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(X.data(), d_B, sizeof(data_type) * B.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_AND_EXIT(hipMemcpyAsync(info.data(), d_info, sizeof(int) * batches, hipMemcpyDeviceToHost, stream));

    CUDA_CHECK_AND_EXIT(hipStreamSynchronize(stream));

    if (std::accumulate(info.begin(), info.end(), 0) != 0) {
        std::cout << "non-zero d_info returned for at least one of the batches after cuSolverDx kernel \n";
        for (int j = 0; j < batches; j++) {
            if (info[j] != 0)
                std::cout << "info[" << j << "]=" << info[j] << std::endl;
        }
        return -1;
    }
    // printf("=====\n");
    // printf(" after cuSolverDx\n");
    // printf("L = \n");
    // common::print_matrix(m, n * batches, L.data(), lda);
    // printf("X = \n");
    // common::print_matrix(b_dim_fast, X.size() / b_dim_fast, X.data(), ldb);

    //=========================
    // cuSolver reference
    //=========================
    common::reference_cusolver_lu<data_type, cuda_data_type>(A,
                                                             B,
                                                             info.data(),
                                                             m,
                                                             n,
                                                             nrhs,
                                                             padded_batches,
                                                             false,
                                                             (arrangement_of_v_a<Solver> == arrangement::col_major),
                                                             (arrangement_of_v_b<Solver> == arrangement::col_major),
                                                             false,
                                                             true,
                                                             nullptr,
                                                             batches);

    // printf("=====\n");
    // printf(" after cuSolver API\n");
    // printf("A = \n");
    // common::print_matrix(m, n * batches, A.data(), lda);
    // printf("B = \n");
    // common::print_matrix(b_dim_fast, X.size() / b_dim_fast, B.data(), ldb);

    /* free resources */
    CUDA_CHECK_AND_EXIT(hipFree(d_A));
    CUDA_CHECK_AND_EXIT(hipFree(d_B));
    CUDA_CHECK_AND_EXIT(hipFree(d_info));

    // check result
    auto total_relative_error = common::check_error<data_type, data_type>(L.data(), A.data(), batches * input_size_a);
    std::cout << "GETRF no pivoting: relative error of A between cuSolverDx and cuSolver results: " << total_relative_error << std::endl;

    total_relative_error = common::check_error<data_type, data_type>(X.data(), B.data(), batches * input_size_b);
    printf("GETRS: relative error of B between cuSolverDx and cuSolver results: = %e\n", total_relative_error);
    if (common::is_error_acceptable<data_type>(total_relative_error)) {
        std::cout << "Success compared to cuSolver getrs Result " << std::endl;
        return 0;
    } else {
        std::cout << "Failure compared to cuSolver getrs Result " << std::endl;
        return 1;
    }
    CUDA_CHECK_AND_EXIT(hipDeviceReset());
    return 0;
}

template<int Arch>
struct gesv_batched_wo_pivot_functor {
    int operator()() { return gesv_batched_wo_pivot<Arch>(); }
};


int main() { return common::run_example_with_sm<gesv_batched_wo_pivot_functor>(); }
