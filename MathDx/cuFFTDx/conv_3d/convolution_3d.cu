#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <limits>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "../common.hpp"
#include "../random.hpp"

#include "io_strided_conv_smem.hpp"
#include "kernels.hpp"
#include "reference.hpp"

// Used for getting stable performance results
inline constexpr unsigned int cufftdx_example_warm_up_runs     = 5;
inline constexpr unsigned int cufftdx_example_performance_runs = 15;

using namespace example;

// This function performs fused 3D FFT convolution with pre- and post-processing, ie. it executes:
// pre-processing, front 3D FFT, filter element-wise function (a.k.a. kernel), back 3D FFT.
// Front and back FFTs can either be inverse and forward FFTs, or more commonly forward and inverse.
template<int Batches, bool IsForwardConv, class FFTXPartial, class FFTYPartial, class FFTZPartial, class LoadFunctor, class FilterFunctor, class StoreFunctor, typename ValueType>
auto cufftdx_3d_convolution(ValueType* input, ValueType* output, hipStream_t stream) {
    using namespace cufftdx;

    using id_op = example::identity;

    // Retrieve precision information from description type
    using precision          = cufftdx::precision_of_t<FFTXPartial>;
    constexpr bool is_double = std::is_same_v<precision, double>;
    using vector_type        = std::conditional_t<is_double, double2, float2>;
    using value_type         = ValueType;

    // Retrieve size information from description types
    static constexpr unsigned int fft_size_x      = cufftdx::size_of<FFTXPartial>::value;
    static constexpr unsigned int fft_size_y      = cufftdx::size_of<FFTYPartial>::value;
    static constexpr unsigned int fft_size_z      = cufftdx::size_of<FFTZPartial>::value;
    static constexpr unsigned int flat_batch_size = fft_size_x * fft_size_y * fft_size_z;

    // Create and configure kernel for the X dimension (strided)
    using FFTX  = decltype(FFTXPartial() + Direction < IsForwardConv ? fft_direction::forward : fft_direction::inverse > ());
    using IFFTX = decltype(FFTXPartial() + Direction < IsForwardConv ? fft_direction::inverse : fft_direction::forward > ());

    using FFTY  = decltype(FFTYPartial() + Direction < IsForwardConv ? fft_direction::forward : fft_direction::inverse > ());
    using IFFTY = decltype(FFTYPartial() + Direction < IsForwardConv ? fft_direction::inverse : fft_direction::forward > ());

    using FFTZ  = decltype(FFTZPartial() + Direction < IsForwardConv ? fft_direction::forward : fft_direction::inverse > ());
    using IFFTZ = decltype(FFTZPartial() + Direction < IsForwardConv ? fft_direction::inverse : fft_direction::forward > ());

    static constexpr auto x_fpb = FFTX::ffts_per_block;
    static constexpr auto y_fpb = FFTY::ffts_per_block;
    static constexpr auto z_fpb = FFTZ::ffts_per_block;

    static constexpr unsigned int x_batches = flat_batch_size / fft_size_x;
    static constexpr unsigned int y_batches = flat_batch_size / fft_size_y;
    static constexpr unsigned int z_batches = flat_batch_size / fft_size_z;

    // This is for C2C only, so front/back FFT executions require same memory accesses
    // hence "Front" parameter is true in all cases
    using io_x = io_strided_conv_smem<dimension::x, true, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;
    using io_y = io_strided_conv_smem<dimension::y, true, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;
    using io_z = io_strided_conv_smem<dimension::z, true, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;

    hipError_t err;

    auto workspace_x = cufftdx::make_workspace<FFTX>(err, stream);
    auto workspace_y = cufftdx::make_workspace<FFTY>(err, stream);
    auto workspace_z = cufftdx::make_workspace<FFTZ>(err, stream);

    // Increase max shared memory if needed (includes extra padding)
    constexpr int x_max_bytes = io_x::get_shared_bytes();
    constexpr int y_max_bytes = io_y::get_shared_bytes();
    constexpr int z_max_bytes = io_z::get_shared_bytes();

    // Define kernels to set maximal shared memory in CUDA runtime
    // Refer to kernels.hpp for a detailed kernel explanation.
    auto set_kernel_shared_size = [](auto kernel, auto size) {
        CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            size));
    };

    auto kernel_z_front = fft_kernel<FFTZ, io_z, LoadFunctor, id_op, value_type>;
    set_kernel_shared_size(kernel_z_front, z_max_bytes);

    // No transform ops for middle dimension
    auto kernel_y_front = fft_kernel<FFTY, io_y, id_op, id_op, value_type>;
    set_kernel_shared_size(kernel_y_front, y_max_bytes);

    auto kernel_x = convolution_kernel<FFTX, IFFTX, FilterFunctor, io_x>;
    set_kernel_shared_size(kernel_x, x_max_bytes);

    // No transform ops for middle dimension
    auto kernel_y_back = fft_kernel<IFFTY, io_y, id_op, id_op, value_type>;
    set_kernel_shared_size(kernel_y_back, y_max_bytes);

    auto kernel_z_back = fft_kernel<IFFTZ, io_z, id_op, StoreFunctor, value_type>;
    set_kernel_shared_size(kernel_z_back, z_max_bytes);

    // Execute cuFFTDx in Z-Y-X order
    auto cufftdx_execution = [&](hipStream_t stream) {
        // CUDA Grid configuration is as follows:
        // Grid --> (Total Subbatches / FPB, Batches, 1)
        // Block --> (Size / EPT, FPB)
        kernel_z_front<<<dim3 {example::div_up(z_batches, z_fpb), Batches, 1}, FFTZ::block_dim, z_max_bytes, stream>>>(
            z_batches,
            input,
            output,
            workspace_z);
        kernel_y_front<<<dim3 {example::div_up(y_batches, y_fpb), Batches, 1}, FFTY::block_dim, y_max_bytes, stream>>>(
            y_batches,
            output,
            output,
            workspace_y);
        // Convolution is performed in the strided dimension to save on strided global memory transfers
        // which occur otherwise.
        kernel_x<<<dim3 {example::div_up(x_batches, x_fpb), Batches, 1}, FFTX::block_dim, x_max_bytes, stream>>>(
            x_batches,
            output,
            output,
            workspace_x);
        kernel_y_back<<<dim3 {example::div_up(y_batches, y_fpb), Batches, 1}, FFTY::block_dim, y_max_bytes, stream>>>(
            y_batches,
            output,
            output,
            workspace_y);
        kernel_z_back<<<dim3 {example::div_up(z_batches, z_fpb), Batches, 1}, FFTZ::block_dim, z_max_bytes, stream>>>(
            z_batches,
            output,
            output,
            workspace_z);
    };

    // Correctness run
    cufftdx_execution(stream);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy results to host
    const size_t             flat_fft_size       = fft_size_x * fft_size_y * fft_size_z;
    const size_t             flat_fft_size_bytes = flat_fft_size * sizeof(vector_type);
    std::vector<vector_type> output_host(Batches * flat_fft_size, {std::numeric_limits<precision>::quiet_NaN(), std::numeric_limits<precision>::quiet_NaN()});
    CUDA_CHECK_AND_EXIT(hipMemcpy(output_host.data(), output, Batches * flat_fft_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Performance measurements
    auto time = example::measure_execution_ms(
        cufftdx_execution,
        cufftdx_example_warm_up_runs,
        cufftdx_example_performance_runs,
        stream);

    // Return results
    return example::fft_results<vector_type> {output_host, (time / cufftdx_example_performance_runs)};
}

template<int Arch>
int conv_3d() {
    using namespace cufftdx;

    // 3D Convolution configuration
    static constexpr unsigned int batches = 100;

    // X - the outermost (most strided) dimension
    static constexpr unsigned int fft_size_x = 81;
    static constexpr unsigned int x_ept      = 9;
    static constexpr unsigned int x_fpb      = 16;

    // Y - the middle dimension, strided
    static constexpr unsigned int fft_size_y = 81;
    static constexpr unsigned int y_ept      = 9;
    static constexpr unsigned int y_fpb      = 16;

    // Z - the contiguous dimension
    static constexpr unsigned int fft_size_z = 128;
    static constexpr unsigned int z_ept      = 16;
    static constexpr unsigned int z_fpb      = 16;

    // Definition of functors for preprocessing, filtering
    // and postprocessing. These will be fused with FFT kernels
    using load_functor   = example::rational_scaler<1, 1>;
    using filter_functor = example::rational_scaler<3, 5>;
    using store_functor  = example::rational_scaler<3, 4>;

    // Is this a forward or inverse convolution
    static constexpr bool is_forward = false;

    // Only FP32 and FP64 supported for cuFFT reference
    constexpr bool is_double_precision = true;
    using precision                    = std::conditional_t<is_double_precision, double, float>;

    // Create cuFFTDx description type summarizing all information
    // passed above regarding the outermost dimension.
    using fftx_partial = decltype(Block() + Size<fft_size_x>() + Type<fft_type::c2c>() +
                                  ElementsPerThread<x_ept>() + FFTsPerBlock<x_fpb>() +
                                  Precision<precision>() + SM<Arch>());

    // Create cuFFTDx description type summarizing all information
    // passed above regarding the middle dimension.
    using ffty_partial = decltype(Block() + Size<fft_size_y>() + Type<fft_type::c2c>() +
                                  ElementsPerThread<y_ept>() + FFTsPerBlock<y_fpb>() +
                                  Precision<precision>() + SM<Arch>());

    // Create cuFFTDx description type summarizing all information
    // passed above regarding the contiguous dimension.
    using fftz_partial = decltype(Block() + Size<fft_size_z>() + Type<fft_type::c2c>() +
                                  ElementsPerThread<z_ept>() + FFTsPerBlock<z_fpb>() +
                                  Precision<precision>() + SM<Arch>());

    // Helper types
    using value_type = cufftdx::complex<precision>;

    // Generate random input data on host
    const unsigned int flat_fft_size = fft_size_x * fft_size_y * fft_size_z;

    auto host_input = example::get_random_complex_data<precision>(batches * flat_fft_size, -1, 1);

    // Allocate managed memory for device input/output
    value_type* input;
    value_type* output;
    const auto  flat_fft_size_bytes = flat_fft_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMalloc(&input, batches * flat_fft_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMalloc(&output, batches * flat_fft_size_bytes));

    // Copy input to the device
    CUDA_CHECK_AND_EXIT(hipMemcpy(input, host_input.data(), batches * flat_fft_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    hipStream_t stream;
    CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream));

    // Run cuFFTDx
    auto cufftdx_results = cufftdx_3d_convolution<batches, is_forward, fftx_partial, ffty_partial, fftz_partial, load_functor, filter_functor, store_functor>(input, output, stream);

    // Run cuFFT
    auto cufft_results = cufft_3d_convolution<false, is_forward, load_functor, filter_functor, store_functor>(fft_size_x, fft_size_y, fft_size_z, batches, input, output, stream, cufftdx_example_warm_up_runs, cufftdx_example_performance_runs);

    // Clean-up
    CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));
    CUDA_CHECK_AND_EXIT(hipFree(input));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    // Check if cuFFTDx results are correct
    auto fft_error =
        example::fft_signal_error::calculate_for_complex_values(cufftdx_results.output, cufft_results.output);

    std::cout << "FFT: (" << fft_size_x << ", " << fft_size_y << ", " << fft_size_z << ")\n";

    bool success = fft_error.l2_relative_error < 0.001;
    std::cout << "Correctness results:\n";
    std::cout << "L2 error: " << fft_error.l2_relative_error << "\n";
    std::cout << "Peak error (index: " << fft_error.peak_error_index << "): " << fft_error.peak_error << "\n";

    // Print performance results
    if (success) {
        std::cout << "\nPerformance results:\n";
        std::cout << std::setw(10) << "cuFFTDx: " << cufftdx_results.avg_time_in_ms << " [ms]\n";
        std::cout << std::setw(10) << "cuFFT + thrust: " << cufft_results.avg_time_in_ms << " [ms]\n";
    }

    if (success) {
        std::cout << "Success\n";
        return 0;
    } else {
        std::cout << "Failure\n";
        return 1;
    }
}

template<unsigned int Arch>
struct conv_3d_functor {
    int operator()() {
        return conv_3d<Arch>();
    }
};

int main(int, char**) {
    return example::sm_runner<conv_3d_functor>();
}
