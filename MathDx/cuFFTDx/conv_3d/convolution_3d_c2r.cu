#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <limits>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "../common.hpp"
#include "../random.hpp"

#include "io_strided_conv_smem.hpp"
#include "kernels.hpp"
#include "reference.hpp"

// Used for getting stable performance results
inline constexpr unsigned int cufftdx_example_warm_up_runs     = 5;
inline constexpr unsigned int cufftdx_example_performance_runs = 15;

using namespace example;

// This function performs fused 3D FFT convolution with pre- and post-processing, ie. it executes:
// pre-processing, front 3D FFT, filter element-wise function (a.k.a. kernel), back 3D FFT.
// Front and back FFTs can either be inverse and forward FFTs, or more commonly forward and inverse.
template<int Batches, bool IsForwardConv, class FFTXPartial, class FFTYPartial, class FFTZPartial, class LoadFunctor, class FilterFunctor, class StoreFunctor, typename ValueType>
auto cufftdx_3d_convolution(ValueType* input, ValueType* output, hipStream_t stream) {
    using namespace cufftdx;

    using id_op = example::identity;

    // Retrieve precision information from description type
    using precision          = cufftdx::precision_of_t<FFTXPartial>;
    constexpr bool is_double = std::is_same_v<precision, double>;
    using vector_type        = std::conditional_t<is_double, double2, float2>;
    using value_type         = ValueType;

    // Create and configure kernel for the X dimension (strided)

    constexpr auto size_z          = cufftdx::size_of<FFTZPartial>::value;
    constexpr bool is_z_power_of_2 = is_power_of_2(size_z);

    constexpr auto z_real_mode = is_z_power_of_2 ? cufftdx::real_mode::folded : cufftdx::real_mode::normal;

    using FFTX  = decltype(FFTXPartial() + Direction<fft_direction::inverse>());
    using IFFTX = decltype(FFTXPartial() + Direction<fft_direction::forward>());

    using FFTY  = decltype(FFTYPartial() + Direction<fft_direction::inverse>());
    using IFFTY = decltype(FFTYPartial() + Direction<fft_direction::forward>());

    using FFTZ  = decltype(FFTZPartial() + Type<fft_type::c2r>() + RealFFTOptions<complex_layout::natural, z_real_mode>());
    using IFFTZ = decltype(FFTZPartial() + Type<fft_type::r2c>() + RealFFTOptions<complex_layout::natural, z_real_mode>());

    // Retrieve size information from description types
    static constexpr unsigned int x_input_length = FFTX::input_length;
    static constexpr unsigned int y_input_length = FFTY::input_length;
    static constexpr unsigned int z_input_length = FFTZ::input_length;

    static constexpr unsigned int flat_input_size = x_input_length * y_input_length * z_input_length;

    static constexpr auto x_fpb = FFTX::ffts_per_block;
    static constexpr auto y_fpb = FFTY::ffts_per_block;
    static constexpr auto z_fpb = FFTZ::ffts_per_block;

    static constexpr unsigned int x_batches = flat_input_size / x_input_length;
    static constexpr unsigned int y_batches = flat_input_size / y_input_length;
    static constexpr unsigned int z_batches = flat_input_size / z_input_length;

    // X is always the convolution dimension
    using io_x_front = io_strided_conv_smem<dimension::x, true, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;
    using io_x_back  = io_strided_conv_smem<dimension::x, false, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;

    using io_y_front = io_strided_conv_smem<dimension::y, true, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;
    using io_y_back  = io_strided_conv_smem<dimension::y, false, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;

    using io_z_front = io_strided_conv_smem<dimension::z, true, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;
    using io_z_back  = io_strided_conv_smem<dimension::z, false, Batches, FFTX, IFFTX, FFTY, IFFTY, FFTZ, IFFTZ>;

    hipError_t err;

    auto workspace_x = cufftdx::make_workspace<FFTX>(err, stream);
    auto workspace_y = cufftdx::make_workspace<FFTY>(err, stream);
    auto workspace_z = cufftdx::make_workspace<FFTZ>(err, stream);

    // Increase max shared memory if needed (includes extra padding)
    constexpr int x_max_bytes = io_x_front::get_shared_bytes();
    constexpr int y_max_bytes = io_y_front::get_shared_bytes();
    constexpr int z_max_bytes = io_z_front::get_shared_bytes();

    // Define kernels to set maximal shared memory in CUDA runtime
    // Refer to kernels.hpp for a detailed kernel explanation.
    auto set_kernel_shared_size = [](auto kernel, auto size) {
        CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel),
            hipFuncAttributeMaxDynamicSharedMemorySize,
            size));
    };

    auto kernel_x_front = fft_kernel<FFTX, io_x_front, LoadFunctor, id_op, value_type>;
    set_kernel_shared_size(kernel_x_front, x_max_bytes);

    // No transform ops for middle dimension
    auto kernel_y_front = fft_kernel<FFTY, io_y_front, id_op, id_op, value_type>;
    set_kernel_shared_size(kernel_y_front, y_max_bytes);

    auto kernel_z = convolution_kernel<FFTZ, IFFTZ, FilterFunctor, io_z_front, io_z_back>;
    set_kernel_shared_size(kernel_z, z_max_bytes);

    // No transform ops for middle dimension
    auto kernel_y_back = fft_kernel<IFFTY, io_y_back, id_op, id_op, value_type>;
    set_kernel_shared_size(kernel_y_back, y_max_bytes);

    auto kernel_x_back = fft_kernel<IFFTX, io_x_back, id_op, StoreFunctor, value_type>;
    set_kernel_shared_size(kernel_x_back, x_max_bytes);

    // Execute cuFFTDx in X-Y-Z order
    auto cufftdx_execution = [&](hipStream_t stream) {
        // CUDA Grid configuration is as follows:
        // Grid --> (Total Subbatches / FPB, Batches, 1)
        // Block --> (Size / EPT, FPB)
        kernel_x_front<<<dim3 {example::div_up(x_batches, x_fpb), Batches, 1}, FFTX::block_dim, x_max_bytes, stream>>>(
            x_batches,
            input,
            output,
            workspace_x);
        kernel_y_front<<<dim3 {example::div_up(y_batches, y_fpb), Batches, 1}, FFTY::block_dim, y_max_bytes, stream>>>(
            y_batches,
            output,
            output,
            workspace_y);
        // Convolution is performed in the strided dimension to save on strided global memory transfers
        // which occur otherwise.
        kernel_z<<<dim3 {example::div_up(z_batches, z_fpb), Batches, 1}, FFTZ::block_dim, z_max_bytes, stream>>>(
            z_batches,
            output,
            output,
            workspace_z);
        kernel_y_back<<<dim3 {example::div_up(y_batches, y_fpb), Batches, 1}, FFTY::block_dim, y_max_bytes, stream>>>(
            y_batches,
            output,
            output,
            workspace_y);
        kernel_x_back<<<dim3 {example::div_up(x_batches, x_fpb), Batches, 1}, FFTX::block_dim, x_max_bytes, stream>>>(
            x_batches,
            output,
            output,
            workspace_x);
    };

    // Correctness run
    cufftdx_execution(stream);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy results to host
    const size_t             flat_input_size_bytes = flat_input_size * sizeof(vector_type);
    std::vector<vector_type> output_host(Batches * flat_input_size, {std::numeric_limits<precision>::quiet_NaN(), std::numeric_limits<precision>::quiet_NaN()});
    CUDA_CHECK_AND_EXIT(hipMemcpy(output_host.data(), output, Batches * flat_input_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Performance measurements
    auto time = example::measure_execution_ms(
        cufftdx_execution,
        cufftdx_example_warm_up_runs,
        cufftdx_example_performance_runs,
        stream);

    // Return results
    return example::fft_results<vector_type> {output_host, (time / cufftdx_example_performance_runs)};
}

template<int Arch>
int conv_3d() {
    using namespace cufftdx;

    // 3D Convolution configuration
    static constexpr unsigned int batches = 200;

    // X - the outermost (most strided) dimension
    static constexpr unsigned int fft_size_x = 64;
    static constexpr unsigned int x_ept      = 8;
    static constexpr unsigned int x_fpb      = 9;

    // Y - the middle dimension, strided
    static constexpr unsigned int fft_size_y = 81;
    static constexpr unsigned int y_ept      = 9;
    static constexpr unsigned int y_fpb      = 16;

    // Z - the contiguous dimension
    static constexpr unsigned int fft_size_z = 128;
    static constexpr unsigned int z_ept      = 16;
    static constexpr unsigned int z_fpb      = 16;


    // Definition of functors for preprocessing, filtering
    // and postprocessing. These will be fused with FFT kernels
    using load_functor   = example::rational_scaler<1, 2>;
    using filter_functor = example::rational_scaler<2, 3>;
    using store_functor  = example::rational_scaler<3, 4>;

    // Only FP32 and FP64 supported for cuFFT reference
    constexpr bool is_double_precision = true;
    using precision                    = std::conditional_t<is_double_precision, double, float>;

    // Create cuFFTDx description type summarizing all information
    // passed above regarding the outermost dimension.
    using fftx_partial = decltype(Block() + Size<fft_size_x>() + Type<fft_type::c2c>() +
                                  ElementsPerThread<x_ept>() + FFTsPerBlock<x_fpb>() +
                                  Precision<precision>() + SM<Arch>());

    // Create cuFFTDx description type summarizing all information
    // passed above regarding the middle dimension.
    using ffty_partial = decltype(Block() + Size<fft_size_y>() + Type<fft_type::c2c>() +
                                  ElementsPerThread<y_ept>() + FFTsPerBlock<y_fpb>() +
                                  Precision<precision>() + SM<Arch>());

    // Create cuFFTDx description type summarizing all information
    // passed above regarding the contiguous dimension.
    using fftz_partial = decltype(Block() + Size<fft_size_z>() +
                                  ElementsPerThread<z_ept>() + FFTsPerBlock<z_fpb>() +
                                  Precision<precision>() + SM<Arch>());

    // Helper types
    using value_type = cufftdx::complex<precision>;

    constexpr int z_input_size = decltype(fftz_partial() + cufftdx::Type<fft_type::c2r>())::input_length;

    // Normally, we could get this information from
    const unsigned int flat_input_size = fft_size_x * fft_size_y * z_input_size;

    // Generate random input data on host
    auto host_input = example::get_random_complex_data<precision>(batches * flat_input_size, -1, 1);

    // Allocate managed memory for device input/output
    value_type* input;
    value_type* output;
    const auto  flat_input_size_bytes = flat_input_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMalloc(&input, batches * flat_input_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMalloc(&output, batches * flat_input_size_bytes));

    // Copy input to the device
    CUDA_CHECK_AND_EXIT(hipMemcpy(input, host_input.data(), batches * flat_input_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    hipStream_t stream;
    CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream));

    // Run cuFFTDx
    auto cufftdx_results = cufftdx_3d_convolution<batches, false, fftx_partial, ffty_partial, fftz_partial, load_functor, filter_functor, store_functor>(input, output, stream);

    // Run cuFFT
    auto cufft_results = cufft_3d_convolution<true, false, load_functor, filter_functor, store_functor>(fft_size_x, fft_size_y, fft_size_z, batches, input, output, stream, cufftdx_example_warm_up_runs, cufftdx_example_performance_runs);

    // Clean-up
    CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));
    CUDA_CHECK_AND_EXIT(hipFree(input));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    // Check if cuFFTDx results are correct
    auto fft_error =
        example::fft_signal_error::calculate_for_complex_values(cufftdx_results.output, cufft_results.output);

    std::cout << "FFT: (" << fft_size_x << ", " << fft_size_y << ", " << fft_size_z << ")\n";

    bool success = fft_error.l2_relative_error < 0.001;
    std::cout << "Correctness results:\n";
    std::cout << "L2 error: " << fft_error.l2_relative_error << "\n";
    std::cout << "Peak error (index: " << fft_error.peak_error_index << "): " << fft_error.peak_error << "\n";

    // Print performance results
    if (success) {
        std::cout << "\nPerformance results:\n";
        std::cout << std::setw(10) << "cuFFTDx: " << cufftdx_results.avg_time_in_ms << " [ms]\n";
        std::cout << std::setw(10) << "cuFFT + thrust: " << cufft_results.avg_time_in_ms << " [ms]\n";
    }

    if (success) {
        std::cout << "Success\n";
        return 0;
    } else {
        std::cout << "Failure\n";
        return 1;
    }
}

template<unsigned int Arch>
struct conv_3d_functor {
    int operator()() {
        return conv_3d<Arch>();
    }
};

int main(int, char**) {
    return example::sm_runner<conv_3d_functor>();
}
