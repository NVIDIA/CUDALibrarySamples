#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "block_io.hpp"
#include "common.hpp"

template<class FFT, class IFFT>
__launch_bounds__(FFT::max_threads_per_block) __global__ void convolution_kernel(typename FFT::value_type* data) {
    using complex_type = typename FFT::value_type;
    using scalar_type  = typename complex_type::value_type;

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Load data from global memory to registers
    example::io<FFT>::load(data, thread_data, local_fft_id);

    // Execute FFT
    extern __shared__ complex_type shared_mem[];
    FFT().execute(thread_data, shared_mem);

    // Scale values
    scalar_type scale = 1.0 / cufftdx::size_of<FFT>::value;
    for (unsigned int i = 0; i < FFT::elements_per_thread; i++) {
        thread_data[i].x *= scale;
        thread_data[i].y *= scale;
    }

    // Execute inverse FFT
    IFFT().execute(thread_data, shared_mem);

    // Save results
    example::io<FFT>::store(thread_data, data, local_fft_id);
}

// This example demonstrates how to use cuFFTDx t operform a convolution using one-dimensional FFTs.
//
// One block is run, it calculates two 128-point convolutions by first doing forward FFT, then
// applying pointwise operation, and ending with inverse FFT.
// Data is generated on host, copied to device buffer, and then results are copied back to host.
template<unsigned int Arch>
void convolution() {
    using namespace cufftdx;

    static constexpr unsigned int ffts_per_block = 2;
    static constexpr unsigned int fft_size       = 128;
    // FFT_base defined common options for FFT and IFFT. FFT_base is not a complete FFT description.
    // In order to complete FFT description directions are specified: forward for FFT, inverse for IFFT.
    using FFT_base     = decltype(Block() + Size<fft_size>() + Type<fft_type::c2c>() + Precision<float>() +
                              ElementsPerThread<8>() + FFTsPerBlock<ffts_per_block>() + SM<Arch>());
    using FFT          = decltype(FFT_base() + Direction<fft_direction::forward>());
    using IFFT         = decltype(FFT_base() + Direction<fft_direction::inverse>());
    using complex_type = typename FFT::value_type;

    // Allocate managed memory for input/output
    complex_type* data;
    auto          size       = ffts_per_block * fft_size;
    auto          size_bytes = size * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&data, size_bytes));
    for (size_t i = 0; i < size; i++) {
        data[i] = complex_type {float(i), -float(i)};
    }

    std::cout << "input [1st FFT]:\n";
    for (size_t i = 0; i < fft_size; i++) {
        std::cout << data[i].x << " " << data[i].y << std::endl;
    }

    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        convolution_kernel<FFT), IFFT>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        FFT::shared_memory_size));

    // Invokes convolution kernel with FFT::block_dim threads in CUDA block
    convolution_kernel<FFT, IFFT><<<1, FFT::block_dim, FFT::shared_memory_size>>>(data);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    std::cout << "output [1st FFT]:\n";
    for (size_t i = 0; i < fft_size; i++) {
        std::cout << data[i].x << " " << data[i].y << std::endl;
    }

    CUDA_CHECK_AND_EXIT(hipFree(data));
    std::cout << "Success" << std::endl;
}

template<unsigned int Arch>
struct convolution_functor {
    void operator()() { return convolution<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<convolution_functor>();
}
