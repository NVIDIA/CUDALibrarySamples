#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include <hipcub/hipcub.hpp>
#include <cub/version.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>

#include "block_io.hpp"
#include "common.hpp"

#if CUB_VERSION < 101300
int main(int, char**) {
    std::cout << "Example disabled, BLOCK_LOAD_STRIPED/BLOCK_STORE_STRIPED is only supported since CUB 1.13 (CUDA 11.5)" << std::endl;
    return 0;
}
#else

template<class FFT>
__launch_bounds__(FFT::max_threads_per_block) __global__ void block_fft_kernel(typename FFT::value_type* data) {
    using complex_type = typename FFT::value_type;

    // FFT::stride == FFT::block_dim.x in most cases
    using BlockLoad  = hipcub::BlockLoad <complex_type, FFT::stride /* BLOCK_DIM_X */, FFT::storage_size, hipcub::BLOCK_LOAD_STRIPED>;
    using BlockStore = hipcub::BlockStore<complex_type, FFT::stride, FFT::storage_size, hipcub::BLOCK_STORE_STRIPED>;

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // ID of FFT in CUDA grid
    unsigned int global_fft_id =
        (FFT::ffts_per_block == 1) ? local_fft_id : ( blockIdx.x * FFT::ffts_per_block + local_fft_id);

    // Load data from global memory to registers
    auto fft_data = data + (global_fft_id * cufftdx::size_of<FFT>::value);
    BlockLoad().Load(fft_data, thread_data, cufftdx::size_of<FFT>::value, complex_type { 0.0, 0.0 });

    // Execute FFT
    extern __shared__ complex_type shared_mem[];
    FFT().execute(thread_data, shared_mem);

    // Save results
    BlockStore().Store(fft_data, thread_data, cufftdx::size_of<FFT>::value);
}

// In this example a one-dimensional complex-to-complex transform is performed by a CUDA block. CUB
// library is used for IO in kernel.
//
// One block is run, it calculates two 128-point C2C float precision FFTs.
// Data is generated on host, copied to device buffer, and then results are copied back to host.
template<unsigned int Arch>
void simple_block_fft() {
    using namespace cufftdx;

    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.
    // Additionally,
    using FFT          = decltype(Block() + Size<128>() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() +
                         Precision<float>() + ElementsPerThread<8>() + FFTsPerBlock<2>() + SM<Arch>());
    #if CUFFTDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using complex_type = example::value_type_t<FFT>;
    #else
    using complex_type = typename FFT::value_type;
    #endif

    // Allocate managed memory for input/output
    complex_type* data;
    auto          size       = FFT::ffts_per_block * cufftdx::size_of<FFT>::value;
    auto          size_bytes = size * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&data, size_bytes));
    for (size_t i = 0; i < size; i++) {
        data[i] = complex_type {float(i), -float(i)};
    }

    std::cout << "input [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << data[i].x << " " << data[i].y << std::endl;
    }

    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        block_fft_kernel<FFT>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        FFT::shared_memory_size));

    // Invokes kernel with FFT::block_dim threads in CUDA block
    block_fft_kernel<FFT><<<1, FFT::block_dim, FFT::shared_memory_size>>>(data);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    std::cout << "output [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << data[i].x << " " << data[i].y << std::endl;
    }
    auto sum = data[0].x;

    CUDA_CHECK_AND_EXIT(hipFree(data));
    if(std::abs(sum - ((cufftdx::size_of<FFT>::value-1) * cufftdx::size_of<FFT>::value / 2)) > 0.1) {
        std::cout << "Failed" << std::endl;
        return;
    }
    std::cout << "Success" << std::endl;
}

template<unsigned int Arch>
struct simple_block_fft_functor {
    void operator()() { return simple_block_fft<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_block_fft_functor>();
}
#endif // CUB_VERSION < 101300
