/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <iostream>
#include <vector>
#include <chrono>
#include <cmath>

#include "block_fft_performance.hpp"

template<unsigned int      Arch,
         unsigned int      FFTSize,
         cufftdx::fft_type FFTType,
         class PrecisionType,
         cufftdx::fft_direction FFTDirection      = cufftdx::fft_direction::forward,
         bool                   UseSuggested      = true,
         unsigned int           ElementsPerThread = 8,
         unsigned int           FFTsPerBlock      = 1>
void block_fft_performance(const hipStream_t& stream, bool verbose) {
    using namespace cufftdx;

    using FFT_base = decltype(Block() + Type<FFTType>() + Precision<PrecisionType>() + SM<Arch>());

    using FFT_with_direction =
        std::conditional_t<FFTType == fft_type::c2c, decltype(FFT_base() + Direction<FFTDirection>()), FFT_base>;

    benchmark_block_fft<FFT_with_direction, FFTSize, ElementsPerThread, FFTsPerBlock, UseSuggested>(stream, verbose);

    if (verbose)
        std::cout << std::endl;
}

template<unsigned int Arch>
struct block_fft_performance_functor {
    void operator()() {
        using namespace cufftdx;

        hipStream_t stream;
        CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream))

        bool default_verbose = false;


        // To specify EPT and FPB values, set UsedSuggested to false.
        // FFTDirection is used if and only if FFTType is C2C.
        // Below is an example of a test run with specified EPT and FPB values.

        block_fft_performance<Arch, 137, fft_type::c2c, float, fft_direction::forward, false, 8, 1>(stream,
                                                                                                    default_verbose);

        block_fft_performance<Arch, 137, fft_type::c2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 251, fft_type::c2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 512, fft_type::c2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 1024, fft_type::c2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 2048, fft_type::c2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 4096, fft_type::c2c, float>(stream, default_verbose);

        block_fft_performance<Arch, 137, fft_type::c2c, float, fft_direction::inverse>(stream, default_verbose);
        block_fft_performance<Arch, 251, fft_type::c2c, float, fft_direction::inverse>(stream, default_verbose);
        block_fft_performance<Arch, 512, fft_type::c2c, float, fft_direction::inverse>(stream, default_verbose);
        block_fft_performance<Arch, 1024, fft_type::c2c, float, fft_direction::inverse>(stream, default_verbose);
        block_fft_performance<Arch, 2048, fft_type::c2c, float, fft_direction::inverse>(stream, default_verbose);
        block_fft_performance<Arch, 4096, fft_type::c2c, float, fft_direction::inverse>(stream, default_verbose);

        block_fft_performance<Arch, 137, fft_type::r2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 251, fft_type::r2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 512, fft_type::r2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 1024, fft_type::r2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 2048, fft_type::r2c, float>(stream, default_verbose);
        block_fft_performance<Arch, 4096, fft_type::r2c, float>(stream, default_verbose);

        block_fft_performance<Arch, 137, fft_type::c2r, float>(stream, default_verbose);
        block_fft_performance<Arch, 251, fft_type::c2r, float>(stream, default_verbose);
        block_fft_performance<Arch, 512, fft_type::c2r, float>(stream, default_verbose);
        block_fft_performance<Arch, 1024, fft_type::c2r, float>(stream, default_verbose);
        block_fft_performance<Arch, 2048, fft_type::c2r, float>(stream, default_verbose);
        block_fft_performance<Arch, 4096, fft_type::c2r, float>(stream, default_verbose);

        CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));
    }
};

int main(int, char**) {
    return example::sm_runner<block_fft_performance_functor>();
}