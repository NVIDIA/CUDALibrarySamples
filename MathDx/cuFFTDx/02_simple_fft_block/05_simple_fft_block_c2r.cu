#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "../common/block_io.hpp"
#include "../common/common.hpp"

template<class FFT>
__launch_bounds__(FFT::max_threads_per_block) __global__
    void block_fft_kernel_c2r(typename FFT::value_type* input_data, cufftdx::precision_of_t<FFT>* output_data) {
    using complex_type = typename FFT::value_type;

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Load data from global memory to registers
    example::io<FFT>::load(input_data, thread_data, local_fft_id);

    // Execute FFT
    extern __shared__ __align__(alignof(float4)) complex_type shared_mem[];
    FFT().execute(thread_data, shared_mem);

    // Save results
    example::io<FFT>::store(thread_data, output_data, local_fft_id);
}

// In this example a one-dimensional complex-to-real transform is performed by a CUDA block.
//
// One block is run, it calculates two 128-point C2R float precision FFTs.
// Data is generated on host, copied to device buffer, and then results are copied back to host.
// Notice different sizes of input and output buffer, and C2R load and store operations in the kernel.
template<unsigned int Arch>
void simple_block_fft_c2r() {
    using namespace cufftdx;

    // R2C and C2R specific properties describing data layout and execution mode for
    // the requested transform.
    using real_fft_options = RealFFTOptions<complex_layout::natural, real_mode::normal>;

    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.
    using FFT          = decltype(Block() + Size<128>() + Type<fft_type::c2r>() + Direction<fft_direction::inverse>() +
                         Precision<float>() + ElementsPerThread<8>() + FFTsPerBlock<2>() + real_fft_options() + SM<Arch>());
    using complex_type = typename FFT::value_type;
    using real_type    = typename complex_type::value_type;

    // Allocate managed memory for input/output
    complex_type* input_data;
    auto          input_size       = FFT::ffts_per_block * FFT::input_length;
    auto          input_size_bytes = input_size * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&input_data, input_size_bytes));
    for (size_t i = 0; i < input_size; i++) {
        input_data[i] = complex_type {float(i), -float(i)};
    }

    real_type* output_data;
    auto       output_size       = FFT::ffts_per_block * FFT::output_length;
    auto       output_size_bytes = output_size * sizeof(real_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&output_data, output_size_bytes));

    std::cout << "input [1st FFT]:\n";
    for (size_t i = 0; i < FFT::input_length; i++) {
        std::cout << input_data[i].x << " " << input_data[i].y << std::endl;
    }

    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        block_fft_kernel_c2r<FFT>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        FFT::shared_memory_size));

    // Invokes kernel with FFT::block_dim threads in CUDA block
    block_fft_kernel_c2r<FFT><<<1, FFT::block_dim, FFT::shared_memory_size>>>(input_data, output_data);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    std::cout << "output [1st FFT]:\n";
    for (size_t i = 0; i < FFT::output_length; i++) {
        std::cout << output_data[i] << std::endl;
    }

    CUDA_CHECK_AND_EXIT(hipFree(input_data));
    CUDA_CHECK_AND_EXIT(hipFree(output_data));
    std::cout << "Success" << std::endl;
}

template<unsigned int Arch>
struct simple_block_fft_c2r_functor {
    void operator()() { return simple_block_fft_c2r<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_block_fft_c2r_functor>();
}
