#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "common.hpp"

template<class FFT>
__global__ void thread_fft_kernel(typename FFT::value_type* data) {
    using complex_type = typename FFT::value_type;

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // Load data from global memory to registers.
    // thread_data should have all input data in order.
    unsigned int index = threadIdx.x * FFT::elements_per_thread;
    for (size_t i = 0; i < FFT::elements_per_thread; i++) {
        thread_data[i] = data[index + i];
    }

    // Execute FFT
    FFT().execute(thread_data);

    // Save results
    for (size_t i = 0; i < FFT::elements_per_thread; i++) {
        data[index + i] = thread_data[i];
    }
}

// In this example a one-dimensional complex-to-complex transform is perform by a CUDA thread.
//
// Four (threads_count) threads are run, and each thread calculates 8-point (fft_size) C2C double precision FFT.
// Data is generated on host, copied to device buffer, and then results are copied back to host.
int main(int, char**) {
    using namespace cufftdx;

    // Number of threads to execute
    static constexpr unsigned int threads_count = 4;

    // FFT is defined, its: size, type, direction, precision. Thread() operator informs that FFT will be executed on thread level.
    using FFT          = decltype(Thread() + Size<8>() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() +
                         Precision<double>());
    using complex_type = typename FFT::value_type;

    // Host data
    std::vector<complex_type> input(cufftdx::size_of<FFT>::value * threads_count);
    for (size_t i = 0; i < input.size(); i++) {
        input[i] = complex_type {double(i), -double(i)};
    }

    std::cout << "input [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << input[i].x << " " << input[i].y << std::endl;
    }

    // Device data
    complex_type* device_buffer;
    auto          size_bytes = input.size() * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMalloc(&device_buffer, size_bytes));
    // Copy host to device
    CUDA_CHECK_AND_EXIT(hipMemcpy(device_buffer, input.data(), size_bytes, hipMemcpyHostToDevice));

    // Invokes kernel with 'threads_count' threads in block, each thread calculates one FFT of size
    thread_fft_kernel<FFT><<<1, threads_count>>>(device_buffer);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy device to host
    std::vector<complex_type> output(input.size());
    CUDA_CHECK_AND_EXIT(hipMemcpy(output.data(), device_buffer, size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipFree(device_buffer));

    std::cout << "output [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << output[i].x << " " << output[i].y << std::endl;
    }

    std::cout << "Success" << std::endl;
}
