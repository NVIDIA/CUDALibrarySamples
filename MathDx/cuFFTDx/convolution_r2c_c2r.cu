#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "block_io.hpp"
#include "common.hpp"

template<class FFTR2C, class FFTC2R>
__launch_bounds__(FFTR2C::max_threads_per_block) __global__ void convolution_kernel(cufftdx::precision_of_t<FFTR2C>* data) {
    using complex_type = typename FFTR2C::value_type;
    using scalar_type  = typename complex_type::value_type;

    // Local array for thread
    complex_type thread_data[FFTR2C::storage_size];

    // ID of FFT in CUDA block, in range [0; FFTR2C::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Load data from global memory to registers
    example::io<FFTR2C>::load_r2c(data, thread_data, local_fft_id);

    // Execute FFT
    extern __shared__ complex_type shared_mem[];
    FFTR2C().execute(thread_data, shared_mem);

    // Scale values
    scalar_type scale = 1.0 / cufftdx::size_of<FFTR2C>::value;
    for (unsigned int i = 0; i < FFTR2C::elements_per_thread; i++) {
        thread_data[i].x *= scale;
        thread_data[i].y *= scale;
    }

    // Execute inverse FFT
    FFTC2R().execute(thread_data, shared_mem);

    // Save results
    example::io<FFTC2R>::store_c2r(thread_data, data, local_fft_id);
}

// This example demonstrates how to use cuFFTDx t operform a convolution using one-dimensional FFTs.
//
// One block is run, it calculates two 128-point convolutions by first doing forward FFT, then
// applying pointwise operation, and ending with inverse FFT.
// Data is generated on host, copied to device buffer, and then results are copied back to host.
template<unsigned int Arch>
void convolution() {
    using namespace cufftdx;

    static constexpr unsigned int ffts_per_block = 2;
    static constexpr unsigned int fft_size       = 128;
    // FFT_base defined common options for FFT and IFFT. FFT_base is not a complete FFT description.
    // In order to complete FFT description directions are specified: forward for FFT, inverse for IFFT.
    using FFT_base  = decltype(Block() + Size<fft_size>() + Precision<float>() +
                              ElementsPerThread<8>() + FFTsPerBlock<ffts_per_block>() + SM<Arch>());
    using FFTR2C    = decltype(FFT_base() + Type<fft_type::r2c>());
    using FFTC2R    = decltype(FFT_base() + Type<fft_type::c2r>());
    using real_type = precision_of_t<FFTR2C>;

    // Allocate managed memory for input/output
    real_type* data;
    auto       size       = ffts_per_block * fft_size;
    auto       size_bytes = size * sizeof(real_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&data, size_bytes));
    for (size_t i = 0; i < size; i++) {
        data[i] = float(i);
    }

    std::cout << "input [1st FFT]:\n";
    for (size_t i = 0; i < fft_size; i++) {
        std::cout << data[i] << std::endl;
    }

    const auto shared_memory_size = std::max(FFTR2C::shared_memory_size, FFTC2R::shared_memory_size);
    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        convolution_kernel<FFTR2C), FFTC2R>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        shared_memory_size));

    // Invokes convolution kernel with FFT::block_dim threads in CUDA block
    convolution_kernel<FFTR2C, FFTC2R><<<1, FFTR2C::block_dim, shared_memory_size>>>(data);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    std::cout << "output [1st FFT]:\n";
    for (size_t i = 0; i < fft_size; i++) {
        std::cout << data[i] << std::endl;
    }

    CUDA_CHECK_AND_EXIT(hipFree(data));
    std::cout << "Success" << std::endl;
}

template<unsigned int Arch>
struct convolution_functor {
    void operator()() { return convolution<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<convolution_functor>();
}
