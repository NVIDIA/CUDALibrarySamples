#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "common.hpp"


template<class FFT>
__launch_bounds__(FFT::max_threads_per_block)
    __global__ void block_fft_kernel(typename FFT::value_type* data, typename FFT::workspace_type workspace) {
    using complex_type = typename FFT::value_type;

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // Local batch id of this FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Global batch id of this FFT in CUDA grid is equal to number of batches per CUDA block (ffts_per_block)
    // times CUDA block id, plus local batch id.
    const unsigned int global_fft_id = (blockIdx.x * FFT::ffts_per_block) + local_fft_id;

    // Load data from global memory to registers
    const unsigned int     offset = cufftdx::size_of<FFT>::value * global_fft_id;
    constexpr unsigned int stride = FFT::stride;
    unsigned int           index  = offset + threadIdx.x;
    for (unsigned int i = 0; i < FFT::elements_per_thread; i++) {
        if ((i * stride + threadIdx.x) < cufftdx::size_of<FFT>::value) {
            thread_data[i] = data[index];
            index += stride;
        }
    }

    // Execute FFT
    extern __shared__ complex_type shared_memory[];
    FFT().execute(thread_data, shared_memory, workspace);

    // Save results
    index = offset + threadIdx.x;
    for (unsigned int i = 0; i < FFT::elements_per_thread; i++) {
        if ((i * stride + threadIdx.x) < cufftdx::size_of<FFT>::value) {
            data[index] = thread_data[i];
            index += stride;
        }
    }
}

// In this example a one-dimensional complex-to-complex transform is performed by a CUDA block.
//
// One block is run, it calculates two 128-point C2C float precision FFTs.
// Data is generated on host, copied to device buffer, and then results are copied back to host.
template<unsigned int Arch>
void introduction_example() {
    using namespace cufftdx;

    // FFT definition
    //
    // Size, precision, type, direction are defined with operators.
    // Block() operator informs that FFT will be executed on block level.
    // Shared memory is required for co-operation between threads.
    //
    // Additionally:
    // * FFTsPerBlock operator defines how many FFTs (batches) are executed in a single CUDA block,
    // * ElementsPerThread operators defines how FFT calculations are mapped into a CUDA block, i.e. how many thread are required, and
    // * SM operator defines targeted CUDA architecture.
    using FFT          = decltype(Size<128>() + Precision<float>() + Type<fft_type::c2c>() + Direction<fft_direction::forward>()
                         + ElementsPerThread<8>() + FFTsPerBlock<2>()
                         + SM<Arch>() + Block());
    using complex_type = typename FFT::value_type;

    // Allocate managed memory for input/output
    complex_type* data;
    auto          size       = FFT::ffts_per_block * cufftdx::size_of<FFT>::value;
    auto          size_bytes = size * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&data, size_bytes));
    // Generate data
    for (size_t i = 0; i < size; i++) {
        data[i] = complex_type {float(i), -float(i)};
    }

    hipError_t error_code = hipSuccess;
    auto workspace = make_workspace<FFT>(error_code);
    CUDA_CHECK_AND_EXIT(error_code);

    std::cout << "input [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << data[i].x << " " << data[i].y << std::endl;
    }

    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        block_fft_kernel<FFT>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        FFT::shared_memory_size));

    // Invokes kernel with FFT::block_dim threads in CUDA block
    block_fft_kernel<FFT><<<1, FFT::block_dim, FFT::shared_memory_size>>>(data, workspace);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    std::cout << "output [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << data[i].x << " " << data[i].y << std::endl;
    }

    CUDA_CHECK_AND_EXIT(hipFree(data));
    std::cout << "Success" << std::endl;
}

template<unsigned int Arch>
struct introduction_example_functor {
    void operator()() { return introduction_example<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<introduction_example_functor>();
}
