#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <limits>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>
#include <hipfft/hipfft.h>

#include "block_io.hpp"
#include "common.hpp"
#include "random.hpp"

// #define CUFFTDX_EXAMPLE_DETAIL_DEBUG_FFT_3D
// #define CUFFTDX_EXAMPLE_DETAIL_DEBUG_FFT_3D_SIMPLE_IO
inline constexpr unsigned int cufftdx_example_warm_up_runs = 5;
inline constexpr unsigned int cufftdx_example_performance_runs = 50;

template<unsigned int MaxThreadsPerBlock,
         class FFTX,
         class FFTY,
         class FFTZ,
         class ComplexType                = typename FFTX::value_type,
         unsigned int RequiredStorageSize = std::max({FFTX::storage_size, FFTY::storage_size, FFTZ::storage_size})>
__launch_bounds__(MaxThreadsPerBlock) __global__
    void cufftdx_3d_fft_single_block_kernel(const ComplexType* input, ComplexType* output) {
    using complex_type                       = ComplexType;
    static constexpr unsigned int fft_size_x = cufftdx::size_of<FFTX>::value;
    static constexpr unsigned int fft_size_y = cufftdx::size_of<FFTY>::value;
    static constexpr unsigned int fft_size_z = cufftdx::size_of<FFTZ>::value;

    // Local array for thread
    constexpr auto required_storage_size = RequiredStorageSize;
    complex_type   thread_data[required_storage_size];

    // Shared memory use for exchanging data between threads
    extern __shared__ complex_type shared_memory[];

    // Load data from global memory to registers.
    static constexpr unsigned int stride_x = fft_size_y * fft_size_z;
    unsigned int                  index    = (threadIdx.x + threadIdx.y * fft_size_y);
    if (threadIdx.x < fft_size_y && threadIdx.y < fft_size_z) {
        for (size_t i = 0; i < FFTX::elements_per_thread; i++) {
            thread_data[i] = input[index];
            index += stride_x;
        }

        // Execute FFT in X dimension
        FFTX().execute(thread_data);

        // Exchange/transpose via shared memory
        index = (threadIdx.x + threadIdx.y * fft_size_y);
        for (size_t i = 0; i < FFTX::elements_per_thread; i++) {
            shared_memory[index] = thread_data[i];
            index += stride_x;
        }
    }

    __syncthreads();
    static constexpr unsigned int stride_y = fft_size_z;
    index                                  = threadIdx.x + threadIdx.y * fft_size_y * fft_size_z;
    if (threadIdx.x < fft_size_z && threadIdx.y < fft_size_x) {
        for (size_t i = 0; i < FFTY::elements_per_thread; i++) {
            thread_data[i] = shared_memory[index];
            index += stride_y;
        }

        // Execute FFT in Y dimension
        FFTY().execute(thread_data);

        // Exchange/transpose via shared memory
        index = threadIdx.x + threadIdx.y * fft_size_y * fft_size_z;
        for (size_t i = 0; i < FFTY::elements_per_thread; i++) {
            shared_memory[index] = thread_data[i];
            index += stride_y;
        }
    }

    __syncthreads();
    if (threadIdx.x < fft_size_x && threadIdx.y < fft_size_y) {
        index = (threadIdx.x + threadIdx.y * fft_size_x) * fft_size_z;
        for (size_t i = 0; i < FFTZ::elements_per_thread; i++) {
            thread_data[i] = shared_memory[index];
            index += 1;
        }
        // Execute FFT in Z dimension
        FFTZ().execute(thread_data);
    }

    // Save results
#ifdef CUFFTDX_EXAMPLE_DETAIL_DEBUG_FFT_3D_SIMPLE_IO
    // Simple IO with poor global memory pattern:
    // Storing the data with stride=1 results in poor global memory
    // write pattern with little or none coalescing
    if (threadIdx.x < fft_size_x && threadIdx.y < fft_size_y) {
        index = (threadIdx.x + threadIdx.y * fft_size_x) * fft_size_z;
        for (size_t i = 0; i < FFTZ::elements_per_thread; i++) {
            output[index] = thread_data[i];
            index += 1;
        }
    }
#else
    // Shared memory IO:
    // Exchanging data via shared memory results in a much better global
    // memory patter with good coalescing
    if (threadIdx.x < fft_size_x && threadIdx.y < fft_size_y) {
        index = (threadIdx.x + threadIdx.y * fft_size_x) * fft_size_z;
        for (size_t i = 0; i < FFTZ::elements_per_thread; i++) {
            shared_memory[index] = thread_data[i];
            index += 1;
        }
    }
    __syncthreads();
    if (threadIdx.x < fft_size_y && threadIdx.y < fft_size_z) {
        index = (threadIdx.x + threadIdx.y * fft_size_y);
        for (size_t i = 0; i < FFTX::elements_per_thread; i++) {
            thread_data[i] = shared_memory[index];
            index += stride_x;
        }

        index = (threadIdx.x + threadIdx.y * fft_size_y);
        for (size_t i = 0; i < FFTX::elements_per_thread; i++) {
            output[index] = thread_data[i];
            index += stride_x;
        }
    }
#endif
}

example::fft_results<float2> cufft_3d_fft(unsigned int  fft_size_x,
                                          unsigned int  fft_size_y,
                                          unsigned int  fft_size_z,
                                          hipfftComplex* input,
                                          hipfftComplex* output,
                                          hipStream_t  stream) {
    // Create cuFFT plan
    hipfftHandle plan;
    CUFFT_CHECK_AND_EXIT(hipfftPlan3d(&plan, fft_size_x, fft_size_y, fft_size_z, HIPFFT_C2C));
    CUFFT_CHECK_AND_EXIT(hipfftSetStream(plan, stream));

    // Execute cuFFT
    auto cufft_execution = [&](hipStream_t /* stream */) {
        CUFFT_CHECK_AND_EXIT(hipfftExecC2C(plan, input, output, HIPFFT_FORWARD))
    };

    // Correctness run
    cufft_execution(stream);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());
    // Copy results to host
    const size_t        flat_fft_size       = fft_size_x * fft_size_y * fft_size_z;
    const size_t        flat_fft_size_bytes = flat_fft_size * sizeof(float2);
    std::vector<float2> output_host(flat_fft_size, {std::numeric_limits<float>::quiet_NaN(), std::numeric_limits<float>::quiet_NaN()});
    CUDA_CHECK_AND_EXIT(hipMemcpy(output_host.data(), output, flat_fft_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Performance measurements
    auto time = example::measure_execution_ms(
        cufft_execution,
        cufftdx_example_warm_up_runs,
        cufftdx_example_performance_runs,
        stream);

    // Clean-up
    CUFFT_CHECK_AND_EXIT(hipfftDestroy(plan));

    // Return results
    return example::fft_results<float2> {output_host, (time / cufftdx_example_performance_runs)};
}

template<unsigned int FFTSizeX, unsigned int FFTSizeY, unsigned int FFTSizeZ>
example::fft_results<float2> cufftdx_3d_fft_single_block(float2* input, float2* output, hipStream_t stream) {
    using namespace cufftdx;

    static constexpr unsigned int fft_size_x = FFTSizeX;
    static constexpr unsigned int fft_size_y = FFTSizeY;
    static constexpr unsigned int fft_size_z = FFTSizeZ;

    using fft_base =
        decltype(Thread() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() + Precision<float>());
    using fft_x        = decltype(fft_base() + Size<fft_size_x>());
    using fft_y        = decltype(fft_base() + Size<fft_size_y>());
    using fft_z        = decltype(fft_base() + Size<fft_size_z>());
    using complex_type = typename fft_x::value_type;

    constexpr unsigned int max_dim               = std::max({fft_size_x, fft_size_y, fft_size_z});
    constexpr dim3         block_dim             = {max_dim, max_dim, 1};
    constexpr unsigned int max_threads_per_block = block_dim.x * block_dim.y * block_dim.z;
    const size_t           shared_memory_size    = (fft_size_x * fft_size_y * fft_size_z) * sizeof(complex_type);

    const auto kernel = cufftdx_3d_fft_single_block_kernel<max_threads_per_block, fft_x, fft_y, fft_z, complex_type>;

    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory_size));

    complex_type* cufftdx_input  = reinterpret_cast<complex_type*>(input);
    complex_type* cufftdx_output = reinterpret_cast<complex_type*>(output);
    auto fft_3d_execution = [&](hipStream_t stream) {
        kernel<<<1, block_dim, shared_memory_size, stream>>>(cufftdx_input, cufftdx_output);
        CUDA_CHECK_AND_EXIT(hipGetLastError());
    };

    // Correctness run
    fft_3d_execution(stream);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());
    // Copy results to host
    static constexpr size_t flat_fft_size       = fft_size_x * fft_size_y * fft_size_z;
    static constexpr size_t flat_fft_size_bytes = flat_fft_size * sizeof(float2);
    std::vector<float2> output_host(flat_fft_size, {std::numeric_limits<float>::quiet_NaN(), std::numeric_limits<float>::quiet_NaN()});
    CUDA_CHECK_AND_EXIT(hipMemcpy(output_host.data(), cufftdx_output, flat_fft_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Performance measurements
    auto time = example::measure_execution_ms(
        [&](hipStream_t stream) {
            fft_3d_execution(stream);
        },
        cufftdx_example_warm_up_runs,
        cufftdx_example_performance_runs,
        stream);

    // Return results
    return example::fft_results<float2>{ output_host, (time / cufftdx_example_performance_runs) };
}

int main(int, char**) {
    // 3D FFT
    static constexpr unsigned int fft_size_x = 16;
    static constexpr unsigned int fft_size_y = 15;
    static constexpr unsigned int fft_size_z = 14;

    // Generate random input data on host
    const unsigned int flat_fft_size = fft_size_x * fft_size_y * fft_size_z;
#ifdef CUFFTDX_EXAMPLE_DETAIL_DEBUG_FFT_3D
    std::vector<float2> host_input(flat_fft_size);
    for (size_t i = 0; i < flat_fft_size; i++) {
        float sign      = (i % 3 == 0) ? -1.0f : 1.0f;
        host_input[i].x = sign * static_cast<float>(i) / 100;
        host_input[i].y = sign * static_cast<float>(i) / 100;
    }
#else
    auto host_input = example::get_random_complex_data<float>(flat_fft_size, -1, 1);
#endif

    // Allocate managed memory for device input/output
    // float2 has the same size and alignment as cuFFTDx fp32 complex type cufftdx::complex<float> and hipfftComplex
    float2*    input;
    float2*    output;
    const auto flat_fft_size_bytes = flat_fft_size * sizeof(float2);
    CUDA_CHECK_AND_EXIT(hipMalloc(&input, flat_fft_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMalloc(&output, flat_fft_size_bytes));

    // Copy input to the device
    CUDA_CHECK_AND_EXIT(hipMemcpy(input, host_input.data(), flat_fft_size_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    hipStream_t stream;
    CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream));

    // Run cuFFT
    auto cufft_results = cufft_3d_fft(fft_size_x, fft_size_y, fft_size_z, input, output, stream);

    // Run cuFFTDx
    auto cufftdx_results = cufftdx_3d_fft_single_block<fft_size_x, fft_size_y, fft_size_z>(input, output, stream);

    // Clean-up
    CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));
    CUDA_CHECK_AND_EXIT(hipFree(input));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    // Check if cuFFTDx results are correct
    auto fft_error = example::fft_signal_error::calculate_for_complex_values(cufftdx_results.output, cufft_results.output);

    std::cout << "FFT: (" << fft_size_x << ", " << fft_size_y << ", " << fft_size_z <<")\n";

#ifdef CUFFTDX_EXAMPLE_DETAIL_DEBUG_FFT_3D
    std::cout << "cuFFT, cuFFTDx\n";
    for (size_t i = 0; i < 8; i++) {
        std::cout << i << ": ";
        std::cout << "(" << cufft_results.output[i].x << ", " << cufft_results.output[i].y << ")";
        std::cout << ", ";
        std::cout << "(" << cufftdx_results.output[i].x << ", " << cufftdx_results.output[i].y << ")";
        std::cout << "\n";
    }
#endif

    bool success = fft_error.l2_relative_error < 0.001;
    std::cout << "Correctness results:\n";
    std::cout << "L2 error: " << fft_error.l2_relative_error << "\n";
    std::cout << "Peak error (index: " << fft_error.peak_error_index << "): " << fft_error.peak_error << "\n";

    // Print performance results
    if(success) {
        std::cout << "\nPerformance results:\n";
        std::cout << std::setw(10) << "cuFFTDx: " << cufftdx_results.avg_time_in_ms << " [ms]\n";
        std::cout << std::setw(10) << "cuFFT: " << cufft_results.avg_time_in_ms << " [ms]\n";
    }

    if (success) {
        std::cout << "Success\n";
        return 0;
    } else {
        std::cout << "Failure\n";
        return 1;
    }
}
