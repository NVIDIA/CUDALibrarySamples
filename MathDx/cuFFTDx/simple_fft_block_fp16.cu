#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "block_io.hpp"
#include "common.hpp"
#include "fp16_common.hpp"

template<class FFT, bool InputInRRIILayout = false, bool OutputInRRIILayout = false>
__launch_bounds__(FFT::max_threads_per_block) __global__ void block_fft_kernel(typename FFT::value_type* data) {
    using complex_type = typename FFT::value_type;

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Load data from global memory to registers
    example::io<FFT>::load<InputInRRIILayout>(data, thread_data, local_fft_id);

    // Execute FFT
    extern __shared__ complex_type shared_mem[];
    FFT().execute(thread_data, shared_mem);

    // Save results
    example::io<FFT>::store<OutputInRRIILayout>(thread_data, data, local_fft_id);
}

// In this example a one-dimensional complex-to-complex transform is performed by a CUDA block.
//
// One block is run, and it calculates four 128-point C2C half precision FFTs.
// Data is generated on host, copied to device buffer, and then results are copied back to host.
//
// Here, we're using complex<half2> with ((Real, Imag), (Real, Imag)) layout as the type of the input/output
// data passed to kernel, and later on the device layout is changed into RRII when values are being loaded.
template<unsigned int Arch>
void simple_block_fft_complex_half2() {
    using namespace cufftdx;

    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.
    // Additionally,
    using FFT          = decltype(Block() + Size<128>() + Type<fft_type::c2c>() + Direction<fft_direction::forward>() +
                         Precision<__half>() + ElementsPerThread<8>() + FFTsPerBlock<4>() + SM<Arch>());
    using complex_type = typename FFT::value_type;

    // Allocate managed memory for input/output
    complex_type* data;
    // For performance reasons half precision cuFFTDx FFTs has an implicit batching of 2 FFTs. This means that:
    // * Used complex type is complex<__half2>, and real type is __half2.
    // * Every thread processes values from two batches simultaneously using __half2 as the base type.
    // * Number of FFTs per block must be a multiple of 2.
    // * Complex data is processed in ((Real1, Real2), (Imag1, Imag2)) layout, where (Real1, Imag1) is a value from
    //   one batch, and (Real2, Imag2) is from a different batch.
    // * Real data is process using __half2 in (Real1, Real2) layout, where Real1 is a value from one batch, and
    //   Real2 is from a different batch.
    constexpr size_t implicit_batching = FFT::implicit_type_batching;
    auto             size              = FFT::ffts_per_block / implicit_batching * cufftdx::size_of<FFT>::value;
    auto             size_bytes        = size * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&data, size_bytes));
    for (size_t i = 0; i < size; i++) {
        float v1 = static_cast<float>(i);
        float v2 = static_cast<float>(i + size);
        // Populate input with complex<half2> values in ((Real, Imag), (Real, Imag)) layout
        data[i] = complex_type {__half2 {v1, -v1}, __half2 {v2, -v2}};
    }

    std::cout << "input [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << __half2float(data[i].x.x) << " " << __half2float(data[i].x.y) << std::endl;
    }

    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        block_fft_kernel<FFT>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        FFT::shared_memory_size));

    // Invokes kernel with FFT::block_dim threads in CUDA block
    block_fft_kernel<FFT><<<1, FFT::block_dim, FFT::shared_memory_size>>>(data);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    std::cout << "output [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << __half2float(data[i].x.x) << " " << __half2float(data[i].x.y) << std::endl;
    }

    CUDA_CHECK_AND_EXIT(hipFree(data));
    std::cout << "Success" << std::endl;
}

template<unsigned int Arch>
struct simple_block_fft_complex_half2_functor {
    void operator()() { return simple_block_fft_complex_half2<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_block_fft_complex_half2_functor>();
}
