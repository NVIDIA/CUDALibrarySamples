#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <limits>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>
#include <hipfft/hipfft.h>

#include "block_io.hpp"
#include "block_io_strided.hpp"
#include "common.hpp"
#include "random.hpp"

// #define CUFFTDX_EXAMPLE_DETAIL_DEBUG_FFT_2D
inline constexpr unsigned int cufftdx_example_warm_up_runs = 5;
inline constexpr unsigned int cufftdx_example_performance_runs = 20;

template<class FFT, class InputType, class OutputType>
__launch_bounds__(FFT::max_threads_per_block) __global__
    void fft_2d_kernel_y(const InputType* input, OutputType* output, typename FFT::workspace_type workspace) {
    using complex_type = typename FFT::value_type;

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Load data from global memory to registers
    example::io<FFT>::load(input, thread_data, local_fft_id);

    // Execute FFT
    extern __shared__ complex_type shared_mem[];
    FFT().execute(thread_data, shared_mem, workspace);

    // Save results
    example::io<FFT>::store(thread_data, output, local_fft_id);
}

template<class FFTF,
         class FFTI,
         unsigned int Stride,
         unsigned int SizeY,
         bool         UseSharedMemoryStridedIO,
         class ComplexType = typename FFTF::value_type>
__launch_bounds__(FFTF::max_threads_per_block) __global__
    void fft_2d_kernel_x(const ComplexType*            input,
                         ComplexType*                  output,
                         typename FFTF::workspace_type workspacef,
                         typename FFTI::workspace_type workspacei) {
    using complex_type = typename FFTF::value_type;

    extern __shared__ complex_type shared_mem[];

    // Local array for thread
    complex_type thread_data[FFTF::storage_size];

    // ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Load data from global memory to registers
    if constexpr (UseSharedMemoryStridedIO) {
        example::io_strided<FFTF>::load_strided<Stride, SizeY>(input, thread_data, shared_mem, local_fft_id);
    } else {
        example::io_strided<FFTF>::load_strided<Stride, SizeY>(input, thread_data, local_fft_id);
    }

    // Execute FFT (part of the 2D R2C FFT)
    FFTF().execute(thread_data, shared_mem, workspacef);

    // Note: You can do any point-wise operation in here.

    // Execute FFT (part of the 2D C2R FFT)
    FFTI().execute(thread_data, shared_mem, workspacei);

    // Save results
    if constexpr (UseSharedMemoryStridedIO) {
        example::io_strided<FFTI>::store_strided<Stride, SizeY>(thread_data, shared_mem, output, local_fft_id);
    } else {
        example::io_strided<FFTI>::store_strided<Stride, SizeY>(thread_data, output, local_fft_id);
    }
}

template<class RealType, class ComplexType>
example::fft_results<RealType> cufft_fft_2d_r2c_c2r(unsigned int fft_size_x,
                                             unsigned int fft_size_y,
                                             RealType*           real_values,
                                             ComplexType*           complex_values,
                                             hipStream_t stream) {
    using complex_type = hipfftComplex;
    static_assert(sizeof(ComplexType) == sizeof(complex_type), "");
    static_assert(std::alignment_of_v<ComplexType> == std::alignment_of_v<complex_type>, "");
    using real_type = hipfftReal;
    static_assert(sizeof(ComplexType) % sizeof(real_type) == 0, "");
    static_assert(std::alignment_of_v<ComplexType> % std::alignment_of_v<real_type> == 0, "");
    static_assert(sizeof(RealType) == sizeof(real_type), "");
    static_assert(std::alignment_of_v<RealType> == std::alignment_of_v<real_type>, "");

    real_type*    cufft_input_r2c  = reinterpret_cast<real_type*>(real_values);
    complex_type* cufft_output_r2c = reinterpret_cast<complex_type*>(complex_values);
    complex_type* cufft_input_c2r  = cufft_output_r2c;
    real_type*    cufft_output_c2r = reinterpret_cast<real_type*>(real_values);

    // Create cuFFT plan
    hipfftHandle plan_r2c;
    hipfftHandle plan_c2r;
    CUFFT_CHECK_AND_EXIT(hipfftPlan2d(&plan_r2c, fft_size_x, fft_size_y, HIPFFT_R2C));
    CUFFT_CHECK_AND_EXIT(hipfftPlan2d(&plan_c2r, fft_size_x, fft_size_y, HIPFFT_C2R));
    CUFFT_CHECK_AND_EXIT(hipfftSetStream(plan_r2c, stream));
    CUFFT_CHECK_AND_EXIT(hipfftSetStream(plan_c2r, stream));

    // Execute cuFFT
    auto cufft_execution = [&](hipStream_t /* stream */) {
        // Out-of-place R2C
        CUFFT_CHECK_AND_EXIT(hipfftExecR2C(plan_r2c, cufft_input_r2c, cufft_output_r2c));
        // Out-of-place C2R
        CUFFT_CHECK_AND_EXIT(hipfftExecC2R(plan_c2r, cufft_input_c2r, cufft_output_c2r));
    };

    // Correctness run
    cufft_execution(stream);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());
    // Copy results to host
    const size_t           flat_fft_size       = fft_size_x * fft_size_y;
    const size_t           flat_fft_size_bytes = flat_fft_size * sizeof(real_type);
    std::vector<real_type> output_host(flat_fft_size, std::numeric_limits<real_type>::quiet_NaN());
    CUDA_CHECK_AND_EXIT(hipMemcpy(output_host.data(), cufft_output_c2r, flat_fft_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Performance measurements
    auto time = example::measure_execution_ms(
        cufft_execution,
        cufftdx_example_warm_up_runs,
        cufftdx_example_performance_runs,
        stream);

    // Clean-up
    CUFFT_CHECK_AND_EXIT(hipfftDestroy(plan_r2c));
    CUFFT_CHECK_AND_EXIT(hipfftDestroy(plan_c2r));

    // Return results
    return example::fft_results<RealType> {output_host, (time / cufftdx_example_performance_runs)};
}

template<class FFTR2CX, class FFTR2CY, class FFTC2RX, class FFTC2RY, bool UseSharedMemoryStridedIO, class RealType, class ComplexType>
example::fft_results<RealType> cufftdx_fft_2d_r2c_c2r(RealType* real_values, ComplexType* complex_values, hipStream_t stream) {
    using FFTX         = FFTR2CX;
    using FFTY         = FFTR2CY;
    using complex_type = typename FFTX::value_type;
    using real_type    = typename complex_type::value_type;

    static constexpr unsigned int fft_size_y = cufftdx::size_of<FFTY>::value;
    static constexpr unsigned int fft_size_x = cufftdx::size_of<FFTX>::value;

    // Checks that FFTX and FFTY are correctly defined
    static_assert(std::is_same_v<cufftdx::precision_of_t<FFTX>, cufftdx::precision_of_t<FFTY>>,
                  "FFTY and FFTX must have the same precision");
    static_assert(std::is_same_v<typename FFTX::value_type, typename FFTY::value_type>,
                  "FFTY and FFTX must operator on the same type");
    static_assert(sizeof(ComplexType) == sizeof(complex_type), "");
    static_assert(std::alignment_of_v<ComplexType> == std::alignment_of_v<complex_type>, "");
    static_assert(sizeof(RealType) == sizeof(real_type), "");
    static_assert(std::alignment_of_v<RealType> == std::alignment_of_v<real_type>, "");
    // Checks below are not caused by any limitation in cuFFTDx, but rather in the example IO functions.
    static_assert((fft_size_x % FFTY::ffts_per_block == 0),
                  "FFTsPerBlock for FFTX must divide Y dimension as IO doesn't check if a batch is in range");

    real_type*    cufftdx_real_values    = reinterpret_cast<real_type*>(real_values);
    complex_type* cufftdx_complex_values = reinterpret_cast<complex_type*>(complex_values);

    // Set shared memory requirements
    auto error_code = hipFuncSetAttribute(reinterpret_cast<const void*>(fft_2d_kernel_y<FFTR2CY), real_type, complex_type>,
                                           hipFuncAttributeMaxDynamicSharedMemorySize,
                                           FFTR2CY::shared_memory_size);
    CUDA_CHECK_AND_EXIT(error_code);
    error_code = hipFuncSetAttribute(reinterpret_cast<const void*>(fft_2d_kernel_y<FFTC2RY), complex_type, real_type>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      FFTC2RY::shared_memory_size);
    CUDA_CHECK_AND_EXIT(error_code);
    unsigned int fft_x_shared_memory_smem_io =
        std::max<unsigned>({FFTR2CX::shared_memory_size,
                            FFTC2RX::shared_memory_size,
                            FFTX::ffts_per_block * fft_size_x * sizeof(complex_type)});
    unsigned int fft_x_shared_memory =
        UseSharedMemoryStridedIO ? fft_x_shared_memory_smem_io : FFTX::shared_memory_size;
    error_code = hipFuncSetAttribute(reinterpret_cast<const void*>(fft_2d_kernel_x<FFTR2CX),
                                                      FFTC2RX,
                                                      ((fft_size_y / 2) + 1),
                                                      ((fft_size_y / 2) + 1),
                                                      UseSharedMemoryStridedIO,
                                                      complex_type>,
                                      hipFuncAttributeMaxDynamicSharedMemorySize,
                                      fft_x_shared_memory);
    CUDA_CHECK_AND_EXIT(error_code);

    // Create workspaces for FFTs
    auto workspace_y_r2c = cufftdx::make_workspace<FFTR2CY>(error_code);
    CUDA_CHECK_AND_EXIT(error_code);
    auto workspace_x_r2c = cufftdx::make_workspace<FFTR2CX>(error_code);
    CUDA_CHECK_AND_EXIT(error_code);
    auto workspace_y_c2r = cufftdx::make_workspace<FFTC2RY>(error_code);
    CUDA_CHECK_AND_EXIT(error_code);
    auto workspace_x_c2r = cufftdx::make_workspace<FFTC2RX>(error_code);
    CUDA_CHECK_AND_EXIT(error_code);

    // Synchronize device before execution
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Out-of-place R2C kernel (R2C Y dimension)
    const auto grid_fft_size_y_r2c = ((fft_size_x + FFTR2CY::ffts_per_block - 1) / FFTR2CY::ffts_per_block);
    const auto grid_fft_size_x     = ((((fft_size_y / 2) + 1) + FFTX::ffts_per_block - 1) / FFTX::ffts_per_block);
    const auto grid_fft_size_y_c2r = ((fft_size_x + FFTC2RY::ffts_per_block - 1) / FFTC2RY::ffts_per_block);
    auto       fft_2d_execution    = [&](hipStream_t stream) {
        fft_2d_kernel_y<FFTY, real_type, complex_type>
            <<<grid_fft_size_y_r2c, FFTY::block_dim, FFTY::shared_memory_size, stream>>>(
                cufftdx_real_values, cufftdx_complex_values, workspace_y_r2c);
        CUDA_CHECK_AND_EXIT(hipGetLastError());

        // In-place C2CF (R2C X dimension) and C2CI (C2R X dimension) kernel
        fft_2d_kernel_x<FFTR2CX,
                        FFTC2RX,
                        ((fft_size_y / 2) + 1),
                        ((fft_size_y / 2) + 1),
                        UseSharedMemoryStridedIO,
                        complex_type><<<grid_fft_size_x, FFTX::block_dim, fft_x_shared_memory, stream>>>(
            cufftdx_complex_values, cufftdx_complex_values, workspace_x_r2c, workspace_x_c2r);
        CUDA_CHECK_AND_EXIT(hipGetLastError());

        // Out-of-place C2R kernel (C2R Y dimension)
        fft_2d_kernel_y<FFTC2RY, complex_type, real_type>
            <<<grid_fft_size_y_c2r, FFTC2RY::block_dim, FFTC2RY::shared_memory_size, stream>>>(
                cufftdx_complex_values, cufftdx_real_values, workspace_y_c2r);
        CUDA_CHECK_AND_EXIT(hipGetLastError());
    };

    // Correctness run
    fft_2d_execution(stream);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());
    // Copy results to host
    static constexpr size_t flat_fft_size       = fft_size_x * fft_size_y;
    static constexpr size_t flat_fft_size_bytes = flat_fft_size * sizeof(real_type);
    std::vector<real_type>  output_host(flat_fft_size, std::numeric_limits<real_type>::quiet_NaN());
    CUDA_CHECK_AND_EXIT(hipMemcpy(output_host.data(), cufftdx_real_values, flat_fft_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Performance measurements
    auto time = example::measure_execution_ms(
        [&](hipStream_t stream) {
            fft_2d_execution(stream);
        },
        cufftdx_example_warm_up_runs,
        cufftdx_example_performance_runs,
        stream);

    // Return results
    return example::fft_results<RealType> {output_host, (time / cufftdx_example_performance_runs)};
}

// Notes:
// * This examples shows how to use cuFFTDx to run multi-dimensional FFT. Final performance will vary depending on the
// FFT definitions (precision, size, type, ept, fpb) and other user customizations.
// * Best possible performance requires adapting parameters in the sample to particular set of parameters and code customizations.
// * Only FP32 supported in this example.
// * cuFFTDx with enabled shared memory IO usually be the faster cuFFTDx option for larger (>512) sizes.
// * The shared memory IO cuFFTDx has high shared memory requirements and will not work for all possible sizes in X dimension.
template<unsigned int Arch>
void fft_2d() {
    using precision_type                     = float;
    using complex_type                       = cufftdx::complex<precision_type>;
    using real_type                          = complex_type::value_type;

    // FFT
    static constexpr unsigned int fft_size_y = 1024;
    static constexpr unsigned int fft_size_x = 1024;
    // Kernel Settings
    static constexpr unsigned int ept_y = 16;
    static constexpr unsigned int fpb_y = 1;
    static constexpr unsigned int ept_x = 16;
    static constexpr unsigned int fpb_x = 8;

    // Other recommended configurations to test:
    // 1:
    // static constexpr unsigned int fft_size_y = 16384;
    // static constexpr unsigned int fft_size_x = 4096;
    // static constexpr unsigned int ept_y = 16;
    // static constexpr unsigned int fpb_y = 1;
    // static constexpr unsigned int ept_x = 16;
    // static constexpr unsigned int fpb_x = 2;
    // 2:
    // static constexpr unsigned int fft_size_y = 2048;
    // static constexpr unsigned int fft_size_x = 2048;
    // static constexpr unsigned int ept_y = 16;
    // static constexpr unsigned int fpb_y = 1;
    // static constexpr unsigned int ept_x = 8;
    // static constexpr unsigned int fpb_x = 4;
    // 3:
    // static constexpr unsigned int fft_size_y = 128;
    // static constexpr unsigned int fft_size_x = 128;
    // static constexpr unsigned int ept_y = 16;
    // static constexpr unsigned int fpb_y = 1;
    // static constexpr unsigned int ept_x = 16;
    // static constexpr unsigned int fpb_x = 8;

    using namespace cufftdx;
    using fft_base  = decltype(Block() + Precision<precision_type>() + SM<Arch>());
    using fft_y_r2c = decltype(fft_base() + Type<fft_type::r2c>() + Size<fft_size_y>() + ElementsPerThread<ept_y>() +
                               FFTsPerBlock<fpb_y>());
    using fft_x_r2c =
        decltype(fft_base() + Type<fft_type::c2c>() + Size<fft_size_x>() + Direction<fft_direction::forward>() +
                 ElementsPerThread<ept_x>() + FFTsPerBlock<fpb_x>());
    using fft_y_c2r = cufftdx::replace_t<fft_y_r2c, Type<fft_type::c2r>>;
    using fft_x_c2r = cufftdx::replace_t<fft_x_r2c, Direction<fft_direction::inverse>>;

    // Host data
    static constexpr size_t flat_fft_size_real          = fft_size_x * fft_size_y;
    static constexpr size_t flat_fft_size_complex       = fft_size_x * ((fft_size_y / 2) + 1);
    static constexpr size_t flat_fft_size_real_bytes    = flat_fft_size_real * sizeof(precision_type);
    static constexpr size_t flat_fft_size_complex_bytes = flat_fft_size_complex * sizeof(complex_type);
#ifdef CUFFTDX_EXAMPLE_DETAIL_DEBUG_FFT_2D
    std::vector<precision_type> input_host(flat_fft_size_real);
    for (size_t i = 0; i < flat_fft_size_real; i++) {
        float sign    = (i % 3 == 0) ? -1.0f : 1.0f;
        input_host[i] = sign * static_cast<float>(i) / flat_fft_size_real;
    }
#else
    auto input_host = example::get_random_real_data<precision_type>(flat_fft_size_real, -1, 1);
#endif

    // Device data
    real_type* real_values;
    complex_type* complex_values;
    CUDA_CHECK_AND_EXIT(hipMalloc(&real_values, flat_fft_size_real_bytes));
    CUDA_CHECK_AND_EXIT(hipMalloc(&complex_values, flat_fft_size_complex_bytes));

    // Copy host to device
    CUDA_CHECK_AND_EXIT(hipMemset(complex_values, 0b11111111, flat_fft_size_complex_bytes));
    CUDA_CHECK_AND_EXIT(hipMemcpy(real_values, input_host.data(), flat_fft_size_real_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    hipStream_t stream;
    CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream));

    // cuFFTDx fused 2D R2C->C2R
    // * the 2nd R2C (dim X) kernel and the 1st (dim X) C2R kernel are fused into one
    // * the final results are stored in real_values
    auto cufftdx_results =
        cufftdx_fft_2d_r2c_c2r<fft_x_r2c, fft_y_r2c, fft_x_c2r, fft_y_c2r, false>(real_values, complex_values, stream);
    CUDA_CHECK_AND_EXIT(hipMemcpy(real_values, input_host.data(), flat_fft_size_real_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // cuFFTDx fused 2D R2C->C2R
    // * the 2nd R2C (dim X) kernel and the 1st (dim X) C2R kernel are fused into one
    // * the final results are stored in real_values
    // * Uses shared memory to speed-up IO in the strided kernel
    auto cufftdx_smemio_results =
        cufftdx_fft_2d_r2c_c2r<fft_x_r2c, fft_y_r2c, fft_x_c2r, fft_y_c2r, true>(real_values, complex_values, stream);
    CUDA_CHECK_AND_EXIT(hipMemcpy(real_values, input_host.data(), flat_fft_size_real_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // cuFFT 2D R2C->C2R
    // * the final results are stored in real_values
    auto cufft_results = cufft_fft_2d_r2c_c2r(fft_size_x, fft_size_y, real_values, complex_values, stream);
    CUDA_CHECK_AND_EXIT(hipMemcpy(real_values, input_host.data(), flat_fft_size_real_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Destroy created CUDA stream
    CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));

    // Free CUDA buffers
    CUDA_CHECK_AND_EXIT(hipFree(real_values));
    CUDA_CHECK_AND_EXIT(hipFree(complex_values));

    std::cout << "FFT: (" << fft_size_x << ", " << fft_size_y << ")\n";

#ifdef CUFFTDX_EXAMPLE_DETAIL_DEBUG_FFT_2D
    std::cout << "cuFFT, cuFFTDx\n";
    for (size_t i = 0; i < 8; i++) {
        std::cout << i << ": ";
        std::cout << "(" << cufft_output_host[i] << ")";
        std::cout << ", ";
        std::cout << "(" << cufftdx_output_host[i] << ")";
        std::cout << "\n";
    }
#endif

    bool success = true;
    // Check if cuFFTDx results are correct
    {
        auto fft_error =
            example::fft_signal_error::calculate_for_real_values(cufftdx_results.output, cufft_results.output);
        std::cout << "cuFFTDx\n";
        std::cout << "L2 error: " << fft_error.l2_relative_error << "\n";
        std::cout << "Peak error (index: " << fft_error.peak_error_index << "): " << fft_error.peak_error << "\n";
        std::cout << "Peak relative error (index: " << fft_error.peak_error_index << "): " << fft_error.peak_error_relative << "\n";
        if (success) {
            success = (fft_error.l2_relative_error < 0.001);
        }
    }
    // Check cuFFTDx with shared memory io
    {
        auto fft_error =
            example::fft_signal_error::calculate_for_real_values(cufftdx_smemio_results.output, cufft_results.output);
        std::cout << "cuFFTDx (shared memory IO)\n";
        std::cout << "L2 error: " << fft_error.l2_relative_error << "\n";
        std::cout << "Peak error (index: " << fft_error.peak_error_index << "): " << fft_error.peak_error << "\n";
        std::cout << "Peak relative error (index: " << fft_error.peak_error_index << "): " << fft_error.peak_error_relative << "\n";
        if (success) {
            success = (fft_error.l2_relative_error < 0.001);
        }
    }

    // Print performance results
    if (success) {
        std::cout << "\nPerformance results:\n";
        std::cout << std::setw(28) << "cuFFTDx: " << cufftdx_results.avg_time_in_ms << " [ms]\n";
        std::cout << std::setw(28) << "cuFFTDx (shared memory IO): " << cufftdx_smemio_results.avg_time_in_ms
                  << " [ms]\n";
        std::cout << std::setw(28) << "cuFFT: " << cufft_results.avg_time_in_ms << " [ms]\n";
    }

    if (success) {
        std::cout << "Success\n";
    } else {
        std::cout << "Failure\n";
        std::exit(1);
    }
}

template<unsigned int Arch>
struct fft_2d_functor {
    void operator()() { return fft_2d<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<fft_2d_functor>();
}
