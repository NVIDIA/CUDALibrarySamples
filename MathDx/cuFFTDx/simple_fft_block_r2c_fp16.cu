#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cufftdx.hpp>

#include "block_io.hpp"
#include "common.hpp"

template<class FFT, class ComplexType = typename FFT::value_type, class ScalarType = typename ComplexType::value_type>
__launch_bounds__(FFT::max_threads_per_block) __global__
    void block_fft_kernel_r2c_fp16(ScalarType* input_data, ComplexType* output_data) {
    using complex_type = ComplexType;

    // Local array for thread
    complex_type thread_data[FFT::storage_size];

    // ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
    const unsigned int local_fft_id = threadIdx.y;
    // Load data from global memory to registers
    example::io<FFT>::load_r2c(input_data, thread_data, local_fft_id);

    // Execute FFT
    extern __shared__ complex_type shared_mem[];
    FFT().execute(thread_data, shared_mem);

    // Save results
    example::io<FFT>::store_r2c<false /* Store to output in RRII layout */>(thread_data, output_data, local_fft_id);
}

// In this example a one-dimensional real-to-complex transform is performed by a CUDA block.
//
// One block is run, and it calculates four 128-point R2C half precision FFTs.
// Data is generated on host, copied to device buffer, and then results are copied back to host.
template<unsigned int Arch>
void simple_block_fft_r2c_fp16() {
    using namespace cufftdx;

    // FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
    // will be executed on block level. Shared memory is required for co-operation between threads.
    using FFT          = decltype(Block() + Size<128>() + Type<fft_type::r2c>() + Direction<fft_direction::forward>() +
                         Precision<__half>() + ElementsPerThread<16>() + FFTsPerBlock<4>() + SM<Arch>());
    using complex_type = typename FFT::value_type;          // complex<__half2>
    using real_type    = typename complex_type::value_type; // __half2

    // Allocate managed memory for input/output
    real_type* input_data;
    // For performance reasons half precision cuFFTDx FFTs has an implicit batching of 2 FFTs. This means that:
    // * Used complex type is complex<__half2>, and real type is __half2.
    // * Every thread processes values from two batches simultaneously using __half2 as the base type.
    // * Number of FFTs per block must be a multiple of 2.
    // * Complex data is processed in ((Real1, Real2), (Imag1, Imag2)) layout, where (Real1, Imag1) is a value from
    //   one batch, and (Real2, Imag2) is from a different batch.
    // * Real data is process using __half2 in (Real1, Real2) layout, where Real1 is a value from one batch, and
    //   Real2 is from a different batch.
    constexpr size_t implicit_batching = FFT::implicit_type_batching;
    auto             input_size        = FFT::ffts_per_block / implicit_batching * cufftdx::size_of<FFT>::value;
    auto             input_size_bytes  = input_size * sizeof(real_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&input_data, input_size_bytes));
    for (size_t i = 0; i < input_size; i++) {
        input_data[i] = __half2 {float(i), float(i + input_size)};
    }
    complex_type* output_data;
    auto          output_size       = FFT::ffts_per_block / implicit_batching * (cufftdx::size_of<FFT>::value / 2 + 1);
    auto          output_size_bytes = output_size * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&output_data, output_size_bytes));

    std::cout << "input [1st FFT]:\n";
    for (size_t i = 0; i < cufftdx::size_of<FFT>::value; i++) {
        std::cout << __half2float(input_data[i].x) << std::endl;
    }

    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        block_fft_kernel_r2c_fp16<FFT>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        FFT::shared_memory_size));

    // Invokes kernel with FFT::block_dim threads in CUDA block
    block_fft_kernel_r2c_fp16<FFT><<<1, FFT::block_dim, FFT::shared_memory_size>>>(input_data, output_data);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    std::cout << "output [1st FFT]:\n";
    for (size_t i = 0; i < (cufftdx::size_of<FFT>::value / 2 + 1); i++) {
        std::cout << __half2float(output_data[i].x.x) << " " << __half2float(output_data[i].x.y) << std::endl;
    }

    CUDA_CHECK_AND_EXIT(hipFree(input_data));
    CUDA_CHECK_AND_EXIT(hipFree(output_data));
    std::cout << "Success" << std::endl;
}

template<unsigned int Arch>
struct simple_block_fft_r2c_fp16_functor {
    void operator()() { return simple_block_fft_r2c_fp16<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_block_fft_r2c_fp16_functor>();
}
