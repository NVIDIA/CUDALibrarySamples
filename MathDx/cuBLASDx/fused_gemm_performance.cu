#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <type_traits>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "block_io.hpp"
#include "reference.hpp"

template<class BLAS1, class BLAS2, class ValueType = typename BLAS1::value_type>
__launch_bounds__(BLAS1::max_threads_per_block) __global__
void fused_gemm_kernel(const ValueType alpha1,
                       const ValueType* a,
                       const ValueType* b,
                       const ValueType  beta1,
                       const ValueType* c,
                       const ValueType  alpha2,
                       const ValueType* d,
                       const ValueType  beta2,
                       const ValueType* f,
                       ValueType*       output) {
    using value_type = ValueType;
    extern __shared__ __align__(16) char smem[];
    constexpr unsigned int block_size = BLAS1::block_dim.x * BLAS1::block_dim.y * BLAS1::block_dim.z;

    static_assert(std::is_same_v<value_type, example::value_type_t<BLAS2>>, "BLAS1 and BLAS2 must have the same type and precision");
    static_assert((BLAS1::c_dim == BLAS2::a_dim), "The dimensions of C matrix are different in BLAS1 and BLAS2");

    // Matrix C is the first in shared memory, because it's reused in the 2nd GEMM. Moreover,
    // matrices A and B might have different sizes than F and D.
    value_type* smem_c = reinterpret_cast<value_type*>(smem);
    value_type* smem_a = reinterpret_cast<value_type*>(smem) + BLAS1::c_size;
    value_type* smem_b = reinterpret_cast<value_type*>(smem) + BLAS1::c_size + BLAS1::a_size;

    example::io<BLAS1>::a_fast_load<block_size>(smem_a, a);
    example::io<BLAS1>::b_fast_load<block_size>(smem_b, b);
    example::io<BLAS1>::c_fast_load<block_size>(smem_c, c);
    __syncthreads();

    BLAS1().execute(alpha1, smem_a, smem_b, beta1, smem_c);
    __syncthreads();

    static_assert((BLAS1::c_size == BLAS2::a_size), "The sizes of C matrix are different in BLAS1 and BLAS2");
    value_type* smem_d = smem_c + BLAS2::a_size;
    value_type* smem_f = smem_c + BLAS2::a_size + BLAS2::b_size;

    example::io<BLAS2>::b_fast_load<block_size>(smem_d, d);
    example::io<BLAS2>::c_fast_load<block_size>(smem_f, f);
    __syncthreads();

    BLAS2().execute(alpha2, smem_c, smem_d, beta2, smem_f);

    __syncthreads();
    example::io<BLAS2>::c_fast_store<block_size>(output, smem_f);
}

template<class BLAS1, class BLAS2, class ValueType = typename BLAS1::value_type>
double measure_cublasdx(unsigned int kernel_warm_up_repeats,
                        unsigned int kernel_repeats,
                        const ValueType alpha1,
                        const ValueType* a,
                        const ValueType* b,
                        const ValueType  beta1,
                        const ValueType* c,
                        const ValueType  alpha2,
                        const ValueType* d,
                        const ValueType  beta2,
                        const ValueType* f,
                        ValueType*       output,
                        hipStream_t     stream) {

    // Increase max dynamic shared memory for the kernel if needed.
    const auto shared_memory = std::max<size_t>(BLAS1::shared_memory_size, BLAS2::shared_memory_size);
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(fused_gemm_kernel<BLAS1), BLAS2>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));

    // Execute kernel.
    double time = example::measure::execution(
        [&](hipStream_t stream) {
            fused_gemm_kernel<BLAS1, BLAS2>
                <<<1, BLAS1::block_dim, shared_memory, stream>>>(alpha1, a, b, beta1, c, alpha2, d, beta2, f, output);
        },
        kernel_warm_up_repeats,
        kernel_repeats,
        stream);

    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    return time;
}

template<class BLAS1, class BLAS2, class ValueType=typename BLAS1::value_type>
double measure_cublas(unsigned int kernel_warm_up_repeats,
                      unsigned int kernel_repeats,
                      ValueType        alpha1,
                      const ValueType* a,
                      const ValueType* b,
                      ValueType        beta1,
                      ValueType*       c,
                      const ValueType  alpha2,
                      const ValueType* d,
                      const ValueType  beta2,
                      ValueType* f,
                      hipStream_t stream) {

    static_assert(std::is_same_v<ValueType, example::value_type_t<BLAS2>>, "BLAS1 and BLAS2 must have the same type and precision");
    static_assert((BLAS1::c_dim == BLAS2::a_dim), "The dimensions of C matrix are different in BLAS1 and BLAS2");

    const unsigned int m1 = cublasdx::size_of<BLAS1>::m;
    const unsigned int n1 = cublasdx::size_of<BLAS1>::n;
    const unsigned int k1 = cublasdx::size_of<BLAS1>::k;

    const unsigned int lda1 = cublasdx::leading_dimension_of<BLAS1>::a;
    const unsigned int ldb1 = cublasdx::leading_dimension_of<BLAS1>::b;
    const unsigned int ldc1 = cublasdx::leading_dimension_of<BLAS1>::c;

    const unsigned int m2 = cublasdx::size_of<BLAS2>::m;
    const unsigned int n2 = cublasdx::size_of<BLAS2>::n;
    const unsigned int k2 = cublasdx::size_of<BLAS2>::k;

    const unsigned int lda2 = cublasdx::leading_dimension_of<BLAS2>::a;
    const unsigned int ldb2 = cublasdx::leading_dimension_of<BLAS2>::b;
    const unsigned int ldc2 = cublasdx::leading_dimension_of<BLAS2>::c;

    static_assert(example::is_complex<ValueType>() && std::is_same_v<float, typename example::get_precision<ValueType>::type>,
        "Type or precision is currently not supported for cuBLAS measurement.");

    //
    // cuBLAS
    //
    hipblasHandle_t handle;
    CUBLAS_CHECK_AND_EXIT(hipblasCreate(&handle));

    const auto a_transpose = example::detail::get_cublas_transpose_mode(cublasdx::transpose_mode_of<BLAS1>::a_transpose_mode);
    const auto b_transpose = example::detail::get_cublas_transpose_mode(cublasdx::transpose_mode_of<BLAS1>::b_transpose_mode);

    const auto c_transpose = example::detail::get_cublas_transpose_mode(cublasdx::transpose_mode_of<BLAS2>::a_transpose_mode);
    const auto d_transpose = example::detail::get_cublas_transpose_mode(cublasdx::transpose_mode_of<BLAS2>::b_transpose_mode);

    hipblasSetStream(handle, stream);

    double time_cublas = example::measure::execution(
        [&](hipStream_t) {
            // C = alpha1 * A * B + beta1 * C
            CUBLAS_CHECK_AND_EXIT(hipblasCgemm(handle,
                a_transpose,
                b_transpose,
                m1,
                n1,
                k1,
                reinterpret_cast<const hipComplex*>(&alpha1),
                reinterpret_cast<const hipComplex*>(a),
                lda1,
                reinterpret_cast<const hipComplex*>(b),
                ldb1,
                reinterpret_cast<const hipComplex*>(&beta1),
                reinterpret_cast<hipComplex*>(c),
                ldc1));
            // F = alpha2 * C * D + beta2 * F
            CUBLAS_CHECK_AND_EXIT(hipblasCgemm(handle,
                c_transpose,
                d_transpose,
                m2,
                n2,
                k2,
                reinterpret_cast<const hipComplex*>(&alpha2),
                reinterpret_cast<const hipComplex*>(c),
                lda2,
                reinterpret_cast<const hipComplex*>(d),
                ldb2,
                reinterpret_cast<const hipComplex*>(&beta2),
                reinterpret_cast<hipComplex*>(f),
                ldc2));
        },
        kernel_warm_up_repeats, kernel_repeats, stream);

    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());
    CUBLAS_CHECK_AND_EXIT(hipblasDestroy(handle));

    return time_cublas;
}

// This example compares the performance of cuBLAS and cuBLASDx for the example described in
// "gemm_fusion.cu". Using cuBLASDx we fuse the chained matrix multiplication into one kernel,
// whereas we compose the final result using cuBLAS using two matrix multiplications.
//
//             1) C = alpha1 * (A * B) + beta1 * C
//             2) F = alpha2 * (C * D) + beta2 * F
//
// * A, B, C, D and F are matrices containing complex single precision floating-point values.
// * (alpha1, beta1) and (alpha2, beta2) and complex single precision floating-point values.
//
// Input data is generated on host using random number generators, and later copied to
// the global memory. Then we compare the performance of the fused kernel in cuBLASDx with the
// performance of the two matrix multiplications needed to compute the result using cuBLAS.
//
// For cuBLASDx the number of threads participating in the GEMM operation is imposed by providing
// BlockDim operator in definition of the GEMM. If BlockDim operator is not used, cuBLASDx automatically
// selects number of threads. Block dimensions are provided via BLAS::block_dim trait.
//
// General Notes:
// * The matrix sizes should be small enough so that a single block is the optimal choice, since
//   cuBLASDx is limited to one using one block whereas cuBLAS can use a larger number of blocks.
//
// cuBLASDx Notes:
// * Both GEMM operations use the same number of threads, however, it's not a requirement.
// * It's important that the dimensions of the first and the 2nd GEMM are set in such a way that
//   the C matrix has the same dimensions in both operations.
template<unsigned int Arch>
int fused_gemm_performance() {
    // Parameters m1, n1, k1 define the dimensions of matrices A, B, and C.
    constexpr unsigned int m1          = 32;
    constexpr unsigned int n1          = 32;
    constexpr unsigned int k1          = 32;

    // Parameters m2, n2, k2 define the dimensions of matrices C, D and F.
    // Note: (m1, n1) and (m2, k2) must be equal as describe the same matrix (matrix C).
    constexpr unsigned int m2          = m1;
    constexpr unsigned int n2          = 32;
    constexpr unsigned int k2          = n1;

    // The logical dimensions of matrix A are: [m1, k1] (m rows, k columns).
    // The logical dimensions of matrix B are: [k1, n1].
    // The logical dimensions of matrix C are: [m1, n1].
    constexpr auto a_transpose_mode = cublasdx::transpose_mode::non_transposed;
    constexpr auto b_transpose_mode = cublasdx::transpose_mode::non_transposed;

    // The logical dimensions of matrix C are: [m2, k2] == [m1, n1].
    // The logical dimensions of matrix D are: [k2, n2].
    // The logical dimensions of matrix F are: [m2, n2].
    constexpr auto c_transpose_mode = cublasdx::transpose_mode::non_transposed;
    constexpr auto d_transpose_mode = cublasdx::transpose_mode::non_transposed;

    // Use the same block size for both GEMM operations, so BLAS1::block_dim == BLAS2::block_dim which
    // simplifies the example.
    constexpr unsigned int block_size = 512;

    // Choose the precision and data type. In this example, we limit ourselves single precision complex
    // data to keep the cuBLAS measurement code simple.
    using precision = float;
    constexpr auto type = cublasdx::type::complex;

    using BLAS1       = decltype(cublasdx::Size<m1, n1, k1>() +
                          cublasdx::Precision<precision>() +
                          cublasdx::Type<type>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::TransposeMode<a_transpose_mode, b_transpose_mode>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());
    using BLAS2       = decltype(cublasdx::Size<m2, n2, k2>() +
                          cublasdx::Precision<precision>() +
                          cublasdx::Type<type>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::TransposeMode<c_transpose_mode, d_transpose_mode>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());
    #if CUBLASDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using value_type = example::value_type_t<BLAS1>;
    #else
    using value_type = typename BLAS1::value_type;
    #endif

    // Set the beta values (beta1 and beta2) for the two GEMMs to 0. since cuBLAS accumulates into the result and
    // we perform multiple repeats.
    // alpha and beta for the first GEMM.
    value_type alpha1 = example::make_value<value_type>(1., 2.);
    value_type beta1  = example::make_value<value_type>(0.);

    // alpha and beta for the second GEMM.
    value_type alpha2 = example::make_value<value_type>(3., 4.);
    value_type beta2  = example::make_value<value_type>(0.);

    // Allocate device memory for a, b, c, d, f and output.
    value_type* inputs;
    value_type* output;
    auto inputs_size       = BLAS1::a_size + BLAS1::b_size + BLAS1::c_size + BLAS2::b_size + BLAS2::c_size;
    auto inputs_size_bytes = inputs_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMalloc(&inputs, inputs_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMalloc(&output, BLAS2::c_size * sizeof(value_type)));

    value_type* a     = inputs;
    value_type* b     = a + (BLAS1::a_size);
    value_type* c     = b + (BLAS1::b_size); // C matrix for BLAS1, A matrix for BLAS2
    value_type* d     = c + (BLAS1::c_size); // D is B matrix for BLAS2
    value_type* f     = d + (BLAS2::b_size); // F is C matrix for BLAS2

    // Fill the A, B, C matrices with random values.
    auto host_a = example::get_random_data<value_type>(0.1, 1.0, BLAS1::a_size);
    auto host_b = example::get_random_data<value_type>(0.1, 1.0, BLAS1::b_size);
    auto host_c = example::get_random_data<value_type>(0.1, 1.0, BLAS1::c_size);
    auto host_d = example::get_random_data<value_type>(1.0, 2.0, BLAS2::b_size);
    auto host_f = example::get_random_data<value_type>(1.0, 10.0, BLAS2::c_size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(a, host_a.data(), BLAS1::a_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(b, host_b.data(), BLAS1::b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(c, host_c.data(), BLAS1::c_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(d, host_d.data(), BLAS2::b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(f, host_f.data(), BLAS2::c_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    const unsigned int kernel_repeats = 100;
    const unsigned int kernel_warm_up_repeats = 1;
    hipStream_t stream;
    CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream))

    // Measure cuBLASDx performance.
    double time_cublasdx =
        measure_cublasdx<BLAS1, BLAS2>(kernel_warm_up_repeats, kernel_repeats, alpha1, a, b, beta1, c, alpha2, d, beta2, f, output, stream);

    // Measure cuBLAS performance.
    double time_cublas =
        measure_cublas<BLAS1, BLAS2>(kernel_warm_up_repeats, kernel_repeats, alpha1, a, b, beta1, c, alpha2, d, beta2, f, stream);

    // Write performance data.
    using cublasdx::size_of;
    std::cout << "m1, n1, k1: " << size_of<BLAS1>::m << ", " << size_of<BLAS1>::n << ", " << size_of<BLAS1>::k
              << std::endl;
    std::cout << "m2, n2, k2: " << size_of<BLAS2>::m << ", " << size_of<BLAS2>::n << ", " << size_of<BLAS2>::k
              << std::endl;
    std::cout << "Type: " << example::type_string<value_type>() << std::endl;
    std::cout << "Precision: " << example::precision_string<value_type>() << std::endl;

    std::cout << "\ncuBLASDx (fused kernel)\n";
    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Avg time [ms]  = " << time_cublasdx / kernel_repeats << "\n";

    std::cout << "\ncuBLAS\n";
    std::cout << "Avg time [ms]  = " << time_cublas / kernel_repeats << "\n";

    // Free resources.
    CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));
    CUDA_CHECK_AND_EXIT(hipFree(inputs));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    return 0;
}

template<unsigned int Arch>
struct fused_gemm_performance_functor {
    int operator()() {
        return fused_gemm_performance<Arch>();
    }
};

int main(int, char**) {
    return example::sm_runner<fused_gemm_performance_functor>();
}
