#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "block_io.hpp"
#include "reference.hpp"

// Number of batches
constexpr unsigned int batches = 2;

template<class BLAS, class ValueType = example::uniform_value_type_t<BLAS>>
__global__ void gemm_kernel(const ValueType* a,
                            const ValueType* b,
                            const ValueType* c,
                            const ValueType  alpha,
                            const ValueType  beta,
                            ValueType*       output) {
    using value_type = ValueType;
    extern __shared__ __align__(16) char smem[];

    // threads (X, 0) calculates the first batch
    // threads (X, 1) calculates the 2nd batch etc.
    // We need to move pointers to corresponding batches
    constexpr auto global_a_size = example::global_memory_size_of<BLAS>::a_size;
    constexpr auto global_b_size = example::global_memory_size_of<BLAS>::b_size;
    constexpr auto global_c_size = example::global_memory_size_of<BLAS>::c_size;

    a += threadIdx.y * global_a_size;
    b += threadIdx.y * global_b_size;
    c += threadIdx.y * global_c_size;
    output += threadIdx.y * global_c_size;

    value_type* smem_a = reinterpret_cast<value_type*>(smem) + threadIdx.y * BLAS::a_size;
    value_type* smem_b = reinterpret_cast<value_type*>(smem) + (batches * BLAS::a_size) + threadIdx.y * BLAS::b_size;
    value_type* smem_c = reinterpret_cast<value_type*>(smem) + (batches * BLAS::a_size) + (batches * BLAS::b_size) + threadIdx.y * BLAS::c_size;

    // Load all batches
    auto a_global_tensor = cublasdx::make_tensor(a, BLAS::get_layout_gmem_a());
    auto b_global_tensor = cublasdx::make_tensor(b, BLAS::get_layout_gmem_b());
    auto c_global_tensor = cublasdx::make_tensor(c, BLAS::get_layout_gmem_c());

    auto a_shared_tensor = cublasdx::make_tensor(smem_a, BLAS::get_layout_smem_a());
    auto b_shared_tensor = cublasdx::make_tensor(smem_b, BLAS::get_layout_smem_b());
    auto c_shared_tensor = cublasdx::make_tensor(smem_c, BLAS::get_layout_smem_c());

    using alignment = cublasdx::alignment_of<BLAS>;
    cublasdx::copy<BLAS, alignment::a>(a_global_tensor, a_shared_tensor);
    cublasdx::copy<BLAS, alignment::b>(b_global_tensor, b_shared_tensor);
    cublasdx::copy<BLAS, alignment::c>(c_global_tensor, c_shared_tensor);
    cublasdx::copy_wait();
    BLAS().execute(alpha, a_shared_tensor, b_shared_tensor, beta, c_shared_tensor);

    __syncthreads();
    auto out_global_tensor = cublasdx::make_tensor(output, BLAS::get_layout_gmem_c());
    cublasdx::copy<BLAS, alignment::c>(c_shared_tensor, out_global_tensor);
}

// This is an example of two fp32 general matrix-matrix multiplications (GEMM) performed in a single CUDA block:
//
//              C(X) = alpha * A(X) * B(X) + beta * C(X)
//
// * X - batch id
// * A, B, and C are matrices containing real single precision floating-point values.
// * alpha and beta are real single precision floating-point values.
//
// Input data is generated on host using random number generators, and later copied to
// the global memory. Next, kernel with GEMM is executed, and then the matrix C (the result)
// is copied back to host memory. The results are verified against cuBLAS.
//
// In this example BlockDim operator added to the GEMM definition defines the number and the layout of threads
// participating in the calculations.
//
// In order to achieve batching on a block level 1D BlockDim operator (BlockDim<64>) is added to the description,
// and launching the kernel with 2D block - dim3(64,2). Threads with the same 2nd dimension (threadIdx.y) participate
// in the same calculations.
//
// Note: Examples demonstrates how to set block dimensions to enable manual batching. The performance of included
// kernel was not checked and it is not optimized.
template<unsigned int Arch>
int simple_gemm() {
    // Parameters m, n, k define the dimensions of matrices A, B, and C
    constexpr unsigned int m = 16;
    constexpr unsigned int n = 16;
    constexpr unsigned int k = 16;

    // Selected CUDA block size (1D)
    constexpr unsigned int block_size = 64;

    // GEMM definition using cuBLASDx operators:
    // 1. The size, the precision, and the type (real or complex) are set.
    // 2. The BLAS function is selected: MM (matrix multiplication).
    // 3. The data arrangements of A, B matrices are set (C is defaulted to column major).
    //    - Optional
    // 4. The data alignment of A, B and C matrices is set to the max accepted value.
    //    - Optional
    // 5. Block operator informs that GEMM should be performed on CUDA block level.
    // 6. BlockDim operator sets layout and number of threads.
    //    - Optional
    // 7. Targeted CUDA compute capability is selected with SM operator.
    using BLAS = decltype(cublasdx::Size<m, n, k>() +
                          cublasdx::Precision<double>() +
                          cublasdx::Type<cublasdx::type::real>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::Arrangement<cublasdx::row_major, cublasdx::col_major>() +
                          cublasdx::Alignment<16, 16, 16>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());

    using value_type = example::uniform_value_type_t<BLAS>;

    // Allocate managed memory for a, b, c, and output
    value_type* inputs;
    value_type* output;

    constexpr auto global_a_size = example::global_memory_size_of<BLAS>::a_size;
    constexpr auto global_b_size = example::global_memory_size_of<BLAS>::b_size;
    constexpr auto global_c_size = example::global_memory_size_of<BLAS>::c_size;
    auto inputs_size       = batches * (global_a_size + global_b_size + global_c_size);

    auto inputs_size_bytes = inputs_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&inputs, inputs_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&output, batches * global_c_size * sizeof(value_type)));

    value_type* a     = inputs;
    value_type* b     = a + (batches * global_a_size);
    value_type* c     = b + (batches * global_b_size);
    value_type  alpha = value_type(1.0);
    value_type  beta  = value_type(2.0);

    // Fill the A, B, C matrices with random values
    auto host_a = example::get_random_data<value_type>(0.1, 1.0, batches * global_a_size);
    auto host_b = example::get_random_data<value_type>(0.1, 1.0, batches * global_b_size);
    auto host_c = example::get_random_data<value_type>(0.1, 1.0, batches * global_c_size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(a, host_a.data(), batches * global_a_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(b, host_b.data(), batches * global_b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(c, host_c.data(), batches * global_c_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    const dim3 block_dim = dim3(block_size, batches);
    const auto shared_memory_size = batches * BLAS::shared_memory_size;

    // Increase max dynamic shared memory for the kernel if needed
    CUDA_CHECK_AND_EXIT(
        hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_kernel<BLAS>), hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory_size));

    // Execute kernel
    gemm_kernel<BLAS><<<1, block_dim, shared_memory_size>>>(a, b, c, alpha, beta, output);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy results of the 1st batch back to host
    std::vector<value_type> host_output1(global_c_size);
    CUDA_CHECK_AND_EXIT(
        hipMemcpy(host_output1.data(), output, global_c_size * sizeof(value_type), hipMemcpyDeviceToHost));
    // Copy results of the 2nd batch back to host
    std::vector<value_type> host_output2(global_c_size);
    CUDA_CHECK_AND_EXIT(
        hipMemcpy(host_output2.data(), output + global_c_size, global_c_size * sizeof(value_type), hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Free device memory
    CUDA_CHECK_AND_EXIT(hipFree(inputs));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    // Calculate reference for the 1st batch
    decltype(host_a) host_a1(host_a.begin(), host_a.begin() + global_a_size);
    decltype(host_b) host_b1(host_b.begin(), host_b.begin() + global_b_size);
    decltype(host_c) host_c1(host_c.begin(), host_c.begin() + global_c_size);
    auto reference_host_output1 = example::reference_gemm<BLAS>(alpha, host_a1, host_b1, beta, host_c1);
    // Calculate reference for the 2nd batch
    decltype(host_a) host_a2(host_a.begin() + global_a_size, host_a.begin() + 2 * global_a_size);
    decltype(host_b) host_b2(host_b.begin() + global_b_size, host_b.begin() + 2 * global_b_size);
    decltype(host_c) host_c2(host_c.begin() + global_c_size, host_c.begin() + 2 * global_c_size);
    auto reference_host_output2 = example::reference_gemm<BLAS>(alpha, host_a2, host_b2, beta, host_c2);

    // Check against reference
    if (example::check(host_output1, reference_host_output1) && example::check(host_output2, reference_host_output2)) {
        std::cout << "Success" << std::endl;
        return 0;
    }
    std::cout << "Failure" << std::endl;
    return 1;
}

template<unsigned int Arch>
struct simple_gemm_functor {
    int operator()() { return simple_gemm<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_gemm_functor>();
}
