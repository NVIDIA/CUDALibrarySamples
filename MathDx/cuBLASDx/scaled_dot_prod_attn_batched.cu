#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <type_traits>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "block_io.hpp"
#include "reduce.hpp"

template <int M, int LD, class ValueType> __device__ __forceinline__
void scale_rows(ValueType *data, int size, ValueType *scale) {
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        int r = i % LD;
        if (r < M) {
            data[i] /= scale[r];
        }
    }
}

template<class BLAS1, class BLAS2, class ValueType = typename BLAS1::value_type>
__launch_bounds__(BLAS1::max_threads_per_block) __global__
void scaled_dot_product_attention_batched(const ValueType* query,
                                          const ValueType* key,
                                          const ValueType* value,
                                          const ValueType* mask,
                                          ValueType*       output) {
    using value_type = ValueType;
    constexpr unsigned int block_size = BLAS1::block_dim.x * BLAS1::block_dim.y * BLAS1::block_dim.z;

    extern __shared__ __align__(16) char smem[];

    static_assert(std::is_same_v<value_type, example::value_type_t<BLAS2>>, "BLAS1 and BLAS2 must have the same type and precision");
    static_assert((BLAS1::c_dim == BLAS2::a_dim), "The dimensions of the C matrix in BLAS1 must be the same as the dimensions of the A matrix in BLAS2");

    // Each block processes one sample from the batch. The inputs and output must be offset to point to the data for the sample.
    query  += blockIdx.x * BLAS1::a_size;
    key    += blockIdx.x * BLAS1::b_size;
    mask   += blockIdx.x * BLAS1::c_size;
    value  += blockIdx.x * BLAS2::b_size;
    output += blockIdx.x * BLAS2::c_size;

    // Matrix C is the first in shared memory, because it's reused in the reduction as well as the 2nd matrix multiplication.
    value_type* smem_c = reinterpret_cast<value_type*>(smem);
    value_type* smem_a = reinterpret_cast<value_type*>(smem) + BLAS1::c_size;
    value_type* smem_b = reinterpret_cast<value_type*>(smem) + BLAS1::c_size + BLAS1::a_size;

    example::io<BLAS1>::a_fast_load<block_size>(smem_a, query);
    example::io<BLAS1>::b_fast_load<block_size>(smem_b, key);
    example::io<BLAS1>::c_fast_load<block_size>(smem_c, mask);
    __syncthreads();

    using cublasdx::size_of;

    // First matrix multiplication C := query @ key.T / sqrt(query.shape[-1]) + mask
    ValueType alpha = rsqrt(ValueType(size_of<BLAS1>::k));  // This can also be precomputed and provided as an argument.
    BLAS1().execute(alpha, smem_a, smem_b, ValueType(1.), smem_c);
    __syncthreads();

    // Compute softmax(C) using the following steps: row-wise reduction, transformation, row-wise reduction, and row scaling.

    constexpr auto ldc = cublasdx::leading_dimension_of<BLAS1>::c;
    value_type* smem_e = smem_c + BLAS1::c_size;
    value_type* smem_w = smem_e + ldc;

    // Find row maximum.
    example::reducers::maximum<ValueType> reducer_max;
    example::reduce_row<size_of<BLAS1>::m, size_of<BLAS1>::n, ldc>(smem_c, reducer_max, smem_w, smem_e);
    __syncthreads();

    // Transform C = exp(C) using row maximum as exponent offset for numerical stability.
    auto transformer = [smem_e, m = size_of<BLAS1>::m, ld=ldc](int i, ValueType v) {
        int r = i % ld;
        return r < m ? example::exp(v - smem_e[r]) : v;
    };
    example::transform(smem_c, BLAS1::c_size, transformer);
    __syncthreads();

    // Calculate E = RowSum(exp(C)).
    example::reducers::addition<ValueType> reducer_add;
    example::reduce_row<size_of<BLAS1>::m, size_of<BLAS1>::n, ldc>(smem_c, reducer_add, smem_w, smem_e);
    __syncthreads();

    // Scale rows to get C = softmax(C): C = C / E[:,None]
    scale_rows<size_of<BLAS1>::m, ldc>(smem_c, BLAS1::c_size, smem_e);
    __syncthreads();

    static_assert((BLAS1::c_size == BLAS2::a_size), "The size of C in BLAS1 must be equal to the size of A in BLAS2");
    value_type* smem_f = smem_c + BLAS2::a_size;
    value_type* smem_g = smem_f + BLAS2::b_size;

    example::io<BLAS2>::b_fast_load<block_size>(smem_f, value);
    __syncthreads();

    // Second matrix multiplication G := C @ value, where C = softmax(query @ key.T / sqrt(query.shape[-1]) + mask)
    BLAS2().execute(ValueType(1.), smem_c, smem_f, ValueType(0.), smem_g);
    __syncthreads();

    example::io<BLAS2>::c_fast_store<block_size>(output, smem_g);
}

template<class BLAS1, class BLAS2, class ValueType = typename BLAS1::value_type>
double measure_cublasdx(unsigned int kernel_warm_up_repeats,
                        unsigned int kernel_repeats,
                        unsigned int batch_size,
                        const ValueType* query,
                        const ValueType* key,
                        const ValueType* value,
                        const ValueType* mask,
                        ValueType*       output,
                        hipStream_t     stream) {

    // Increase max dynamic shared memory for the kernel if needed.
    // The memory required is the maximum of the memory required for the row reduction and the two matrix multiplications
    // C := Q @ K.T / sqrt(Q.shape[-1]) + M and R := softmax(C) @ V.
    const size_t redn_smem_size = (BLAS1::c_size + cublasdx::leading_dimension_of<BLAS1>::c + BLAS1::block_dim.x) * sizeof(ValueType);
    const auto shared_memory = std::max<size_t>(std::max<size_t>(BLAS1::shared_memory_size, redn_smem_size), BLAS2::shared_memory_size);
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(scaled_dot_product_attention_batched<BLAS1), BLAS2>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));

    // Execute kernel. Each sample in the the batch is executed in a different CUDA block.
    double time = example::measure::execution(
        [&](hipStream_t stream) {
            scaled_dot_product_attention_batched<BLAS1, BLAS2><<<batch_size, BLAS1::block_dim, shared_memory, stream>>>(query, key, value, mask, output);
        },
        kernel_warm_up_repeats, kernel_repeats, stream);

    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    return time;
}

// This example illustrates how to compute multi-head attention using batched "scaled dot product attention"
// calculations. The entire computation is implemented in a single fused kernel using cuBLASDx.
//
// The inputs are three matrices "query", "key", and "value" along with a mask, with the following dimensions:
//     query(N, S, E),
//     key(N, S, E),
//     value(N, S, E),
//     mask(N, S, S),
// where N is the batch size, S is the maximum sequence length and E is the embedding size per attention head.
// The product of E with the number of heads is the embedding dimension chosen for the words in the vocabulary.
// Note that the use of batch here encompasses the attention heads as well as the number of samples (it's the
// product of the number of attention heads and the number of samples).
//
// The result r of scaled dot product attention is a matrix of size (N, S, E):
//     r = softmax(query @ key.T / sqrt(query.shape[-1]) + mask) @ value
// where @ represents batched matrix multiplication. This operation essentially reweights the values matrix
// according to the (multiple, one for each head) attention for each sample in the batch.
//
// Notes:
// * The sizes of query, key, and value for N=1  must be such that the data should not only fit within shared
//   memory but also small enough such that a single block is the _optimal_ choice. This is because cuBLASDx is
//   limited to using one block whereas libraries like cuBLAS can use a larger number of blocks.
//
// * Each item in the batch will be processed using a single CUDA block and the items can be processed in
//   parallel (subject to the number of SMs available on the selected device).
//
template<unsigned int Arch>
int scaled_dot_product_attention_batched_performance() {
    // Define the maximum sequence length (number of tokens) S.
    constexpr unsigned int S = 9;

    // Define the embedding dimension E.
    constexpr unsigned int E = 64;

    // Define the batch size N.
    constexpr unsigned int N = 64;

    // Parameters m1, n1, k1 define the dimensions of matrices "query", "key", and "mask".
    constexpr unsigned int m1          = S;
    constexpr unsigned int n1          = S;
    constexpr unsigned int k1          = E;

    // Parameters m2, n2, k2 define the dimensions of matrices "query @ key.T" and "value".
    // Note: (m1, n1) and (m2, k2) must be equal as describe the same matrix.
    constexpr unsigned int m2          = m1;
    constexpr unsigned int n2          = E;
    constexpr unsigned int k2          = n1;

    // Use the same block size for both GEMM operations as well as the reduction, which
    // simplifies the example.
    constexpr unsigned int block_size = 256;

    // Choose the precision (__half, float, double). The data type can only be real.
    using precision = float;

    using BASE        = decltype(cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::Precision<precision>() +
                          cublasdx::Type<cublasdx::type::real>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());
    using BLAS1_      = decltype(BASE() +
                          cublasdx::Size<m1, n1, k1>() +
                          cublasdx::TransposeMode<cublasdx::transpose_mode::non_transposed, cublasdx::transpose_mode::transposed>());
    // Use cuBLASDx suggested leading dimensions. Since we perform two matrix multiplications, the C matrix from the first has to be
    // consistent (dimensions and padding) with the A matrix of the second. Therefore create the second matrix multiplication
    // descriptor first and use its lda to set the first matrix multiplication's ldc.
    using LD1         = cublasdx::suggested_leading_dimension_of<BLAS1_, Arch>;
    using BLAS2_      = decltype(BASE() +
                          cublasdx::Size<m2, n2, k2>() +
                          cublasdx::TransposeMode<cublasdx::transpose_mode::non_transposed, cublasdx::transpose_mode::non_transposed>());
    using LD2         = cublasdx::suggested_leading_dimension_of<BLAS2_, Arch>;
    using BLAS2       = decltype(BLAS2_() + typename LD2::type());
    using BLAS1       = decltype(BLAS1_() + cublasdx::LeadingDimension<LD1::lda, LD1::ldb, LD2::lda>());
    #if CUBLASDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using value_type = example::value_type_t<BLAS1>;
    #else
    using value_type = typename BLAS1::value_type;
    #endif

    // Allocate device memory for query, key, value, mask, and output.
    value_type* inputs;
    value_type* output;
    auto inputs_size       = N * (BLAS1::a_size + BLAS1::b_size + BLAS1::c_size + BLAS2::b_size);
    auto inputs_size_bytes = inputs_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMalloc(&inputs, inputs_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMalloc(&output, N * BLAS2::c_size * sizeof(value_type)));

    value_type* query     = inputs;                      // A matrix for BLAS1
    value_type* key       = query + N * (BLAS1::a_size); // B matrix for BLAS1
    value_type* mask      = key   + N * (BLAS1::b_size); // C matrix for BLAS1
    value_type* value     = mask  + N * (BLAS1::c_size); // B matrix for BLAS2

    // Fill the query, key, and value matrices with random values.
    auto host_query = example::get_random_data<value_type>(0.1, 0.5, N * BLAS1::a_size);
    auto host_key   = example::get_random_data<value_type>(0.1, 0.5, N * BLAS1::b_size);
    auto host_value = example::get_random_data<value_type>(0.1, 0.5, N * BLAS2::b_size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(query, host_query.data(), N * BLAS1::a_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(key, host_key.data(), N * BLAS1::b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(value, host_value.data(), N * BLAS2::b_size * sizeof(value_type), hipMemcpyHostToDevice));
    // Set the mask to 0. (no mask).
    CUDA_CHECK_AND_EXIT(hipMemset(mask, 0, N * BLAS1::c_size * sizeof(value_type)));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    const unsigned int kernel_repeats = 100;
    const unsigned int kernel_warm_up_repeats = 1;
    hipStream_t stream;
    CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream))

    // Measure cuBLASDx performance.
    double time_cublasdx =
        measure_cublasdx<BLAS1, BLAS2>(kernel_warm_up_repeats, kernel_repeats, N, query, key, value, mask, output, stream);

    // Write performance data.
    using cublasdx::size_of;
    std::cout << "m1, n1, k1: " << size_of<BLAS1>::m << ", " << size_of<BLAS1>::n << ", " << size_of<BLAS1>::k
              << std::endl;
    const auto [lda1, ldb1, ldc1] = cublasdx::leading_dimension_of_v<BLAS1>;
    std::cout << "Leading dimensions (lda1, ldb1, ldc1): " << lda1 << ", " << ldb1 << ", " << ldc1 << std::endl;
    std::cout << "m2, n2, k2: " << size_of<BLAS2>::m << ", " << size_of<BLAS2>::n << ", " << size_of<BLAS2>::k
              << std::endl;
    const auto [lda2, ldb2, ldc2] = cublasdx::leading_dimension_of_v<BLAS2>;
    std::cout << "Leading dimensions (lda2, ldb2, ldc2): " << lda2 << ", " << ldb2 << ", " << ldc2 << std::endl;
    std::cout << "Type: " << example::type_string<value_type>() << std::endl;
    std::cout << "Precision: " << example::precision_string<value_type>() << std::endl;
    std::cout << "Batch Size: " << N << std::endl;

    std::cout << "\ncuBLASDx (fused kernel for batched scaled dot product attention calculation)\n";
    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Avg time [ms]  = " << time_cublasdx / kernel_repeats << "\n";

    // Free resources.
    CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));
    CUDA_CHECK_AND_EXIT(hipFree(inputs));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    return 0;
}

template<unsigned int Arch>
struct scaled_dot_product_attention_batched_performance_functor {
    int operator()() {
        return scaled_dot_product_attention_batched_performance<Arch>();
    }
};

int main(int, char**) {
    return example::sm_runner<scaled_dot_product_attention_batched_performance_functor>();
}
