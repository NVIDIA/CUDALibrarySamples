#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>

#include <cublasdx.hpp>
#include <cufftdx.hpp>

#include "common.hpp"
#include "block_io.hpp"
#include "reference.hpp"

// Batch size (number of signals to process)
constexpr unsigned int batch_size = 2;

template <class T, class U>
void copy(T* source, U* destination, unsigned int size) {
    static_assert(example::is_complex<T>() && example::is_complex<U>(), "Expect complex types.");
    for (unsigned int i = 0; i < size; ++i) {
        destination[i].x  = source[i].real();
        destination[i].y  = source[i].imag();
    }
}

template<class FFT, class BLAS, class ValueType = cublasdx::complex<float>>
void reference(const ValueType* a,
               ValueType* b,
               ValueType* c,
               const ValueType  alpha,
               const ValueType  beta,
               ValueType*       output,
               hipStream_t     stream) {
    constexpr auto m = cublasdx::size_of<BLAS>::m;
    constexpr auto n = cublasdx::size_of<BLAS>::n;
    constexpr auto k = cublasdx::size_of<BLAS>::k;

    // Prepare cuFFT
    const unsigned int fft_size = cublasdx::size_of<BLAS>::k;

    hipfftHandle plan;
    CUFFT_CHECK_AND_EXIT(hipfftPlan1d(&plan, fft_size, HIPFFT_C2C, batch_size));
    CUFFT_CHECK_AND_EXIT(hipfftSetStream(plan, stream));

    // Run cuFFT
    CUFFT_CHECK_AND_EXIT(
        hipfftExecC2C(plan, reinterpret_cast<hipfftComplex*>(b), reinterpret_cast<hipfftComplex*>(b), HIPFFT_FORWARD));

    // Prepare cuBLAS
    hipblasHandle_t handle;
    CUBLAS_CHECK_AND_EXIT(hipblasCreate(&handle));
    CUBLAS_CHECK_AND_EXIT(hipblasSetStream(handle, stream));
    constexpr bool is_a_transposed = (cublasdx::arrangement_of<BLAS>::a == cublasdx::row_major);
    constexpr bool is_b_transposed = (cublasdx::arrangement_of<BLAS>::b == cublasdx::row_major);
    const auto a_transpose = example::detail::get_cublas_transpose_mode(cublasdx::arrangement_of<BLAS>::a);
    const auto b_transpose = example::detail::get_cublas_transpose_mode(cublasdx::arrangement_of<BLAS>::b);
    static_assert(cublasdx::arrangement_of<BLAS>::c == cublasdx::arrangement::col_major, "Only column-major C matrix supported");

    // Run cuBLAS
    copy(c, output, m * n);
    CUBLAS_CHECK_AND_EXIT(hipblasCgemm(handle,
                                      a_transpose,
                                      b_transpose,
                                      m,
                                      n,
                                      k,
                                      reinterpret_cast<const hipFloatComplex*>(&alpha),
                                      reinterpret_cast<const hipFloatComplex*>(a),
                                      is_a_transposed ? k : m,
                                      reinterpret_cast<const hipFloatComplex*>(b),
                                      is_b_transposed ? n : k,
                                      reinterpret_cast<const hipFloatComplex*>(&beta),
                                      reinterpret_cast<hipFloatComplex*>(output),
                                      m));

    CUBLAS_CHECK_AND_EXIT(hipblasDestroy(handle));
    CUFFT_CHECK_AND_EXIT(hipfftDestroy(plan));
}

template<class FFT, class BLAS, class ValueType = typename example::uniform_value_type_t<BLAS>>
__launch_bounds__(FFT::max_threads_per_block) __global__ void gemm_fft_fp16_kernel(const ValueType* a,
                                                                                   const ValueType* b,
                                                                                   const ValueType* c,
                                                                                   const ValueType  alpha,
                                                                                   const ValueType  beta,
                                                                                   ValueType*       output) {
    #if CUBLASDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using blas_complex_type = example::uniform_value_type_t<BLAS>;
    using fft_complex_type = example::value_type_t<FFT>;
    #else
    using blas_complex_type = example::uniform_value_type_t<BLAS>;
    using fft_complex_type  = typename FFT::value_type;
    #endif

    using complex_type = blas_complex_type;
    using value_type = ValueType;
    constexpr unsigned int block_size = BLAS::block_dim.x * BLAS::block_dim.y * BLAS::block_dim.z;

    extern __shared__ complex_type smem[];

    auto [smem_a, smem_b, smem_c] = BLAS::slice_shared_memory(reinterpret_cast<char*>(smem));

    // Compute FFT(B, axis=0).
    fft_complex_type thread_data[FFT::storage_size];

    // Load data from global memory into registers for FFT, converting to RRII form from RIRI.
    example::load<FFT::elements_per_thread, cufftdx::size_of<FFT>::value, FFT::stride>(b, thread_data);
    __syncthreads();

    // Execute batched FFT on registers.
    FFT().execute(thread_data, reinterpret_cast<fft_complex_type *>(smem));
    __syncthreads();

    // Store register data into smem_b, converting back to RIRI form from RRII.
    example::store<FFT::elements_per_thread, cufftdx::size_of<FFT>::value, FFT::stride>(thread_data, smem_b);

    // Compute C := alpha * A @ FFT(B, axis=0) + beta * C.

    // Load A and C from global to shared memory, B is already in shared memory after the FFT
    auto a_global_tensor = cublasdx::make_tensor(a, BLAS::get_layout_gmem_a());
    auto c_global_tensor = cublasdx::make_tensor(c, BLAS::get_layout_gmem_c());

    auto a_shared_tensor = cublasdx::make_tensor(smem_a, BLAS::get_layout_smem_a());
    auto b_shared_tensor = cublasdx::make_tensor(smem_b, BLAS::get_layout_smem_b());
    auto c_shared_tensor = cublasdx::make_tensor(smem_c, BLAS::get_layout_smem_c());

    using alignment = cublasdx::alignment_of<BLAS>;
    cublasdx::copy<BLAS, alignment::a>(a_global_tensor, a_shared_tensor);
    cublasdx::copy<BLAS, alignment::c>(c_global_tensor, c_shared_tensor);
    cublasdx::copy_wait();

    // Execute GEMM: C = alpha * A @ FFT(B, axis=0) + beta * C.
    BLAS().execute(alpha, a_shared_tensor, b_shared_tensor, beta, c_shared_tensor);
    __syncthreads();

    // Store the results.
    auto out_global_tensor = cublasdx::make_tensor(output, BLAS::get_layout_gmem_c());
    cublasdx::copy<BLAS, alignment::c>(c_shared_tensor, out_global_tensor);
}

// In this example cuBLASDx and cuFFTDx libraries are combined to perform GEMM and FFT in one pipeline for complex half-precision
// data. A key goal is to illustrate the differences in how cuFFTDx and cuBLASDx handle complex half-precision.
//
// The kernel computes the following operations:
//     1) FFT(B, axis=0)
//     2) C = alpha * A @ B + beta * C
//
// This sequence of operations can represent reweighting of signals in the frequency domain, for example.
//
// We first load B from global memory into registers, interleaving data from the two batches in RRII format. The FFT operation is
// implicitly batched in this case, with the two batches being processed in one execution. The results are then stored into shared
// memory in preparation for the GEMM, with the data being converted back to RIRI format. Next the matrices A and C are loaded
// into shared memory, and the GEMM (which is in fact a batched GEMV) is executed. The results are stored back to global memory.
//
// Important notes:
// * Results are verified against cuFFT and cuBLAS.
// * This example is written only for a batch size of 2 and for complex half-precision type.
// * The type used by cuFFTDx is complex<__half2> while the type used by cuBLASDx is complex<half>.
// * Shared memory required by the kernel is the max of the amount required by FFT and GEMM.
template<unsigned int Arch>
int gemm_fft_fp16() {
    using precision_type = __half;
    constexpr unsigned int m = 64;
    constexpr unsigned int n = batch_size;
    constexpr unsigned int k = m;

    static_assert(batch_size == 2, "This example only supports a batch size of 2.");
    static_assert(std::is_same_v<precision_type, __half>, "This example only supports half-precision.");

    using FFT  = decltype(cufftdx::Block() + cufftdx::Size<k>() + cufftdx::Type<cufftdx::fft_type::c2c>() +
                          cufftdx::Direction<cufftdx::fft_direction::forward>() + cufftdx::Precision<precision_type>() +
                          cufftdx::ElementsPerThread<2>() + cufftdx::FFTsPerBlock<batch_size>() + cufftdx::SM<Arch>());

    using BLAS = decltype(cublasdx::Size<m, n, k>() +
                          cublasdx::Precision<precision_type>() +
                          cublasdx::Type<cublasdx::type::complex>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::Arrangement<cublasdx::col_major, cublasdx::col_major>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<FFT::block_dim.x>() +
                          cublasdx::SM<Arch>());

    #if CUBLASDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using fft_complex_type = example::value_type_t<FFT>;
    using blas_complex_type = example::uniform_value_type_t<BLAS>;
    #else
    using fft_complex_type = typename FFT::value_type;
    using blas_complex_type = example::uniform_value_type_t<BLAS>;
    #endif
    using complex_type = blas_complex_type;

    // Check that FFT matches GEMM dimensions.
    static_assert(cufftdx::size_of<FFT>::value == cublasdx::size_of<BLAS>::k,
                  "FFT must have the same size as the GEMM k dimension");
    // Checking that block dims match
    static_assert((FFT::block_dim.x == BLAS::block_dim.x) && (FFT::block_dim.y == BLAS::block_dim.y) &&
                  (FFT::block_dim.z == BLAS::block_dim.z),
                  "FFT must require the same CUDA block dimenions as GEMM");

    // Allocate managed memory.
    complex_type* buffer;
    complex_type* a;
    complex_type* b;
    complex_type* c;
    complex_type* output;

    constexpr auto global_a_size = example::global_memory_size_of<BLAS>::a_size;
    constexpr auto global_b_size = example::global_memory_size_of<BLAS>::b_size;
    constexpr auto global_c_size = example::global_memory_size_of<BLAS>::c_size;

    auto          size = (global_a_size + // a
                          global_b_size + // b
                          global_c_size + // c
                          global_c_size   // output
                         );
    auto          size_bytes = size * sizeof(complex_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&buffer, size_bytes));
    a                = buffer;
    b                = a + global_a_size;
    c                = b + global_b_size;
    output           = c + global_c_size;

    complex_type alpha = {float(1), float(0)};
    complex_type beta  = {float(0), float(0)};

    // Fill the a, b, c matrices.
    {
        float base = cublasdx::size_of<BLAS>::m * cublasdx::size_of<BLAS>::k;
        for (size_t i = 0; i < global_a_size; i++) {
            a[i] = complex_type {float(i) / base, float(i) / base};
        }
        for (size_t i = 0; i < global_b_size; i++) {
            b[i] = complex_type {float(i) / base, float(i) / base};
        }
        for (size_t i = 0; i < global_c_size; i++) {
            c[i] = complex_type {float(1) / base, float(1) / base};
        }
    }

    // Create stream
    hipStream_t stream;
    CUDA_CHECK_AND_EXIT(hipStreamCreate(&stream));

    // Prefetch memory to device
    {
        int device;
        CUDA_CHECK_AND_EXIT(hipGetDevice(&device));
        CUDA_CHECK_AND_EXIT(hipMemPrefetchAsync(buffer, size_bytes, device, stream));
        CUDA_CHECK_AND_EXIT(hipStreamSynchronize(stream));
        CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());
    }

    // Get max shared memory required by FFT and GEMM
    constexpr auto shared_memory_size = std::max({FFT::shared_memory_size, BLAS::shared_memory_size});
    // Increase max shared memory if needed
    CUDA_CHECK_AND_EXIT(hipFuncSetAttribute(reinterpret_cast<const void*>(
        gemm_fft_fp16_kernel<FFT), BLAS>,
        hipFuncAttributeMaxDynamicSharedMemorySize,
        shared_memory_size));

    // Invokes cuBLASDx+cuFFTDx kernel with FFT::block_dim threads in CUDA block
    gemm_fft_fp16_kernel<FFT, BLAS><<<1, FFT::block_dim, shared_memory_size, stream>>>(a, b, c, alpha, beta, output);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Compute reference results using cuBLAS and cuFFT (with float32 precision).
    using reference_complex_type = cublasdx::complex<float>;
    auto  reference_size_bytes   = size * sizeof(reference_complex_type);

    reference_complex_type* reference_buffer;
    reference_complex_type* reference_a;
    reference_complex_type* reference_b;
    reference_complex_type* reference_c;
    reference_complex_type* reference_output;

    CUDA_CHECK_AND_EXIT(hipMallocManaged(&reference_buffer, reference_size_bytes));
    reference_a                = reference_buffer;
    reference_b                = reference_a + global_a_size;
    reference_c                = reference_b + global_b_size;
    reference_output           = reference_c + global_c_size;

    reference_complex_type reference_alpha{alpha.real(), alpha.imag()};
    reference_complex_type reference_beta{beta.real(), beta.imag()};

    // Copy a, b, and c to the corresponding reference data buffers.
    copy(a, reference_a, global_a_size);
    copy(b, reference_b, global_b_size);
    copy(c, reference_c, global_c_size);

    // cuBLAS+cuFFT
    reference<FFT, BLAS>(reference_a, reference_b, reference_c, reference_alpha, reference_beta, reference_output, stream);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Print results.
    std::cout << std::fixed << std::showpos << std::setprecision(4);
    std::cout << "[cuBLASDx + cuFFTDx] (float16):\n";
    std::cout << "     Batch 1             Batch 2   \n";
    for (size_t i = 0; i <  m; i++) {
        for (size_t j = 0; j < n; j++) {
            auto index = i + j * m;
            std::cout << "[" << float(output[index].real()) << ", " << float(output[index].imag()) << "]  ";
        }
        std::cout << "\n";
    }
    std::cout << "[cuBLAS + cuFFT] (float32):\n";
    std::cout << "     Batch 1             Batch 2   \n";
    for (size_t i = 0; i <  m; i++) {
        for (size_t j = 0; j < n; j++) {
            auto index = i + j * m;
            std::cout << "[" << float(reference_output[index].x) << ", " << float(reference_output[index].y) << "]  ";
        }
        std::cout << "\n";
    }

    CUDA_CHECK_AND_EXIT(hipStreamDestroy(stream));
    CUDA_CHECK_AND_EXIT(hipFree(buffer));
    CUDA_CHECK_AND_EXIT(hipFree(reference_buffer));

    std::cout << "Success" << std::endl;
    return 0;
}

template<unsigned int Arch>
struct gemm_fft_fp16_functor {
    int operator()() { return gemm_fft_fp16<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<gemm_fft_fp16_functor>();
}
