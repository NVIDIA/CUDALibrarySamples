#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "block_io.hpp"
#include "reference.hpp"

template<class BLAS, class AValueType = typename BLAS::a_value_type,
                     class BValueType = typename BLAS::b_value_type,
                     class CValueType = typename BLAS::c_value_type>
__launch_bounds__(BLAS::max_threads_per_block) //
    __global__                                 //
    void gemm_kernel(const AValueType* a,
                     const BValueType* b,
                     const CValueType* c,
                     const CValueType  alpha,
                     const CValueType  beta,
                     CValueType*       output) {
    extern __shared__ __align__(16) char smem[];

    auto a_global_tensor = cublasdx::make_tensor(a, BLAS::get_layout_gmem_a());
    auto b_global_tensor = cublasdx::make_tensor(b, BLAS::get_layout_gmem_b());
    auto c_global_tensor = cublasdx::make_tensor(c, BLAS::get_layout_gmem_c());

    using a_engine = typename decltype(cublasdx::make_tensor(std::declval<decltype(a)>(), BLAS::get_layout_gmem_a()))::engine_type;
    static_assert(std::is_same_v<a_engine, typename decltype(a_global_tensor)::engine_type>, "");


    auto [smem_a, smem_b] = cublasdx::slice_shared_memory_ab<BLAS>(smem);
    auto a_shared_tensor = cublasdx::make_tensor(smem_a, BLAS::get_layout_smem_a());
    auto b_shared_tensor = cublasdx::make_tensor(smem_b, BLAS::get_layout_smem_b());


    using alignment = cublasdx::alignment_of<BLAS>;
    cublasdx::copy<BLAS, alignment::a>(a_global_tensor, a_shared_tensor);
    cublasdx::copy<BLAS, alignment::b>(b_global_tensor, b_shared_tensor);
    cublasdx::copy_wait();

    auto [c_frag, partitioner] = BLAS().execute(a_shared_tensor, b_shared_tensor);

    auto d_frag = partitioner.make_accumulator_fragment();
    cublasdx::copy_fragment<alignment::c>(c_global_tensor, d_frag, partitioner);
    cublasdx::axpby(alpha, c_frag, beta, d_frag);

    auto out_global_tensor = cublasdx::make_tensor(output, BLAS::get_layout_gmem_c());
    cublasdx::copy_fragment<alignment::c>(d_frag, out_global_tensor, partitioner);
}

// This is an example of int8 / int8 / int32 general matrix-matrix multiplication (GEMM) performed
// in a single CUDA block with use of Tensor Cores:
//
//              C = alpha * A * B + beta * C
//
// * A, B, and C are matrices containing:
//    A --> int8_t
//    B --> int8_t
//    C --> int32_t
// * alpha and beta are real int32_t values.
//
// Input data is generated on host using random number generators, and later copied to
// the global memory. Next, kernel with GEMM is executed, and then the matrix C (the result)
// is copied back to host memory. The results are verified against cuBLAS.
//
// In this example the number of threads participating in the GEMM operation is imposed by providing
// BlockDim operator in definition of the GEMM. If BlockDim operator is not used, cuBLASDx automatically
// selects number of threads. Block dimensions are provided via BLAS::block_dim trait.
template<unsigned int Arch>
int simple_gemm() {
    // Parameters m, n, k define the dimensions of matrices A, B, and C
    constexpr unsigned int m = 16;
    constexpr unsigned int n = 32;
    constexpr unsigned int k = 64;

    // Selected CUDA block size (1D)
    constexpr unsigned int block_size = 256;

    // GEMM definition using cuBLASDx operators:
    // 1. The size, the precision, and the type (real or complex) are set.
    // 2. The BLAS function is selected: MM (matrix multiplication).
    // 3. Block operator informs that GEMM should be performed on CUDA block level.
    // 4. BlockDim operator sets CUDA block dimensions that the kernel will be executed with.
    // 5. Targeted CUDA compute capability is selected with SM operator.
    using BLAS = decltype(cublasdx::Size<m, n, k>() +
                          cublasdx::Precision<int8_t, int8_t, int32_t>() +
                          cublasdx::Type<cublasdx::type::real>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());

    using a_value_type = typename BLAS::a_value_type;
    using b_value_type = typename BLAS::b_value_type;
    using c_value_type = typename BLAS::c_value_type;

    // Allocate managed memory for a, b, c, and output
    a_value_type* input_a;
    b_value_type* input_b;
    c_value_type* input_c;
    c_value_type* output_c;

    constexpr auto global_a_size = example::global_memory_size_of<BLAS>::a_size;
    constexpr auto global_b_size = example::global_memory_size_of<BLAS>::b_size;
    constexpr auto global_c_size = example::global_memory_size_of<BLAS>::c_size;

    CUDA_CHECK_AND_EXIT(hipMallocManaged(&input_a,  global_a_size * sizeof(a_value_type)));
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&input_b,  global_b_size * sizeof(b_value_type)));
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&input_c,  global_c_size * sizeof(c_value_type)));
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&output_c, global_c_size * sizeof(c_value_type)));

    c_value_type  alpha = c_value_type(1.0);
    c_value_type  beta  = c_value_type(2.0);

    // Fill the A, B, C matrices with random values
    auto host_a = example::get_random_data<a_value_type>(-10, 10, global_a_size);
    auto host_b = example::get_random_data<b_value_type>(-10, 10, global_b_size);
    auto host_c = example::get_random_data<c_value_type>(-100, 100, global_c_size);

    CUDA_CHECK_AND_EXIT(hipMemcpy(input_a, host_a.data(), global_a_size * sizeof(a_value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(input_b, host_b.data(), global_b_size * sizeof(b_value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(input_c, host_c.data(), global_c_size * sizeof(c_value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Increase max dynamic shared memory for the kernel if needed
    CUDA_CHECK_AND_EXIT(
        hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_kernel<BLAS>), hipFuncAttributeMaxDynamicSharedMemorySize, cublasdx::get_shared_storage_size<BLAS>()));

    // Execute kernel
    gemm_kernel<BLAS><<<1, BLAS::block_dim, cublasdx::get_shared_storage_size<BLAS>()>>>(input_a, input_b, input_c, alpha, beta, output_c);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy results back to host
    std::vector<c_value_type> host_output(global_c_size);
    CUDA_CHECK_AND_EXIT(
        hipMemcpy(host_output.data(), output_c, global_c_size * sizeof(c_value_type), hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Free device memory
    CUDA_CHECK_AND_EXIT(hipFree(input_a));
    CUDA_CHECK_AND_EXIT(hipFree(input_b));
    CUDA_CHECK_AND_EXIT(hipFree(input_c));
    CUDA_CHECK_AND_EXIT(hipFree(output_c));

    // Calculate reference
    auto reference_host_output = example::reference_gemm<BLAS>(alpha, host_a, host_b, beta, host_c);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());

    // Check against reference
    if (example::check_error<BLAS>(host_output, reference_host_output)) {
        std::cout << "Success" << std::endl;
        return 0;
    }
    std::cout << "Failure" << std::endl;
    return 1;
}

struct simple_gemm_functor {
    template<int Arch>
    int operator()(std::integral_constant<int, Arch>) {
        return simple_gemm<Arch>();
    }
};

int main(int, char**) {
    return example::sm_runner(simple_gemm_functor{});
}
