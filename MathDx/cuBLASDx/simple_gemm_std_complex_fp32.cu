#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <cuda/std/complex>
#if _LIBCUDACXX_CUDA_API_VERSION < 001007000
int main(int, char**) {
    std::cout << "Example disabled, cuBLASDx requires cuda::std::complex from libcu++ 1.7.0 (CTK 11.6) or newer" << std::endl;
    return 0;
}
#else

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "reference.hpp"

template<class BLAS, class ValueType = typename example::uniform_value_type_t<BLAS>>
__launch_bounds__(BLAS::max_threads_per_block) __global__ void gemm_kernel(const ValueType* a,
                                                                           const ValueType* b,
                                                                           const ValueType* c,
                                                                           const ValueType  alpha,
                                                                           const ValueType  beta,
                                                                           ValueType*       output) {
    using value_type = ValueType;
    extern __shared__ __align__(16) char smem[];

    // Note: here we can not use slice_shared_memory since
    // BLAS::a_value_type = cublasdx::complex<precision>, not cuda::std::complex<precision>;
    value_type* smem_a = reinterpret_cast<value_type*>(smem);
    value_type* smem_b = reinterpret_cast<value_type*>(smem) + BLAS::a_size;
    value_type* smem_c = reinterpret_cast<value_type*>(smem) + BLAS::a_size + BLAS::b_size;

    auto a_global_tensor = cublasdx::make_tensor(a, BLAS::get_layout_gmem_a());
    auto b_global_tensor = cublasdx::make_tensor(b, BLAS::get_layout_gmem_b());
    auto c_global_tensor = cublasdx::make_tensor(c, BLAS::get_layout_gmem_c());

    auto a_shared_tensor = cublasdx::make_tensor(smem_a, BLAS::get_layout_smem_a());
    auto b_shared_tensor = cublasdx::make_tensor(smem_b, BLAS::get_layout_smem_b());
    auto c_shared_tensor = cublasdx::make_tensor(smem_c, BLAS::get_layout_smem_c());

    using alignment = cublasdx::alignment_of<BLAS>;
    cublasdx::copy<BLAS, alignment::a>(a_global_tensor, a_shared_tensor);
    cublasdx::copy<BLAS, alignment::b>(b_global_tensor, b_shared_tensor);
    cublasdx::copy<BLAS, alignment::c>(c_global_tensor, c_shared_tensor);
    cublasdx::copy_wait();

    BLAS().execute(alpha, a_shared_tensor, b_shared_tensor, beta, c_shared_tensor);

    __syncthreads();
    auto out_global_tensor = cublasdx::make_tensor(output, BLAS::get_layout_gmem_c());
    cublasdx::copy<BLAS, alignment::c>(c_shared_tensor, out_global_tensor);
}

// This is an example of a complex general matrix-matrix multiplication (GEMM) performed in a single
// CUDA block using cuda::std::complex instead of cublasdx::complex.
//
//              C = alpha * A * B + beta * C
//
// * A, B, and C are matrices containing complex floating-point values of the specified precision.
// * alpha and beta are complex floating-point values also of the specified precision.
//
// Input data is generated on host using random number generators, and later copied to
// the global memory. Next, kernel with GEMM is executed, and then the matrix C (the result)
// is copied back to host memory. The results are verified against cuBLAS.
//
// In this example the number of threads participating in the GEMM operation is automatically
// selected by cuBLASDx. Setting operator BlockDim in the GEMM definition can be used to impose the
// number of threads that the GEMM will be performed with. Block dimensions are provided via
// BLAS::block_dim trait.
template<unsigned int Arch>
int simple_gemm() {
    // Parameters m, n, k define the dimensions of matrices A, B, and C
    constexpr unsigned int m          = 32;
    constexpr unsigned int n          = 32;
    constexpr unsigned int k          = 32;

    // Specify the precision for the complex type.
    using precision = float;

    // Use cuda::std::complex as the complex type.
    using value_type = cuda::std::complex<precision>;

    // If matrix A is column-major (or not transposed in BLAS nomenclature) its logical dimensions are: [m, k] (m rows, k columns)
    // If matrix B is column-major its logical dimensions are: [k, n]
    // If matrix A is row-major (or transposed in BLAS nomenclature) its logical dimensions are: [k, m]
    // If matrix B is row-major its logical dimensions are: [n, k]
    // The dimensions of matrix C are: [m, n]
    constexpr auto arrangement_a = cublasdx::col_major;
    constexpr auto arrangement_b = cublasdx::row_major;

    // GEMM definition using cuBLASDx operators:
    // 1. The size, the precision, and the type (real or complex) are set.
    // 2. The BLAS function is selected: MM (matrix multiplication).
    // 3. Block operator informs that GEMM should be performed on CUDA block level.
    // 4. Targeted CUDA compute capability is selected with SM operator.
    using BLAS = decltype(cublasdx::Size<m, n, k>() +
                          cublasdx::Precision<precision>() +
                          cublasdx::Type<cublasdx::type::complex>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::Arrangement<arrangement_a, arrangement_b>() +
                          cublasdx::Block() +
                          cublasdx::SM<Arch>());

    // Allocate managed memory for a, b, c, and output
    value_type* inputs;
    value_type* output;

    constexpr auto global_a_size = example::global_memory_size_of<BLAS>::a_size;
    constexpr auto global_b_size = example::global_memory_size_of<BLAS>::b_size;
    constexpr auto global_c_size = example::global_memory_size_of<BLAS>::c_size;

    auto inputs_size       = global_a_size + global_b_size + global_c_size;
    auto inputs_size_bytes = inputs_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&inputs, inputs_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&output, global_c_size * sizeof(value_type)));

    value_type* a     = inputs;
    value_type* b     = a + (global_a_size);
    value_type* c     = b + (global_b_size);
    value_type  alpha = value_type(1.0, 1.0);
    value_type  beta  = value_type(2.0, 2.0);

    // Fill the A, B, C matrices with random values
    auto host_a = example::get_random_data<value_type>(0.1, 1.0, global_a_size);
    auto host_b = example::get_random_data<value_type>(0.1, 1.0, global_b_size);
    auto host_c = example::get_random_data<value_type>(0.1, 1.0, global_c_size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(a, host_a.data(), global_a_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(b, host_b.data(), global_b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(c, host_c.data(), global_c_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Increase max dynamic shared memory for the kernel if needed
    CUDA_CHECK_AND_EXIT(
        hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_kernel<BLAS), value_type>, hipFuncAttributeMaxDynamicSharedMemorySize, BLAS::shared_memory_size));

    // Execute kernel
    gemm_kernel<BLAS, value_type><<<1, BLAS::block_dim, BLAS::shared_memory_size>>>(a, b, c, alpha, beta, output);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy results back to host
    std::vector<value_type> host_output(global_c_size);
    CUDA_CHECK_AND_EXIT(
        hipMemcpy(host_output.data(), output, global_c_size * sizeof(value_type), hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Free device memory
    CUDA_CHECK_AND_EXIT(hipFree(inputs));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    // Calculate reference
    auto reference_host_output = example::reference_gemm<BLAS>(alpha, host_a, host_b, beta, host_c);

    // Check against reference
    if (example::check(host_output, reference_host_output)) {
        std::cout << "Success" << std::endl;
        return 0;
    }
    std::cout << "Failure" << std::endl;
    return 1;
}

template<unsigned int Arch>
struct simple_gemm_functor {
    int operator()() { return simple_gemm<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_gemm_functor>();
}
#endif
