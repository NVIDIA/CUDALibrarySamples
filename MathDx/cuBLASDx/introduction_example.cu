#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "reference.hpp"

template<class GEMM>
__global__ void gemm_kernel(const typename GEMM::c_value_type  alpha,
                            const typename GEMM::a_value_type* a,
                            const typename GEMM::b_value_type* b,
                            const typename GEMM::c_value_type  beta,
                            typename GEMM::c_value_type* c) {
    extern __shared__ __align__(16) char smem[];

    // Make global memory tensor
    auto a_global_tensor = cublasdx::make_tensor(a, GEMM::get_layout_gmem_a());
    auto b_global_tensor = cublasdx::make_tensor(b, GEMM::get_layout_gmem_b());
    auto c_global_tensor = cublasdx::make_tensor(c, GEMM::get_layout_gmem_c());

    // Make shared memory tensor
    auto [smem_a, smem_b, smem_c] = GEMM::slice_shared_memory(smem);
    auto a_shared_tensor = cublasdx::make_tensor(smem_a, GEMM::get_layout_smem_a());
    auto b_shared_tensor = cublasdx::make_tensor(smem_b, GEMM::get_layout_smem_b());
    auto c_shared_tensor = cublasdx::make_tensor(smem_c, GEMM::get_layout_smem_c());

    // Load data from global memory tensor to shared memory tensor
    using alignment = cublasdx::alignment_of<GEMM>;
    cublasdx::copy<GEMM, alignment::a>(a_global_tensor, a_shared_tensor);
    cublasdx::copy<GEMM, alignment::b>(b_global_tensor, b_shared_tensor);
    cublasdx::copy<GEMM, alignment::c>(c_global_tensor, c_shared_tensor);
    cublasdx::copy_wait();

    // Execute GEMM
    GEMM().execute(alpha, a_shared_tensor, b_shared_tensor, beta, c_shared_tensor);
    __syncthreads();

    // Store data from shared memory tensor to global memory tensor
    cublasdx::copy<GEMM, alignment::c>(c_shared_tensor, c_global_tensor);
}

template<unsigned int Arch>
int introduction_example() {
    using GEMM = decltype(cublasdx::Size<32, 32, 32>()
                  + cublasdx::Precision<double>()
                  + cublasdx::Type<cublasdx::type::real>()
                  + cublasdx::Arrangement<cublasdx::row_major, cublasdx::col_major>()
                  + cublasdx::Function<cublasdx::function::MM>()
                  + cublasdx::SM<700>()
                  + cublasdx::Block()
                  + cublasdx::BlockDim<256>());

    using value_type = typename example::uniform_value_type_t<GEMM>;

    constexpr auto global_a_size = example::global_memory_size_of<GEMM>::a_size;
    constexpr auto global_b_size = example::global_memory_size_of<GEMM>::b_size;
    constexpr auto global_c_size = example::global_memory_size_of<GEMM>::c_size;

    // Allocate managed memory for A, B, C matrices in one go
    value_type* abc;
    auto        size       = global_a_size + global_b_size + global_c_size;
    auto        size_bytes = size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&abc, size_bytes));
    // Generate data
    for (size_t i = 0; i < size; i++) {
        abc[i] = double(i / size);
    }

    value_type* a = abc;
    value_type* b = abc + global_a_size;
    value_type* c = abc + global_a_size + global_b_size;

    // Invokes kernel with GEMM::block_dim threads in CUDA block
    gemm_kernel<GEMM><<<1, GEMM::block_dim, GEMM::shared_memory_size>>>(1.0, a, b, 1.0, c);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    CUDA_CHECK_AND_EXIT(hipFree(abc));
    std::cout << "Success" << std::endl;
    return 0;
}

template<unsigned int Arch>
struct introduction_example_functor {
    int operator()() { return introduction_example<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<introduction_example_functor>();
}
