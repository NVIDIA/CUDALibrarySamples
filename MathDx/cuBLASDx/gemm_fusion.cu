#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <type_traits>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "block_io.hpp"
#include "reference.hpp"

template<class BLAS1, class BLAS2, class ValueType = typename BLAS1::value_type>
__launch_bounds__(BLAS1::max_threads_per_block) //
    __global__                                 //
    void gemm_kernel(const ValueType alpha1,
                     const ValueType* a,
                     const ValueType* b,
                     const ValueType  beta1,
                     const ValueType* c,
                     const ValueType  alpha2,
                     const ValueType* d,
                     const ValueType  beta2,
                     const ValueType* f,
                     ValueType*       output) {
    using value_type = ValueType;
    extern __shared__ __align__(16) char smem[];

    static_assert(std::is_same_v<value_type, example::value_type_t<BLAS2>>, "BLAS1 and BLAS2 must have the same type and precision");
    static_assert((BLAS1::c_dim == BLAS2::a_dim), "The dimensions of C matrix are different in BLAS1 and BLAS2");

    // Matrix C is the first in shared memory, because it's reused in the 2nd GEMM. Moreover,
    // matrices A and B might have different sizes than F and D.
    value_type* smem_c = reinterpret_cast<value_type*>(smem);
    value_type* smem_a = reinterpret_cast<value_type*>(smem) + BLAS1::c_size;
    value_type* smem_b = reinterpret_cast<value_type*>(smem) + BLAS1::c_size + BLAS1::a_size;

    example::io<BLAS1>::load(smem_a, a, BLAS1::a_size);
    example::io<BLAS1>::load(smem_b, b, BLAS1::b_size);
    example::io<BLAS1>::load(smem_c, c, BLAS1::c_size);
    __syncthreads();

    BLAS1().execute(alpha1, smem_a, smem_b, beta1, smem_c);
    __syncthreads();

    static_assert((BLAS1::c_size == BLAS2::a_size), "The sizes of C matrix are different in BLAS1 and BLAS2");
    value_type* smem_d = smem_c + BLAS2::a_size;
    value_type* smem_f = smem_c + BLAS2::a_size + BLAS2::b_size;
    example::io<BLAS2>::load(smem_d, d, BLAS2::b_size);
    example::io<BLAS2>::load(smem_f, f, BLAS2::c_size);
    __syncthreads();

    BLAS2().execute(alpha2, smem_c, smem_d, beta2, smem_f);

    __syncthreads();
    example::io<BLAS2>::store(output, smem_f, BLAS2::c_size);
}

// This is an example of two fp16 general matrix-matrix multiplications (GEMM) fused together
// and performed in one kernel in a single CUDA block:
//
//             1) C = alpha1 * (A * B) + beta1 * C
//             2) F = alpha2 * (C * D) + beta2 * F
//
// * A, B, C, D and F are matrices containing real half precision floating-point values.
// * (alpha1, beta1) and (alpha2, beta2) and real half precision floating-point values.
//
// Input data is generated on host using random number generators, and later copied to
// the global memory. Next, kernel with GEMM is executed, and then the matrix F (the result)
// is copied back to host memory. The results are verified against cuBLAS.
//
// In this example the number of threads participating in the GEMM operation is imposed by providing
// BlockDim operator in definition of the GEMM. If BlockDim operator is not used, cuBLASDx automatically
// selects number of threads. Block dimensions are provided via BLAS::block_dim trait.
//
// Notes:
// * Both GEMM operations use the same number of threads, however, it's not a requirement.
// * It's important that the dimensions of the first and the 2nd GEMM are set in such a way that
//   the C matrix has the same dimensions in both operations.
template<unsigned int Arch>
int simple_gemm() {
    // Parameters m1, n1, k1 define the dimensions of matrices A, B, and C
    constexpr unsigned int m1          = 64;
    constexpr unsigned int n1          = 64;
    constexpr unsigned int k1          = 64;

    // Parameters m2, n2, k2 define the dimensions of matrices C, D and F
    // Note: (m1, n1) and (m2, k2) must be equal as describe the same matrix (matrix C)
    constexpr unsigned int m2          = m1;
    constexpr unsigned int n2          = 128;
    constexpr unsigned int k2          = n1;

    // The logical dimensions of matrix A are: [m1, k1] (m rows, k columns)
    // The logical dimensions of matrix B are: [k1, n1]
    // The logical dimensions of matrix C are: [m1, n1]
    constexpr auto a_transpose_mode = cublasdx::transpose_mode::non_transposed;
    constexpr auto b_transpose_mode = cublasdx::transpose_mode::non_transposed;

    // The logical dimensions of matrix C are: [m2, k2] == [m1, n1]
    // The logical dimensions of matrix D are: [k2, n2]
    // The logical dimensions of matrix F are: [m2, n2]
    constexpr auto c_transpose_mode = cublasdx::transpose_mode::non_transposed;
    constexpr auto d_transpose_mode = cublasdx::transpose_mode::non_transposed;

    // Use the same block size for both GEMM operations, so BLAS1::block_dim == BLAS2::block_dim which
    // simplifies the example.
    constexpr unsigned int block_size = 128;

    using BLAS1       = decltype(cublasdx::Size<m1, n1, k1>() +
                          cublasdx::Precision<__half>() +
                          cublasdx::Type<cublasdx::type::real>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::TransposeMode<a_transpose_mode, b_transpose_mode>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());
    using BLAS2       = decltype(cublasdx::Size<m2, n2, k2>() +
                          cublasdx::Precision<__half>() +
                          cublasdx::Type<cublasdx::type::real>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::TransposeMode<c_transpose_mode, d_transpose_mode>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());
    #if CUBLASDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using value_type = example::value_type_t<BLAS1>;
    #else
    using value_type = typename BLAS1::value_type;
    #endif

    // alpha and beta for the first GEMM
    value_type alpha1 = 1.0;
    value_type beta1  = 0.0;

    // alpha and beta for the 2nd GEMM
    value_type alpha2 = 1.0;
    value_type beta2  = 1.0;

    // Allocate managed memory for a, b, c, d, f and output
    value_type* inputs;
    value_type* output;
    auto inputs_size       = BLAS1::a_size + BLAS1::b_size + BLAS1::c_size + BLAS2::b_size + BLAS2::c_size;
    auto inputs_size_bytes = inputs_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMalloc(&inputs, inputs_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMalloc(&output, BLAS2::c_size * sizeof(value_type)));

    value_type* a     = inputs;
    value_type* b     = a + (BLAS1::a_size);
    value_type* c     = b + (BLAS1::b_size); // C matrix for BLAS1, A matrix for BLAS2
    value_type* d     = c + (BLAS1::c_size); // D is B matrix for BLAS2
    value_type* f     = d + (BLAS2::b_size); // F is C matrix for BLAS2

    // Fill the A, B, C matrices with random values
    auto host_a = example::get_random_data<value_type>(0.1, 1.0, BLAS1::a_size);
    auto host_b = example::get_random_data<value_type>(0.1, 1.0, BLAS1::b_size);
    auto host_c = example::get_random_data<value_type>(0.1, 1.0, BLAS1::c_size);
    auto host_d = example::get_random_data<value_type>(1.0, 2.0, BLAS2::b_size);
    auto host_f = example::get_random_data<value_type>(1.0, 10.0, BLAS2::c_size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(a, host_a.data(), BLAS1::a_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(b, host_b.data(), BLAS1::b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(c, host_c.data(), BLAS1::c_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(d, host_d.data(), BLAS2::b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(f, host_f.data(), BLAS2::c_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Increase max dynamic shared memory for the kernel if needed
    const auto shared_memory = std::max<size_t>(BLAS1::shared_memory_size, BLAS2::shared_memory_size);
    CUDA_CHECK_AND_EXIT(
        hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_kernel<BLAS1), BLAS2>, hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory));

    // Execute kernel
    gemm_kernel<BLAS1, BLAS2><<<1, BLAS1::block_dim, shared_memory>>>(alpha1, a, b, beta1, c, alpha2, d, beta2, f, output);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy results back to host
    std::vector<value_type> host_output(BLAS2::c_size);
    CUDA_CHECK_AND_EXIT(
        hipMemcpy(host_output.data(), output, BLAS2::c_size * sizeof(value_type), hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Free device memory
    CUDA_CHECK_AND_EXIT(hipFree(inputs));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    // Calculate reference
    // 1st GEMM
    auto blas1_reference_host_output = example::reference_gemm<BLAS1, value_type>(alpha1, host_a, host_b, beta1, host_c);
    // 2nd GEMM
    std::vector<value_type> blas2_host_c(blas1_reference_host_output.size());
    blas2_host_c.assign(blas1_reference_host_output.begin(), blas1_reference_host_output.end());
    auto reference_host_output = example::reference_gemm<BLAS2, value_type>(alpha2, blas2_host_c, host_d, beta2, host_f);

    // Check against reference
    if (example::check(host_output, reference_host_output)) {
        std::cout << "Success" << std::endl;
        return 0;
    }
    std::cout << "Failure" << std::endl;
    return 1;
}

template<unsigned int Arch>
struct simple_gemm_functor {
    int operator()() { return simple_gemm<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_gemm_functor>();
}
