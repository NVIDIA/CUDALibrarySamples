#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "block_io.hpp"
#include "reference.hpp"

template<class BLAS, class ValueType = typename BLAS::value_type>
__launch_bounds__(BLAS::max_threads_per_block) //
    __global__                                 //
    void gemm_kernel(const ValueType* a,
                     const ValueType* b,
                     const ValueType* c,
                     const ValueType  alpha,
                     const ValueType  beta,
                     ValueType*       output) {
    using value_type = ValueType;
    extern __shared__ __align__(16) char smem[];
    constexpr unsigned int block_size = BLAS::block_dim.x * BLAS::block_dim.y * BLAS::block_dim.z;

    value_type* smem_a = reinterpret_cast<value_type*>(smem);
    value_type* smem_b = reinterpret_cast<value_type*>(smem) + BLAS::a_size;
    value_type* smem_c = reinterpret_cast<value_type*>(smem) + BLAS::a_size + BLAS::b_size;

    example::io<BLAS>::a_fast_load<block_size>(smem_a, a);
    example::io<BLAS>::b_fast_load<block_size>(smem_b, b);
    example::io<BLAS>::c_fast_load<block_size>(smem_c, c);
    __syncthreads();

    BLAS().execute(alpha, smem_a, smem_b, beta, smem_c);

    __syncthreads();
    example::io<BLAS>::c_fast_store<block_size>(output, smem_c);
}

// This is an example of fp32 general matrix-matrix multiplication (GEMM) performed
// in a single CUDA block:
//
//              C = alpha * A * B + beta * C
//
// * A, B, and C are matrices containing real single precision floating-point values.
// * alpha and beta are real single precision floating-point values.
//
// Input data is generated on host using random number generators, and later copied to
// the global memory. Next, kernel with GEMM is executed, and then the matrix C (the result)
// is copied back to host memory. The results are verified against cuBLAS.
//
// In this example the number of threads participating in the GEMM operation is imposed by providing
// BlockDim operator in definition of the GEMM. If BlockDim operator is not used, cuBLASDx automatically
// selects number of threads. Block dimensions are provided via BLAS::block_dim trait.
template<unsigned int Arch>
int simple_gemm() {
    // Parameters m, n, k define the dimensions of matrices A, B, and C
    constexpr unsigned int m = 32;
    constexpr unsigned int n = 16;
    constexpr unsigned int k = 64;

    // If matrix A is not transposed its logical dimensions are: [m, k] (m rows, k columns)
    // If matrix B is not transposed its logical dimensions are: [k, n]
    // If matrix A is transposed its logical dimensions are: [k, m]
    // If matrix B is transposed its logical dimensions are: [n, k]
    // The dimensions of matrix C are: [m, n]
    constexpr auto a_transpose_mode = cublasdx::transpose_mode::non_transposed;
    constexpr auto b_transpose_mode = cublasdx::transpose_mode::transposed;

    // Selected CUDA block size (1D)
    constexpr unsigned int block_size = 256;

    // GEMM definition using cuBLASDx operators:
    // 1. The size, the precision, and the type (real or complex) are set.
    // 2. The BLAS function is selected: MM (matrix multiplication).
    // 3. The transpose modes of A and B matrices are set.
    // 4. Block operator informs that GEMM should be performed on CUDA block level.
    // 5. BlockDim operator sets CUDA block dimensions that the kernel will be executed with.
    // 6. Targeted CUDA compute capability is selected with SM operator.
    using BLAS       = decltype(cublasdx::Size<m, n, k>() +
                          cublasdx::Precision<float>() +
                          cublasdx::Type<cublasdx::type::real>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::TransposeMode<a_transpose_mode, b_transpose_mode>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());
    #if CUBLASDX_EXAMPLE_DETAIL_NVCC_12_2_BUG_WORKAROUND
    using value_type = example::value_type_t<BLAS>;
    #else
    using value_type = typename BLAS::value_type;
    #endif

    // Allocate managed memory for a, b, c, and output
    value_type* inputs;
    value_type* output;
    // BLAS::a_size/b_size/c_size include padding (take into account the leading dimension if set)
    auto inputs_size       = BLAS::a_size + BLAS::b_size + BLAS::c_size;
    auto inputs_size_bytes = inputs_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&inputs, inputs_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&output, BLAS::c_size * sizeof(value_type)));

    value_type* a     = inputs;
    value_type* b     = a + (BLAS::a_size);
    value_type* c     = b + (BLAS::b_size);
    value_type  alpha = value_type(1.0);
    value_type  beta  = value_type(2.0);

    // Fill the A, B, C matrices with random values
    auto host_a = example::get_random_data<value_type>(0.1, 1.0, BLAS::a_size);
    auto host_b = example::get_random_data<value_type>(0.1, 1.0, BLAS::b_size);
    auto host_c = example::get_random_data<value_type>(0.1, 1.0, BLAS::c_size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(a, host_a.data(), BLAS::a_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(b, host_b.data(), BLAS::b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(c, host_c.data(), BLAS::c_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Increase max dynamic shared memory for the kernel if needed
    CUDA_CHECK_AND_EXIT(
        hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_kernel<BLAS>), hipFuncAttributeMaxDynamicSharedMemorySize, BLAS::shared_memory_size));

    // Execute kernel
    gemm_kernel<BLAS><<<1, BLAS::block_dim, BLAS::shared_memory_size>>>(a, b, c, alpha, beta, output);
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy results back to host
    std::vector<value_type> host_output(BLAS::c_size);
    CUDA_CHECK_AND_EXIT(
        hipMemcpy(host_output.data(), output, BLAS::c_size * sizeof(value_type), hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Free device memory
    CUDA_CHECK_AND_EXIT(hipFree(inputs));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    // Calculate reference
    auto reference_host_output = example::reference_gemm<BLAS>(alpha, host_a, host_b, beta, host_c);

    // Check against reference
    if (example::check(host_output, reference_host_output)) {
        std::cout << "Success" << std::endl;
        return 0;
    }
    std::cout << "Failure" << std::endl;
    return 1;
}

template<unsigned int Arch>
struct simple_gemm_functor {
    int operator()() { return simple_gemm<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_gemm_functor>();
}
