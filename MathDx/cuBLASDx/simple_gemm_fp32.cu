#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#include "common.hpp"
#include "block_io.hpp"
#include "reference.hpp"

template<class BLAS, class ValueType = typename example::uniform_value_type_t<BLAS>>
__launch_bounds__(BLAS::max_threads_per_block) //
    __global__                                 //
    void gemm_kernel(const ValueType* a,
                     const ValueType* b,
                     const ValueType* c,
                     const ValueType  alpha,
                     const ValueType  beta,
                     ValueType*       output) {
    using value_type = ValueType;
    extern __shared__ __align__(16) char smem[];

    auto a_global_tensor = cublasdx::make_tensor(a, BLAS::get_layout_gmem_a());
    auto b_global_tensor = cublasdx::make_tensor(b, BLAS::get_layout_gmem_b());
    auto c_global_tensor = cublasdx::make_tensor(c, BLAS::get_layout_gmem_c());

    auto [smem_a, smem_b, smem_c] = BLAS::slice_shared_memory(smem);
    auto a_shared_tensor = cublasdx::make_tensor(smem_a, BLAS::get_layout_smem_a());
    auto b_shared_tensor = cublasdx::make_tensor(smem_b, BLAS::get_layout_smem_b());
    auto c_shared_tensor = cublasdx::make_tensor(smem_c, BLAS::get_layout_smem_c());

    using alignment = cublasdx::alignment_of<BLAS>;
    cublasdx::copy<BLAS, alignment::a>(a_global_tensor, a_shared_tensor);
    cublasdx::copy<BLAS, alignment::b>(b_global_tensor, b_shared_tensor);
    cublasdx::copy<BLAS, alignment::c>(c_global_tensor, c_shared_tensor);
    cublasdx::copy_wait();

    BLAS().execute(alpha, a_shared_tensor, b_shared_tensor, beta, c_shared_tensor);

    __syncthreads();

    auto out_global_tensor = cublasdx::make_tensor(output, BLAS::get_layout_gmem_c());
    cublasdx::copy<BLAS, alignment::c>(c_shared_tensor, out_global_tensor);
}

// This is an example of fp32 general matrix-matrix multiplication (GEMM) performed
// in a single CUDA block:
//
//              C = alpha * A * B + beta * C
//
// * A, B, and C are matrices containing real single precision floating-point values.
// * alpha and beta are real single precision floating-point values.
//
// Input data is generated on host using random number generators, and later copied to
// the global memory. Next, kernel with GEMM is executed, and then the matrix C (the result)
// is copied back to host memory. The results are verified against cuBLAS.
//
// In this example the number of threads participating in the GEMM operation is imposed by providing
// BlockDim operator in definition of the GEMM. If BlockDim operator is not used, cuBLASDx automatically
// selects number of threads. Block dimensions are provided via BLAS::block_dim trait.
template<unsigned int Arch>
int simple_gemm() {
    // Parameters m, n, k define the dimensions of matrices A, B, and C
    constexpr unsigned int m = 8;
    constexpr unsigned int n = 16;
    constexpr unsigned int k = 32;

    // Selected CUDA block size (1D)
    constexpr unsigned int block_size = 256;

    // GEMM definition using cuBLASDx operators:
    // 1. The size, the precision, and the type (real or complex) are set.
    // 2. The BLAS function is selected: MM (matrix multiplication).
    // 3. Block operator informs that GEMM should be performed on CUDA block level.
    // 4. BlockDim operator sets CUDA block dimensions that the kernel will be executed with.
    // 5. Targeted CUDA compute capability is selected with SM operator.
    using BLAS = decltype(cublasdx::Size<m, n, k>() +
                          cublasdx::Precision<float>() +
                          cublasdx::Type<cublasdx::type::real>() +
                          cublasdx::Function<cublasdx::function::MM>() +
                          cublasdx::Block() +
                          cublasdx::BlockDim<block_size>() +
                          cublasdx::SM<Arch>());

    using value_type = typename example::uniform_value_type_t<BLAS>;

    // Allocate managed memory for a, b, c, and output
    value_type* inputs;
    value_type* output;

    constexpr auto global_a_size = example::global_memory_size_of<BLAS>::a_size;
    constexpr auto global_b_size = example::global_memory_size_of<BLAS>::b_size;
    constexpr auto global_c_size = example::global_memory_size_of<BLAS>::c_size;

    auto inputs_size       = global_a_size + global_b_size + global_c_size;
    auto inputs_size_bytes = inputs_size * sizeof(value_type);
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&inputs, inputs_size_bytes));
    CUDA_CHECK_AND_EXIT(hipMallocManaged(&output, global_c_size * sizeof(value_type)));

    value_type* a     = inputs;
    value_type* b     = a + (global_a_size);
    value_type* c     = b + (global_b_size);
    value_type  alpha = value_type(1.0);
    value_type  beta  = value_type(2.0);

    // Fill the A, B, C matrices with random values
    auto host_a = example::get_random_data<value_type>(0.1, 1.0, global_a_size);
    auto host_b = example::get_random_data<value_type>(0.1, 1.0, global_b_size);
    auto host_c = example::get_random_data<value_type>(0.1, 1.0, global_c_size);
    CUDA_CHECK_AND_EXIT(hipMemcpy(a, host_a.data(), global_a_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(b, host_b.data(), global_b_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipMemcpy(c, host_c.data(), global_c_size * sizeof(value_type), hipMemcpyHostToDevice));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Increase max dynamic shared memory for the kernel if needed
    CUDA_CHECK_AND_EXIT(
        hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_kernel<BLAS>), hipFuncAttributeMaxDynamicSharedMemorySize, BLAS::shared_memory_size));

    // Execute kernel
    gemm_kernel<BLAS><<<1, BLAS::block_dim, BLAS::shared_memory_size>>>(a, b, c, alpha, beta, output);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Copy results back to host
    std::vector<value_type> host_output(global_c_size);
    CUDA_CHECK_AND_EXIT(
        hipMemcpy(host_output.data(), output, global_c_size * sizeof(value_type), hipMemcpyDeviceToHost));
    CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

    // Free device memory
    CUDA_CHECK_AND_EXIT(hipFree(inputs));
    CUDA_CHECK_AND_EXIT(hipFree(output));

    // Calculate reference
    auto reference_host_output = example::reference_gemm<BLAS>(alpha, host_a, host_b, beta, host_c);
    CUDA_CHECK_AND_EXIT(hipPeekAtLastError());

    // Check against reference
    if (example::check(host_output, reference_host_output)) {
        std::cout << "Success" << std::endl;
        return 0;
    }
    std::cout << "Failure" << std::endl;
    return 1;
}

template<unsigned int Arch>
struct simple_gemm_functor {
    int operator()() { return simple_gemm<Arch>(); }
};

int main(int, char**) {
    return example::sm_runner<simple_gemm_functor>();
}
