#include <hiptensor.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>
#include <cstdint>
#include <unordered_map>
#include <chrono>

bool CHECK_success(hipError_t status)
{
    return status == hipSuccess;
}

const char* CHECK_pretty(hipError_t status)
{
    return hipGetErrorName(status);
}

bool CHECK_success(hiptensorStatus_t status)
{
    return status == HIPTENSOR_STATUS_SUCCESS;
}

const char* CHECK_pretty(hiptensorStatus_t status)
{
    return hiptensorGetErrorString(status);
}

template<typename T>
T product(const std::vector<T> &values)
{
    T result = 1;
    for (auto& value : values)
    {
        result *= value;
    }
    return result;
}

template<typename T, typename U>
std::vector<T> multiply(const std::vector<T> &lhs, const std::vector<U> &rhs)
{
    std::vector<T> result;
    assert(lhs.size() == rhs.size() || lhs.empty() || rhs.empty());
    for (size_t i = 0; i < lhs.size(); i++)
    {
        result.push_back((lhs.empty() ? 1 : lhs[i]) * (rhs.empty() ? 1 : rhs[i]));
    }
    return result;
}

template<typename T, typename U>
std::vector<T> discretize(const std::vector<T> &in, const std::vector<U> &block)
{
    if (in.empty())
    {
        return in;
    }
    if (block.empty())
    {
        return in;
    }

    std::vector<T> result;
    assert(in.size() == block.size());
    for (size_t i = 0; i < in.size(); i++)
    {
        U b = block[i];
        result.push_back(b * ((in[i] + b - 1) / b));
    }
    return result;
}

#define CHECK(x) do { auto CHECK_err = (x); if (! CHECK_success(CHECK_err)) { \
  printf("Error (%s:%d): \"%s\" returned %s (%d)\n", __FILE__, __LINE__, \
    #x, CHECK_pretty(CHECK_err), CHECK_err); exit(-1);} } while(0)

template<typename K, typename V, typename K2>
std::vector<V> collect(const std::unordered_map<K, V> &map, const std::vector<K2> &index)
{
    std::vector<V> result;
    for (auto& elem : index)
    {
        result.push_back(map.at(elem));
    }
    return result;
}

void printDeviceInfo(int deviceId)
{
    struct hipDeviceProp_t prop;
    int currentDeviceId = 0;
    CHECK(hipGetDevice(&currentDeviceId));
    CHECK(hipSetDevice(deviceId));
    CHECK(hipGetDeviceProperties(&prop, deviceId));
    printf( "device %d (%s): SMs %2d  Capabilities %d.%d, SmClock %.1f Mhz, MemSize (MB) %d, MemClock %.1f Mhz\n",
            deviceId,
            prop.name,
            prop.multiProcessorCount, prop.major, prop.minor,
            (float)prop.clockRate*1e-3,
            (int)(prop.totalGlobalMem/(1024*1024)),
            (float)prop.memoryClockRate*1e-3);
    CHECK(hipSetDevice(currentDeviceId));
}

int main(int argc, char** argv)
{
    printf("This sample uses the following GPUs:\n");
    std::vector<int32_t> devices;
    if (argc == 1)
    {
        int numDevices;
        CHECK(hipGetDeviceCount(&numDevices));
        for (int i = 0; i < numDevices; i++)
        {
            printDeviceInfo(i);
            devices.push_back(i);
        }
    }
    else
    {
        for (int i = 1; i < argc; i++)
        {
            const int deviceId = atoi(argv[i]);
            printDeviceInfo(deviceId);
            devices.push_back(deviceId);
        }
    }
    cutensorMgHandle_t handle;
    printf("Initializing cutensorMg handle ... ");
    CHECK(cutensorMgCreate(&handle, devices.size(), devices.data()));
    printf("done.\n");
    
    std::unordered_map<int32_t, int64_t> extent;
    extent['i'] = 4096;
    extent['j'] = 4096;
    extent['k'] = 4096;

    std::unordered_map<int32_t, int64_t> blocksize;
    blocksize['i'] = 2048;
    blocksize['j'] = 2048;
    blocksize['k'] = 2048;

    std::unordered_map<int32_t, int32_t> deviceCount;
    deviceCount['i'] = 2;
    deviceCount['j'] = 2;
    deviceCount['k'] = 2;

    std::vector<int32_t> modesA {'i', 'k'};
    std::vector<int32_t> modesB {'k', 'j'};
    std::vector<int32_t> modesC {'i', 'j'};

    hipDataType kDataType = HIP_R_32F;
    const int64_t kElementSize = 4;

    printf("Creating distributed tensor descriptors ... ");

    auto fillUp = [](const std::vector<int32_t> &devices, const int32_t n)
    {
        std::vector<int32_t> ret; 
        int32_t numDevices = devices.size();
        for(int i=0; i < n; ++i)
        {
            ret.push_back(devices[i%numDevices]);
        }
        return ret;
    };

    cutensorMgTensorDescriptor_t descA;
    std::vector<int64_t> extentA = collect(extent, modesA);
    std::vector<int64_t> blocksizeA = collect(blocksize, modesA);
    std::vector<int32_t> deviceCountA = collect(deviceCount, modesA);
    std::vector<int32_t> devicesA = fillUp(devices, product(deviceCountA));
    assert(product(deviceCountA) == devicesA.size());
    CHECK(cutensorMgCreateTensorDescriptor(handle, &descA, modesA.size(),
        extentA.data(), NULL, blocksizeA.data(), NULL,
        deviceCountA.data(), devicesA.size(), devicesA.data(), kDataType));

    cutensorMgTensorDescriptor_t descB;
    std::vector<int64_t> extentB = collect(extent, modesB);
    std::vector<int64_t> blocksizeB = collect(blocksize, modesB);
    std::vector<int32_t> deviceCountB = collect(deviceCount, modesB);
    std::vector<int32_t> devicesB = fillUp(devices, product(deviceCountB));
    assert(product(deviceCountB) == devicesB.size());
    CHECK(cutensorMgCreateTensorDescriptor(handle, &descB, modesB.size(),
        extentB.data(), NULL, blocksizeB.data(), NULL,
        deviceCountB.data(), devicesB.size(), devicesB.data(), kDataType));

    cutensorMgTensorDescriptor_t descC;
    std::vector<int64_t> extentC = collect(extent, modesC);
    std::vector<int64_t> blocksizeC = collect(blocksize, modesC);
    std::vector<int32_t> deviceCountC = collect(deviceCount, modesC);
    std::vector<int32_t> devicesC = fillUp(devices, product(deviceCountC));
    assert(product(deviceCountC) == devicesC.size());
    CHECK(cutensorMgCreateTensorDescriptor(handle, &descC, modesC.size(),
        extentC.data(), NULL, blocksizeC.data(), NULL,
        deviceCountC.data(), devicesC.size(), devicesC.data(), kDataType));

    printf("done.\n");

    printf("Querying workspace size (per GPU) ... ");

    const hiptensorComputeDescriptor_t kComputeType = HIPTENSOR_COMPUTE_DESC_32F;
    const hiptensorWorksizePreference_t kWorksizePreference = 
        CUTENSOR_WORKSPACE_RECOMMENDED;

    cutensorMgContractionDescriptor_t contractionDesc;
    CHECK(cutensorMgCreateContractionDescriptor(handle, &contractionDesc,
                descA, modesA.data(),
                descB, modesB.data(),
                descC, modesC.data(),
                descC, modesC.data(),
                kComputeType));

    cutensorMgContractionFind_t contractionFind;
    CHECK(cutensorMgCreateContractionFind(handle, &contractionFind,
                CUTENSORMG_ALGO_DEFAULT));

    std::vector<int64_t> workspaceSize(devices.size());
    int64_t workspaceHostSize;
    CHECK(cutensorMgContractionGetWorkspace(handle,
        contractionDesc, contractionFind, kWorksizePreference, workspaceSize.data(), &workspaceHostSize));

    printf("done.\n");

    printf("Initializing contraction plan ... \n");
 
    cutensorMgContractionPlan_t plan;
    CHECK(cutensorMgCreateContractionPlan(handle, &plan,
                contractionDesc, contractionFind, workspaceSize.data(), workspaceHostSize));

    printf("done.\n");

    printf("Allocating data ... ");

    int64_t elementsA = product(discretize(extentA, multiply(deviceCountA, blocksizeA))) / product(deviceCountA);
    std::vector<void*> memoryA;
    for (auto& device : devicesA)
    {
        void* memory;
        CHECK(hipSetDevice(device));
        CHECK(hipMalloc(&memory, elementsA * kElementSize));
        memoryA.push_back(memory);
    }

    int64_t elementsB = product(discretize(extentB, multiply(deviceCountB, blocksizeB))) / product(deviceCountB);
    std::vector<void*> memoryB;
    for (auto& device : devicesB)
    {
        void* memory;
        CHECK(hipSetDevice(device));
        CHECK(hipMalloc(&memory, elementsB * kElementSize));
        memoryB.push_back(memory);
    }

    int64_t elementsC = product(discretize(extentC, multiply(deviceCountC, blocksizeC))) / product(deviceCountC);
    std::vector<void*> memoryC;
    for (auto& device : devicesC)
    {
        void* memory;
        CHECK(hipSetDevice(device));
        CHECK(hipMalloc(&memory, elementsC * kElementSize));
        memoryC.push_back(memory);
    }

    std::vector<hipStream_t> streams;
    for (auto& device : devices)
    {
        hipStream_t stream;
        CHECK(hipSetDevice(device));
        CHECK(hipStreamCreate(&stream));
        streams.push_back(stream);
    }

    /*
     * Allocate workspace
     */
    // host
    void* workspaceHost = nullptr;
    CHECK(hipHostMalloc(&workspaceHost, workspaceHostSize));

    // devices
    std::vector<void*> workspace;
    for (int i = 0; i < devices.size(); i++)
    {
        void* memory;
        CHECK(hipSetDevice(devices[i]));
        CHECK(hipMalloc(&memory, workspaceSize[i]));
        workspace.push_back(memory);
    }

    printf("done.\n");

    printf("Performing distributed tensor contraction ...\n");

    float kAlpha = 1;
    float kBeta = 0;


    int currentDeviceId = -1;
    CHECK(hipGetDevice(&currentDeviceId));

    float minElapsed = 0;
    const int nRep = 3; // for stable timings
    for (int rep = 0; rep < nRep; rep++)
    {
        const auto start = std::chrono::steady_clock::now();
        CHECK(cutensorMgContraction(handle, plan, &kAlpha,
            const_cast<const void**>(memoryA.data()),
            const_cast<const void**>(memoryB.data()), &kBeta, 
            const_cast<const void**>(memoryC.data()), memoryC.data(),
            workspace.data(), workspaceHost, streams.data()));

        for (auto& deviceId : devices)
        {
            CHECK(hipSetDevice(deviceId));
            CHECK(hipDeviceSynchronize());
        }

        const auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur = end - start;
        if (minElapsed == 0 || minElapsed > dur.count()) {
            minElapsed = dur.count();
        }
    }
    CHECK(hipSetDevice(currentDeviceId));

    printf("execution took: %.2e millisec.\n", minElapsed);

    printf("Free resources ...\n");

    for (auto& stream : streams)
    {
        CHECK(hipStreamSynchronize(stream));
        CHECK(hipStreamDestroy(stream));
    }

    for (auto& memory : memoryA)
    {
        CHECK(hipFree(memory));
    }

    for (auto& memory : memoryB)
    {
        CHECK(hipFree(memory));
    }

    for (auto& memory : memoryC)
    {
        CHECK(hipFree(memory));
    }

    CHECK(hipHostFree(workspaceHost));

    CHECK(cutensorMgDestroyContractionDescriptor(contractionDesc));
    CHECK(cutensorMgDestroyContractionFind(contractionFind));
    CHECK(cutensorMgDestroyContractionPlan(plan));

    CHECK(cutensorMgDestroyTensorDescriptor(descA));
    CHECK(cutensorMgDestroyTensorDescriptor(descB));
    CHECK(cutensorMgDestroyTensorDescriptor(descC));

    CHECK(cutensorMgDestroy(handle));
    printf("Done: everything has completed successfully.\n");
}
