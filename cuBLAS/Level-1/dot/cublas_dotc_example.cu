/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = hipDoubleComplex;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    /*
     *   A = | 1.1 + 1.2j | 2.3 + 2.4j | 3.5 + 3.6j | 4.7 + 4.8j |
     *   B = | 5.1 + 5.2j | 6.3 + 6.4j | 7.5 + 7.6j | 8.7 + 8.8j |
     */

    const std::vector<data_type> A = {{1.1, 1.2}, {2.3, 2.4}, {3.5, 3.6}, {4.7, 4.8}};
    const std::vector<data_type> B = {{5.1, 5.2}, {6.3, 6.4}, {7.5, 7.6}, {8.7, 8.8}};
    const int incx = 1;
    const int incy = 1;

    data_type result = {0.0, 0.0};

    data_type *d_A = nullptr;
    data_type *d_B = nullptr;

    printf("A\n");
    print_vector(A.size(), A.data());
    printf("=====\n");

    printf("B\n");
    print_vector(B.size(), B.data());
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    CUBLAS_CHECK(hipblasZdotc(cublasH, A.size(), d_A, incx, d_B, incy, &result));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   result = 178.44+-1.60j
     */

    printf("Result\n");
    printf("%0.2f+%0.2fj\n", result.x, result.y);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}