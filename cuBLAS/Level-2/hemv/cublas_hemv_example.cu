/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = hipDoubleComplex;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 2;
    const int n = 2;
    const int lda = m;

    /*
     *   A = | 1.1 + 1.2j | 2.3 + 2.4j |
     *       | 3.5 + 3.6j | 4.7 + 4.8j |
     *   x = | 5.1 + 6.2j | 7.3 + 8.4j |
     */

    const std::vector<data_type> A = {{1.1, 1.2}, {3.5, 3.6}, {2.3, 2.4}, {4.7, 4.8}};
    const std::vector<data_type> x = {{5.1, 6.2}, {7.3, 8.4}};
    std::vector<data_type> y(m);
    const data_type alpha = {1.0, 1.0};
    const data_type beta = {0.0, 0.0};
    const int incx = 1;
    const int incy = 1;

    data_type *d_A = nullptr;
    data_type *d_x = nullptr;
    data_type *d_y = nullptr;

    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

    printf("A\n");
    print_matrix(m, n, A.data(), lda);
    printf("=====\n");

    printf("x\n");
    print_vector(x.size(), x.data());
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(data_type) * x.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_y), sizeof(data_type) * y.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_x, x.data(), sizeof(data_type) * x.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    CUBLAS_CHECK(hipblasZhemv(cublasH, uplo, n, &alpha, d_A, lda, d_x, incx, &beta, d_y, incy));

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(y.data(), d_y, sizeof(data_type) * y.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   y = | -41.42 + 45.90j 19.42 + 102.42j |
     */

    printf("y\n");
    print_vector(y.size(), y.data());
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}