/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = hipDoubleComplex;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 2;
    const int n = 2;
    const int lda = m;

    /*
     *   AP = | 1.1 + 1.2j | 2.3 + 2.4j |
     *        | 3.5 + 3.6j | 4.7 + 4.8j |
     *   x  = | 5.1 + 6.2j | 7.3 + 8.4j |
     *   y  = | 1.1 + 2.2j | 3.3 + 4.4j |
     */

    std::vector<data_type> AP = {{1.1, 1.2}, {3.5, 3.6}, {2.3, 2.4}, {4.7, 4.8}};
    const std::vector<data_type> x = {{5.1, 6.2}, {7.3, 8.4}};
    const std::vector<data_type> y = {{1.1, 2.2}, {3.3, 4.4}};
    const data_type alpha = {1.0, 1.0};
    const int incx = 1;
    const int incy = 1;

    data_type *d_AP = nullptr;
    data_type *d_x = nullptr;
    data_type *d_y = nullptr;

    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

    printf("AP\n");
    print_matrix(m, n, AP.data(), lda);
    printf("=====\n");

    printf("x\n");
    print_vector(x.size(), x.data());
    printf("=====\n");

    printf("y\n");
    print_vector(y.size(), y.data());
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_AP), sizeof(data_type) * AP.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_x), sizeof(data_type) * x.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_y), sizeof(data_type) * y.size()));

    CUDA_CHECK(hipMemcpyAsync(d_AP, AP.data(), sizeof(data_type) * AP.size(),
                               hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_x, x.data(), sizeof(data_type) * x.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_y, y.data(), sizeof(data_type) * y.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    CUBLAS_CHECK(hipblasZhpr2(cublasH, uplo, n, &alpha, d_x, incx, d_y, incy, d_AP));

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(AP.data(), d_AP, sizeof(data_type) * AP.size(),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   AP = | 48.40 +  0.00j | 133.20 + 0.00j |
     *        | 82.92 + 26.04j |   4.70 + 4.80j |
     */

    printf("AP\n");
    print_matrix(m, n, AP.data(), lda);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_AP));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
