/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 2;
    const int n = 2;
    const int k = 2;
    const int lda = 2;
    const int ldb = 2;
    const int ldc = 2;
    /*
     *   A = | 1.0 | 2.0 |
     *       | 3.0 | 4.0 |
     *
     *   B = | 5.0 | 6.0 |
     *       | 7.0 | 8.0 |
     */

    const std::vector<data_type> A = {1.0, 3.0, 2.0, 4.0};
    const std::vector<data_type> B = {5.0, 7.0, 6.0, 8.0};
    std::vector<data_type> C(m * n);
    const data_type alpha = 1.0;
    const data_type beta = 2.0;

    data_type *d_A = nullptr;
    data_type *d_B = nullptr;
    data_type *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    printf("A\n");
    print_matrix(m, k, A.data(), lda);
    printf("=====\n");

    printf("B\n");
    print_matrix(k, n, B.data(), ldb);
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(data_type) * C.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    CUBLAS_CHECK(
        hipblasDgeam(cublasH, transa, transb, m, n, &alpha, d_A, lda, &beta, d_B, ldb, d_C, ldc));

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(C.data(), d_C, sizeof(data_type) * C.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   C = | 11.0 | 14.0 |
     *       | 17.0 | 20.0 |
     */

    printf("C\n");
    print_matrix(m, n, C.data(), ldc);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
