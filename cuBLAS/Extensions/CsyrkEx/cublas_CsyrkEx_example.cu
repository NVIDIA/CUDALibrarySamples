/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = hipComplex;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 2;
    const int n = 2;
    const int k = 2;
    const int lda = 2;
    const int ldc = 2;

    /*
     *   A = | 1.1 + 1.2j | 2.3 + 2.4j |
     *       | 3.5 + 3.6j | 4.7 + 4.8j |
     */

    const std::vector<data_type> A = {{1.1, 1.2}, {3.5, 3.6}, {3.5, 3.6}, {4.7, 4.8}};
    std::vector<data_type> C(m * n);

    const data_type alpha = {1.0, 1.0};
    const data_type beta = {0.0, 0.0};

    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
    hipblasOperation_t trans = HIPBLAS_OP_N;

    data_type *d_A = nullptr;
    data_type *d_C = nullptr;

    printf("A\n");
    print_matrix(n, k, A.data(), lda);
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(data_type) * C.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    CUBLAS_CHECK(cublasCsyrkEx(cublasH, uplo, trans, n, k, &alpha, d_A,
                               traits<data_type>::cuda_data_type, lda, &beta, d_C,
                               traits<data_type>::cuda_data_type, ldc));

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(C.data(), d_C, sizeof(data_type) * C.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   C = | -28.78 + 26.90j | -43.18 + 40.58j |
     *       | -28.78 + 26.90j | -43.18 + 40.58j |
     */

    printf("C\n");
    print_matrix(n, k, C.data(), ldc);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}