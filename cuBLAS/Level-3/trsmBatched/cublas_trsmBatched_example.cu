/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 2;
    const int n = 2;
    const int k = 2;
    const int lda = 2;
    const int ldb = 2;
    const int batch_count = 2;

    /*
     *   A = | 1.0 | 2.0 | 5.0 | 6.0 |
     *       | 3.0 | 4.0 | 7.0 | 8.0 |
     *
     *   B = | 5.0 | 6.0 |  9.0 | 10.0 |
     *       | 7.0 | 8.0 | 11.0 | 12.0 |
     */

    const std::vector<std::vector<data_type>> A_array = {{1.0, 3.0, 2.0, 4.0},
                                                         {5.0, 7.0, 6.0, 8.0}};
    std::vector<std::vector<data_type>> B_array = {{5.0, 7.0, 6.0, 8.0}, {9.0, 11.0, 10.0, 12.0}};
    const data_type alpha = 1.0;

    data_type **d_A_array = nullptr;
    data_type **d_B_array = nullptr;

    std::vector<data_type *> d_A(batch_count, nullptr);
    std::vector<data_type *> d_B(batch_count, nullptr);

    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasDiagType_t diag = HIPBLAS_DIAG_NON_UNIT;

    printf("A[0]\n");
    print_matrix(m, k, A_array[0].data(), lda);
    printf("=====\n");

    printf("A[1]\n");
    print_matrix(m, k, A_array[1].data(), lda);
    printf("=====\n");

    printf("B[0] (in)\n");
    print_matrix(k, n, B_array[0].data(), ldb);
    printf("=====\n");

    printf("B[1] (in)\n");
    print_matrix(k, n, B_array[1].data(), ldb);
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    for (int i = 0; i < batch_count; i++) {
        CUDA_CHECK(
            hipMalloc(reinterpret_cast<void **>(&d_A[i]), sizeof(data_type) * A_array[i].size()));
        CUDA_CHECK(
            hipMalloc(reinterpret_cast<void **>(&d_B[i]), sizeof(data_type) * B_array[i].size()));
    }

    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A_array), sizeof(data_type *) * batch_count));
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_B_array), sizeof(data_type *) * batch_count));

    for (int i = 0; i < batch_count; i++) {
        CUDA_CHECK(hipMemcpyAsync(d_A[i], A_array[i].data(), sizeof(data_type) * A_array[i].size(),
                                   hipMemcpyHostToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync(d_B[i], B_array[i].data(), sizeof(data_type) * B_array[i].size(),
                                   hipMemcpyHostToDevice, stream));
    }

    CUDA_CHECK(hipMemcpyAsync(d_A_array, d_A.data(), sizeof(data_type *) * batch_count,
                               hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_B_array, d_B.data(), sizeof(data_type *) * batch_count,
                               hipMemcpyHostToDevice, stream));

    /* step 3: compute */
    CUBLAS_CHECK(hipblasDtrsmBatched(cublasH, side, uplo, transa, diag, m, n, &alpha, d_A_array, lda,
                                    d_B_array, ldb, batch_count));

    /* step 4: copy data to host */
    for (int i = 0; i < batch_count; i++) {
        CUDA_CHECK(hipMemcpyAsync(B_array[i].data(), d_B[i], sizeof(data_type) * B_array[i].size(),
                                   hipMemcpyDeviceToHost, stream));
    }

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   B = | 1.50 | 2.00 | 0.15 | 0.20 |
     *       | 1.75 | 2.00 | 1.38 | 1.50 |
     */

    printf("B[0] (out)\n");
    print_matrix(k, n, B_array[0].data(), ldb);
    printf("=====\n");

    printf("B[1] (out)\n");
    print_matrix(k, n, B_array[1].data(), ldb);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_B_array));
    for (int i = 0; i < batch_count; i++) {
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_B[i]));
    }
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}