/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

using data_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    const int m = 2;
    const int n = 2;
    const int k = 2;
    const int lda = 2;
    const int ldb = 2;
    const int ldc = 2;
    const int batch_count = 2;

    const long long int strideA = m * k;
    const long long int strideB = k * n;
    const long long int strideC = m * n;

    /*
     *   A = | 1.0 | 2.0 | 5.0 | 6.0 |
     *       | 3.0 | 4.0 | 7.0 | 8.0 |
     *
     *   B = | 5.0 | 6.0 |  9.0 | 10.0 |
     *       | 7.0 | 8.0 | 11.0 | 12.0 |
     */

    const std::vector<data_type> A = {1.0, 3.0, 2.0, 4.0, 5.0, 7.0, 6.0, 8.0};
    const std::vector<data_type> B = {5.0, 7.0, 6.0, 8.0, 9.0, 11.0, 10.0, 12.0};
    std::vector<data_type> C(m * n * batch_count);
    const data_type alpha = 1.0;
    const data_type beta = 0.0;

    data_type *d_A = nullptr;
    data_type *d_B = nullptr;
    data_type *d_C = nullptr;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    printf("A[0]\n");
    print_matrix(m, k, A.data(), lda);
    printf("=====\n");

    printf("A[1]\n");
    print_matrix(m, k, A.data() + (m * k), lda);
    printf("=====\n");

    printf("B[0]\n");
    print_matrix(k, n, B.data(), ldb);
    printf("=====\n");

    printf("B[1]\n");
    print_matrix(k, n, B.data() + (k * n), ldb);
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(data_type) * C.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    CUBLAS_CHECK(hipblasDgemmStridedBatched(cublasH, transa, transb, m, n, k, &alpha, d_A, lda,
                                           strideA, d_B, ldb, strideB, &beta, d_C, ldc, strideC,
                                           batch_count));

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(C.data(), d_C, sizeof(data_type) * C.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   C = | 19.0 | 22.0 | 111.0 | 122.0 |
     *       | 43.0 | 50.0 | 151.0 | 166.0 |
     */

    printf("C[0]\n");
    print_matrix(m, n, C.data(), ldc);
    printf("=====\n");

    printf("C[1]\n");
    print_matrix(m, n, C.data() + (m * n), ldc);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}