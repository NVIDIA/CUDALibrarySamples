/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


/* 
 * Example showing the use of LTO callbacks with CUFFT to perform 
 * truncation with zero padding.
 * 
*/


#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>

struct cb_params {
	unsigned window_N;
	unsigned signal_size;
};

// This is the store callback routine. It filters high frequencies
// based on a truncation window specified by the user
// NOTE: unlike the non-LTO version, the callback device function
// must have the name cufftJITCallbackLoadComplex, it cannot be aliased
__device__ hipfftComplex cufftJITCallbackLoadComplex(void *input,
                                                    size_t index,
                                                    void *info,
                                                    void *sharedmem) {
  
	const cb_params* params = static_cast<const cb_params*>(info);
	hipfftComplex* cb_output = static_cast<hipfftComplex*>(input);
	const unsigned sample   = index % params->signal_size;

	return (sample < params->window_N) ? cb_output[index] : hipfftComplex{0.f, 0.f};
}

