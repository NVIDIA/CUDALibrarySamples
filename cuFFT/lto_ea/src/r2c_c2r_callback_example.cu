/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


/* 
 * Example showing the use of LTO callbacks with CUFFT to perform 
 * normalization and truncation with zero padding.
 * 
*/

#include <hip/hip_runtime_api.h>
#include <hipfft/hipfftXt.h>
#include "common.h"
#include "r2c_c2r_reference.h"

// Struct to pass data to callback
struct cb_params {
  unsigned window_N;
  unsigned signal_size;
};

// This is the store callback routine. It filters high frequencies
// based on a truncation window specified by the user
// NOTE: unlike the LTO version, the callback function can have
// any name
__device__ hipfftComplex windowing_callback(void *input,
                                           size_t index,
                                           void *info,
                                           void *sharedmem) {
 	const cb_params* params = static_cast<const cb_params*>(info);
	hipfftComplex* cb_output = static_cast<hipfftComplex*>(input);
	const unsigned sample   = index % params->signal_size;

	return (sample < params->window_N) ? cb_output[index] : hipfftComplex{0.f, 0.f};
}

__device__ hipfftCallbackLoadC device_callback_ptr = windowing_callback;

// Problem input parameters
constexpr unsigned batches              = 830;
constexpr unsigned signal_size          = 328;
constexpr unsigned window_size          =  32;
constexpr unsigned complex_signal_size  = signal_size / 2 + 1;

// Precision threshold
constexpr float threshold = 1e-6;

static_assert(window_size < (signal_size/2 + 1), "The window size must be smaller than the signal size in complex space");

int test_r2c_window_c2r() {

	// Padded array for in-place transforms
	float  input_signals[batches][2 * complex_signal_size] = {};
	float output_signals[batches][2 * complex_signal_size];
	float      reference[batches][2 * complex_signal_size];

	init_input_signals(batches, signal_size, &input_signals[0][0]);

	const size_t complex_size_bytes = batches * complex_signal_size * 2 * sizeof(float);

	// Allocate and copy input from host to GPU
	float *device_signals;
	CHECK_ERROR(hipMalloc((void **)&device_signals, complex_size_bytes));
	CHECK_ERROR(hipMemcpy(device_signals, input_signals, complex_size_bytes, hipMemcpyHostToDevice));

	// Define a structure used to pass in the window size
	cb_params host_params;
	host_params.window_N    = window_size;
	host_params.signal_size = complex_signal_size;

	// Allocate and copy callback parameters from host to GPU
	cb_params *device_params;
	CHECK_ERROR(hipMalloc((void **)&device_params, sizeof(cb_params)));
	CHECK_ERROR(hipMemcpy(device_params, &host_params, sizeof(cb_params), hipMemcpyHostToDevice));

	// Create a CUFFT plan for the forward transform, and a cuFFT plan for the inverse transform with load callback
	hipfftHandle forward_plan, inverse_plan_cb;
	size_t work_size;

	CHECK_ERROR(hipfftCreate(&forward_plan));
	CHECK_ERROR(hipfftCreate(&inverse_plan_cb));

	CHECK_ERROR(hipfftMakePlan1d(forward_plan, signal_size, HIPFFT_R2C, batches, &work_size));
	CHECK_ERROR(hipfftMakePlan1d(inverse_plan_cb, signal_size, HIPFFT_C2R, batches, &work_size));

	// NOTE: The host needs to get a copy of the device pointer to the callback. Not required for LTO callback
	hipfftCallbackLoadC host_callback_ptr;
	CHECK_ERROR(hipMemcpyFromSymbol(&host_callback_ptr, HIP_SYMBOL(device_callback_ptr), sizeof(host_callback_ptr)));

	// Now associate the load callback with the plan.
	CHECK_ERROR(hipfftXtSetCallback(inverse_plan_cb, (void **)&host_callback_ptr, HIPFFT_CB_LD_COMPLEX, (void **)&device_params));

	// Transform signal forward
	printf("Transforming signal hipfftExecR2C\n");
	CHECK_ERROR(hipfftExecR2C(forward_plan,    (hipfftReal *)device_signals, (hipfftComplex *)device_signals));

	// Apply window via load callback and inverse-transform the signal
	printf("Transforming signal hipfftExecC2R\n");
	CHECK_ERROR(hipfftExecC2R(inverse_plan_cb, (hipfftComplex *)device_signals, (hipfftReal *)device_signals));

	// Copy device memory to host
	CHECK_ERROR(hipMemcpy(output_signals, device_signals, complex_size_bytes, hipMemcpyDeviceToHost));

	// Destroy CUFFT context
	CHECK_ERROR(hipfftDestroy(forward_plan));
	CHECK_ERROR(hipfftDestroy(inverse_plan_cb));

	// Cleanup memory
	CHECK_ERROR(hipFree(device_signals));
	CHECK_ERROR(hipFree(device_params));

	// Compute reference
	if(reference_r2c_window_c2r(batches, signal_size, window_size, &input_signals[0][0], &reference[0][0]) != PASS_VALUE) {
		printf("Failed to compute the reference");
		return ERROR_VALUE;
	};

	double l2_error = compute_error<float>(&reference[0][0], &output_signals[0][0], batches, signal_size);
	printf("L2 error: %e\n", l2_error);

	return (l2_error < threshold) ? PASS_VALUE : ERROR_VALUE;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  struct hipDeviceProp_t properties;
  int device;
  CHECK_ERROR(hipGetDevice(&device));
  CHECK_ERROR(hipGetDeviceProperties(&properties, device));
  if (!(properties.major >= 5)) {
    printf("cuFFT with LTO requires CUDA architecture SM5.0 or higher\n");
    return ERROR_VALUE;
  }

  return test_r2c_window_c2r();
}

