/*
 * Copyright 2024 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"


int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    hipsolverDnParams_t params = NULL;

    using data_type = hipDoubleComplex;

    const int64_t n = 3;
    const int64_t lda = 3; // lda >= n
    const int64_t ldr = n; // for validation

    /*
     *  A = | 2.0 + 1.0j | -1.0 + 0.0j | 1.0 + 2.0j |
     *      | 2.0 + 1.0j | -3.0 + 1.0j | 2.0 + 3.0j |
     *      | 1.0 + 2.0j | -1.0 + 2.0j | 0.0 + 1.0j |
     */
    std::vector<data_type> A(n*lda);
    A[0] = data_type{ 2.,1.};
    A[1] = data_type{ 2.,1.};
    A[2] = data_type{1., 2.};
    A[0 + lda] = data_type{ -1.,0.};
    A[1 + lda] = data_type{ -3.,1.};
    A[2 + lda] = data_type{ -1.,2.};
    A[0 + 2*lda] = data_type{ 1.,2.};
    A[1 + 2*lda] = data_type{ 2.,3.};
    A[2 + 2*lda] = data_type{ 0.,1.};

    std::vector<data_type> W(n);

    /* Compute only right eigenvectors.*/
    hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_VECTOR;
    const int64_t ldvl = 1; // ldvl >= 1 if jobvl = HIPSOLVER_EIG_MODE_NOVECTOR
    const int64_t ldvr = 3; // ldvr >= n if jobvr = HIPSOLVER_EIG_MODE_VECTOR
    std::vector<data_type> VR(ldvr * n);

    data_type *d_A = nullptr;
    data_type *d_W = nullptr;
    data_type *d_VR = nullptr;
    data_type *d_VL = nullptr;
    data_type *d_R = nullptr; // for validation R = A * VR - VR * W

    int *d_info = nullptr;
    int info = 0;
    const data_type h_minus_one = data_type{-1.0,0.0};
    const data_type h_one = data_type{1.0,0.0};
    size_t workspaceInBytesOnDevice = 0;
    size_t workspaceInBytesOnHost = 0;
    void *d_work = nullptr;
    void *h_work = nullptr;

    std::printf("A = (matlab base -1) \n");
    print_matrix(n, n, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(data_type) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_VR), sizeof(data_type) * VR.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(),
                               hipMemcpyHostToDevice, stream));

    /* step 3: query size of workspace and allocate host and device buffers*/
    CUSOLVER_CHECK(cusolverDnXgeev_bufferSize(
        cusolverH,
        params,
        jobvl,
        jobvr,
        n,
        traits<data_type>::cuda_data_type,
        d_A,
        lda,
        traits<data_type>::cuda_data_type,
        d_W,
        traits<data_type>::cuda_data_type,
        d_VL,
        ldvl,
        traits<data_type>::cuda_data_type,
        d_VR,
        ldvr,
        traits<data_type>::cuda_data_type,
        &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    CUDA_CHECK(hipHostMalloc(&h_work, workspaceInBytesOnHost)); // pinned host memory for best performance
    CUDA_CHECK(hipMalloc(&d_work, workspaceInBytesOnDevice));

    /* step 4: compute eigenvalues and eigenvectors */
    CUSOLVER_CHECK(cusolverDnXgeev(
        cusolverH,
        params,
        jobvl, jobvr,
        n,
        traits<data_type>::cuda_data_type,
        d_A,
        lda,
        traits<data_type>::cuda_data_type,
        d_W,
        traits<data_type>::cuda_data_type,
        d_VL,
        ldvl,
        traits<data_type>::cuda_data_type,
        d_VR,
        ldvr,
        traits<data_type>::cuda_data_type,
        d_work,
        workspaceInBytesOnDevice,
        h_work,
        workspaceInBytesOnHost,
        d_info));

    CUDA_CHECK(hipMemcpyAsync(VR.data(), d_VR, sizeof(data_type) * VR.size(),
               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(W.data(), d_W, sizeof(data_type) * W.size(),
               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int),
               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
    } else if (0 < info) {
        std::printf("info = %d : Xgeev did not converge \n", info);
    } else {
        std::printf("info = %d : Xgeev converged\n", info);

        std::printf("eigenvalues = (matlab base -1) \n");
        print_matrix(n, 1, W.data(), 1);
        std::printf("=====\n");

        std::printf("VR = \n");
        print_matrix(n, n, VR.data(), ldvr);
        std::printf("=====\n");

        /* step 5: Verify the results */
        // Xgeev overwrites d_A. Reset d_A to the original input matrix.
        CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(),
                                hipMemcpyHostToDevice, stream));
        // R = VR * diag(W)
        CUBLAS_CHECK(hipblasZdgmm(cublasH, HIPBLAS_SIDE_RIGHT, n, n, d_VR, ldvr,
            d_W, 1, d_R, ldr));
        // R = R - A * VR
        CUBLAS_CHECK(hipblasZgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, n, n, &h_minus_one, d_A, lda, d_VR, ldvr, &h_one, d_R, ldr));
        double dR_nrm = 0.0;
        CUBLAS_CHECK(hipblasDznrm2(cublasH, n*ldr, d_R, 1, &dR_nrm));
        CUDA_CHECK(hipStreamSynchronize(stream));
        std::printf("|A*VR - VR*diag(W)| = %E \n", dR_nrm);
    }

    return EXIT_SUCCESS;
}