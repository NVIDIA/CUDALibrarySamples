/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"


int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    hipsolverDnParams_t params = NULL;

    using data_type = hipDoubleComplex;

    const int64_t n = 3;
    const int64_t lda = 3; // lda >= n
    const int64_t ldr = n; // for validation

    /*
     *  A = | 2.0 + 1.0j | -1.0 + 0.0j | 1.0 + 2.0j |
     *      | 2.0 + 1.0j | -3.0 + 1.0j | 2.0 + 3.0j |
     *      | 1.0 + 2.0j | -1.0 + 2.0j | 0.0 + 1.0j |
     */
    std::vector<data_type> A(n*lda);
    A[0] = data_type{ 2.,1.};
    A[1] = data_type{ 2.,1.};
    A[2] = data_type{1., 2.};
    A[0 + lda] = data_type{ -1.,0.};
    A[1 + lda] = data_type{ -3.,1.};
    A[2 + lda] = data_type{ -1.,2.};
    A[0 + 2*lda] = data_type{ 1.,2.};
    A[1 + 2*lda] = data_type{ 2.,3.};
    A[2 + 2*lda] = data_type{ 0.,1.};

    std::vector<data_type> W(n);

    /* Compute only right eigenvectors.*/
    hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_VECTOR;
    const int64_t ldvl = 1; // ldvl >= 1 if jobvl = HIPSOLVER_EIG_MODE_NOVECTOR
    const int64_t ldvr = 3; // ldvr >= n if jobvr = HIPSOLVER_EIG_MODE_VECTOR
    std::vector<data_type> VR(ldvr * n);

    data_type *d_A = nullptr;
    data_type *d_W = nullptr;
    data_type *d_VR = nullptr;
    data_type *d_VL = nullptr;
    data_type *d_R = nullptr; // for validation R = A * VR - VR * W

    int *d_info = nullptr;
    int info = 0;
    const data_type h_minus_one = data_type{-1.0,0.0};
    const data_type h_one = data_type{1.0,0.0};
    size_t workspaceInBytesOnDevice = 0;
    size_t workspaceInBytesOnHost = 0;
    void *d_work = nullptr;
    void *h_work = nullptr;

    std::printf("A = (matlab base -1) \n");
    print_matrix(n, n, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(data_type) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_VR), sizeof(data_type) * VR.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(),
                               hipMemcpyHostToDevice, stream));

    /* step 3: query size of workspace and allocate host and device buffers*/
    CUSOLVER_CHECK(cusolverDnXgeev_bufferSize(
        cusolverH,
        params,
        jobvl,
        jobvr,
        n,
        traits<data_type>::cuda_data_type,
        d_A,
        lda,
        traits<data_type>::cuda_data_type,
        d_W,
        traits<data_type>::cuda_data_type,
        d_VL,
        ldvl,
        traits<data_type>::cuda_data_type,
        d_VR,
        ldvr,
        traits<data_type>::cuda_data_type,
        &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    CUDA_CHECK(hipHostMalloc(&h_work, workspaceInBytesOnHost)); // pinned host memory for best performance
    CUDA_CHECK(hipMalloc(&d_work, workspaceInBytesOnDevice));

    /* step 4: compute eigenvalues and eigenvectors */
    CUSOLVER_CHECK(cusolverDnXgeev(
        cusolverH,
        params,
        jobvl, jobvr,
        n,
        traits<data_type>::cuda_data_type,
        d_A,
        lda,
        traits<data_type>::cuda_data_type,
        d_W,
        traits<data_type>::cuda_data_type,
        d_VL,
        ldvl,
        traits<data_type>::cuda_data_type,
        d_VR,
        ldvr,
        traits<data_type>::cuda_data_type,
        d_work,
        workspaceInBytesOnDevice,
        h_work,
        workspaceInBytesOnHost,
        d_info));

    CUDA_CHECK(hipMemcpyAsync(VR.data(), d_VR, sizeof(data_type) * VR.size(),
               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(W.data(), d_W, sizeof(data_type) * W.size(),
               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int),
               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
    } else if (0 < info) {
        std::printf("info = %d : Xgeev did not converge \n", info);
    } else {
        std::printf("info = %d : Xgeev converged\n", info);

        std::printf("eigenvalues = (matlab base -1) \n");
        print_matrix(n, 1, W.data(), 1);
        std::printf("=====\n");

        std::printf("VR = \n");
        print_matrix(n, n, VR.data(), ldvr);
        std::printf("=====\n");

        /* step 5: Verify the results */
        // Xgeev overwrites d_A. Reset d_A to the original input matrix.
        CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(),
                                hipMemcpyHostToDevice, stream));
        // R = VR * diag(W)
        CUBLAS_CHECK(hipblasZdgmm(cublasH, HIPBLAS_SIDE_RIGHT, n, n, d_VR, ldvr,
            d_W, 1, d_R, ldr));
        // R = R - A * VR
        CUBLAS_CHECK(hipblasZgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, n, n, &h_minus_one, d_A, lda, d_VR, ldvr, &h_one, d_R, ldr));
        double dR_nrm = 0.0;
        CUBLAS_CHECK(hipblasDznrm2(cublasH, n*ldr, d_R, 1, &dR_nrm));
        CUDA_CHECK(hipStreamSynchronize(stream));
        std::printf("|A*VR - VR*diag(W)| = %E \n", dR_nrm);
    }

    return EXIT_SUCCESS;
}