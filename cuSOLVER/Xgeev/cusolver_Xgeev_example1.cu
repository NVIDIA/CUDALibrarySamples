/*
 * Copyright 2024 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"


int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    hipsolverDnParams_t params = NULL;

    using data_type = double;

    const int64_t n = 3;
    const int64_t lda = 3; // lda >= n
    const int64_t ldr = n; // for validation

    /*
     *  A = | 1.0 | 2.0 | -3.0 |
     *      | 7.0 | 4.0 | -2.0 |
     *      | 4.0 | 2.0 |  1.0 |
     */
    const std::vector<data_type> A = {1.0, 7.0, 4.0, 2.0, 4.0, 2.0, -3.0, -2.0, 1.0};

    /* The real parts and imaginary parts of the eigenvalues are stored consecutively in 2*n vector W*/
    std::vector<data_type> W(2 * n, 0);
    data_type *WR = W.data();     // the first n entries of W are the real parts
    data_type *WI = W.data() + n; // the last n entries of W are the imaginary parts

    /* Compute only right eigenvectors.*/
    hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_VECTOR;
    const int64_t ldvl = 1; // ldvl >= 1 if jobvl = HIPSOLVER_EIG_MODE_NOVECTOR
    const int64_t ldvr = 3; // ldvr >= n if jobvr = HIPSOLVER_EIG_MODE_VECTOR
    std::vector<data_type> VR(ldvr * n, 0);

    data_type *d_A = nullptr;
    data_type *d_W = nullptr;
    data_type *d_VR = nullptr;
    data_type *d_VL = nullptr;
    data_type *d_R = nullptr; // for validation R = A * VR - VR * W

    int *d_info = nullptr;
    int info = 0;
    const data_type h_minus_one = -1.0;
    const data_type h_one = 1.0;
    size_t workspaceInBytesOnDevice = 0;
    size_t workspaceInBytesOnHost = 0;
    void *d_work = nullptr;
    void *h_work = nullptr;

    std::printf("A = (matlab base -1) \n");
    print_matrix(n, n, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(data_type) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_VR), sizeof(data_type) * VR.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(),
                               hipMemcpyHostToDevice, stream));

    /* step 3: query size of workspace and allocate host and device buffers*/
    CUSOLVER_CHECK(cusolverDnXgeev_bufferSize(
        cusolverH,
        params,
        jobvl,
        jobvr,
        n,
        traits<data_type>::cuda_data_type,
        d_A,
        lda,
        traits<data_type>::cuda_data_type,
        d_W,
        traits<data_type>::cuda_data_type,
        d_VL,
        ldvl,
        traits<data_type>::cuda_data_type,
        d_VR,
        ldvr,
        traits<data_type>::cuda_data_type,
        &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    CUDA_CHECK(hipHostMalloc(&h_work, workspaceInBytesOnHost)); // pinned host memory for best performance
    CUDA_CHECK(hipMalloc(&d_work, workspaceInBytesOnDevice));

    /* step 4: compute eigenvalues and eigenvectors */
    CUSOLVER_CHECK(cusolverDnXgeev(
        cusolverH,
        params,
        jobvl, jobvr,
        n,
        traits<data_type>::cuda_data_type,
        d_A,
        lda,
        traits<data_type>::cuda_data_type,
        d_W,
        traits<data_type>::cuda_data_type,
        d_VL,
        ldvl,
        traits<data_type>::cuda_data_type,
        d_VR,
        ldvr,
        traits<data_type>::cuda_data_type,
        d_work,
        workspaceInBytesOnDevice,
        h_work,
        workspaceInBytesOnHost,
        d_info));

    CUDA_CHECK(hipMemcpyAsync(VR.data(), d_VR, sizeof(data_type) * VR.size(),
               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(W.data(), d_W, sizeof(data_type) * W.size(),
               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int),
               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
    } else if (0 < info) {
        std::printf("info = %d : Xgeev did not converge \n", info);
    } else {
        std::printf("info = %d : Xgeev converged\n", info);

        std::printf("eigenvalues = (matlab base -1) \n");
        for (int64_t k = 1; k <= n; k++) {
            printf("WR[%ld] + 1i* WI[%ld] = %E + 1i*%E\n", k, k, WR[k-1], WI[k-1]);
        }
        std::printf("=====\n");

        std::printf("VR = (matlab base -1) \n");
        print_matrix(n, n, VR.data(), ldvr);
        std::printf("=====\n");

        /* step 5: Verify the results */

        // Xgeev overwrites d_A. Reset d_A to the original input matrix.
        CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(),
                                   hipMemcpyHostToDevice, stream));
        // R = VR * diag(WR)
        CUBLAS_CHECK(hipblasDdgmm(cublasH, HIPBLAS_SIDE_RIGHT, n, n, d_VR, ldvr,
            d_W, 1, d_R, ldr));
        // R = R - A * VR
        CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, n, n, &h_minus_one, d_A, lda, d_VR, ldvr, &h_one, d_R, ldr));
        // Update residual matrix by contributions of the imaginary parts.
        for (int64_t k = 0; k < n; k++) {
            if (WI[k] != 0) {
                // R(:,k)   = R(:,k)   - VR(:,k+1) * WI(k)
                CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    n, 1, 1, &h_minus_one, d_VR + (k+1) * ldvr, ldvr, d_W+n+k, 1,
                    &h_one, d_R + k * ldvr, ldr));
                // R(:,k+1) = R(:,k+1) + VR(:,k)   * WI(k)
                CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    n, 1, 1, &h_one, d_VR + k * ldvr, ldvr, d_W+n+k, 1,
                    &h_one, d_R + (k+1) * ldvr, ldr));
                k++;
            }
        }
        double dR_nrm = 0.0;
        CUBLAS_CHECK(hipblasDnrm2(cublasH, n*ldr, d_R, 1, &dR_nrm));
        CUDA_CHECK(hipStreamSynchronize(stream));
        std::printf("|A*VR - VR*diag(W)| = %E \n", dR_nrm);
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_VR));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipHostFree(h_work));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_R));
    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}