/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"


int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;
    hipsolverDnParams_t params = NULL;

    using data_type = double;

    const int64_t n = 3;
    const int64_t lda = 3; // lda >= n
    const int64_t ldr = n; // for validation

    /*
     *  A = | 1.0 | 2.0 | -3.0 |
     *      | 7.0 | 4.0 | -2.0 |
     *      | 4.0 | 2.0 |  1.0 |
     */
    const std::vector<data_type> A = {1.0, 7.0, 4.0, 2.0, 4.0, 2.0, -3.0, -2.0, 1.0};

    /* The real parts and imaginary parts of the eigenvalues are stored consecutively in 2*n vector W*/
    std::vector<data_type> W(2 * n, 0);
    data_type *WR = W.data();     // the first n entries of W are the real parts
    data_type *WI = W.data() + n; // the last n entries of W are the imaginary parts

    /* Compute only right eigenvectors.*/
    hipsolverEigMode_t jobvl = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipsolverEigMode_t jobvr = HIPSOLVER_EIG_MODE_VECTOR;
    const int64_t ldvl = 1; // ldvl >= 1 if jobvl = HIPSOLVER_EIG_MODE_NOVECTOR
    const int64_t ldvr = 3; // ldvr >= n if jobvr = HIPSOLVER_EIG_MODE_VECTOR
    std::vector<data_type> VR(ldvr * n, 0);

    data_type *d_A = nullptr;
    data_type *d_W = nullptr;
    data_type *d_VR = nullptr;
    data_type *d_VL = nullptr;
    data_type *d_R = nullptr; // for validation R = A * VR - VR * W

    int *d_info = nullptr;
    int info = 0;
    const data_type h_minus_one = -1.0;
    const data_type h_one = 1.0;
    size_t workspaceInBytesOnDevice = 0;
    size_t workspaceInBytesOnHost = 0;
    void *d_work = nullptr;
    void *h_work = nullptr;

    std::printf("A = (matlab base -1) \n");
    print_matrix(n, n, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(data_type) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_VR), sizeof(data_type) * VR.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(),
                               hipMemcpyHostToDevice, stream));

    /* step 3: query size of workspace and allocate host and device buffers*/
    CUSOLVER_CHECK(cusolverDnXgeev_bufferSize(
        cusolverH,
        params,
        jobvl,
        jobvr,
        n,
        traits<data_type>::cuda_data_type,
        d_A,
        lda,
        traits<data_type>::cuda_data_type,
        d_W,
        traits<data_type>::cuda_data_type,
        d_VL,
        ldvl,
        traits<data_type>::cuda_data_type,
        d_VR,
        ldvr,
        traits<data_type>::cuda_data_type,
        &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    CUDA_CHECK(hipHostMalloc(&h_work, workspaceInBytesOnHost)); // pinned host memory for best performance
    CUDA_CHECK(hipMalloc(&d_work, workspaceInBytesOnDevice));

    /* step 4: compute eigenvalues and eigenvectors */
    CUSOLVER_CHECK(cusolverDnXgeev(
        cusolverH,
        params,
        jobvl, jobvr,
        n,
        traits<data_type>::cuda_data_type,
        d_A,
        lda,
        traits<data_type>::cuda_data_type,
        d_W,
        traits<data_type>::cuda_data_type,
        d_VL,
        ldvl,
        traits<data_type>::cuda_data_type,
        d_VR,
        ldvr,
        traits<data_type>::cuda_data_type,
        d_work,
        workspaceInBytesOnDevice,
        h_work,
        workspaceInBytesOnHost,
        d_info));

    CUDA_CHECK(hipMemcpyAsync(VR.data(), d_VR, sizeof(data_type) * VR.size(),
               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(W.data(), d_W, sizeof(data_type) * W.size(),
               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int),
               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
    } else if (0 < info) {
        std::printf("info = %d : Xgeev did not converge \n", info);
    } else {
        std::printf("info = %d : Xgeev converged\n", info);

        std::printf("eigenvalues = (matlab base -1) \n");
        for (int64_t k = 1; k <= n; k++) {
            printf("WR[%ld] + 1i* WI[%ld] = %E + 1i*%E\n", k, k, WR[k-1], WI[k-1]);
        }
        std::printf("=====\n");

        std::printf("VR = (matlab base -1) \n");
        print_matrix(n, n, VR.data(), ldvr);
        std::printf("=====\n");

        /* step 5: Verify the results */

        // Xgeev overwrites d_A. Reset d_A to the original input matrix.
        CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(),
                                   hipMemcpyHostToDevice, stream));
        // R = VR * diag(WR)
        CUBLAS_CHECK(hipblasDdgmm(cublasH, HIPBLAS_SIDE_RIGHT, n, n, d_VR, ldvr,
            d_W, 1, d_R, ldr));
        // R = R - A * VR
        CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, n, n, &h_minus_one, d_A, lda, d_VR, ldvr, &h_one, d_R, ldr));
        // Update residual matrix by contributions of the imaginary parts.
        for (int64_t k = 0; k < n; k++) {
            if (WI[k] != 0) {
                // R(:,k)   = R(:,k)   - VR(:,k+1) * WI(k)
                CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    n, 1, 1, &h_minus_one, d_VR + (k+1) * ldvr, ldvr, d_W+n+k, 1,
                    &h_one, d_R + k * ldvr, ldr));
                // R(:,k+1) = R(:,k+1) + VR(:,k)   * WI(k)
                CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    n, 1, 1, &h_one, d_VR + k * ldvr, ldvr, d_W+n+k, 1,
                    &h_one, d_R + (k+1) * ldvr, ldr));
                k++;
            }
        }
        double dR_nrm = 0.0;
        CUBLAS_CHECK(hipblasDnrm2(cublasH, n*ldr, d_R, 1, &dR_nrm));
        CUDA_CHECK(hipStreamSynchronize(stream));
        std::printf("|A*VR - VR*diag(W)| = %E \n", dR_nrm);
    }

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_VR));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipHostFree(h_work));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_R));
    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}