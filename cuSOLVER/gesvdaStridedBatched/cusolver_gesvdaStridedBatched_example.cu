#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int batchSize = 2;
    const int m = 3;
    const int n = 2;
    const int lda = m;
    const int ldu = m;
    const int ldv = n;
    const int rank = n;
    const long long int strideA = static_cast<long long int>(lda * n);
    const long long int strideS = n;
    const long long int strideU = static_cast<long long int>(ldu * n);
    const long long int strideV = static_cast<long long int>(ldv * n);

    /*
     *        | 1 2  |       | 10 9 |
     *   A0 = | 4 5  |, A1 = |  8 7 |
     *        | 2 1  |       |  6 5 |
     */

    const std::vector<float> A = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0, 10.0, 8.0, 6.0, 9.0, 7.0, 5.0};
    std::vector<float> U(strideU * batchSize, 0); /* left singular vectors  */
    std::vector<float> V(strideV * batchSize, 0); /* right singular vectors */
    std::vector<float> S(strideS * batchSize, 0); /* numerical singular value */

    /* exact singular values */
    const std::vector<float> S_exact = {7.065283497082729, 1.040081297712078, 18.839649186929730,
                                        0.260035600289472};

    float *d_A = nullptr;  /* device copy of A */
    float *d_S = nullptr;  /* singular values */
    float *d_U = nullptr;  /* left singular vectors */
    float *d_V = nullptr;  /* right singular vectors */
    int *d_info = nullptr; /* error info */

    int lwork = 0;           /* size of workspace */
    float *d_work = nullptr; /* device workspace for getrf */

    std::vector<int> info(batchSize, 0);     /* host copy of error info */
    std::vector<double> RnrmF(batchSize, 0); /* residual norm */

    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */

    std::printf("A0 = (matlab base-1)\n");
    print_matrix(m, n, A.data(), lda);
    std::printf("=====\n");

    std::printf("A1 = (matlab base-1)\n");
    print_matrix(m, n, A.data() + strideA, lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(float) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(float) * S.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(float) * U.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(float) * V.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * info.size()));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(float) * A.size(), hipMemcpyHostToDevice, stream));

    /* step 3: query working space of SVD */
    CUSOLVER_CHECK(hipsolverDnSgesvdaStridedBatched_bufferSize(
        cusolverH, jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
                         /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        rank,            /* number of singular values */
        m,               /* nubmer of rows of Aj, 0 <= m */
        n,               /* number of columns of Aj, 0 <= n  */
        d_A,             /* Aj is m-by-n */
        lda,             /* leading dimension of Aj */
        strideA,         /* >= lda*n */
        d_S,             /* Sj is rank-by-1, singular values in descending order */
        strideS,         /* >= rank */
        d_U,             /* Uj is m-by-rank */
        ldu,             /* leading dimension of Uj, ldu >= max(1,m) */
        strideU,         /* >= ldu*rank */
        d_V,             /* Vj is n-by-rank */
        ldv,             /* leading dimension of Vj, ldv >= max(1,n) */
        strideV,         /* >= ldv*rank */
        &lwork, batchSize /* number of matrices */
        ));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(float) * lwork));

    /* step 4: compute SVD */
    CUSOLVER_CHECK(hipsolverDnSgesvdaStridedBatched(
        cusolverH, jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
                         /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        rank,            /* number of singular values */
        m,               /* nubmer of rows of Aj, 0 <= m */
        n,               /* number of columns of Aj, 0 <= n  */
        d_A,             /* Aj is m-by-n */
        lda,             /* leading dimension of Aj */
        strideA,         /* >= lda*n */
        d_S,             /* Sj is rank-by-1 */
                         /* the singular values in descending order */
        strideS,         /* >= rank */
        d_U,             /* Uj is m-by-rank */
        ldu,             /* leading dimension of Uj, ldu >= max(1,m) */
        strideU,         /* >= ldu*rank */
        d_V,             /* Vj is n-by-rank */
        ldv,             /* leading dimension of Vj, ldv >= max(1,n) */
        strideV,         /* >= ldv*rank */
        d_work, lwork, d_info, RnrmF.data(), batchSize /* number of matrices */
        ));

    CUDA_CHECK(
        hipMemcpyAsync(U.data(), d_U, sizeof(float) * U.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_V, sizeof(float) * V.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(S.data(), d_S, sizeof(float) * S.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(info.data(), d_info, sizeof(int) * info.size(),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 > info[0]) {
        std::printf("%d-th parameter is wrong \n", -info[0]);
        exit(1);
    }
    for (int idx = 0; idx < batchSize; idx++) {
        if (0 == info[idx]) {
            std::printf("%d-th matrix, gesvda converges \n", idx);
        } else {
            std::printf("WARNING: info[%d] = %d : gesvda does not converge \n", idx, info[idx]);
        }
    }

    std::printf("S0 = (matlab base-1)\n");
    print_matrix(rank, 1, S.data(), n);
    std::printf("=====\n");

    std::printf("U0 = (matlab base-1)\n");
    print_matrix(m, rank, U.data(), ldu);
    std::printf("=====\n");

    std::printf("V) = (matlab base-1)\n");
    print_matrix(n, rank, V.data(), ldv);
    std::printf("=====\n");

    float ds_sup = 0;
    for (int j = 0; j < n; j++) {
        float err = fabs(S[j] - S_exact[j]);
        ds_sup = (ds_sup > err) ? ds_sup : err;
    }
    std::printf("|S0 - S0_exact|_sup = %E \n", ds_sup);

    std::printf("residual |A0 - U0*S0*V0**H|_F = %E \n", RnrmF[0]);

    std::printf("S1 = (matlab base-1)\n");
    print_matrix(rank, 1, S.data() + strideS, n);
    std::printf("=====\n");

    std::printf("U1 = (matlab base-1)\n");
    print_matrix(m, rank, U.data() + strideU, ldu);
    std::printf("=====\n");

    std::printf("V1 = (matlab base-1)\n");
    print_matrix(n, rank, V.data() + strideV, ldv);
    std::printf("=====\n");

    ds_sup = 0;
    for (int j = 0; j < n; j++) {
        float err = fabs(S[strideS + j] - S_exact[strideS + j]);
        ds_sup = (ds_sup > err) ? ds_sup : err;
    }
    std::printf("|S1 - S1_exact|_sup = %E \n", ds_sup);

    std::printf("residual |A1 - U1*S1*V1**H|_F = %E \n", RnrmF[1]);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_S));
    CUDA_CHECK(hipFree(d_U));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
