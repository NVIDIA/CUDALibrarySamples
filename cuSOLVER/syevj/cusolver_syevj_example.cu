#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;

    const int m = 3;
    const int lda = m;
    /*
     *       | 3.5 0.5 0 |
     *   A = | 0.5 3.5 0 |
     *       | 0   0   2 |
     */

    const std::vector<double> A = {3.5, 0.5, 0, 0.5, 3.5, 0.0, 0.0, 0.0, 2.0};
    const std::vector<double> lambda = {2.0, 3.0, 4.0};

    std::vector<double> V(lda * m); // eigenvectors
    std::vector<double> W(m);       // eigenvalues

    double *d_A = nullptr;
    double *d_W = nullptr;
    int *devInfo = nullptr;
    double *d_work = nullptr;
    int lwork = 0;
    int info_gpu = 0;

    /* configuration of syevj  */
    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    /* numerical results of syevj  */
    double residual = 0;
    int executed_sweeps = 0;

    printf("tol = %E, default value is machine zero \n", tol);
    printf("max. sweeps = %d, default value is 100\n", max_sweeps);

    printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: configuration of syevj */
    CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));

    /* default value of tolerance is machine zero */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params, tol));

    /* default value of max. sweeps is 100 */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps));

    /* step 3: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * lda * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&devInfo), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * lda * m, hipMemcpyHostToDevice, stream));

    /* step 4: query working space of syevj */
    CUSOLVER_CHECK(
        hipsolverDnDsyevj_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W, &lwork, syevj_params));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 5: compute eigen-pair   */
    CUSOLVER_CHECK(hipsolverDnDsyevj(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork, devInfo,
                                    syevj_params));

    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_A, sizeof(double) * lda * m, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(W.data(), d_W, sizeof(double) * m, hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 == info_gpu) {
        printf("syevj converges \n");
    } else if (0 > info_gpu) {
        printf("%d-th parameter is wrong \n", -info_gpu);
        exit(1);
    } else {
        printf("WARNING: info = %d : syevj does not converge \n", info_gpu);
    }

    printf("Eigenvalue = (matlab base-1), ascending order\n");
    for (int i = 0; i < m; i++) {
        printf("W[%d] = %E\n", i + 1, W[i]);
    }

    printf("V = (matlab base-1)\n");
    print_matrix(m, m, V.data(), lda);
    printf("=====\n");

    /* step 6: check eigenvalues */
    double lambda_sup = 0;
    for (int i = 0; i < m; i++) {
        double error = fabs(lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error) ? lambda_sup : error;
    }
    printf("|lambda - W| = %E\n", lambda_sup);

    CUSOLVER_CHECK(hipsolverDnXsyevjGetSweeps(cusolverH, syevj_params, &executed_sweeps));

    CUSOLVER_CHECK(hipsolverDnXsyevjGetResidual(cusolverH, syevj_params, &residual));

    printf("residual |A - V*W*V**H|_F = %E \n", residual);
    printf("number of executed sweeps = %d \n", executed_sweeps);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(devInfo));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
