/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    using data_type = double;

    const int64_t m = 3;
    const int64_t lda = m;
    const int64_t ldb = m;

    /*
     *     | 1     2     3 |
     * A = | 2     5     5 | = L0 * L0**T
     *     | 3     5    12 |
     *
     *            | 1.0000         0         0 |
     * where L0 = | 2.0000    1.0000         0 |
     *            | 3.0000   -1.0000    1.4142 |
     *
     */

    const std::vector<data_type> A = {1.0, 2.0, 3.0, 2.0, 5.0, 5.0, 3.0, 5.0, 12.0};
    const std::vector<data_type> B = {1.0, 2.0, 3.0};
    std::vector<data_type> X(m, 0);
    std::vector<data_type> L(lda * m, 0);
    int info = 0;

    data_type *d_A = nullptr; /* device copy of A */
    data_type *d_B = nullptr; /* device copy of B */
    int *d_info = nullptr;    /* error info */

    size_t d_lwork = 0;     /* size of workspace */
    void *d_work = nullptr; /* device workspace */
    size_t h_lwork = 0;     /* size of workspace */
    void *h_work = nullptr; /* host workspace */

    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");

    std::printf("B = (matlab base-1)\n");
    print_matrix(m, 1, B.data(), ldb);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: query working space */
    CUSOLVER_CHECK(hipsolverDnXpotrf_bufferSize(
        cusolverH, NULL, uplo, m, traits<data_type>::cuda_data_type, d_A, lda,
        traits<data_type>::cuda_data_type, &d_lwork, &h_lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(data_type) * d_lwork));

    /* step 4: Cholesky factorization */
    CUSOLVER_CHECK(hipsolverDnXpotrf(cusolverH, NULL, uplo, m, traits<data_type>::cuda_data_type,
                                    d_A, lda, traits<data_type>::cuda_data_type, d_work, d_lwork,
                                    h_work, h_lwork, d_info));

    CUDA_CHECK(hipMemcpyAsync(L.data(), d_A, sizeof(data_type) * A.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after Xpotrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    std::printf("L = (matlab base-1)\n");
    print_matrix(m, m, L.data(), lda);
    std::printf("=====\n");

    /*
     * step 5: solve A*X = B
     *       | 1 |       | -0.3333 |
     *   B = | 2 |,  X = |  0.6667 |
     *       | 3 |       |  0      |
     *
     */

    CUSOLVER_CHECK(hipsolverDnXpotrs(cusolverH, NULL, uplo, m, 1, /* nrhs */
                                    traits<data_type>::cuda_data_type, d_A, lda,
                                    traits<data_type>::cuda_data_type, d_B, ldb, d_info));

    CUDA_CHECK(hipMemcpyAsync(X.data(), d_B, sizeof(data_type) * X.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("X = (matlab base-1)\n");
    print_matrix(m, 1, X.data(), ldb);
    std::printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
