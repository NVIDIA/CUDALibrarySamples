#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;

    const int m = 3;
    const int lda = m;
    const int batchSize = 2;
    /*
     *        |  1  -1   0 |
     *   A0 = | -1   2   0 |
     *        |  0   0   0 |
     *
     *   A0 = V0 * W0 * V0**T
     *
     *   W0 = diag(0, 0.3820, 2.6180)
     *
     *        |  3   4  0 |
     *   A1 = |  4   7  0 |
     *        |  0   0  0 |
     *
     *   A1 = V1 * W1 * V1**T
     *
     *   W1 = diag(0, 0.5279, 9.4721)
     *
     */

    std::vector<double> A(lda * m * batchSize, 0); /* V = [A0 ; A1] */
    std::vector<double> V(lda * m * batchSize, 0); /* V = [V0 ; V1] */
    std::vector<double> W(m * batchSize, 0);       /* W = [W0 ; W1] */
    std::vector<int> info(batchSize, 0);           /* info = [info0 ; info1] */

    double *d_A = nullptr;    /* lda-by-m-by-batchSize */
    double *d_W = nullptr;    /* m-by-batchSize */
    int *d_info = nullptr;    /* batchSize */
    double *d_work = nullptr; /* device workspace for syevjBatched */
    int lwork = 0;            /* size of workspace */

    /* configuration of syevj  */
    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const int sort_eig = 0;                                  /* don't sort eigenvalues */
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    double *A0 = A.data();
    double *A1 = A.data() + lda * m;
    /*
     *        |  1  -1   0 |
     *   A0 = | -1   2   0 |
     *        |  0   0   0 |
     *   A0 is column-major
     */
    A0[0 + 0 * lda] = 1.0;
    A0[1 + 0 * lda] = -1.0;
    A0[2 + 0 * lda] = 0.0;

    A0[0 + 1 * lda] = -1.0;
    A0[1 + 1 * lda] = 2.0;
    A0[2 + 1 * lda] = 0.0;

    A0[0 + 2 * lda] = 0.0;
    A0[1 + 2 * lda] = 0.0;
    A0[2 + 2 * lda] = 0.0;
    /*
     *        |  3   4  0 |
     *   A1 = |  4   7  0 |
     *        |  0   0  0 |
     *   A1 is column-major
     */
    A1[0 + 0 * lda] = 3.0;
    A1[1 + 0 * lda] = 4.0;
    A1[2 + 0 * lda] = 0.0;

    A1[0 + 1 * lda] = 4.0;
    A1[1 + 1 * lda] = 7.0;
    A1[2 + 1 * lda] = 0.0;

    A1[0 + 2 * lda] = 0.0;
    A1[1 + 2 * lda] = 0.0;
    A1[2 + 2 * lda] = 0.0;

    std::printf("A0 = (matlab base-1)\n");
    print_matrix(m, m, A0, lda);
    std::printf("=====\n");

    std::printf("A1 = (matlab base-1)\n");
    print_matrix(m, m, A1, lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: configuration of syevj */
    CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));

    /* default value of tolerance is machine zero */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params, tol));

    /* default value of max. sweeps is 100 */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps));

    /* disable sorting */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig));

    /* step 3: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * info.size()));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));
    /* step 4: query working space of syevj */
    CUSOLVER_CHECK(hipsolverDnDsyevjBatched_bufferSize(cusolverH, jobz, uplo, m, d_A, lda, d_W,
                                                      &lwork, syevj_params, batchSize));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 5: compute eigen-pair   */
    CUSOLVER_CHECK(hipsolverDnDsyevjBatched(cusolverH, jobz, uplo, m, d_A, lda, d_W, d_work, lwork,
                                           d_info, syevj_params, batchSize));

    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_A, sizeof(double) * A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(W.data(), d_W, sizeof(double) * W.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(info.data(), d_info, sizeof(int) * info.size(),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int i = 0; i < batchSize; i++) {
        if (0 == info[i]) {
            std::printf("matrix %d: syevj converges \n", i);
        } else if (0 > info[i]) {
            /* only info[0] shows if some input parameter is wrong.
             * If so, the error is HIPSOLVER_STATUS_INVALID_VALUE.
             */
            std::printf("Error: %d-th parameter is wrong \n", -info[i]);
            exit(1);
        } else { /* info = m+1 */
                 /* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
            std::printf("WARNING: matrix %d, info = %d : sygvj does not converge \n", i, info[i]);
        }
    }

    /* Step 6: show eigenvalues and eigenvectors */
    double *W0 = W.data();
    double *W1 = W.data() + m;

    std::printf("==== \n");
    for (int i = 0; i < m; i++) {
        std::printf("W0[%d] = %f\n", i, W0[i]);
    }
    std::printf("==== \n");
    for (int i = 0; i < m; i++) {
        std::printf("W1[%d] = %f\n", i, W1[i]);
    }
    std::printf("==== \n");

    double *V0 = V.data();
    double *V1 = V.data() + lda * m;

    std::printf("V0 = (matlab base-1)\n");
    print_matrix(m, m, V0, lda);
    std::printf("V1 = (matlab base-1)\n");
    print_matrix(m, m, V1, lda);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}