#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;

    const int m = 3;
    const int lda = m;
    /*
     *       | 3.5 0.5 0 |
     *   A = | 0.5 3.5 0 |
     *       | 0   0   2 |
     *
     *       | 10  2   3 |
     *   B = | 2  10   5 |
     *       | 3   5  10 |
     */

    const std::vector<double> A = {3.5, 0.5, 0, 0.5, 3.5, 0.0, 0.0, 0.0, 2.0};
    const std::vector<double> B = {10.0, 2.0, 3.0, 2.0, 10.0, 5.0, 3.0, 5.0, 10.0};
    const std::vector<double> lambda = {0.158660256604, 0.370751508101882, 0.6};

    std::vector<double> V(lda * m, 0); // eigenvectors
    std::vector<double> W(m, 0);       // eigenvalues

    double *d_A = nullptr;
    double *d_B = nullptr;
    double *d_W = nullptr;
    int *d_info = nullptr;
    double *d_work = nullptr;
    int lwork = 0;
    int info = 0;

    /* configuration of syevj  */
    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1;     // A*x = (lambda)*B*x
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    /* numerical results of syevj  */
    double residual = 0;
    int executed_sweeps = 0;

    std::printf("tol = %E, default value is machine zero \n", tol);
    std::printf("max. sweeps = %d, default value is 100\n", max_sweeps);

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");

    std::printf("B = (matlab base-1)\n");
    print_matrix(m, m, B.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: configuration of syevj */
    CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));

    /* default value of tolerance is machine zero */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params, tol));

    /* default value of max. sweeps is 100 */
    CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps));

    /* step 3: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(
        hipMemcpyAsync(d_B, B.data(), sizeof(double) * B.size(), hipMemcpyHostToDevice, stream));

    /* step 4: query working space of sygvj */
    CUSOLVER_CHECK(hipsolverDnDsygvj_bufferSize(cusolverH, itype, jobz, uplo, m, d_A, lda, d_B,
                                               lda, /* ldb */
                                               d_W, &lwork, syevj_params));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 5: compute spectrum of (A,B) */
    CUSOLVER_CHECK(hipsolverDnDsygvj(cusolverH, itype, jobz, uplo, m, d_A, lda, d_B, lda, /* ldb */
                                    d_W, d_work, lwork, d_info, syevj_params));

    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_A, sizeof(double) * A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(W.data(), d_W, sizeof(double) * W.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 == info) {
        std::printf("sygvj converges \n");
    } else if (0 > info) {
        std::printf("Error: %d-th parameter is wrong \n", -info);
        exit(1);
    } else if (m >= info) {
        std::printf("Error: leading minor of order %d of B is not positive definite\n", -info);
        exit(1);
    } else { /* info = m+1 */
        std::printf("WARNING: info = %d : sygvj does not converge \n", info);
    }

    std::printf("Eigenvalue = (matlab base-1), ascending order\n");
    int idx = 1;
    for (auto const &i : W) {
        std::printf("W[%i] = %E\n", idx, i);
        idx++;
    }

    std::printf("V = (matlab base-1)\n");
    print_matrix(m, m, V.data(), lda);
    std::printf("=====\n");

    /* step 6: check eigenvalues */
    double lambda_sup = 0;
    for (int i = 0; i < m; i++) {
        double error = fabs(lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error) ? lambda_sup : error;
    }
    std::printf("|lambda - W| = %E\n", lambda_sup);

    CUSOLVER_CHECK(hipsolverDnXsyevjGetSweeps(cusolverH, syevj_params, &executed_sweeps));

    CUSOLVER_CHECK(hipsolverDnXsyevjGetResidual(cusolverH, syevj_params, &residual));

    std::printf("residual |A - V*W*V**H|_F = %E \n", residual);
    std::printf("number of executed sweeps = %d \n", executed_sweeps);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}