/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cassert>
#include <cstring>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, const char *argv[]) {
    bool verbose = false;

    // Matrix size
    const int N = 1024;

    // Numer of right hand sides
    const int nrhs = 1;

    // Use double precision matrix and half precision factorization
    typedef double T;
    const cusolverPrecType_t matrix_precision = CUSOLVER_R_64F;
    // make sure that you specify matrix precision that matches to the data type
    assert(traits<T>::cusolver_precision_type == matrix_precision);
    const cusolverPrecType_t compute_lower_precision = CUSOLVER_R_16F;

    // Use GMRES refinement solver
    const cusolverIRSRefinement_t refinement_solver = CUSOLVER_IRS_REFINE_GMRES;

    T *hA;
    int lda;
    T *hB;
    int ldb;
    T *hX;
    int ldx;

    hipStream_t stream;
    hipEvent_t event_start, event_end;
    hipsolverHandle_t handle;
    cusolverDnIRSParams_t gesv_params;
    cusolverDnIRSInfos_t gesv_info;

    std::cout << "Generating matrix A on host..." << std::endl;
    generate_random_matrix<T>(N, N, &hA, &lda);
    std::cout << "make A diagonal dominant..." << std::endl;
    make_diag_dominant_matrix<T>(N, N, hA, lda);
    std::cout << "Generating matrix B on host..." << std::endl;
    generate_random_matrix<T>(nrhs, N, &hB, &ldb);
    std::cout << "Generating matrix X on host..." << std::endl;
    generate_random_matrix<T>(nrhs, N, &hX, &ldx);

    if (verbose) {
        std::cout << "A: \n";
        print_matrix(N, N, hA, lda);
        std::cout << "B: \n";
        print_matrix(nrhs, N, hB, ldb);
        std::cout << "X: \n";
        print_matrix(nrhs, N, hX, ldx);
    }

    std::cout << "Initializing CUDA..." << std::endl;
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK(hipEventCreate(&event_start));
    CUDA_CHECK(hipEventCreate(&event_end));
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    CUSOLVER_CHECK(hipsolverSetStream(handle, stream));

    std::cout << "Setting up gesv() parameters..." << std::endl;
    // create solver parameters
    CUSOLVER_CHECK(cusolverDnIRSParamsCreate(&gesv_params));
    // set matrix precision and factorization precision
    CUSOLVER_CHECK(cusolverDnIRSParamsSetSolverPrecisions(gesv_params, matrix_precision,
                                                          compute_lower_precision));
    // set refinement solver
    CUSOLVER_CHECK(cusolverDnIRSParamsSetRefinementSolver(gesv_params, refinement_solver));
    // create solve info structure
    CUSOLVER_CHECK(cusolverDnIRSInfosCreate(&gesv_info));

    // matrix on device
    T *dA;
    int ldda = ALIGN_TO(N * sizeof(T), device_alignment) / sizeof(T);
    // right hand side on device
    T *dB;
    int lddb = ALIGN_TO(N * sizeof(T), device_alignment) / sizeof(T);
    // solution on device
    T *dX;
    int lddx = ALIGN_TO(N * sizeof(T), device_alignment) / sizeof(T);

    // pivot sequence on device
    int *dipiv;
    // info indicator on device
    int *dinfo;
    // work buffer
    void *dwork;
    // size of work buffer
    size_t dwork_size;
    // number of refinement iterations returned by solver
    int iter;

    std::cout << "Allocating memory on device..." << std::endl;
    // allocate data
    CUDA_CHECK(hipMalloc(&dA, ldda * N * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dB, lddb * nrhs * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dX, lddx * nrhs * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dipiv, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&dinfo, sizeof(int)));

    // copy input data
    CUDA_CHECK(hipMemcpy2D(dA, ldda * sizeof(T), hA, lda * sizeof(T), N * sizeof(T), N,
                            hipMemcpyDefault));
    CUDA_CHECK(hipMemcpy2D(dB, lddb * sizeof(T), hB, ldb * sizeof(T), N * sizeof(T), nrhs,
                            hipMemcpyDefault));

    // get required device work buffer size
    CUSOLVER_CHECK(cusolverDnIRSXgesv_bufferSize(handle, gesv_params, N, nrhs, &dwork_size));
    std::cout << "Workspace is " << dwork_size << " bytes" << std::endl;
    CUDA_CHECK(hipMalloc(&dwork, dwork_size));

    std::cout << "Solving matrix on device..." << std::endl;
    CUDA_CHECK(hipEventRecord(event_start, stream));

    hipsolverStatus_t gesv_status =
        cusolverDnIRSXgesv(handle, gesv_params, gesv_info, N, nrhs, dA, ldda, dB, lddb, dX, lddx,
                           dwork, dwork_size, &iter, dinfo);
    CUSOLVER_CHECK(gesv_status);

    CUDA_CHECK(hipEventRecord(event_end, stream));
    // check solve status
    int info = 0;
    CUDA_CHECK(
        hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "Solve info is: " << info << ", iter is: " << iter << std::endl;

    CUDA_CHECK(hipMemcpy2D(hX, ldx * sizeof(T), dX, lddx * sizeof(T), N * sizeof(T), nrhs,
                            hipMemcpyDefault));
    if (verbose) {
        std::cout << "X:\n";
        print_matrix(nrhs, N, hX, ldx);
    }

    CUDA_CHECK(hipGetLastError());

    float solve_time = 0.f;
    CUDA_CHECK(hipEventElapsedTime(&solve_time, event_start, event_end));
    std::cout << "Solved matrix " << N << "x" << N << " with " << nrhs << " right hand sides in "
              << solve_time << "ms" << std::endl;

    std::cout << "Releasing resources..." << std::endl;
    CUDA_CHECK(hipFree(dwork));
    CUDA_CHECK(hipFree(dinfo));
    CUDA_CHECK(hipFree(dipiv));
    CUDA_CHECK(hipFree(dX));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dA));

    free(hA);
    free(hB);
    free(hX);

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));
    CUDA_CHECK(hipEventDestroy(event_start));
    CUDA_CHECK(hipEventDestroy(event_end));
    CUDA_CHECK(hipStreamDestroy(stream));

    std::cout << "Done!" << std::endl;

    return 0;
}
