/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstring>
#include <functional>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, const char *argv[]) {
    bool verbose = false;

    // Matrix size
    const int N = 1024;

    // Numer of right hand sides
    const int nrhs = 1;

    // Use double precision matrix and half precision factorization
    typedef double T;
    // Select appropriate functions for chosen precisions
    auto cusolver_gesv_buffersize = cusolverDnDHgesv_bufferSize;
    auto cusolver_gesv = cusolverDnDHgesv;

    T *hA;
    int lda;
    T *hB;
    int ldb;
    T *hX;
    int ldx;

    hipStream_t stream;
    hipEvent_t event_start, event_end;
    hipsolverHandle_t handle;

    std::cout << "Generating matrix A on host..." << std::endl;
    generate_random_matrix<T>(N, N, &hA, &lda);
    std::cout << "make A diagonal dominant..." << std::endl;
    make_diag_dominant_matrix<T>(N, N, hA, lda);
    std::cout << "Generating matrix B on host..." << std::endl;
    generate_random_matrix<T>(nrhs, N, &hB, &ldb);
    std::cout << "Generating matrix X on host..." << std::endl;
    generate_random_matrix<T>(nrhs, N, &hX, &ldx);

    if (verbose) {
        std::cout << "A: \n";
        print_matrix(N, N, hA, lda);
        std::cout << "B: \n";
        print_matrix(nrhs, N, hB, ldb);
        std::cout << "X: \n";
        print_matrix(nrhs, N, hX, ldx);
    }

    std::cout << "Initializing CUDA..." << std::endl;
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK(hipEventCreate(&event_start));
    CUDA_CHECK(hipEventCreate(&event_end));
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    CUSOLVER_CHECK(hipsolverSetStream(handle, stream));

    // matrix on device
    T *dA;
    int ldda = ALIGN_TO(N * sizeof(T), device_alignment) / sizeof(T);
    // right hand side on device
    T *dB;
    int lddb = ALIGN_TO(N * sizeof(T), device_alignment) / sizeof(T);
    // solution on device
    T *dX;
    int lddx = ALIGN_TO(N * sizeof(T), device_alignment) / sizeof(T);

    // pivot sequence on device
    int *dipiv;
    // info indicator on device
    int *dinfo;
    // work buffer
    void *dwork;
    // size of work buffer
    size_t dwork_size;
    // number of refinement iterations returned by solver
    int iter;

    std::cout << "Allocating memory on device..." << std::endl;
    // allocate data
    CUDA_CHECK(hipMalloc(&dA, ldda * N * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dB, lddb * nrhs * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dX, lddx * nrhs * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dipiv, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&dinfo, sizeof(int)));

    // copy input data
    CUDA_CHECK(hipMemcpy2D(dA, ldda * sizeof(T), hA, lda * sizeof(T), N * sizeof(T), N,
                            hipMemcpyDefault));
    CUDA_CHECK(hipMemcpy2D(dB, lddb * sizeof(T), hB, ldb * sizeof(T), N * sizeof(T), nrhs,
                            hipMemcpyDefault));

    // get required device work buffer size
    CUSOLVER_CHECK(cusolver_gesv_buffersize(handle, N, nrhs, dA, ldda, dipiv, dB, lddb, dX, lddx,
                                            dwork, &dwork_size));
    std::cout << "Workspace is " << dwork_size << " bytes" << std::endl;
    CUDA_CHECK(hipMalloc(&dwork, dwork_size));

    std::cout << "Solving matrix on device..." << std::endl;
    CUDA_CHECK(hipEventRecord(event_start, stream));

    hipsolverStatus_t gesv_status = cusolver_gesv(handle, N, nrhs, dA, ldda, dipiv, dB, lddb, dX,
                                                 lddx, dwork, dwork_size, &iter, dinfo);
    CUSOLVER_CHECK(gesv_status);

    CUDA_CHECK(hipEventRecord(event_end, stream));
    // check solve status
    int info = 0;
    CUDA_CHECK(
        hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    std::cout << "Solve info is: " << info << ", iter is: " << iter << std::endl;

    CUDA_CHECK(hipMemcpy2D(hX, ldx * sizeof(T), dX, lddx * sizeof(T), N * sizeof(T), nrhs,
                            hipMemcpyDefault));
    if (verbose) {
        std::cout << "X:\n";
        print_matrix(nrhs, N, hX, ldx);
    }

    CUDA_CHECK(hipGetLastError());

    float solve_time = 0.f;
    CUDA_CHECK(hipEventElapsedTime(&solve_time, event_start, event_end));

    std::cout << "Releasing resources..." << std::endl;
    CUDA_CHECK(hipFree(dwork));
    CUDA_CHECK(hipFree(dinfo));
    CUDA_CHECK(hipFree(dipiv));
    CUDA_CHECK(hipFree(dX));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dA));

    free(hA);
    free(hB);
    free(hX);

    CUSOLVER_CHECK(hipsolverDnDestroy(handle));
    CUDA_CHECK(hipEventDestroy(event_start));
    CUDA_CHECK(hipEventDestroy(event_end));
    CUDA_CHECK(hipStreamDestroy(stream));

    std::cout << "Done!" << std::endl;

    return 0;
}
