#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream{};

    const int m = 3;
    const int lda = m;
    const int ldb = m;
    const int nrhs = 1; // number of right hand side vectors

    /*       | 1 2 3 |
     *   A = | 4 5 6 |
     *       | 2 1 1 |
     *
     *   x = (1 1 1)'
     *   b = (6 15 4)'
     */

    const std::vector<double> A = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0, 3.0, 6.0, 1.0};
    // std::vector<double> X = {1.0, 1.0, 1.0}; // exact solution
    const std::vector<double> B = {6.0, 15.0, 4.0};
    std::vector<double> XC(ldb * nrhs, 0); // solution matrix from GPU

    /* device memory */
    double *d_A = nullptr;
    double *d_tau = nullptr;
    double *d_B = nullptr;
    int *d_info = nullptr;
    double *d_work = nullptr;

    int lwork_geqrf = 0;
    int lwork_ormqr = 0;
    int lwork = 0;
    int info = 0;

    const double one = 1;

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");
    std::printf("B = (matlab base-1)\n");
    print_matrix(m, nrhs, B.data(), ldb);
    std::printf("=====\n");

    /* step 1: create cudense/cublas handle */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy A and B to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_tau), sizeof(double) * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(
        hipMemcpyAsync(d_B, B.data(), sizeof(double) * B.size(), hipMemcpyHostToDevice, stream));

    /* step 3: query working space of geqrf and ormqr */
    CUSOLVER_CHECK(hipsolverDnDgeqrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork_geqrf));

    CUSOLVER_CHECK(hipsolverDnDormqr_bufferSize(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, nrhs, m,
                                               d_A, lda, d_tau, d_B, ldb, &lwork_ormqr));

    lwork = std::max(lwork_geqrf, lwork_ormqr);

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 4: compute QR factorization */
    CUSOLVER_CHECK(hipsolverDnDgeqrf(cusolverH, m, m, d_A, lda, d_tau, d_work, lwork, d_info));

    /* check if QR is good or not */
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after geqrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    /* step 5: compute Q^T*B */
    CUSOLVER_CHECK(hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, nrhs, m, d_A, lda,
                                    d_tau, d_B, ldb, d_work, lwork, d_info));

    /* check if QR is good or not */
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after ormqr: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    /* step 6: compute x = R \ Q^T*B */
    CUBLAS_CHECK(hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
                             HIPBLAS_DIAG_NON_UNIT, m, nrhs, &one, d_A, lda, d_B, ldb));

    CUDA_CHECK(hipMemcpyAsync(XC.data(), d_B, sizeof(double) * XC.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("X = (matlab base-1)\n");
    print_matrix(m, nrhs, XC.data(), ldb);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_tau));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}