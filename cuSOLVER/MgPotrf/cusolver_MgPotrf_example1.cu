#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolverMg_utils.h"
#include "cusolver_utils.h"

/* compute |x|_inf */
template <typename T> static T vec_nrm_inf(int n, const T *x) {
    T max_nrm = 0.0;
    for (int row = 1; row <= n; row++) {
        T xi = x[IDX1F(row)];
        max_nrm = (max_nrm > fabs(xi)) ? max_nrm : fabs(xi);
    }
    return max_nrm;
}

/* A is 1D laplacian, return A(N:-1:1, :) */
template <typename T> static void gen_1d_laplacian(int N, T *A, int lda) {
    for (int J = 1; J <= N; J++) {
        A[IDX2F(J, J, lda)] = 2.0;
        if ((J - 1) >= 1) {
            A[IDX2F(J, J - 1, lda)] = -1.0;
        }
        if ((J + 1) <= N) {
            A[IDX2F(J, J + 1, lda)] = -1.0;
        }
    }
}

/* Generate matrix B := A * X */
template <typename T>
static void gen_ref_B(int N, int NRHS, double *A, int lda, double *X, int ldx, double *B, int ldb) {
    for (int J = 1; J <= NRHS; J++) {
        for (int I = 1; I <= N; I++) {
            for (int K = 1; K <= N; K++) {
                T Aik = A[IDX2F(I, K, lda)];
                T Xk = X[IDX2F(K, J, ldx)];
                B[IDX2F(I, J, ldb)] += (Aik * Xk);
            }
        }
    }
}

int main(int argc, char *argv[]) {
    cusolverMgHandle_t cusolverH = NULL;

    using data_type = double;

    /* maximum number of GPUs */
    const int MAX_NUM_DEVICES = 16;

    int nbGpus = 0;
    std::vector<int> deviceList(MAX_NUM_DEVICES);

    const int NRHS = 2;
    const int N = 8;

    const int IA = 1;
    const int JA = 1;
    const int T_A = 256; /* tile size of A */
    const int lda = N;

    const int IB = 1;
    const int JB = 1;
    const int T_B = 10; /* tile size of B */
    const int ldb = N;

    int info = 0;

    cudaLibMgMatrixDesc_t descrA;
    cudaLibMgMatrixDesc_t descrB;
    cudaLibMgGrid_t gridA;
    cudaLibMgGrid_t gridB;
    cusolverMgGridMapping_t mapping = CUDALIBMG_GRID_MAPPING_COL_MAJOR;

    int64_t lwork_potrf = 0;
    int64_t lwork_potrs = 0;
    int64_t lwork = 0; /* workspace: number of elements per device */

    std::printf("Test 1D Laplacian of order %d\n", N);

    std::printf("Step 1: Create Mg handle and select devices \n");
    CUSOLVER_CHECK(cusolverMgCreate(&cusolverH));

    CUDA_CHECK(hipGetDeviceCount(&nbGpus));

    nbGpus = (nbGpus < MAX_NUM_DEVICES) ? nbGpus : MAX_NUM_DEVICES;
    std::printf("\tThere are %d GPUs \n", nbGpus);
    for (int j = 0; j < nbGpus; j++) {
        deviceList[j] = j;
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, j));
        std::printf("\tDevice %d, %s, cc %d.%d \n", j, prop.name, prop.major, prop.minor);
    }

    CUSOLVER_CHECK(cusolverMgDeviceSelect(cusolverH, nbGpus, deviceList.data()));

    std::printf("step 2: Enable peer access.\n");
    enablePeerAccess(nbGpus, deviceList.data());

    std::printf("Step 3: Allocate host memory A \n");
    std::vector<data_type> A(lda * N, 0);
    std::vector<data_type> B(ldb * NRHS, 0);
    std::vector<data_type> X(ldb * NRHS, 0);

    std::printf("Step 4: Prepare 1D Laplacian for A and X = ones(N,NRHS) \n");
    gen_1d_laplacian<data_type>(N, &A[IDX2F(IA, JA, lda)], lda);

#ifdef SHOW_FORMAT
    std::printf("A = matlab base-1\n");
    print_matrix(N, N, A.data(), lda);
#endif

    /* X = ones(N,1) */
    for (int row = 1; row <= N; row++) {
        for (int col = 1; col <= NRHS; col++) {
            X[IDX2F(row, col, ldb)] = 1.0;
        }
    }

#ifdef SHOW_FORMAT
    std::printf("X = matlab base-1\n");
    print_matrix(N, NRHS, X.data(), lda, HIPBLAS_OP_T);
#endif

    /* Set B := A * X */
    printf("Step 5: Create RHS for reference solution on host B = A*X \n");
    gen_ref_B<data_type>(N, NRHS, A.data(), /* input */
                         lda, X.data(),     /* input */
                         ldb,               /* same leading dimension as B */
                         B.data(),          /* output */
                         ldb);

#ifdef SHOW_FORMAT
    std::printf("B = matlab base-1\n");
    print_matrix(N, NRHS, B.data(), ldb, HIPBLAS_OP_T);
#endif

    std::printf("Step 6: Create matrix descriptors for A and D \n");

    CUSOLVER_CHECK(cusolverMgCreateDeviceGrid(&gridA, 1, nbGpus, deviceList.data(), mapping));
    CUSOLVER_CHECK(cusolverMgCreateDeviceGrid(&gridB, 1, nbGpus, deviceList.data(), mapping));

    /* (global) A is N-by-N */
    CUSOLVER_CHECK(cusolverMgCreateMatrixDesc(&descrA, N, /* nubmer of rows of (global) A */
                                              N,          /* number of columns of (global) A */
                                              N,          /* number or rows in a tile */
                                              T_A,        /* number of columns in a tile */
                                              traits<data_type>::cuda_data_type, gridA));

    /* (global) B is N-by-NRHS */
    CUSOLVER_CHECK(cusolverMgCreateMatrixDesc(&descrB, N, /* nubmer of rows of (global) B */
                                              NRHS,       /* number of columns of (global) B */
                                              N,          /* number or rows in a tile */
                                              T_B,        /* number of columns in a tile */
                                              traits<data_type>::cuda_data_type, gridB));

    std::printf("Step 7: Allocate distributed matrices A and B \n");

    std::vector<data_type *> array_d_A(nbGpus, nullptr);
    std::vector<data_type *> array_d_B(nbGpus, nullptr);

    /* A := 0 */
    createMat<data_type>(nbGpus, deviceList.data(), N, /* number of columns of global A */
                         T_A,                          /* number of columns per column tile */
                         lda,                          /* leading dimension of local A */
                         array_d_A.data());

    /* B := 0 */
    createMat<data_type>(nbGpus, deviceList.data(), NRHS, /* number of columns of global B */
                         T_B,                             /* number of columns per column tile */
                         ldb,                             /* leading dimension of local B */
                         array_d_B.data());

    std::printf("Step 8: Prepare data on devices \n");
    memcpyH2D<data_type>(nbGpus, deviceList.data(), N, N,
                         /* input */
                         A.data(), lda,
                         /* output */
                         N,                /* number of columns of global A */
                         T_A,              /* number of columns per column tile */
                         lda,              /* leading dimension of local A */
                         array_d_A.data(), /* host pointer array of dimension nbGpus */
                         IA, JA);

    memcpyH2D<data_type>(nbGpus, deviceList.data(), N, NRHS,
                         /* input */
                         B.data(), ldb,
                         /* output */
                         1,                /* number of columns of global A */
                         T_B,              /* number of columns per column tile */
                         ldb,              /* leading dimension of local A */
                         array_d_B.data(), /* host pointer array of dimension nbGpus */
                         IB, JB);

    std::printf("Step 9: Allocate workspace space \n");
    CUSOLVER_CHECK(
        cusolverMgPotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, N,
                                   reinterpret_cast<void **>(array_d_A.data()), IA, /* base-1 */
                                   JA,                                              /* base-1 */
                                   descrA, traits<data_type>::cuda_data_type, &lwork_potrf));

    CUSOLVER_CHECK(cusolverMgPotrs_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, N, NRHS, /* NRHS */
                                              reinterpret_cast<void **>(array_d_A.data()), IA, JA,
                                              descrA, reinterpret_cast<void **>(array_d_B.data()),
                                              IB, JB, descrB, traits<data_type>::cuda_data_type,
                                              &lwork_potrs));

    lwork = std::max(lwork_potrf, lwork_potrs);
    std::printf("\tAllocate device workspace, lwork = %lld \n", static_cast<long long>(lwork));

    std::vector<data_type *> array_d_work(nbGpus, nullptr);

    /* array_d_work[j] points to device workspace of device j */
    workspaceAlloc(nbGpus, deviceList.data(),
                   sizeof(data_type) * lwork, /* number of bytes per device */
                   reinterpret_cast<void **>(array_d_work.data()));

    /* sync all devices */
    CUDA_CHECK(hipDeviceSynchronize());

    std::printf("Step 10: Solve A*X = B by POTRF and POTRS \n");
    CUSOLVER_CHECK(cusolverMgPotrf(
        cusolverH, HIPBLAS_FILL_MODE_LOWER, N, reinterpret_cast<void **>(array_d_A.data()), IA, JA,
        descrA, traits<data_type>::cuda_data_type, reinterpret_cast<void **>(array_d_work.data()),
        lwork, &info /* host */
        ));

    /* sync all devices */
    CUDA_CHECK(hipDeviceSynchronize());

    /* check if A is singular */
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    CUSOLVER_CHECK(cusolverMgPotrs(cusolverH, HIPBLAS_FILL_MODE_LOWER, N, NRHS, /* NRHS */
                                   reinterpret_cast<void **>(array_d_A.data()), IA, JA, descrA,
                                   reinterpret_cast<void **>(array_d_B.data()), IB, JB, descrB,
                                   traits<data_type>::cuda_data_type,
                                   reinterpret_cast<void **>(array_d_work.data()), lwork,
                                   &info /* host */
                                   ));

    /* sync all devices */
    CUDA_CHECK(hipDeviceSynchronize());

    /* check if parameters are valid */
    if (0 > info) {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    std::printf("Step 11: Solution vector B\n");

    memcpyD2H<data_type>(nbGpus, deviceList.data(), N, NRHS,
                         /* input */
                         NRHS, /* number of columns of global B */
                         T_B,  /* number of columns per column tile */
                         ldb,  /* leading dimension of local B */
                         array_d_B.data(), IB, JB,
                         /* output */
                         B.data(), /* N-by-1 */
                         ldb);

#ifdef SHOW_FORMAT
    /* B is N-by-NRHS */
    std::printf("Computed solution (B)\n");
    print_matrix(N, NRHS, B.data(), ldb, HIPBLAS_OP_T);
    /* X is N-by-NRHS */
    std::printf("Reference solution (X)\n");
    print_matrix(N, NRHS, X.data(), ldb, HIPBLAS_OP_T);
#endif

    std::printf("Step 12: Measure residual error |b - A*x| \n");
    data_type max_err = 0;
    for (int col = 1; col <= NRHS; col++) {
        std::printf("errors for X[:,%d] \n", col);
        for (int row = 1; row <= N; row++) {
            data_type xij = X[IDX2F(row, col, ldb)];
            data_type bij = B[IDX2F(row, col, ldb)];
            data_type err = fabs(xij - bij);
            max_err = (err > max_err) ? err : max_err;
        }
        data_type x_nrm_inf = vec_nrm_inf(N, &X[IDX2F(1, col, ldb)]);
        data_type b_nrm_inf = vec_nrm_inf(N, &B[IDX2F(1, col, ldb)]);
        data_type A_nrm_inf = 4.0;
        data_type rel_err = max_err / (A_nrm_inf * x_nrm_inf + b_nrm_inf);
        std::printf("\t|b - A*x|_inf = %E\n", max_err);
        std::printf("\t|x|_inf = %E\n", x_nrm_inf);
        std::printf("\t|b|_inf = %E\n", b_nrm_inf);
        std::printf("\t|A|_inf = %E\n", A_nrm_inf);
        /* relative error is around machine zero  */
        /* the user can use |b - A*x|/(N*|A|*|x|+|b|) as well */
        std::printf("\t|b - A*x|/(|A|*|x|+|b|) = %E\n\n", rel_err);
    }

    std::printf("step 12: Free resources \n");
    destroyMat(nbGpus, deviceList.data(), N, /* number of columns of global A */
               T_A,                          /* number of columns per column tile */
               reinterpret_cast<void **>(array_d_A.data()));
    destroyMat(nbGpus, deviceList.data(), 1, /* number of columns of global B */
               T_B,                          /* number of columns per column tile */
               reinterpret_cast<void **>(array_d_B.data()));

    workspaceFree(nbGpus, deviceList.data(), reinterpret_cast<void **>(array_d_work.data()));

    return EXIT_SUCCESS;
}
