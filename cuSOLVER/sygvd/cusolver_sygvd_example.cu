#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int m = 3;
    const int lda = m;
    /*
     *       | 3.5 0.5 0 |
     *   A = | 0.5 3.5 0 |
     *       | 0   0   2 |
     *
     *       | 10  2   3 |
     *   B = | 2  10   5 |
     *       | 3   5  10 |
     */

    const std::vector<double> A = {3.5, 0.5, 0, 0.5, 3.5, 0.0, 0.0, 0.0, 2.0};
    const std::vector<double> B = {10.0, 2.0, 3.0, 2.0, 10.0, 5.0, 3.0, 5.0, 10.0};
    const std::vector<double> lambda = {0.158660256604, 0.370751508101882, 0.6};

    std::vector<double> V(lda * m, 0); // eigenvectors
    std::vector<double> W(m, 0);       // eigenvalues

    double *d_A = nullptr;
    double *d_B = nullptr;
    double *d_W = nullptr;
    int *d_info = nullptr;
    double *d_work = nullptr;
    int lwork = 0;
    int info = 0;

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");

    std::printf("B = (matlab base-1)\n");
    print_matrix(m, m, B.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(
        hipMemcpyAsync(d_B, B.data(), sizeof(double) * B.size(), hipMemcpyHostToDevice, stream));

    // step 3: query working space of sygvd
    hipsolverEigType_t itype = HIPSOLVER_EIG_TYPE_1;     // A*x = (lambda)*B*x
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    CUSOLVER_CHECK(hipsolverDnDsygvd_bufferSize(cusolverH, itype, jobz, uplo, m, d_A, lda, d_B, lda,
                                               d_W, &lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    // step 4: compute spectrum of (A,B)
    CUSOLVER_CHECK(hipsolverDnDsygvd(cusolverH, itype, jobz, uplo, m, d_A, lda, d_B, lda, d_W,
                                    d_work, lwork, d_info));

    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_A, sizeof(double) * A.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(W.data(), d_W, sizeof(double) * W.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after sygvd: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    std::printf("eigenvalue = (matlab base-1), ascending order\n");
    int idx = 1;
    for (auto const &i : W) {
        std::printf("W[%i] = %E\n", idx, i);
        idx++;
    }

    std::printf("V = (matlab base-1)\n");
    print_matrix(m, m, V.data(), lda);
    std::printf("=====\n");

    // step 4: check eigenvalues
    double lambda_sup = 0;
    for (int i = 0; i < m; i++) {
        double error = fabs(lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error) ? lambda_sup : error;
    }
    std::printf("|lambda - W| = %E\n", lambda_sup);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}