#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const int m = 3;
    const int lda = m;
    /*
     *       | 3.5 0.5 0.0 |
     *   A = | 0.5 3.5 0.0 |
     *       | 0.0 0.0 2.0 |
     *
     */
    const std::vector<double> A = {3.5, 0.5, 0.0, 0.5, 3.5, 0.0, 0.0, 0.0, 2.0};
    const std::vector<double> lambda = {2.0, 3.0, 4.0};

    std::vector<double> V(lda * m, 0); // eigenvectors
    std::vector<double> W(m, 0);       // eigenvalues

    double *d_A = nullptr;
    double *d_W = nullptr;
    int *d_info = nullptr;

    int info = 0;

    double vl = 0.0;
    double vu = 0.0;
    int il = 1;
    int iu = 2;
    int h_meig = 0;

    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace*/

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));

    // step 3: query working space of syevd
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipsolverEigRange_t range = HIPSOLVER_EIG_RANGE_I;   // eigenvalues/eigenvectors in the half-open
                                                       // interval (vl,vu] will be found
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    CUSOLVER_CHECK(hipsolverDnDsyevdx_bufferSize(cusolverH, jobz, range, uplo, m, d_A, lda, vl, vu,
                                                il, iu, &h_meig, d_W, &lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    // step 4: compute spectrum
    CUSOLVER_CHECK(hipsolverDnDsyevdx(cusolverH, jobz, range, uplo, m, d_A, lda, vl, vu, il, iu,
                                     &h_meig, d_W, d_work, lwork, d_info));

    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_A, sizeof(double) * V.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(W.data(), d_W, sizeof(double) * W.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after syevd: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    std::printf("eigenvalue = (matlab base-1), ascending order\n");
    int idx = 1;
    for (auto const &i : W) {
        std::printf("W[%i] = %E\n", idx, i);
        idx++;
    }

    std::printf("V = (matlab base-1)\n");
    print_matrix(m, m, V.data(), lda);
    std::printf("=====\n");

    std::printf("Eigenvalues found = %d\n", h_meig);

    // step 4: check eigenvalues
    double lambda_sup = 0;
    for (int i = 0; i < m; i++) {
        double error = fabs(lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error) ? lambda_sup : error;
    }
    std::printf("|lambda - W| = %E\n", lambda_sup);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
