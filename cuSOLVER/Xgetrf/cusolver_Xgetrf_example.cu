/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    using data_type = double;

    const int64_t m = 3;
    const int64_t lda = m;
    const int64_t ldb = m;
    /*       | 1 2 3  |
     *   A = | 4 5 6  |
     *       | 7 8 10 |
     *
     * without pivoting: A = L*U
     *       | 1 0 0 |      | 1  2  3 |
     *   L = | 4 1 0 |, U = | 0 -3 -6 |
     *       | 7 2 1 |      | 0  0  1 |
     *
     * with pivoting: P*A = L*U
     *       | 0 0 1 |
     *   P = | 1 0 0 |
     *       | 0 1 0 |
     *
     *       | 1       0     0 |      | 7  8       10     |
     *   L = | 0.1429  1     0 |, U = | 0  0.8571  1.5714 |
     *       | 0.5714  0.5   1 |      | 0  0       -0.5   |
     */

    const std::vector<data_type> A = {1.0, 4.0, 7.0, 2.0, 5.0, 8.0, 3.0, 6.0, 10.0};
    const std::vector<data_type> B = {1.0, 2.0, 3.0};
    std::vector<data_type> X(m, 0);
    std::vector<data_type> LU(lda * m, 0);
    std::vector<int64_t> Ipiv(m, 0);
    int info = 0;

    data_type *d_A = nullptr;  /* device copy of A */
    data_type *d_B = nullptr;  /* device copy of B */
    int64_t *d_Ipiv = nullptr; /* pivoting sequence */
    int *d_info = nullptr;     /* error info */

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr;              /* device workspace for getrf */
    size_t workspaceInBytesOnHost = 0;   /* size of workspace */
    void *h_work = nullptr;              /* host workspace for getrf */

    const int pivot_on = 1;
    const int algo = 0;

    if (pivot_on) {
        std::printf("pivot is on : compute P*A = L*U \n");
    } else {
        std::printf("pivot is off: compute A = L*U (not numerically stable)\n");
    }

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");

    std::printf("B = (matlab base-1)\n");
    print_matrix(m, 1, B.data(), ldb);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* Create advanced params */
    hipsolverDnParams_t params;
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));
    if (algo == 0) {
        std::printf("Using New Algo\n");
        CUSOLVER_CHECK(hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_0));
    } else {
        std::printf("Using Legacy Algo\n");
        CUSOLVER_CHECK(hipsolverDnSetAdvOptions(params, HIPSOLVERDN_GETRF, HIPSOLVER_ALG_1));
    }

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Ipiv), sizeof(int64_t) * Ipiv.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: query working space of getrf */
    CUSOLVER_CHECK(
        hipsolverDnXgetrf_bufferSize(cusolverH, params, m, m, traits<data_type>::cuda_data_type, d_A,
                                    lda, traits<data_type>::cuda_data_type, &workspaceInBytesOnDevice,
                                    &workspaceInBytesOnHost));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice));

    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    /* step 4: LU factorization */
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnXgetrf(cusolverH, params, m, m, traits<data_type>::cuda_data_type,
                                        d_A, lda, d_Ipiv, traits<data_type>::cuda_data_type, d_work,
                                        workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnXgetrf(cusolverH, params, m, m, traits<data_type>::cuda_data_type,
                                        d_A, lda, nullptr, traits<data_type>::cuda_data_type,
                                        d_work, workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info));
    }

    if (pivot_on) {
        CUDA_CHECK(hipMemcpyAsync(Ipiv.data(), d_Ipiv, sizeof(int64_t) * Ipiv.size(),
                                   hipMemcpyDeviceToHost, stream));
    }
    CUDA_CHECK(hipMemcpyAsync(LU.data(), d_A, sizeof(data_type) * A.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after Xgetrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    if (pivot_on) {
        std::printf("pivoting sequence, matlab base-1\n");
        for (int j = 0; j < m; j++) {
            std::printf("Ipiv(%d) = %lu\n", j + 1, Ipiv[j]);
        }
    }
    std::printf("L and U = (matlab base-1)\n");
    print_matrix(m, m, LU.data(), lda);
    std::printf("=====\n");

    /*
     * step 5: solve A*X = B
     *       | 1 |       | -0.3333 |
     *   B = | 2 |,  X = |  0.6667 |
     *       | 3 |       |  0      |
     *
     */
    if (pivot_on) {
        CUSOLVER_CHECK(hipsolverDnXgetrs(cusolverH, params, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        traits<data_type>::cuda_data_type, d_A, lda, d_Ipiv,
                                        traits<data_type>::cuda_data_type, d_B, ldb, d_info));
    } else {
        CUSOLVER_CHECK(hipsolverDnXgetrs(cusolverH, params, HIPBLAS_OP_N, m, 1, /* nrhs */
                                        traits<data_type>::cuda_data_type, d_A, lda, nullptr,
                                        traits<data_type>::cuda_data_type, d_B, ldb, d_info));
    }

    CUDA_CHECK(hipMemcpyAsync(X.data(), d_B, sizeof(data_type) * X.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("X = (matlab base-1)\n");
    print_matrix(m, 1, X.data(), ldb);
    std::printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_Ipiv));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));
    free(h_work);

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}