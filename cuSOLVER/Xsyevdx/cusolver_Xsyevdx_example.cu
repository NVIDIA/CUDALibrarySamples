#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverDnParams_t params = NULL;

    using data_type = double;

    const int m = 3;
    const int lda = m;
    /*
     *       | 3.5 0.5 0.0 |
     *   A = | 0.5 3.5 0.0 |
     *       | 0.0 0.0 2.0 |
     *
     */
    const std::vector<data_type> A = {3.5, 0.5, 0.0, 0.5, 3.5, 0.0, 0.0, 0.0, 2.0};
    const std::vector<data_type> lambda = {2.0, 3.0, 4.0};

    std::vector<data_type> V(lda * m, 0); // eigenvectors
    std::vector<data_type> W(m, 0);       // eigenvalues

    data_type *d_A = nullptr;
    data_type *d_W = nullptr;
    data_type vl;
    data_type vu;
    int64_t h_meig = 0;
    int *d_info = nullptr;

    int info = 0;

    size_t workspaceInBytesOnDevice = 0; /* size of workspace */
    void *d_work = nullptr; /* device workspace */
    size_t workspaceInBytesOnHost = 0; /* size of workspace */
    void *h_work = nullptr; /* host workspace for */

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUSOLVER_CHECK(hipsolverDnCreateParams(&params));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(data_type) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));

    // step 3: query working space of syevd
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipsolverEigRange_t range = HIPSOLVER_EIG_RANGE_ALL;

    CUSOLVER_CHECK(cusolverDnXsyevdx_bufferSize(
        cusolverH, params, jobz, range, uplo, m, traits<data_type>::cuda_data_type, d_A, lda, &vl,
        &vu, 0L, 0L, &h_meig, traits<data_type>::cuda_data_type, d_W,
        traits<data_type>::cuda_data_type, &workspaceInBytesOnDevice, &workspaceInBytesOnHost));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), workspaceInBytesOnDevice));

    if (0 < workspaceInBytesOnHost) {
        h_work = reinterpret_cast<void *>(malloc(workspaceInBytesOnHost));
        if (h_work == nullptr) {
            throw std::runtime_error("Error: h_work not allocated.");
        }
    }

    // step 4: compute spectrum
    CUSOLVER_CHECK(cusolverDnXsyevdx(
        cusolverH, params, jobz, range, uplo, m, traits<data_type>::cuda_data_type, d_A, lda, &vl,
        &vu, 0L, 0L, &h_meig, traits<data_type>::cuda_data_type, d_W, traits<data_type>::cuda_data_type,
        d_work, workspaceInBytesOnDevice, h_work, workspaceInBytesOnHost, d_info));

    CUDA_CHECK(hipMemcpyAsync(V.data(), d_A, sizeof(data_type) * V.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(W.data(), d_W, sizeof(data_type) * W.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after Xsyevdx: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    std::printf("eigenvalue = (matlab base-1), ascending order\n");
    int idx = 1;
    for (auto const &i : W) {
        std::printf("W[%i] = %E\n", idx, i);
        idx++;
    }

    std::printf("V = (matlab base-1)\n");
    print_matrix(m, m, V.data(), lda);
    std::printf("=====\n");

    // step 4: check eigenvalues
    data_type lambda_sup = 0;
    for (int i = 0; i < m; i++) {
        data_type error = fabs(lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error) ? lambda_sup : error;
    }
    std::printf("|lambda - W| = %E\n", lambda_sup);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));
    free(h_work);

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}