#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    using data_type = double;

    const int m = 3;
    const int lda = m;
    /*
     *       | 3.5 0.5 0.0 |
     *   A = | 0.5 3.5 0.0 |
     *       | 0.0 0.0 2.0 |
     *
     */
    const std::vector<data_type> A = {3.5, 0.5, 0.0, 0.5, 3.5, 0.0, 0.0, 0.0, 2.0};
    const std::vector<data_type> lambda = {2.0, 3.0, 4.0};

    std::vector<data_type> V(lda * m, 0); // eigenvectors
    std::vector<data_type> W(m, 0);       // eigenvalues

    data_type *d_A = nullptr;
    data_type *d_W = nullptr;
    int *d_info = nullptr;

    int info = 0;

    size_t d_lwork = 0;     /* size of workspace */
    void *d_work = nullptr; /* device workspace */
    size_t h_lwork = 0;     /* size of workspace */
    void *h_work = nullptr; /* host workspace for */

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(data_type) * W.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));

    // step 3: query working space of syevd
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    CUSOLVER_CHECK(cusolverDnXsyevd_bufferSize(
        cusolverH, NULL, jobz, uplo, m, traits<data_type>::cuda_data_type, d_A, lda,
        traits<data_type>::cuda_data_type, d_W, traits<data_type>::cuda_data_type, &d_lwork,
        &h_lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(data_type) * d_lwork));

    // step 4: compute spectrum
    CUSOLVER_CHECK(cusolverDnXsyevd(
        cusolverH, NULL, jobz, uplo, m, traits<data_type>::cuda_data_type, d_A, lda,
        traits<data_type>::cuda_data_type, d_W, traits<data_type>::cuda_data_type, d_work, d_lwork,
        h_work, h_lwork, d_info));

    CUDA_CHECK(hipMemcpyAsync(V.data(), d_A, sizeof(data_type) * V.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(W.data(), d_W, sizeof(data_type) * W.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after Xsyevdx: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    std::printf("eigenvalue = (matlab base-1), ascending order\n");
    int idx = 1;
    for (auto const &i : W) {
        std::printf("W[%i] = %E\n", idx, i);
        idx++;
    }

    std::printf("V = (matlab base-1)\n");
    print_matrix(m, m, V.data(), lda);
    std::printf("=====\n");

    // step 4: check eigenvalues
    data_type lambda_sup = 0;
    for (int i = 0; i < m; i++) {
        data_type error = fabs(lambda[i] - W[i]);
        lambda_sup = (lambda_sup > error) ? lambda_sup : error;
    }
    std::printf("|lambda - W| = %E\n", lambda_sup);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
