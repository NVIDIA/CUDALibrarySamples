#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream{};

    const int m = 3;
    const int n = 2;
    const int lda = m;

    /*
     *       | 1 2 |
     *   A = | 4 5 |
     *       | 2 1 |
     */

    const std::vector<double> A = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
    std::vector<double> Q(lda * n, 0); // orthonormal columns
    std::vector<double> R(n * n, 0);   // R = I - Q**T*Q

    /* device memory */
    double *d_A = nullptr;
    double *d_tau = nullptr;
    int *d_info = nullptr;
    double *d_work = nullptr;

    double *d_R = nullptr;

    int lwork_geqrf = 0;
    int lwork_orgqr = 0;
    int lwork = 0;
    int info = 0;

    const double h_one = 1;
    const double h_minus_one = -1;

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, n, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cudense/cublas handle */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy A and B to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_tau), sizeof(double) * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_R), sizeof(double) * R.size()));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));

    /* step 3: query working space of geqrf and orgqr */
    CUSOLVER_CHECK(hipsolverDnDgeqrf_bufferSize(cusolverH, m, n, d_A, lda, &lwork_geqrf));

    CUSOLVER_CHECK(hipsolverDnDorgqr_bufferSize(cusolverH, m, n, n, d_A, lda, d_tau, &lwork_orgqr));

    lwork = std::max(lwork_geqrf, lwork_orgqr);

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 4: compute QR factorization */
    CUSOLVER_CHECK(hipsolverDnDgeqrf(cusolverH, m, n, d_A, lda, d_tau, d_work, lwork, d_info));

    /* check if QR is successful or not */
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after geqrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    /* step 5: compute Q */
    CUSOLVER_CHECK(hipsolverDnDorgqr(cusolverH, m, n, n, d_A, lda, d_tau, d_work, lwork, d_info));

    /* check if orgqr is successful or not */
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after orgqr: info = %d\n", info);
    if (0 > info) {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    CUDA_CHECK(
        hipMemcpyAsync(Q.data(), d_A, sizeof(double) * A.size(), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("Q = (matlab base-1)\n");
    print_matrix(m, n, Q.data(), lda);

    // step 6: measure R = I - Q**T*Q

    // Set R = I.
    std::fill(R.begin(), R.end(), 0.0);
    for (int i = 0; i < n; i++) {
        R[i + i * n] = 1.0;
    }
    CUDA_CHECK(
        hipMemcpyAsync(d_R, R.data(), sizeof(double) * R.size(), hipMemcpyHostToDevice, stream));

    CUBLAS_CHECK(hipblasDgemm(cublasH,
                             HIPBLAS_OP_T,  // Q**T
                             HIPBLAS_OP_N,  // Q
                             n,            // number of rows of R
                             n,            // number of columns of R
                             m,            // number of columns of Q**T
                             &h_minus_one, /* host pointer */
                             d_A,          // Q**T
                             lda,
                             d_A,         // Q
                             lda, &h_one, /* hostpointer */
                             d_R, n));

    double dR_nrm2 = 0.0;
    CUBLAS_CHECK(hipblasDnrm2(cublasH, R.size(), d_R, 1, &dR_nrm2));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("|I - Q**T*Q| = %E\n", dR_nrm2);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_tau));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_R));

    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}