#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolverMg_utils.h"
#include "cusolver_utils.h"

template <typename T> static void gen_1d_laplacian(int N, T *A, int lda) {
    memset(A, 0, sizeof(T) * lda * N);
    for (int J = 1; J <= N; J++) {
        /* A(J,J) = 2 */
        A[IDX2F(J, J, lda)] = 2.0;
        if ((J - 1) >= 1) {
            /* A(J, J-1) = -1*/
            A[IDX2F(J, J - 1, lda)] = -1.0;
        }
        if ((J + 1) <= N) {
            /* A(J, J+1) = -1*/
            A[IDX2F(J, J + 1, lda)] = -1.0;
        }
    }
}

int main(int argc, char *argv[]) {
    cusolverMgHandle_t cusolverH = NULL;

    using data_type = double;

    /* maximum number of GPUs */
    const int MAX_NUM_DEVICES = 16;

    int nbGpus = 0;
    std::vector<int> deviceList(MAX_NUM_DEVICES);

    const int N = 2111;
    const int IA = 1;
    const int JA = 1;
    const int T_A = 256; /* tile size */
    const int lda = N;

    int info = 0;

    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;

    cudaLibMgMatrixDesc_t descrA;
    cudaLibMgGrid_t gridA;
    cusolverMgGridMapping_t mapping = CUDALIBMG_GRID_MAPPING_COL_MAJOR;

    int64_t lwork = 0; /* workspace: number of elements per device */

    std::printf("Test 1D Laplacian of order %d\n", N);

    std::printf("Step 1: Create Mg handle and select devices \n");
    CUSOLVER_CHECK(cusolverMgCreate(&cusolverH));

    CUDA_CHECK(hipGetDeviceCount(&nbGpus));

    nbGpus = (nbGpus < MAX_NUM_DEVICES) ? nbGpus : MAX_NUM_DEVICES;
    std::printf("\tThere are %d GPUs \n", nbGpus);
    for (int j = 0; j < nbGpus; j++) {
        deviceList[j] = j;
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, j));
        std::printf("\tDevice %d, %s, cc %d.%d \n", j, prop.name, prop.major, prop.minor);
    }

    CUSOLVER_CHECK(cusolverMgDeviceSelect(cusolverH, nbGpus, deviceList.data()));

    std::printf("Step 2: Enable peer access \n");
    enablePeerAccess(nbGpus, deviceList.data());

    std::printf("Step 3: Allocate host memory A \n");
    std::vector<data_type> A(lda * N, 0);
    std::vector<data_type> D(N, 0);

    std::printf("Step 4: Prepare 1D Laplacian \n");
    gen_1d_laplacian<data_type>(N, &A[IDX2F(IA, JA, lda)], lda);

#ifdef SHOW_FORMAT
    std::printf("A = matlab base-1\n");
    print_matrix(N, N, A.data(), lda);
#endif

    std::printf("Step 5: Create matrix descriptors for A and D \n");

    CUSOLVER_CHECK(cusolverMgCreateDeviceGrid(&gridA, 1, nbGpus, deviceList.data(), mapping));

    /* (global) A is N-by-N */
    CUSOLVER_CHECK(cusolverMgCreateMatrixDesc(&descrA, N, /* number of rows of (global) A */
                                              N,          /* number of columns of (global) A */
                                              N,          /* number or rows in a tile */
                                              T_A,        /* number of columns in a tile */
                                              traits<data_type>::cuda_data_type, gridA));

    std::printf("Step 6: Allocate distributed matrices A and D \n");

    std::vector<data_type *> array_d_A(nbGpus, nullptr);

    const int A_num_blks = (N + T_A - 1) / T_A;
    const int blks_per_device = (A_num_blks + nbGpus - 1) / nbGpus;

    for (int p = 0; p < nbGpus; p++) {
        CUDA_CHECK(hipSetDevice(deviceList[p]));
        CUDA_CHECK(hipMalloc(&(array_d_A[p]), sizeof(double) * lda * T_A * blks_per_device));
    }

    printf("Step 7: Prepare data on devices \n");
    /* The following setting only works for IA = JA = 1 */
    for (int k = 0; k < A_num_blks; k++) {
        /* k = ibx * nbGpus + p */
        const int p = (k % nbGpus);
        const int ibx = (k / nbGpus);
        double *h_Ak = A.data() + (size_t)lda * T_A * k;
        double *d_Ak = array_d_A[p] + (size_t)lda * T_A * ibx;
        const int width = std::min(T_A, (N - T_A * k));
        CUDA_CHECK(hipMemcpy(d_Ak, h_Ak, sizeof(double) * lda * width, hipMemcpyHostToDevice));
    }
    /* sync all devices */
    CUDA_CHECK(hipDeviceSynchronize());

    std::printf("Step 8: Allocate workspace space \n");
    CUSOLVER_CHECK(cusolverMgSyevd_bufferSize(
        cusolverH, (hipsolverEigMode_t)jobz, HIPBLAS_FILL_MODE_LOWER, /* only support lower mode */
        N, reinterpret_cast<void **>(array_d_A.data()), IA,         /* base-1 */
        JA,                                                         /* base-1 */
        descrA, reinterpret_cast<void *>(D.data()), traits<data_type>::cuda_data_type,
        traits<data_type>::cuda_data_type, &lwork));

    std::printf("\tAllocate device workspace, lwork = %lld \n", static_cast<long long>(lwork));

    std::vector<data_type *> array_d_work(nbGpus, nullptr);

    /* array_d_work[j] points to device workspace of device j */
    workspaceAlloc(nbGpus, deviceList.data(),
                   sizeof(data_type) * lwork, /* number of bytes per device */
                   reinterpret_cast<void **>(array_d_work.data()));

    /* sync all devices */
    CUDA_CHECK(hipDeviceSynchronize());

    std::printf("Step 9: Compute eigenvalues and eigenvectors \n");
    CUSOLVER_CHECK(cusolverMgSyevd(
        cusolverH, (hipsolverEigMode_t)jobz, HIPBLAS_FILL_MODE_LOWER, /* only support lower mode */
        N, reinterpret_cast<void **>(array_d_A.data()),             /* exit: eigenvectors */
        IA, JA, descrA, reinterpret_cast<void **>(D.data()),        /* exit: eigenvalues */
        traits<data_type>::cuda_data_type, traits<data_type>::cuda_data_type,
        reinterpret_cast<void **>(array_d_work.data()), lwork, &info /* host */
        ));

    /* sync all devices */
    CUDA_CHECK(hipDeviceSynchronize());

    /* check if SYEVD converges */
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    std::printf("Step 10: Copy eigenvectors to A and eigenvalues to D \n");

    memcpyD2H<data_type>(nbGpus, deviceList.data(), N, N,
                         /* input */
                         N,   /* number of columns of global A */
                         T_A, /* number of columns per column tile */
                         lda, /* leading dimension of local A */
                         array_d_A.data(), IA, JA,
                         /* output */
                         A.data(), /* N-y-N eigenvectors */
                         lda);

#ifdef SHOW_FORMAT
    /* D is 1-by-N */
    std::printf("Eigenvalue D = \n");
    print_matrix(1, N, D.data(), 1);
#endif

    std::printf("Step 11: Verify eigenvalues \n");
    std::printf("     lambda(k) = 4 * sin(pi/2 *k/(N+1))^2 for k = 1:N \n");
    data_type max_err_D = 0;
    for (int k = 1; k <= N; k++) {
        const data_type pi = 4 * atan(1.0);
        const data_type h = 1.0 / (static_cast<data_type>(N) + 1);
        const data_type factor = sin(pi / 2.0 * (static_cast<data_type>(k)) * h);
        const data_type lambda = 4.0 * factor * factor;
        const data_type err = fabs(D[IDX1F(k)] - lambda);
        max_err_D = (max_err_D > err) ? max_err_D : err;
    }
    std::printf("\n|D - lambda|_inf = %E\n\n", max_err_D);

    std::printf("Step 12: Free resources \n");
    workspaceFree(nbGpus, deviceList.data(), reinterpret_cast<void **>(array_d_work.data()));

    destroyMat(nbGpus, deviceList.data(), N, /* number of columns of global A */
               T_A,                          /* number of columns per column tile */
               reinterpret_cast<void **>(array_d_A.data()));

    CUSOLVER_CHECK(cusolverMgDestroyMatrixDesc(descrA));
    CUSOLVER_CHECK(cusolverMgDestroyGrid(gridA));
    CUSOLVER_CHECK(cusolverMgDestroy(cusolverH));

    return EXIT_SUCCESS;
}