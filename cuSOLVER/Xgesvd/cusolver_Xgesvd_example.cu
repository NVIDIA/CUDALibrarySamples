#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    using data_type = double;

    const int64_t m = 3;
    const int64_t n = 2;
    const int64_t lda = m;
    /*       | 1 2  |
     *   A = | 4 5  |
     *       | 2 1  |
     */

    const std::vector<data_type> A = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
    std::vector<data_type> U(lda * m, 0);
    std::vector<data_type> VT(lda * n, 0);
    std::vector<data_type> S(n, 0);
    std::vector<data_type> S_exact(n, 0);

    data_type *d_A = nullptr;
    data_type *d_S = nullptr;
    data_type *d_U = nullptr;
    data_type *d_VT = nullptr;
    int *d_info = nullptr;
    data_type *d_work = nullptr;
    data_type *d_rwork = nullptr;
    data_type *d_W = nullptr; // W = S*VT

    int lwork = 0;
    int info = 0;
    const data_type h_one = 1;
    const data_type h_minus_one = -1;

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, n, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(data_type) * S.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(data_type) * U.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_VT), sizeof(data_type) * VT.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(data_type) * lda * n));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * lda * n, hipMemcpyHostToDevice,
                               stream));

    /* step 3: query working space of SVD */
    CUSOLVER_CHECK(hipsolverDnDgesvd_bufferSize(cusolverH, m, n, &lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(data_type) * lwork));

    /* step 4: compute SVD */
    signed char jobu = 'A';  // all m columns of U
    signed char jobvt = 'A'; // all n columns of VT

    CUSOLVER_CHECK(hipsolverDnDgesvd(cusolverH, jobu, jobvt, m, n, d_A, lda, d_S, d_U,
                                    lda, // ldu
                                    d_VT,
                                    lda, // ldvt,
                                    d_work, lwork, d_rwork, d_info));

    CUDA_CHECK(hipMemcpyAsync(U.data(), d_U, sizeof(data_type) * U.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(VT.data(), d_VT, sizeof(data_type) * VT.size(),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(S.data(), d_S, sizeof(data_type) * S.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after Xgesvd: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    std::printf("=====\n");

    std::printf("S = (matlab base-1)\n");
    print_matrix(n, 1, S.data(), lda);
    std::printf("=====\n");

    std::printf("U = (matlab base-1)\n");
    print_matrix(m, m, U.data(), lda);
    std::printf("=====\n");

    std::printf("VT = (matlab base-1)\n");
    print_matrix(n, n, VT.data(), lda);
    std::printf("=====\n");

    // step 5: measure error of singular value
    double ds_sup = 0;
    for (int j = 0; j < n; j++) {
        double err = fabs(S[j] - S_exact[j]);
        ds_sup = (ds_sup > err) ? ds_sup : err;
    }
    std::printf("|S - S_exact| = %E \n", ds_sup);

    // step 6: |A - U*S*VT|
    // W = S*VT
    CUBLAS_CHECK(hipblasDdgmm(cublasH, HIPBLAS_SIDE_LEFT, n, n, d_VT, lda, d_S, 1, d_W, lda));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));

    CUBLAS_CHECK(hipblasDgemm(cublasH,
                                HIPBLAS_OP_N,  // U
                                HIPBLAS_OP_N,  // W
                                m,            // number of rows of A
                                n,            // number of columns of A
                                n,            // number of columns of U
                                &h_minus_one, /* host pointer */
                                d_U,          // U
                                lda,
                                d_W,         // W
                                lda, &h_one, /* hostpointer */
                                d_A, lda));

    double dR_fro = 0.0;
    CUBLAS_CHECK(hipblasDnrm2(cublasH, A.size(), d_A, 1, &dR_fro));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("|A - U*S*VT| = %E \n", dR_fro);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_S));
    CUDA_CHECK(hipFree(d_U));
    CUDA_CHECK(hipFree(d_VT));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_rwork));
    CUDA_CHECK(hipFree(d_W));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
