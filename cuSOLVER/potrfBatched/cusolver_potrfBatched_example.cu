/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    const int batchSize = 2;
    const int nrhs = 1;
    const int m = 3;
    const int lda = m;
    const int ldb = m;
    /*
     *      | 1     2     3 |
     * A0 = | 2     5     5 | = L0 * L0**T
     *      | 3     5    12 |
     *
     *            | 1.0000         0         0 |
     * where L0 = | 2.0000    1.0000         0 |
     *            | 3.0000   -1.0000    1.4142 |
     *
     *      | 1     2     3 |
     * A1 = | 2     4     5 | is not s.p.d., failed at row 2
     *      | 3     5    12 |
     *
     */

    const std::vector<double> A0 = {1.0, 2.0, 3.0, 2.0, 5.0, 5.0, 3.0, 5.0, 12.0};
    const std::vector<double> A1 = {1.0, 2.0, 3.0, 2.0, 4.0, 5.0, 3.0, 5.0, 12.0};
    const std::vector<double> B0 = {1.0, 1.0, 1.0};
    std::vector<double> X0(m, 0);             /* X0 = A0\B0 */
    std::vector<int> infoArray(batchSize, 0); /* host copy of error info */

    std::vector<double> L0(lda * m); /* cholesky factor of A0 */

    std::vector<double *> Aarray(batchSize, nullptr);
    std::vector<double *> Barray(batchSize, nullptr);

    double **d_Aarray = nullptr;
    double **d_Barray = nullptr;
    int *d_infoArray = nullptr;

    std::printf("A0 = (matlab base-1)\n");
    print_matrix(m, m, A0.data(), lda);
    std::printf("=====\n");

    std::printf("A1 = (matlab base-1)\n");
    print_matrix(m, m, A1.data(), lda);
    std::printf("=====\n");

    std::printf("B0 = (matlab base-1)\n");
    print_matrix(m, 1, B0.data(), ldb);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */
    for (int j = 0; j < batchSize; j++) {
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&Aarray[j]), sizeof(double) * lda * m));
        CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&Barray[j]), sizeof(double) * ldb * nrhs));
    }
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_infoArray), sizeof(int) * infoArray.size()));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Aarray), sizeof(double *) * Aarray.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Barray), sizeof(double *) * Barray.size()));

    CUDA_CHECK(hipMemcpyAsync(Aarray[0], A0.data(), sizeof(double) * A0.size(),
                               hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(Aarray[1], A1.data(), sizeof(double) * A1.size(),
                               hipMemcpyHostToDevice, stream));

    CUDA_CHECK(
        hipMemcpyAsync(Barray[0], B0.data(), sizeof(double) * B0.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(
        hipMemcpyAsync(Barray[1], B0.data(), sizeof(double) * B0.size(), hipMemcpyHostToDevice, stream));

    CUDA_CHECK(hipMemcpyAsync(d_Aarray, Aarray.data(), sizeof(double) * Aarray.size(),
                               hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_Barray, Barray.data(), sizeof(double) * Barray.size(),
                               hipMemcpyHostToDevice, stream));

    /* step 3: Cholesky factorization */
    CUSOLVER_CHECK(
        hipsolverDnDpotrfBatched(cusolverH, uplo, m, d_Aarray, lda, d_infoArray, batchSize));

    CUDA_CHECK(hipMemcpyAsync(infoArray.data(), d_infoArray, sizeof(int) * infoArray.size(),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(L0.data(), Aarray[0], sizeof(double) * lda * m,
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int j = 0; j < batchSize; j++) {
        std::printf("info[%d] = %d\n", j, infoArray[j]);
    }

    assert(0 == infoArray[0]);
    /* A1 is singular */
    assert(2 == infoArray[1]);

    std::printf("L = (matlab base-1), upper triangle is don't care \n");
    print_matrix(m, m, L0.data(), lda);
    std::printf("=====\n");

    /*
     * step 4: solve A0*X0 = B0
     *        | 1 |        | 10.5 |
     *   B0 = | 1 |,  X0 = | -2.5 |
     *        | 1 |        | -1.5 |
     */
    CUSOLVER_CHECK(hipsolverDnDpotrsBatched(cusolverH, uplo, m, nrhs, /* only support rhs = 1*/
                                           d_Aarray, lda, d_Barray, ldb, d_infoArray, batchSize));

    CUDA_CHECK(hipMemcpyAsync(infoArray.data(), d_infoArray, sizeof(int), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(
        hipMemcpyAsync(X0.data(), Barray[0], sizeof(double) * X0.size(), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after potrsBatched: infoArray[0] = %d\n", infoArray[0]);
    if (0 > infoArray[0]) {
        std::printf("%d-th parameter is wrong \n", -infoArray[0]);
        exit(1);
    }

    std::printf("X0 = (matlab base-1)\n");
    print_matrix(m, 1, X0.data(), ldb);
    std::printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_Aarray));
    CUDA_CHECK(hipFree(d_Barray));
    CUDA_CHECK(hipFree(d_infoArray));
    for (int j = 0; j < batchSize; j++) {
        CUDA_CHECK(hipFree(Aarray[j]));
        CUDA_CHECK(hipFree(Barray[j]));
    }

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}