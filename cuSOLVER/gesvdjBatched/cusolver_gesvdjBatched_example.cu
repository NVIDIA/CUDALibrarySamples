#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverGesvdjInfo_t gesvdj_params = NULL;

    const int m = 3;   /* 1 <= m <= 32 */
    const int n = 2;   /* 1 <= n <= 32 */
    const int lda = m; /* lda >= m */
    const int ldu = m; /* ldu >= m */
    const int ldv = n; /* ldv >= n */
    const int batchSize = 2;
    const int minmn = (m < n) ? m : n; /* min(m,n) */

    /*
     *        |  1  -1  |
     *   A0 = | -1   2  |
     *        |  0   0  |
     *
     *   A0 = U0 * S0 * V0**T
     *   S0 = diag(2.6180, 0.382)
     *
     *        |  3   4  |
     *   A1 = |  4   7  |
     *        |  0   0  |
     *
     *   A1 = U1 * S1 * V1**T
     *   S1 = diag(9.4721, 0.5279)
     */

    std::vector<double> A(lda * n * batchSize, 0); /* A = [A0 ; A1] */
    std::vector<double> U(ldu * m * batchSize, 0); /* U = [U0 ; U1] */
    std::vector<double> V(ldv * n * batchSize, 0); /* V = [V0 ; V1] */
    std::vector<double> S(minmn * batchSize, 0);   /* S = [S0 ; S1] */
    std::vector<int> info(batchSize, 0);             /* info = [info0 ; info1] */

    double *d_A = nullptr; /* lda-by-n-by-batchSize */
    double *d_U = nullptr; /* ldu-by-m-by-batchSize */
    double *d_V = nullptr; /* ldv-by-n-by-batchSize */
    double *d_S = nullptr; /* minmn-by-batchSize */
    int *d_info = nullptr; /* batchSize */

    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace for getrf */

    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const int sort_svd = 0;                                  /* don't sort singular values */
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute singular vectors */

    double *A0 = A.data();
    double *A1 = A.data() + lda * n; /* Aj is m-by-n */

    /*
     *        |  1  -1  |
     *   A0 = | -1   2  |
     *        |  0   0  |
     *   A0 is column-major
     */
    A0[0 + 0 * lda] = 1.0;
    A0[1 + 0 * lda] = -1.0;
    A0[2 + 0 * lda] = 0.0;

    A0[0 + 1 * lda] = -1.0;
    A0[1 + 1 * lda] = 2.0;
    A0[2 + 1 * lda] = 0.0;

    /*
     *        |  3   4  |
     *   A1 = |  4   7  |
     *        |  0   0  |
     *   A1 is column-major
     */
    A1[0 + 0 * lda] = 3.0;
    A1[1 + 0 * lda] = 4.0;
    A1[2 + 0 * lda] = 0.0;

    A1[0 + 1 * lda] = 4.0;
    A1[1 + 1 * lda] = 7.0;
    A1[2 + 1 * lda] = 0.0;

    std::printf("m = %d, n = %d \n", m, n);
    std::printf("tol = %E, default value is machine zero \n", tol);
    std::printf("max. sweeps = %d, default value is 100\n", max_sweeps);

    std::printf("A0 = (matlab base-1)\n");
    print_matrix(m, n, A.data(), lda);
    std::printf("=====\n");

    std::printf("A1 = (matlab base-1)\n");
    print_matrix(m, n, A.data() + lda * n, lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: configuration of gesvdj */
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));

    /* default value of tolerance is machine zero */
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol));

    /* default value of max. sweeps is 100 */
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps));

    /* disable sorting */
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetSortEig(gesvdj_params, sort_svd));

    /* step 3: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(double) * U.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(double) * V.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * S.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * info.size()));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));

    /* step 4: query working space of gesvdjBatched */
    CUSOLVER_CHECK(hipsolverDnDgesvdjBatched_bufferSize(cusolverH, jobz, m, n, d_A, lda, d_S, d_U,
                                                       ldu, d_V, ldv, &lwork, gesvdj_params,
                                                       batchSize));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 5: compute singular values of A0 and A1 */
    CUSOLVER_CHECK(hipsolverDnDgesvdjBatched(cusolverH, jobz, m, n, d_A, lda, d_S, d_U, ldu, d_V,
                                            ldv, d_work, lwork, d_info, gesvdj_params, batchSize));

    CUDA_CHECK(
        hipMemcpyAsync(U.data(), d_U, sizeof(double) * U.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_V, sizeof(double) * V.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(S.data(), d_S, sizeof(double) * S.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(info.data(), d_info, sizeof(int) * info.size(),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int i = 0; i < batchSize; i++) {
        if (0 == info[i]) {
            std::printf("matrix %d: gesvdj converges \n", i);
        } else if (0 > info[i]) {
            /* only info[0] shows if some input parameter is wrong.
             * If so, the error is HIPSOLVER_STATUS_INVALID_VALUE.
             */
            std::printf("Error: %d-th parameter is wrong \n", -info[i]);
            exit(1);
        } else { /* info = m+1 */
                 /* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
            std::printf("WARNING: matrix %d, info = %d : gesvdj does not converge \n", i, info[i]);
        }
    }

    /* Step 6: show singular values and singular vectors */
    double *S0 = S.data();
    double *S1 = S.data() + minmn;
    std::printf("==== \n");
    for (int i = 0; i < minmn; i++) {
        std::printf("S0(%d) = %20.16E\n", i + 1, S0[i]);
    }
    std::printf("==== \n");
    for (int i = 0; i < minmn; i++) {
        std::printf("S1(%d) = %20.16E\n", i + 1, S1[i]);
    }
    std::printf("==== \n");

    double *U0 = U.data();
    double *U1 = U.data() + ldu * m; /* Uj is m-by-m */
    std::printf("U0 = (matlab base-1)\n");
    print_matrix(m, m, U0, ldu);
    std::printf("U1 = (matlab base-1)\n");
    print_matrix(m, m, U1, ldu);

    double *V0 = V.data();
    double *V1 = V.data() + ldv * n; /* Vj is n-by-n */
    std::printf("V0 = (matlab base-1)\n");
    print_matrix(n, n, V0, ldv);
    std::printf("V1 = (matlab base-1)\n");
    print_matrix(n, n, V1, ldv);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_U));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_S));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}