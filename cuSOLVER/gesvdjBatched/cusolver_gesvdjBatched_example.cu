#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverGesvdjInfo_t gesvdj_params = NULL;

    const int m = 3;   /* 1 <= m <= 32 */
    const int n = 2;   /* 1 <= n <= 32 */
    const int lda = m; /* lda >= m */
    const int ldu = m; /* ldu >= m */
    const int ldv = n; /* ldv >= n */
    const int batchSize = 2;
    const int minmn = (m < n) ? m : n; /* min(m,n) */

    /*
     *        |  1  -1  |
     *   A0 = | -1   2  |
     *        |  0   0  |
     *
     *   A0 = U0 * S0 * V0**T
     *   S0 = diag(2.6180, 0.382)
     *
     *        |  3   4  |
     *   A1 = |  4   7  |
     *        |  0   0  |
     *
     *   A1 = U1 * S1 * V1**T
     *   S1 = diag(9.4721, 0.5279)
     */

    std::vector<double> A(lda * n * batchSize, 0); /* A = [A0 ; A1] */
    std::vector<double> U(ldu * m * batchSize, 0); /* U = [U0 ; U1] */
    std::vector<double> V(ldv * n * batchSize, 0); /* V = [V0 ; V1] */
    std::vector<double> S(minmn * batchSize, 0);   /* S = [S0 ; S1] */
    std::vector<int> info(batchSize, 0);             /* info = [info0 ; info1] */

    double *d_A = nullptr; /* lda-by-n-by-batchSize */
    double *d_U = nullptr; /* ldu-by-m-by-batchSize */
    double *d_V = nullptr; /* ldv-by-n-by-batchSize */
    double *d_S = nullptr; /* minmn-by-batchSize */
    int *d_info = nullptr; /* batchSize */

    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace for getrf */

    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const int sort_svd = 0;                                  /* don't sort singular values */
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute singular vectors */

    double *A0 = A.data();
    double *A1 = A.data() + lda * n; /* Aj is m-by-n */

    /*
     *        |  1  -1  |
     *   A0 = | -1   2  |
     *        |  0   0  |
     *   A0 is column-major
     */
    A0[0 + 0 * lda] = 1.0;
    A0[1 + 0 * lda] = -1.0;
    A0[2 + 0 * lda] = 0.0;

    A0[0 + 1 * lda] = -1.0;
    A0[1 + 1 * lda] = 2.0;
    A0[2 + 1 * lda] = 0.0;

    /*
     *        |  3   4  |
     *   A1 = |  4   7  |
     *        |  0   0  |
     *   A1 is column-major
     */
    A1[0 + 0 * lda] = 3.0;
    A1[1 + 0 * lda] = 4.0;
    A1[2 + 0 * lda] = 0.0;

    A1[0 + 1 * lda] = 4.0;
    A1[1 + 1 * lda] = 7.0;
    A1[2 + 1 * lda] = 0.0;

    std::printf("m = %d, n = %d \n", m, n);
    std::printf("tol = %E, default value is machine zero \n", tol);
    std::printf("max. sweeps = %d, default value is 100\n", max_sweeps);

    std::printf("A0 = (matlab base-1)\n");
    print_matrix(m, n, A.data(), lda);
    std::printf("=====\n");

    std::printf("A1 = (matlab base-1)\n");
    print_matrix(m, n, A.data() + lda * n, lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: configuration of gesvdj */
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));

    /* default value of tolerance is machine zero */
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol));

    /* default value of max. sweeps is 100 */
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps));

    /* disable sorting */
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetSortEig(gesvdj_params, sort_svd));

    /* step 3: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(double) * U.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(double) * V.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * S.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * info.size()));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice, stream));

    /* step 4: query working space of gesvdjBatched */
    CUSOLVER_CHECK(hipsolverDnDgesvdjBatched_bufferSize(cusolverH, jobz, m, n, d_A, lda, d_S, d_U,
                                                       ldu, d_V, ldv, &lwork, gesvdj_params,
                                                       batchSize));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 5: compute singular values of A0 and A1 */
    CUSOLVER_CHECK(hipsolverDnDgesvdjBatched(cusolverH, jobz, m, n, d_A, lda, d_S, d_U, ldu, d_V,
                                            ldv, d_work, lwork, d_info, gesvdj_params, batchSize));

    CUDA_CHECK(
        hipMemcpyAsync(U.data(), d_U, sizeof(double) * U.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_V, sizeof(double) * V.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(S.data(), d_S, sizeof(double) * S.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(info.data(), d_info, sizeof(int) * info.size(),
                               hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    for (int i = 0; i < batchSize; i++) {
        if (0 == info[i]) {
            std::printf("matrix %d: gesvdj converges \n", i);
        } else if (0 > info[i]) {
            /* only info[0] shows if some input parameter is wrong.
             * If so, the error is HIPSOLVER_STATUS_INVALID_VALUE.
             */
            std::printf("Error: %d-th parameter is wrong \n", -info[i]);
            exit(1);
        } else { /* info = m+1 */
                 /* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
            std::printf("WARNING: matrix %d, info = %d : gesvdj does not converge \n", i, info[i]);
        }
    }

    /* Step 6: show singular values and singular vectors */
    double *S0 = S.data();
    double *S1 = S.data() + minmn;
    std::printf("==== \n");
    for (int i = 0; i < minmn; i++) {
        std::printf("S0(%d) = %20.16E\n", i + 1, S0[i]);
    }
    std::printf("==== \n");
    for (int i = 0; i < minmn; i++) {
        std::printf("S1(%d) = %20.16E\n", i + 1, S1[i]);
    }
    std::printf("==== \n");

    double *U0 = U.data();
    double *U1 = U.data() + ldu * m; /* Uj is m-by-m */
    std::printf("U0 = (matlab base-1)\n");
    print_matrix(m, m, U0, ldu);
    std::printf("U1 = (matlab base-1)\n");
    print_matrix(m, m, U1, ldu);

    double *V0 = V.data();
    double *V1 = V.data() + ldv * n; /* Vj is n-by-n */
    std::printf("V0 = (matlab base-1)\n");
    print_matrix(n, n, V0, ldv);
    std::printf("V1 = (matlab base-1)\n");
    print_matrix(n, n, V1, ldv);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_U));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_S));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
