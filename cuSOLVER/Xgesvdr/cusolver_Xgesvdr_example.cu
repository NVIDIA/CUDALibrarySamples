#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverDnParams_t params_gesvdr = NULL;

    using data_type = double;

    /* Input matrix dimensions */
    const int64_t m = 5;
    const int64_t n = 5;
    const int64_t lda = m;
    const int64_t ldu = m;
    const int64_t ldv = n;

    /* rank of matrix A */
    const int64_t min_mn = std::min(m, n);

    /* Compute left/right eigenvectors */
    signed char jobu = 'S';
    signed char jobv = 'S';

    /* Number of iterations */
    const int64_t iters = 2;
    const int64_t rank = std::min(2, *reinterpret_cast<int *>(const_cast<int64_t *>(&n)));
    const int64_t p = std::min(2, static_cast<int>(n - rank));

    std::printf("%lu, %lu\n", rank, p);

    const std::vector<data_type> A = {0.76420743, 0.61411544, 0.81724151, 0.42040879, 0.03446089,
                                      0.03697287, 0.85962444, 0.67584086, 0.45594666, 0.02074835,
                                      0.42018265, 0.39204509, 0.12657948, 0.90250559, 0.23076218,
                                      0.50339844, 0.92974961, 0.21213988, 0.63962457, 0.58124562,
                                      0.58325673, 0.11589871, 0.39831112, 0.21492685, 0.00540355};
    const std::vector<data_type> S_ref{2.36539241, 0.81117785, 0.68562255, 0.41390509, 0.01519322};
    std::vector<data_type> S_gpu(m, 0);

    data_type *d_A = nullptr;
    data_type *d_U = nullptr;
    data_type *d_S = nullptr;
    data_type *d_V = nullptr;
    int *d_info = nullptr;
    int info = 0;

    size_t d_lwork = 0;     /* size of workspace */
    void *d_work = nullptr; /* device workspace for getrf */
    size_t h_lwork = 0;     /* size of workspace */
    void *h_work = nullptr; /* host workspace for getrf */

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, n, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    CUSOLVER_CHECK(hipsolverDnCreateParams(&params_gesvdr));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(data_type) * ldu * m));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(data_type) * ldv * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(data_type) * S_ref.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * lda * n, hipMemcpyHostToDevice,
                               stream));

    std::printf("m = %ld, n = %ld, rank = %ld, p = %ld, iters = %ld\n", m, n, rank, p, iters);
    if ((rank + p) > n) {
        throw std::runtime_error("Error: (rank + p) > n ");
    }

    /* step 3: query working space of SVD */
    CUSOLVER_CHECK(cusolverDnXgesvdr_bufferSize(
        cusolverH, params_gesvdr, jobu, jobv, m, n, rank, p, iters,
        traits<data_type>::cuda_data_type, d_A, lda, traits<data_type>::cuda_data_type, d_S,
        traits<data_type>::cuda_data_type, d_U, ldu, traits<data_type>::cuda_data_type, d_V, ldv,
        traits<data_type>::cuda_data_type, &d_lwork, &h_lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(data_type) * d_lwork));

    if (0 < h_lwork) {
        h_work = reinterpret_cast<void *>(malloc(h_lwork));
        if (d_work == nullptr) {
            throw std::runtime_error("Error: d_work not allocated.");
        }
    }

    /* step 4: compute SVD */
    CUSOLVER_CHECK(cusolverDnXgesvdr(
        cusolverH, params_gesvdr, jobu, jobv, m, n, rank, p, iters,
        traits<data_type>::cuda_data_type, d_A, lda, traits<data_type>::cuda_data_type, d_S,
        traits<data_type>::cuda_data_type, d_U, ldu, traits<data_type>::cuda_data_type, d_V, ldv,
        traits<data_type>::cuda_data_type, d_work, d_lwork, h_work, h_lwork, d_info));

    CUDA_CHECK(hipMemcpyAsync(S_gpu.data(), d_S, sizeof(data_type) * S_gpu.size(),
                               hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /* check info value */
    std::printf("after Xgesvdr: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    double max_err = 0;
    double max_relerr = 0;
    for (int i = 0; i < rank; i++) {
        const double lambda_ref = S_ref[i];
        const double lambda_gpu = S_gpu[i];
        const double AbsErr = fabs(lambda_ref - lambda_gpu);
        const double RelErr = AbsErr / lambda_ref;

        max_err = std::max(max_err, AbsErr) ? max_err : AbsErr;
        max_relerr = std::max(max_relerr, RelErr) ? max_relerr : RelErr;

        std::printf("S_ref[%d]=%f  S_gpu=[%d]=%f  AbsErr=%E  RelErr=%E\n", i, lambda_ref, i,
                    lambda_gpu, AbsErr, RelErr);
    }
    std::printf("\n");

    double eps = 1.E-8;
    std::printf("max_err = %E, max_relerr = %E, eps = %E\n", max_err, max_relerr, eps);

    if (max_relerr > eps) {
        std::printf("Error: max_relerr is bigger than eps\n");
        std::printf("try to increase oversampling or iters\n");
        std::printf("otherwise, reduce eps\n");
    } else {
        std::printf("Success: max_relerr is smaller than eps\n");
    }

    /* free resources */
    free(h_work);

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_U));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_S));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroyParams(params_gesvdr));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
