#include "hip/hip_runtime.h"
/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int main(int argc, char *argv[]) {
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    hipsolverGesvdjInfo_t gesvdj_params = NULL;

    const int m = 3;                   /* 1 <= m <= 32 */
    const int n = 2;                   /* 1 <= n <= 32 */
    const int lda = m;                 /* lda >= m */
    const int ldu = m;                 /* ldu >= m */
    const int ldv = n;                 /* ldv >= n */
    const int minmn = (m < n) ? m : n; /* min(m,n) */

    /*
     *       | 1 2 |
     *   A = | 4 5 |
     *       | 2 1 |
     */

    std::vector<double> A = {1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
    std::vector<double> U(ldu * m, 0); /* m-by-m unitary matrix, left singular vectors  */
    std::vector<double> V(ldv * n, 0); /* n-by-n unitary matrix, right singular vectors */
    std::vector<double> S(minmn, 0);   /* numerical singular value */
    std::vector<double> S_exact = {7.065283497082729,
                                   1.040081297712078}; /* exact singular values */
    int info = 0;                                      /* host copy of error info */

    double *d_A = nullptr;
    double *d_S = nullptr; /* singular values */
    double *d_U = nullptr; /* left singular vectors */
    double *d_V = nullptr; /* right singular vectors */

    int *d_info = nullptr;

    int lwork = 0;            /* size of workspace */
    double *d_work = nullptr; /* device workspace for getrf */

    /* configuration of gesvdj  */
    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvectors.
    const int econ = 0;                                      /* econ = 1 for economy size */

    /* numerical results of gesvdj  */
    double residual = 0;
    int executed_sweeps = 0;

    std::printf("m = %d, n = %d \n", m, n);
    std::printf("tol = %E, default value is machine zero \n", tol);
    std::printf("max. sweeps = %d, default value is 100\n", max_sweeps);
    std::printf("econ = %d \n", econ);

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, n, A.data(), lda);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: configuration of gesvdj */
    CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));

    /* default value of tolerance is machine zero */
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, tol));

    /* default value of max. sweeps is 100 */
    CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, max_sweeps));

    /* step 3: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_S), sizeof(double) * S.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_U), sizeof(double) * U.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(double) * V.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(
        hipMemcpyAsync(d_A, A.data(), sizeof(double) * lda * n, hipMemcpyHostToDevice, stream));

    /* step 4: query working space of SVD */
    CUSOLVER_CHECK(hipsolverDnDgesvdj_bufferSize(
        cusolverH, jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
                         /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        econ,            /* econ = 1 for economy size */
        m,               /* nubmer of rows of A, 0 <= m */
        n,               /* number of columns of A, 0 <= n  */
        d_A,             /* m-by-n */
        lda,             /* leading dimension of A */
        d_S,             /* min(m,n) */
                         /* the singular values in descending order */
        d_U,             /* m-by-m if econ = 0 */
                         /* m-by-min(m,n) if econ = 1 */
        ldu,             /* leading dimension of U, ldu >= max(1,m) */
        d_V,             /* n-by-n if econ = 0  */
                         /* n-by-min(m,n) if econ = 1  */
        ldv,             /* leading dimension of V, ldv >= max(1,n) */
        &lwork, gesvdj_params));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork));

    /* step 5: compute SVD*/
    CUSOLVER_CHECK(hipsolverDnDgesvdj(
        cusolverH, jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
                         /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        econ,            /* econ = 1 for economy size */
        m,               /* nubmer of rows of A, 0 <= m */
        n,               /* number of columns of A, 0 <= n  */
        d_A,             /* m-by-n */
        lda,             /* leading dimension of A */
        d_S,             /* min(m,n)  */
                         /* the singular values in descending order */
        d_U,             /* m-by-m if econ = 0 */
                         /* m-by-min(m,n) if econ = 1 */
        ldu,             /* leading dimension of U, ldu >= max(1,m) */
        d_V,             /* n-by-n if econ = 0  */
                         /* n-by-min(m,n) if econ = 1  */
        ldv,             /* leading dimension of V, ldv >= max(1,n) */
        d_work, lwork, d_info, gesvdj_params));

    CUDA_CHECK(
        hipMemcpyAsync(U.data(), d_U, sizeof(double) * U.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(V.data(), d_V, sizeof(double) * V.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(
        hipMemcpyAsync(S.data(), d_S, sizeof(double) * S.size(), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    if (0 == info) {
        std::printf("gesvdj converges \n");
    } else if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    } else {
        std::printf("WARNING: info = %d : gesvdj does not converge \n", info);
    }

    std::printf("S = singular values (matlab base-1)\n");
    print_matrix(minmn, 1, S.data(), minmn);
    std::printf("=====\n");

    std::printf("U = left singular vectors (matlab base-1)\n");
    print_matrix(m, m, U.data(), ldu);
    std::printf("=====\n");

    std::printf("V = right singular vectors (matlab base-1)\n");
    print_matrix(n, n, V.data(), ldv);
    std::printf("=====\n");

    /* step 6: measure error of singular value */
    double ds_sup = 0;
    for (int j = 0; j < minmn; j++) {
        double err = fabs(S[j] - S_exact[j]);
        ds_sup = (ds_sup > err) ? ds_sup : err;
    }
    std::printf("|S - S_exact|_sup = %E \n", ds_sup);

    CUSOLVER_CHECK(hipsolverDnXgesvdjGetSweeps(cusolverH, gesvdj_params, &executed_sweeps));

    CUSOLVER_CHECK(hipsolverDnXgesvdjGetResidual(cusolverH, gesvdj_params, &residual));

    std::printf("residual |A - U*S*V**H|_F = %E \n", residual);
    std::printf("number of executed sweeps = %d \n", executed_sweeps);

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_U));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_S));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
