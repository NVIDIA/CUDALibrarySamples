#include <numeric>
#include <vector>
#include <complex>
#include <random>
#include <cstdlib>
#include <cstdio>
#include <hipfft/hipfft.h>
#include <cufftMp.h>
#include <mpi.h>

#include "../common/error_checks.hpp"
#include "../common/scaling.cuh"
#include "../common/generate_random.hpp"
#include "../iterators/box_iterator.hpp"

/**
 * This samples illustrates a basic use of cuFFTMp using the built-in, optimized, data distributions.
 * 
 * It assumes the CPU data is initially distributed according to HIPFFT_XT_FORMAT_INPLACE, a.k.a. X-Slabs.
 * Given a global array of size X * Y * Z, every MPI rank owns approximately (X / ngpus) * Y * Z entries.
 * More precisely, 
 * - The first (X % ngpus) MPI rank each own (X / ngpus + 1) planes of size Y * Z,
 * - The remaining MPI rank each own (X / ngpus) planes of size Y * Z
 * 
 * The CPU data is then copied on GPU and a forward transform is applied.
 * 
 * After that transform, GPU data is distributed according to HIPFFT_XT_FORMAT_INPLACE_SHUFFLED, a.k.a. Y-Slabs.
 * Given a global array of size X * Y * Z, every MPI rank owns approximately X * (Y / ngpus) * Z entries.
 * More precisely, 
 * - The first (Y % ngpus) MPI rank each own (Y / ngpus + 1) planes of size X * Z,
 * - The remaining MPI rank each own (Y / ngpus) planes of size X * Z
 * 
 * A scaling kerel is applied, on the distributed GPU data (distributed according to HIPFFT_XT_FORMAT_INPLACE)
 * This kernel prints some elements to illustrate the HIPFFT_XT_FORMAT_INPLACE_SHUFFLED data distribution and
 * normalize entries by (nx * ny * nz)
 * 
 * Finally, a backward transform is applied.
 * After this, data is again distributed according to HIPFFT_XT_FORMAT_INPLACE, same as the input data.
 * 
 * Data is finally copied back to CPU and compared to the input data. They should be almost identical.
 */

void run_c2c_fwd_inv(size_t nx, size_t ny, size_t nz, std::complex<float>* cpu_data, int rank, int size, MPI_Comm comm) {

    hipfftHandle plan = 0;
    hipStream_t stream = nullptr;

    CUDA_CHECK(hipStreamCreate(&stream));

    CUFFT_CHECK(hipfftCreate(&plan));

    CUFFT_CHECK(cufftMpAttachComm(plan, CUFFT_COMM_MPI, &comm));

    CUFFT_CHECK(hipfftSetStream(plan, stream));

    size_t workspace;
    CUFFT_CHECK(hipfftMakePlan3d(plan, nx, ny, nz, HIPFFT_C2C, &workspace));

    // Allocate memory, copy CPU data to GPU
    // Data is distributed as X-Slabs
    hipLibXtDesc *desc;
    CUFFT_CHECK(hipfftXtMalloc(plan, &desc, HIPFFT_XT_FORMAT_INPLACE));
    CUFFT_CHECK(hipfftXtMemcpy(plan, (void*)desc, (void*)cpu_data, HIPFFT_COPY_HOST_TO_DEVICE));

    // Run C2C Fwd
    CUFFT_CHECK(hipfftXtExecDescriptor(plan, desc, desc, HIPFFT_FORWARD));

    // Data is now distributed as Y-Slabs
    // We run a kernel on the distributed data, using the BoxIterator's for convenience
    auto[begin_d, end_d] = BoxIterators(HIPFFT_XT_FORMAT_INPLACE_SHUFFLED, HIPFFT_C2C, 
                                        rank, size, nx, ny, nz, (hipfftComplex*)desc->descriptor->data[0]);
    const size_t num_elements = std::distance(begin_d, end_d);
    const size_t num_threads  = 128;
    const size_t num_blocks   = (num_elements + num_threads - 1) / num_threads;
    scaling_kernel<<<num_blocks, num_threads, 0, stream>>>(begin_d, end_d, rank, size, nx, ny, nz);
    
    // Run C2C Bwd
    CUFFT_CHECK(hipfftXtExecDescriptor(plan, desc, desc, HIPFFT_BACKWARD));

    // Copy back and free
    // Data is distributed as X-Slabs again
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUFFT_CHECK(hipfftXtMemcpy(plan, (void*)cpu_data, (void*)desc, HIPFFT_COPY_DEVICE_TO_HOST));
    CUFFT_CHECK(hipfftXtFree(desc));

    CUFFT_CHECK(hipfftDestroy(plan));

    CUDA_CHECK(hipStreamDestroy(stream));
};

int main(int argc, char** argv) {

    MPI_Init(&argc, &argv);

    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    int ndevices;
    CUDA_CHECK(hipGetDeviceCount(&ndevices));
    CUDA_CHECK(hipSetDevice(rank % ndevices));

    size_t nx = (argc >= 2 ? atoi(argv[1]) : 8*size);  // any value >= size is OK
    size_t ny = (argc >= 2 ? atoi(argv[1]) : 8*size);  // any value >= size is OK
    size_t nz = (argc >= 2 ? atoi(argv[1]) : 8*size);  // any value >= size is OK

    // We start with X-Slabs
    // Ranks 0 ... (nx % size - 1) have 1 more element in the X dimension
    // and every rank own all elements in the Y and Z dimensions.
    int ranks_cutoff = nx % size;
    size_t my_nx = (nx / size) + (rank < ranks_cutoff ? 1 : 0);
    size_t my_ny =  ny;
    size_t my_nz =  nz;
    
    printf("Hello from rank %d/%d using GPU %d transform of size %zu x %zu x %zu, local size %zu x %zu x %zu\n", rank, size, rank % ndevices, nx, ny, nz, my_nx, my_ny, my_nz);

    // Generate local, distributed, data
    std::vector<std::complex<float>> data(my_nx * my_ny * my_nz);
    generate_random(data, rank);
    std::vector<std::complex<float>> ref = data;

    // Run Forward and Inverse FFT
    run_c2c_fwd_inv(nx, ny, nz, data.data(), rank, size, MPI_COMM_WORLD);

    // Compute error
    double error = compute_error(ref, data, buildCufftBox3d(HIPFFT_XT_FORMAT_INPLACE, HIPFFT_C2C, rank, size, nx, ny, nz));

    MPI_Finalize();

    return assess_error(error);
}
