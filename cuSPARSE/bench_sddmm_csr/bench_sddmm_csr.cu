/*
 * Copyright 1993-2021 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <cusp/csr_matrix.h>
#include <utils/generate_random_data.h>

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return EXIT_FAILURE;                                       \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            return EXIT_FAILURE;                                           \
        }                                                                  \
    }

int main(void)
{
    // Host problem definition
    int A_num_rows = 4;
    int A_num_cols = 4;
    int B_num_rows = A_num_cols;
    int B_num_cols = 3;
    // int   C_nnz        = 9;
    float sparsity = 0.1f;
    int C_nnz = A_num_rows * B_num_cols * sparsity;
    int lda = A_num_cols;
    int ldb = B_num_cols;
    int A_size = lda * A_num_rows;
    int B_size = ldb * B_num_rows;
    // initializing data
    // float hA[]         = { 1.0f,   2.0f,  3.0f,  4.0f,
    //                        5.0f,   6.0f,  7.0f,  8.0f,
    //                        9.0f,  10.0f, 11.0f, 12.0f,
    //                        13.0f, 14.0f, 15.0f, 16.0f };
    // float hB[]         = {  1.0f,  2.0f,  3.0f,
    //                         4.0f,  5.0f,  6.0f,
    //                         7.0f,  8.0f,  9.0f,
    //                        10.0f, 11.0f, 12.0f };
    // int   hC_offsets[] = { 0, 3, 4, 7, 9 };
    // int   hC_columns[] = { 0, 1, 2, 1, 0, 1, 2, 0, 2 };
    // float hC_values[]  = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
    //                        0.0f, 0.0f, 0.0f, 0.0f };
    float *hA = (float *)malloc(A_size * sizeof(float));
    float *hB = (float *)malloc(B_size * sizeof(float));
    generate_random_matrix(hA, A_size);
    generate_random_matrix(hB, B_size);
    cusp::csr_matrix<int, float, cusp::host_memory> hC = generate_random_sparse_matrix<cusp::csr_matrix<int, float, cusp::host_memory>>(A_num_rows, B_num_cols, C_nnz);

    cusp::csr_matrix<int, float, cusp::device_memory> dC(hC);

    float alpha = 1.0f;
    float beta = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    // int   *dC_offsets, *dC_columns;
    // float *dC_values,
    float *dB, *dA;
    CHECK_CUDA(hipMalloc((void **)&dA, A_size * sizeof(float)))
    CHECK_CUDA(hipMalloc((void **)&dB, B_size * sizeof(float)))
    // CHECK_CUDA( hipMalloc((void**) &dC_offsets,
    //                        (A_num_rows + 1) * sizeof(int)) )
    // CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz * sizeof(int))   )
    // CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz * sizeof(float)) )

    CHECK_CUDA(hipMemcpy(dA, hA, A_size * sizeof(float),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dB, hB, B_size * sizeof(float),
                          hipMemcpyHostToDevice))
    // CHECK_CUDA( hipMemcpy(dC_offsets, hC_offsets,
    //                        (A_num_rows + 1) * sizeof(int),
    //                        hipMemcpyHostToDevice) )
    // CHECK_CUDA( hipMemcpy(dC_columns, hC_columns, C_nnz * sizeof(int),
    //                        hipMemcpyHostToDevice) )
    // CHECK_CUDA( hipMemcpy(dC_values, hC_values, C_nnz * sizeof(float),
    //                        hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------

    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void *dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle))
    // Create dense matrix A
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                       HIP_R_32F, HIPSPARSE_ORDER_ROW))
    // Create dense matrix B
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                       HIP_R_32F, HIPSPARSE_ORDER_ROW))
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                     // dC_offsets, dC_columns, dC_values,
                                     (void *)thrust::raw_pointer_cast(dC.row_offsets.data()),
                                     (void *)thrust::raw_pointer_cast(dC.column_indices.data()),
                                     (void *)thrust::raw_pointer_cast(dC.values.data()),
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseSDDMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize))
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

    // execute preprocess (optional)
    CHECK_CUSPARSE(hipsparseSDDMM_preprocess(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer))
    // execute SpMM
    CHECK_CUSPARSE(hipsparseSDDMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer))
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matC))
    CHECK_CUSPARSE(hipsparseDestroy(handle))
    //--------------------------------------------------------------------------
    // device result check
    // CHECK_CUDA( hipMemcpy(hC_values, (void*)thrust::raw_pointer_cast(dC.values.data()), C_nnz * sizeof(float),
    //                        hipMemcpyDeviceToHost) )
    // int correct = 1;
    // for (int i = 0; i < C_nnz; i++) {
    //     if (hC_values[i] != hC_result[i]) {
    //         correct = 0; // direct floating point comparison is not reliable
    //         printf("%d: %f != %f\n", i, hC_values[i], hC_result[i]);
    //         break;
    //     }
    //     else{
    //         printf("%d: %f == %f\n", i, hC_values[i], hC_result[i]);
    //     }
    // }
    // if (correct)
    //     printf("sddmm_csr_example test PASSED\n");
    // else
    //     printf("sddmm_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA(hipFree(dBuffer))
    CHECK_CUDA(hipFree(dA))
    CHECK_CUDA(hipFree(dB))
    // CHECK_CUDA( hipFree(dC_offsets) )
    // CHECK_CUDA( hipFree(dC_columns) )
    // CHECK_CUDA( hipFree(dC_values) )
    free(hA);
    free(hB);
    return EXIT_SUCCESS;
}
