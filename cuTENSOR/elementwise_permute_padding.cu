/*  
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <algorithm>
#include <iostream>
#include <string>
#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                                   \
{ auto const __err = x;                                                   \
  if( __err != HIPTENSOR_STATUS_SUCCESS )                                  \
  { printf("Error: %d %s\n", __LINE__, hiptensorGetErrorString(__err)); exit(-1); } \
};

#define HANDLE_CUDA_ERROR(x)                                          \
{ auto const __err = x;                                               \
  if( __err != hipSuccess )                                          \
  { printf("Error: %d %s\n", __LINE__, hipGetErrorString(__err)); exit(-1); } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, nullptr);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, nullptr);
    }

    float seconds() 
    {
        hipEventRecord(stop_, nullptr);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return static_cast<float>(time * 1e-3);
    }

private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef float floatTypeA;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hiptensorDataType_t          const typeA       = HIPTENSOR_R_32F;
    hiptensorDataType_t          const typeC       = HIPTENSOR_R_32F;
    hiptensorComputeDescriptor_t const descCompute = HIPTENSOR_COMPUTE_DESC_32F;

    floatTypeCompute alpha = (floatTypeCompute)1.0f;

    /**********************
     * This example illustrates the use case where an input tensor A (in host memory) is
     * permuted from an NCHW data layout to NHWC while moving the data from host to device
     * memory C. It also adds padding for 'w' and 'h' modes:
     *
     * C_{c',w',h,n} = alpha * A_{w,h,c,n}
     **********************/

    std::vector<int> modeC{'c','w','h','n'};
    std::vector<int> modeA{'w','h','c','n'};
    int nmodeA = modeA.size();
    int nmodeC = modeC.size();

    std::unordered_map<int, int64_t> extent;
    extent['h'] = 128;
    extent['w'] = 32;
    extent['c'] = 128;
    extent['n'] = 128;

    std::unordered_map<int, int> paddingLeft;
    paddingLeft['h'] = 1;
    paddingLeft['w'] = 1;
    paddingLeft['c'] = 0;
    paddingLeft['n'] = 0;

    std::unordered_map<int, int> paddingRight;
    paddingLeft['h'] = 1;
    paddingLeft['w'] = 1;
    paddingLeft['c'] = 0;
    paddingLeft['n'] = 0;

    floatTypeC paddingValue = 0.0f;

    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentC;
    std::vector<int> paddingLeftVec;
    std::vector<int> paddingRightVec;
    for (auto mode : modeC)
    {
        extentC.push_back(extent[mode]);
        paddingLeftVec.push_back(paddingLeft[mode]);
        paddingRightVec.push_back(paddingRight[mode]);
    }


    /**********************
     * Allocating data
     **********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode] + paddingLeft[mode] + paddingRight[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    printf("Total memory: %.2f GiB\n", (sizeA + sizeC)/1024./1024./1024);

    void *A_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));

    uint32_t const kAlignment = 128;  // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);

    floatTypeA *A, *C;
    HANDLE_CUDA_ERROR(hipHostMalloc((void**) &A, sizeof(floatTypeA) * elementsA));
    HANDLE_CUDA_ERROR(hipHostMalloc((void**) &C, sizeof(floatTypeC) * elementsC));

    if (A == nullptr || C == nullptr)
    {
        printf("Error: Host allocation of A or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (size_t i = 0; i < elementsA; i++)
    {
        A[i] = (((float) rand())/RAND_MAX)*100;
    }
    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(A_d, sizeA, A, sizeA, sizeA, 1, hipMemcpyDefault, nullptr));

    /*************************
     * CUTENSOR
     *************************/

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t  descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descA,
                                                nmodeA,
                                                extentA.data(),
                                                nullptr /* stride */,
                                                typeA,
                                                kAlignment));

    hiptensorTensorDescriptor_t  descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descC,
                                                nmodeC,
                                                extentC.data(),
                                                nullptr /* stride */,
                                                typeC,
                                                kAlignment));

    /*******************************
     * Create Permutation Descriptor
     *******************************/

    hiptensorOperationDescriptor_t  desc;
    HANDLE_ERROR(hiptensorCreatePermutation(handle,
                                           &desc,
                                           descA,
                                           modeA.data(),
                                           HIPTENSOR_OP_IDENTITY,
                                           descC,
                                           modeC.data(),
                                           descCompute));

    /*******************************
     * Set Padding Information
     *******************************/
    HANDLE_ERROR(hiptensorOperationDescriptorSetAttribute(handle,
                                                         desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_LEFT,
                                                         paddingLeftVec.data(),
                                                         sizeof(int) * nmodeC));

    HANDLE_ERROR(hiptensorOperationDescriptorSetAttribute(handle,
                                                         desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_RIGHT,
                                                         paddingRightVec.data(),
                                                         sizeof(int) * nmodeC));

    HANDLE_ERROR(hiptensorOperationDescriptorSetAttribute(handle,
                                                         desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_PADDING_VALUE,
                                                         &paddingValue,
                                                         sizeof(paddingValue)));

    /*****************************
     * Optional (but recommended): ensure that the scalar type is correct.
     *****************************/

    hiptensorDataType_t scalarType;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle, desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                                                         (void*)&scalarType,
                                                         sizeof(scalarType)));

    assert(scalarType == HIPTENSOR_R_32F);


    /**************************
    * Set the algorithm to use
    ***************************/

    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    hiptensorPlanPreference_t  planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(handle,
                                              &planPref,
                                              algo,
                                              HIPTENSOR_JIT_MODE_NONE));

    /**************************
     * Create Plan
     **************************/

    hiptensorPlan_t  plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                                    &plan,
                                    desc,
                                    planPref,
                                    0 /* workspaceSizeLimit */));

    /**********************
     * Run
     **********************/

    double minTimeCUTENSOR = 1e100;
    for (int i = 0; i < 3; i++)
    {
        GPUTimer timer;
        timer.start();

        HANDLE_ERROR(hiptensorPermute(handle,
                        plan,
                        &alpha, A_d, C_d, nullptr /* stream */));

        auto time = timer.seconds();
        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }

    /*************************/

    double transferedBytes = 2.0 * sizeC;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GB/s\n", transferedBytes / minTimeCUTENSOR);

    HANDLE_ERROR(hiptensorDestroy(handle));
    HANDLE_ERROR(hiptensorDestroyPlan(plan));
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    HANDLE_ERROR(hiptensorDestroyPlanPreference(planPref));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));

    if (A) hipHostFree(A);
    if (C) hipHostFree(C);
    if (A_d) hipFree(A_d);
    if (C_d) hipFree(C_d);

    return 0;
}
