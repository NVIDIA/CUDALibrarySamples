/*  
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <iostream>
#include <string>
#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); return err; } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); return err; } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef float floatTypeA;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hipDataType typeA = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hipDataType typeCompute = HIP_R_32F;

    /**********************
     * This example illustrates the use case where an input tensor A (in host memory) is
     * permuted from an NCHW data layout to NHWC while moving the data from host to device
     * memory C:
     *
     * C_{c,w,h,n} = A_{w,h,c,n}
     **********************/

    std::vector<int> modeC{'c','w','h','n'};
    std::vector<int> modeA{'w','h','c','n'};
    int nmodeA = modeA.size();
    int nmodeC = modeC.size();

    std::unordered_map<int, int64_t> extent;
    extent['h'] = 128;
    extent['w'] = 32;
    extent['c'] = 128;
    extent['n'] = 128;

    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);

    /**********************
     * Allocating data
     **********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeC = sizeof(floatTypeC) * elementsC;

    void *A_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));

    floatTypeA *A, *C;
    HANDLE_CUDA_ERROR(hipHostMalloc((void**) &A, sizeof(floatTypeA) * elementsA));
    HANDLE_CUDA_ERROR(hipHostMalloc((void**) &C, sizeof(floatTypeC) * elementsC));

    /*******************
     * Initialize data
     *******************/

    for (size_t i = 0; i < elementsA; i++)
    {
        A[i] = (((float) rand())/RAND_MAX)*100;
    }
    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(A_d, sizeA, A, sizeA, sizeA, 1, hipMemcpyDefault, 0));

    /*************************
     * CUTENSOR
     *************************/

    hiptensorStatus_t err;
    hiptensorHandle_t handle;
    HANDLE_ERROR(cutensorInit(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL /* stride */,
                 typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL /* stride */,
                 typeC, HIPTENSOR_OP_IDENTITY));

    double minTimeCUTENSOR = 1e100;
    for (int i = 0; i < 3; i++)
    {
        GPUTimer timer;
        timer.start();
        const floatTypeCompute one = 1.0f;

        err = hiptensorPermutation(&handle,
                 &one, A_d, &descA, modeA.data(),
                       C_d, &descC, modeC.data(),
                 typeCompute, 0 /* stream */);

        auto time = timer.seconds();
        if (err != HIPTENSOR_STATUS_SUCCESS)
            printf("ERROR: %s\n", hiptensorGetErrorString(err));
        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }

    /*************************/

    double transferedBytes = 2.0 * sizeC;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GB/s\n", transferedBytes / minTimeCUTENSOR);

    if (A) hipHostFree(A);
    if (C) hipHostFree(C);
    if (A_d) hipFree(A_d);
    if (C_d) hipFree(C_d);

    return 0;
}
