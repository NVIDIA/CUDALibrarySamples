/*  
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdio.h>
#include <stdlib.h>

#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); return err; } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); return err; } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hipDataType typeA = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta  = (floatTypeCompute)0.f;

    /**********************
     * Computing (partial) reduction : C_{m,v} = alpha * A_{m,h,k,v} + beta * C_{m,v}
     *********************/

    std::vector<int32_t> modeA{'m','h','k','v'};
    std::vector<int32_t> modeC{'m','v'};
    int32_t nmodeA = modeA.size();
    int32_t nmodeC = modeC.size();

    std::unordered_map<int32_t, int64_t> extent;
    extent['m'] = 196;
    extent['v'] = 64;
    extent['h'] = 256;
    extent['k'] = 64;

    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);

    /**********************
     * Allocating data
     *********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    printf("Total memory: %.2f GiB\n",(sizeA + sizeC)/1024./1024./1024);

    void *A_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&C_d, sizeC));

    floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
    floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

    if (A == NULL || C == NULL)
    {
        printf("Error: Host allocation of A, B, or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = (((float) rand())/RAND_MAX - 0.5)*100;

    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));

    /*************************
     * cuTENSOR
     *************************/ 

    hiptensorHandle_t handle;
    HANDLE_ERROR(cutensorInit(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL /* stride */,
                 typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL /* stride */,
                 typeC, HIPTENSOR_OP_IDENTITY));

    const hiptensorOperator_t opReduce = HIPTENSOR_OP_ADD;

    /**********************
     * Querry workspace
     **********************/

    uint64_t worksize = 0;
    HANDLE_ERROR(cutensorReductionGetWorkspaceSize(&handle, 
                 A_d, &descA, modeA.data(),
                 C_d, &descC, modeC.data(),
                 C_d, &descC, modeC.data(),
                 opReduce, typeCompute, &worksize));
    void *work = nullptr;
    if (worksize > 0)
    {
        if (hipSuccess != hipMalloc(&work, worksize))
        {
            work = nullptr;
            worksize = 0;
        }
    } 

    /**********************
     * Run
     **********************/

    double minTimeCUTENSOR = 1e100;
    hiptensorStatus_t err;
    for(int i=0; i < 3; ++i)
    {
        HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));
        HANDLE_CUDA_ERROR(hipDeviceSynchronize());

        // Set up timing
        GPUTimer timer;
        timer.start();

        err = hiptensorReduction(&handle, 
                (const void*)&alpha, A_d, &descA, modeA.data(),
                (const void*)&beta,  C_d, &descC, modeC.data(), 
                                     C_d, &descC, modeC.data(), 
                opReduce, typeCompute, work, worksize, 0 /* stream */);

        // Synchronize and measure timing
        auto time = timer.seconds();

        if (err != HIPTENSOR_STATUS_SUCCESS)
        {
            printf("ERROR: %s in line %d\n", hiptensorGetErrorString(err), __LINE__);
        }
        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }

    /*************************/

    double transferedBytes = sizeC + sizeA;
    transferedBytes += ((float) beta != 0.f) ? sizeC : 0;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GB/s\n", transferedBytes / minTimeCUTENSOR);

    if (A) free(A);
    if (C) free(C);
    if (A_d) hipFree(A_d);
    if (C_d) hipFree(C_d);
    if (work) hipFree(work);

    return 0;
}
