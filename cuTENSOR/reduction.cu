/*  
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); exit(-1); } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); exit(-1); } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hiptensorDataType_t typeA = HIPTENSOR_R_32F;
    hiptensorDataType_t typeC = HIPTENSOR_R_32F;
    const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;

    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta  = (floatTypeCompute)0.f;

    /**********************
     * Computing (partial) reduction : C_{m,v} = alpha * A_{m,h,k,v} + beta * C_{m,v}
     *********************/

    std::vector<int32_t> modeA{'m','h','k','v'};
    std::vector<int32_t> modeC{'m','v'};
    int32_t nmodeA = modeA.size();
    int32_t nmodeC = modeC.size();

    std::unordered_map<int32_t, int64_t> extent;
    extent['m'] = 196;
    extent['v'] = 64;
    extent['h'] = 256;
    extent['k'] = 64;

    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);

    /**********************
     * Allocating data
     *********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    printf("Total memory: %.2f GiB\n",(sizeA + sizeC)/1024./1024./1024);

    void *A_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**)&C_d, sizeC));

    const uint32_t kAlignment = 256; // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);

    floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
    floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

    if (A == NULL || C == NULL)
    {
        printf("Error: Host allocation of A, B, or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = (((float) rand())/RAND_MAX - 0.5)*100;

    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));

    /*************************
     * cuTENSOR
     *************************/ 

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL /* stride */,
                 typeA, kAlignment));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL /* stride */,
                 typeC, kAlignment));

    const hiptensorOperator_t opReduce = HIPTENSOR_OP_ADD;

    /*******************************
     * Create Reduction Descriptor
     *******************************/

    hiptensorOperationDescriptor_t desc;
    HANDLE_ERROR(hiptensorCreateReduction(
                 handle, &desc,
                 descA, modeA.data(), HIPTENSOR_OP_IDENTITY,
                 descC, modeC.data(), HIPTENSOR_OP_IDENTITY,
                 descC, modeC.data(),
                 opReduce, descCompute));

    /**************************
    * Set the algorithm to use
    ***************************/

    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    hiptensorPlanPreference_t planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(
                               handle,
                               &planPref,
                               algo,
                               HIPTENSOR_JIT_MODE_NONE));

    /**********************
     * Query workspace estimate
     **********************/

    uint64_t workspaceSizeEstimate = 0;
    const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
    HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle,
                                          desc,
                                          planPref,
                                          workspacePref,
                                          &workspaceSizeEstimate));

    /**************************
     * Create Contraction Plan
     **************************/

    hiptensorPlan_t plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                 &plan,
                 desc,
                 planPref,
                 workspaceSizeEstimate));

    /**************************
     * Optional: Query information about the created plan
     **************************/

    // query actually used workspace
    uint64_t actualWorkspaceSize = 0;
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
        plan,
        HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
        &actualWorkspaceSize,
        sizeof(actualWorkspaceSize)));

    // At this point the user knows exactly how much memory is need by the operation and
    // only the smaller actual workspace needs to be allocated
    assert(actualWorkspaceSize <= workspaceSizeEstimate);

    void *work = nullptr;
    if (actualWorkspaceSize > 0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
    }

    /**********************
     * Run
     **********************/

    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    double minTimeCUTENSOR = 1e100;
    for(int i=0; i < 3; ++i)
    {
        HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));
        HANDLE_CUDA_ERROR(hipDeviceSynchronize());

        // Set up timing
        GPUTimer timer;
        timer.start();

        HANDLE_ERROR(hiptensorReduce(handle, plan,
                (const void*)&alpha, A_d,
                (const void*)&beta,  C_d, 
                                     C_d, work, actualWorkspaceSize, stream));

        // Synchronize and measure timing
        auto time = timer.seconds();
        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }

    /*************************/

    double transferedBytes = sizeC + sizeA;
    transferedBytes += ((float) beta != 0.f) ? sizeC : 0;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GB/s\n", transferedBytes / minTimeCUTENSOR);

    HANDLE_ERROR(hiptensorDestroy(handle));
    HANDLE_ERROR(hiptensorDestroyPlan(plan));
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
    HANDLE_CUDA_ERROR(hipStreamDestroy(stream));

    if (A) free(A);
    if (C) free(C);
    if (A_d) hipFree(A_d);
    if (C_d) hipFree(C_d);
    if (work) hipFree(work);

    return 0;
}
