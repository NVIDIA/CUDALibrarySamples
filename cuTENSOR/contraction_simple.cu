/*  
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdlib.h>
#include <stdio.h>

#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); return err; } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); return err; } \
};

/* This routine computes the tensor contraction \f[ D = alpha * A * B + beta * C \f] using the staged-API */
hiptensorStatus_t cutensorContractionSimple(const hiptensorHandle_t* handle,
                                           const void* alpha, const void *A, const hiptensorTensorDescriptor_t* descA, const int32_t modeA[],
                                                              const void *B, const hiptensorTensorDescriptor_t* descB, const int32_t modeB[],
                                           const void* beta,  const void *C, const hiptensorTensorDescriptor_t* descC, const int32_t modeC[],
                                                                    void *D, const hiptensorTensorDescriptor_t* descD, const int32_t modeD[],
                                           hiptensorComputeDescriptor_t typeCompute, hiptensorAlgo_t algo, hiptensorWorksizePreference_t workPref,
                                           hipStream_t stream)
{
    /**********************************************
     * Retrieve the memory alignment for each tensor
     **********************************************/ 

     uint32_t alignmentRequirementA;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                  A, descA, &alignmentRequirementA));

     uint32_t alignmentRequirementB;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                  B, descB, &alignmentRequirementB));

     uint32_t alignmentRequirementC;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                  C, descC, &alignmentRequirementC));

     uint32_t alignmentRequirementD;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(handle,
                  D, descD, &alignmentRequirementD));

    /*******************************
     * Create Contraction Descriptor
     *******************************/

    cutensorContractionDescriptor_t desc;
    HANDLE_ERROR(cutensorInitContractionDescriptor(handle, 
                 &desc,
                 descA, modeA, alignmentRequirementA,
                 descB, modeB, alignmentRequirementB,
                 descC, modeC, alignmentRequirementC,
                 descD, modeD, alignmentRequirementD,
                 typeCompute));

    /**************************
    * Set the algorithm to use
    ***************************/

    cutensorContractionFind_t find;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 handle, &find, 
                 algo));

    /**********************
     * Query workspace
     **********************/

    size_t worksize = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(handle,
                 &desc,
                 &find,
                 workPref, &worksize));

    void *work = nullptr;
    if (worksize > 0)
    {
        if(hipSuccess != hipMalloc(&work, worksize))
        {
            work = nullptr;
            worksize = 0;
        }
    } 

    /**************************
     * Create Contraction Plan
     **************************/

    hiptensorContractionPlan_t plan;
    HANDLE_ERROR(cutensorInitContractionPlan(handle,
                 &plan,
                 &desc,
                 &find,
                 worksize));

    /**********************
     * Run
     **********************/

    HANDLE_ERROR(hiptensorContraction(handle,
                 &plan,
                 (void*) &alpha, A, B,
                 (void*) &beta,  C, D, 
                 work, worksize, stream));

    return HIPTENSOR_STATUS_SUCCESS;
}


int main()
{
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hipDataType typeA = HIP_R_32F;
    hipDataType typeB = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

    floatTypeCompute alpha = (floatTypeCompute) 1.1f;
    floatTypeCompute beta  = (floatTypeCompute) 0.f;

    /**********************
     * Computing: C_{m,u,n,v} = alpha * A_{m,h,k,n} B_{u,k,v,h} + beta * C_{m,u,n,v}
     **********************/

    std::vector<int> modeC{'m','u','n','v'};
    std::vector<int> modeA{'m','h','k','n'};
    std::vector<int> modeB{'u','k','v','h'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    std::unordered_map<int, int64_t> extent;
    extent['m'] = 96;
    extent['n'] = 96;
    extent['u'] = 96;
    extent['v'] = 64;
    extent['h'] = 64;
    extent['k'] = 64;

    double gflops = (2.0 * extent['m'] * extent['n'] * extent['u'] * extent['v'] * extent['k'] * extent['h']) /1e9;
    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);

    /**********************
     * Allocating data
     **********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeB = sizeof(floatTypeB) * elementsB;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC)/1024./1024./1024);

    void *A_d, *B_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &B_d, sizeB));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));

    floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
    floatTypeB *B = (floatTypeB*) malloc(sizeof(floatTypeB) * elementsB);
    floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

    if (A == NULL || B == NULL || C == NULL)
    {
        printf("Error: Host allocation of A, B, or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsB; i++)
        B[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = (((float) rand())/RAND_MAX - 0.5)*100;

    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

    /*************************
     * cuTENSOR
     *************************/ 

    hiptensorHandle_t handle;
    HANDLE_ERROR(cutensorInit(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL /* stride */,
                 typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descB,
                 nmodeB,
                 extentB.data(),
                 NULL /* stride */,
                 typeB, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL /* stride */,
                 typeC, HIPTENSOR_OP_IDENTITY));

    HANDLE_ERROR(cutensorContractionSimple(&handle,
                 (void*)&alpha, A_d, &descA, modeA.data(),
                                B_d, &descB, modeB.data(),
                 (void*)&beta,  C_d, &descC, modeC.data(),
                                C_d, &descC, modeC.data(),
                 typeCompute, HIPTENSOR_ALGO_DEFAULT,
                 CUTENSOR_WORKSPACE_RECOMMENDED, 0 /* stream */));

    return 0;
}
