/*  
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <chrono>
#include <complex>
#include <stdlib.h>
#include <stdio.h>
#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

// Handle cuTENSOR errors
#define HANDLE_ERROR(x) {                                                                \
  const auto err = x;                                                                    \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                                   \
  { printf("Error: %s in line %d\n", hiptensorGetErrorString(err), __LINE__); exit(-1); } \
};

// Handle CUDA errors
#define HANDLE_CUDA_ERROR(x) {                                                       \
  const auto err = x;                                                                \
  if( err != hipSuccess )                                                           \
  { printf("Error: %s in line %d\n", hipGetErrorString(err), __LINE__); exit(-1); } \
};

class CPUTimer
{
public:
    void start()
    {
        start_ = std::chrono::steady_clock::now();
    }

    double seconds()
    {
        end_ = std::chrono::steady_clock::now();
        elapsed_ = end_ - start_;
        //return in ms
        return elapsed_.count() * 1000;
    }

private:
    typedef std::chrono::steady_clock::time_point tp;
    tp start_;
    tp end_;
    std::chrono::duration<double> elapsed_;
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef std::complex<float> TypeA;
    typedef std::complex<float> TypeB;
    typedef std::complex<float> TypeC;
    typedef std::complex<float> TypeScalar;

    auto alpha = TypeScalar(1.1, 0.0);
    auto beta  = TypeScalar(0.0, 0.0);

    hiptensorDataType_t typeA = HIPTENSOR_C_32F;
    hiptensorDataType_t typeB = HIPTENSOR_C_32F;
    hiptensorDataType_t typeC = HIPTENSOR_C_32F;
    hiptensorComputeDescriptor_t descCompute = CUTENSOR_COMPUTE_DESC_3XTF32;

    /**********************
     * Computing: C_{0,1,2,3,4,6,8,9,25,26,10,12,14,27,15,28,17,19,29,20,21,30,23,24} =
     *                   \alpha A_{0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24}
     *                   B_{25,26,27,28,29,30,5,7,11,13,16,18,22}
     *                   + \beta C_{0,1,2,3,4,6,8,9,25,26,10,12,14,27,15,28,17,19,29,20,21,30,23,24}
     **********************/

    /* ***************************** */

    // Create vector of modes
    std::vector<int> modeC{0,1,2,3,4,6,8,9,25,26,10,12,14,27,15,28,17,19,29,20,21,30,23,24};
    std::vector<int> modeA{0,2,1,4,3,5,6,7,8,9,10,11,12,13,14,15,16,17,18,20,19,21,22,23,24};
    std::vector<int> modeB{25,26,27,28,29,30,5,7,11,13,16,18,22};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    // Extents
    std::unordered_map<int, int64_t> extent;
    for (auto i = 0; i <= 30; i++)
        extent[i] = 2;

    // Create a vector of extents for each tensor
    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);

    /**********************
     * Allocating data
     **********************/

    // Number of elements of each tensor
    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    // Size in bytes
    size_t sizeA = sizeof(TypeA) * elementsA;
    size_t sizeB = sizeof(TypeB) * elementsB;
    size_t sizeC = sizeof(TypeC) * elementsC;
    printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC)/1024./1024./1024);

    // Allocate on device
    void *A_d, *B_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &B_d, sizeB));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));

    TypeA *A = (TypeA*) malloc(sizeof(TypeA) * elementsA);
    TypeB *B = (TypeB*) malloc(sizeof(TypeB) * elementsB);
    TypeC *C = (TypeC*) malloc(sizeof(TypeC) * elementsC);

    if (A == nullptr || B == nullptr || C == nullptr)
    {
        printf("Error: Host allocation of A, B, or C.\n");
        exit(-1);
    }

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsB; i++)
        B[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = (((float) rand())/RAND_MAX - 0.5)*100;

    // Copy to device
    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

    const uint32_t kAlignment = 128; // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(B_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);

    /*************************
     * cuTENSOR
     *************************/ 

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    // Read kernel cache from file (if the file was generated by a prior execution)
    auto readKernelCacheStatus = hiptensorReadKernelCacheFromFile(handle, "kernelCache.bin");

    if (readKernelCacheStatus == HIPTENSOR_STATUS_IO_ERROR)
        printf("No kernel cache found. It will be generated before the end of this execution.\n");
    else if (readKernelCacheStatus == HIPTENSOR_STATUS_SUCCESS)
        printf("Kernel cache found and read successfully.\n");
    else
        HANDLE_ERROR(readKernelCacheStatus);

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descA,
                                                nmodeA,
                                                extentA.data(),
                                                NULL,/*stride*/
                                                typeA, kAlignment));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descB,
                                                nmodeB,
                                                extentB.data(),
                                                NULL,/*stride*/
                                                typeB, kAlignment));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descC,
                                                nmodeC,
                                                extentC.data(),
                                                NULL,/*stride*/
                                                typeC, kAlignment));

    /*******************************
     * Create Contraction Descriptor
     *******************************/

    hiptensorOperationDescriptor_t desc;
    HANDLE_ERROR(hiptensorCreateContraction(handle, 
                                           &desc,
                                           descA, modeA.data(), /* unary operator A*/HIPTENSOR_OP_IDENTITY,
                                           descB, modeB.data(), /* unary operator B*/HIPTENSOR_OP_IDENTITY,
                                           descC, modeC.data(), /* unary operator C*/HIPTENSOR_OP_IDENTITY,
                                           descC, modeC.data(),
                                           descCompute));

    /*****************************
     * Optional (but recommended): ensure that the scalar type is correct.
     *****************************/

    hiptensorDataType_t scalarType;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle,
                                                         desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
                                                         (void*)&scalarType,
                                                         sizeof(scalarType)));

    assert(scalarType == HIPTENSOR_C_32F);

    /**************************
    * Set the algorithm to use -- without just-in-time compilation
    ***************************/

    const hiptensorAlgo_t algo = CUTENSOR_ALGO_GETT;

    hiptensorPlanPreference_t planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(handle,
                                              &planPref,
                                              algo,
                                              HIPTENSOR_JIT_MODE_NONE));

    /**********************
     * Query workspace estimate
     **********************/

    uint64_t workspaceSizeEstimate = 0;
    const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
    HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle,
                                               desc,
                                               planPref,
                                               workspacePref,
                                               &workspaceSizeEstimate));

    /**************************
     * Create Contraction Plan -- without just-in-time compilation
     **************************/

    hiptensorPlan_t plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                                    &plan,
                                    desc,
                                    planPref,
                                    workspaceSizeEstimate));

    /**************************
     * Optional: Query information about the created plan
     **************************/

    // query actually used workspace
    uint64_t actualWorkspaceSize = 0;
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
                                              plan,
                                              HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
                                              &actualWorkspaceSize,
                                              sizeof(actualWorkspaceSize)));

    // At this point the user knows exactly how much memory is need by the operation and
    // only the smaller actual workspace needs to be allocated
    assert(actualWorkspaceSize <= workspaceSizeEstimate);

    void *work = nullptr;
    if (actualWorkspaceSize > 0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
    }

    /**********************
     * Execute the tensor contraction
     **********************/

    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    double minTimeCUTENSOR = 1e100;
    for (int i=0; i < 3; ++i)
    {
        hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice);

        // Set up timing
        GPUTimer timer;
        timer.start();

        HANDLE_ERROR(hiptensorContract(handle,
                                      plan,
                                      (void*) &alpha, A_d, B_d,
                                      (void*) &beta,  C_d, C_d,
                                      work, actualWorkspaceSize, stream))

        // Synchronize and measure timing
        auto time = timer.seconds();

        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }

    /*************************/

    /**************************
    * Set the algorithm to use -- with just-in-time compilation
    ***************************/

    hiptensorPlanPreference_t planPrefJit;
    HANDLE_ERROR(hiptensorCreatePlanPreference(handle,
                                              &planPrefJit,
                                              algo,
                                              HIPTENSOR_JIT_MODE_DEFAULT));

    /**********************
     * Query workspace estimate
     **********************/

    uint64_t workspaceSizeEstimateJit = 0;
    const hiptensorWorksizePreference_t workspacePrefJit = HIPTENSOR_WORKSPACE_DEFAULT;
    HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle,
                                               desc,
                                               planPrefJit,
                                               workspacePrefJit,
                                               &workspaceSizeEstimateJit));

    /**************************
     * Create Contraction Plan -- with just-in-time compilation
     **************************/

    hiptensorPlan_t planJit;
    CPUTimer jitPlanTimer;
    jitPlanTimer.start();
    // This is where the kernel is actually compiled
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                                    &planJit,
                                    desc,
                                    planPrefJit,
                                    workspaceSizeEstimateJit));
    auto jitPlanTime = jitPlanTimer.seconds();

    /**************************
     * Optional: Query information about the created plan
     **************************/

    // query actually used workspace
    uint64_t actualWorkspaceSizeJit = 0;
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
                                              planJit,
                                              HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
                                              &actualWorkspaceSizeJit,
                                              sizeof(actualWorkspaceSizeJit)));

    // At this point the user knows exactly how much memory is need by the operation and
    // only the smaller actual workspace needs to be allocated
    assert(actualWorkspaceSizeJit <= workspaceSizeEstimateJit);

    void *workJit = nullptr;
    if (actualWorkspaceSizeJit > 0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&workJit, actualWorkspaceSizeJit));
        assert(uintptr_t(workJit) % 128 == 0); // workspace must be aligned to 128 byte-boundary
    }

    /**********************
     * Execute the tensor contraction using the JIT compiled kernel
     **********************/

    double minTimeCUTENSORJit = 1e100;
    for (int i=0; i < 3; ++i)
    {
        hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice);

        // Set up timing
        GPUTimer timer;
        timer.start();

        HANDLE_ERROR(hiptensorContract(handle,
                                      planJit,
                                      (void*) &alpha, A_d, B_d,
                                      (void*) &beta,  C_d, C_d,
                                      workJit, actualWorkspaceSizeJit, stream))

        // Synchronize and measure timing
        auto time = timer.seconds();

        minTimeCUTENSORJit = (minTimeCUTENSORJit < time) ? minTimeCUTENSORJit : time;
    }

    /*************************/

    float flops;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle,
                                                         desc,
                                                         HIPTENSOR_OPERATION_DESCRIPTOR_FLOPS,
                                                         (void*)&flops,
                                                         sizeof(flops)));
    auto gflops = flops / 1e9;
    auto gflopsPerSec = gflops / minTimeCUTENSOR;
    auto gflopsPerSecJit = gflops / minTimeCUTENSORJit;

    printf("cuTENSOR    : %6.0f GFLOPs/s\n", gflopsPerSec);
    printf("cuTENSOR JIT: %6.0f GFLOPs/s\n", gflopsPerSecJit);
    printf("Speedup: %.1fx\n", gflopsPerSecJit / gflopsPerSec);
    printf("JIT Compilation time: %.1f seconds ", jitPlanTime / 1e3);
    if (readKernelCacheStatus == HIPTENSOR_STATUS_SUCCESS)
        printf("(Kernel cache file was read successfully; Compilation was not required)\n");
    else
        printf("\n");

    // Write kernel cache to file
    HANDLE_ERROR(hiptensorWriteKernelCacheToFile(handle, "kernelCache.bin"))
    printf("Kernel cache written to file. Will be read in next execution.\n");

    HANDLE_ERROR(hiptensorDestroy(handle));
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descB));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
    HANDLE_CUDA_ERROR(hipStreamDestroy(stream));
    HANDLE_ERROR(hiptensorDestroyPlanPreference(planPref));
    HANDLE_ERROR(hiptensorDestroyPlan(plan));
    HANDLE_ERROR(hiptensorDestroyPlanPreference(planPrefJit));
    HANDLE_ERROR(hiptensorDestroyPlan(planJit));

    if (A) free(A);
    if (B) free(B);
    if (C) free(C);
    if (A_d) hipFree(A_d);
    if (B_d) hipFree(B_d);
    if (C_d) hipFree(C_d);
    if (work) hipFree(work);
    if (workJit) hipFree(workJit);

    printf("Successful completion\n");
    return 0;
}
