/*  
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdlib.h>
#include <stdio.h>

#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); return err; } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); return err; } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hipDataType typeA = HIP_R_32F;
    hipDataType typeB = HIP_R_32F;
    hipDataType typeC = HIP_R_32F;
    hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta  = (floatTypeCompute)0.f;

    /**********************
     * Computing: C_{m,u,n,v} = alpha * A_{m,h,k,n} B_{u,k,v,h} + beta * C_{m,u,n,v}
     **********************/

    std::vector<int> modeC{'m','u','n','v'};
    std::vector<int> modeA{'m','h','k','n'};
    std::vector<int> modeB{'u','k','v','h'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    std::unordered_map<int, int64_t> extent;
    extent['m'] = 96;
    extent['n'] = 96;
    extent['u'] = 96;
    extent['v'] = 64;
    extent['h'] = 64;
    extent['k'] = 64;

    double gflops = (2.0 * extent['m'] * extent['n'] * extent['u'] * extent['v'] * extent['k'] * extent['h']) /1e9;

    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);

    /**********************
     * Allocating data
     **********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeB = sizeof(floatTypeB) * elementsB;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC)/1024./1024./1024);

    void *A_d, *B_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &B_d, sizeB));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));

    floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
    floatTypeB *B = (floatTypeB*) malloc(sizeof(floatTypeB) * elementsB);
    floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

    if (A == NULL || B == NULL || C == NULL)
    {
        printf("Error: Host allocation of A or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsB; i++)
        B[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = (((float) rand())/RAND_MAX - 0.5)*100;

    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

    /*************************
     * cuTENSOR
     *************************/ 

    hiptensorHandle_t handle;
    HANDLE_ERROR(cutensorInit(&handle));

    /**********************
     * Setup planCache
     **********************/
    constexpr int32_t numCachelines = 1024;
    size_t sizeCache = numCachelines * sizeof(cutensorPlanCacheline_t);
    printf("Allocating: %.2f kB for the cache\n", sizeCache / 1000.);
    cutensorPlanCacheline_t* cachelines = (cutensorPlanCacheline_t*) malloc(sizeCache);
    HANDLE_ERROR( cutensorHandleAttachPlanCachelines(&handle, cachelines, numCachelines) );

    const char cacheFilename[] = "./cache.bin";
    uint32_t numCachelinesRead = 0;
    hiptensorStatus_t status = cutensorHandleReadCacheFromFile(&handle, cacheFilename, &numCachelinesRead);
    if (status == HIPTENSOR_STATUS_SUCCESS)
    {
        printf("%d cachelines have been successfully read from file (%s).\n", numCachelinesRead, cacheFilename);
    }
    else if (status == HIPTENSOR_STATUS_IO_ERROR)
    {
        printf("File (%s) doesn't seem to exist.\n", cacheFilename);
    }
    else if (status == HIPTENSOR_STATUS_INSUFFICIENT_WORKSPACE)
    {
        printf("Cannot read cache: Please attach at least %d cachelines to the handle.\n", numCachelinesRead);
    }

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL,/*stride*/
                 typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descB,
                 nmodeB,
                 extentB.data(),
                 NULL,/*stride*/
                 typeB, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( &handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL,/*stride*/
                 typeC, HIPTENSOR_OP_IDENTITY));

    /**********************************************
     * Retrieve the memory alignment for each tensor
     **********************************************/ 

     uint32_t alignmentRequirementA;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
                  A_d,
                  &descA,
                  &alignmentRequirementA));

     uint32_t alignmentRequirementB;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
                  B_d,
                  &descB,
                  &alignmentRequirementB));

     uint32_t alignmentRequirementC;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
                  C_d,
                  &descC, 
                  &alignmentRequirementC));

    /*******************************
     * Create Contraction Descriptor
     *******************************/

    cutensorContractionDescriptor_t desc;
    HANDLE_ERROR(cutensorInitContractionDescriptor(&handle, 
                 &desc,
                 &descA, modeA.data(), alignmentRequirementA,
                 &descB, modeB.data(), alignmentRequirementB,
                 &descC, modeC.data(), alignmentRequirementC,
                 &descC, modeC.data(), alignmentRequirementC,
                 typeCompute));

    /**************************
    * Set the algorithm to use
    ***************************/

    cutensorContractionFind_t find;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 &handle, &find, 
                 HIPTENSOR_ALGO_DEFAULT));

    const hiptensorCacheMode_t cacheMode = HIPTENSOR_CACHE_MODE_PEDANTIC;
    HANDLE_ERROR(cutensorContractionFindSetAttribute(
        &handle,
        &find,
        CUTENSOR_CONTRACTION_FIND_CACHE_MODE,
        &cacheMode,
        sizeof(hiptensorCacheMode_t)));

    const hiptensorAutotuneMode_t autotuneMode = HIPTENSOR_AUTOTUNE_MODE_INCREMENTAL;
    HANDLE_ERROR(cutensorContractionFindSetAttribute(
        &handle,
        &find,
        CUTENSOR_CONTRACTION_FIND_AUTOTUNE_MODE,
        &autotuneMode ,
        sizeof(hiptensorAutotuneMode_t)));

    const uint32_t incCount = 4;
    HANDLE_ERROR(cutensorContractionFindSetAttribute(
        &handle,
        &find,
        CUTENSOR_CONTRACTION_FIND_INCREMENTAL_COUNT,
        &incCount,
        sizeof(uint32_t)));

    /**********************
     * Query workspace
     **********************/

    uint64_t worksize = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(&handle,
                 &desc,
                 &find,
                 HIPTENSOR_WORKSPACE_MAX, &worksize)); // TODO

    void *work = nullptr;
    if (worksize > 0)
    {
        if (hipSuccess != hipMalloc(&work, worksize))
        {
            work = nullptr;
            worksize = 0;
        }
    } 

    /**************************
     * Create Contraction Plan
     **************************/

    hiptensorContractionPlan_t plan;

    /**********************
     * Run
     **********************/

    double minTimeCUTENSOR = 1e100;
    // warm-up GPU (without caching) (optional, but recommended for more accurate measurements later on)
    for (int i=0; i < 4; ++i)
    {

        cutensorContractionFind_t find_copy = find;

        const hiptensorCacheMode_t cacheMode = HIPTENSOR_CACHE_MODE_NONE;
        HANDLE_ERROR(cutensorContractionFindSetAttribute(
                    &handle,
                    &find_copy,
                    CUTENSOR_CONTRACTION_FIND_CACHE_MODE,
                    &cacheMode,
                    sizeof(hiptensorCacheMode_t)));

        // To take advantage of the incremental-autotuning (via the cache), it's important to re-initialize the plan
        HANDLE_ERROR(cutensorInitContractionPlan(&handle,
                    &plan,
                    &desc,
                    &find_copy,
                    worksize));

        HANDLE_ERROR(hiptensorContraction(&handle,
                                  &plan,
                                  (void*) &alpha, A_d, B_d,
                                  (void*) &beta,  C_d, C_d, 
                                  work, worksize, 0 /* stream */));
    }
    hipDeviceSynchronize();
    printf("Warm-up completed.\n");


    for (int i=0; i < incCount + 1; ++i) // last iteration will hit the cache
    {
        hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        // Set up timing
        GPUTimer timer;
        timer.start();

        // To take advantage of the incremental-autotuning (via the cache), it's important to re-initialize the plan
        HANDLE_ERROR(cutensorInitContractionPlan(&handle,
                    &plan,
                    &desc,
                    &find,
                    worksize));

        hiptensorStatus_t err = hiptensorContraction(&handle,
                                  &plan,
                                  (void*) &alpha, A_d, B_d,
                                  (void*) &beta,  C_d, C_d, 
                                  work, worksize, 0 /* stream */);

        // Synchronize and measure timing
        auto time = timer.seconds();

        if (err != HIPTENSOR_STATUS_SUCCESS)
        {
            printf("ERROR: %s in %s:%d\n", hiptensorGetErrorString(err), __FILE__, __LINE__);
            break;
        }
        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }

    /*************************/

    double transferedBytes = sizeC + sizeA + sizeB;
    transferedBytes += ((float) beta != 0.f) ? sizeC : 0;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GFLOPs/s %.2f GB/s\n", gflops / minTimeCUTENSOR, transferedBytes/ minTimeCUTENSOR);


    /*
     * Optional: Write cache to disk
     */
    HANDLE_ERROR( cutensorHandleWriteCacheToFile(&handle, cacheFilename) );
    printf("Cache has been successfully written to file (%s).\n", cacheFilename);

    // Detach cache and free-up resources
    HANDLE_ERROR( cutensorHandleDetachPlanCachelines(&handle) );

    if (A) free(A);
    if (B) free(B);
    if (C) free(C);
    if (cachelines) free(cachelines);
    if (A_d) hipFree(A_d);
    if (B_d) hipFree(B_d);
    if (C_d) hipFree(C_d);
    if (work) hipFree(work);

    return 0;
}
