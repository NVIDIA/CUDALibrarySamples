/*  
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <algorithm>
#include <iostream>
#include <string>
#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                                   \
{ auto const __err = x;                                                   \
  if( __err != HIPTENSOR_STATUS_SUCCESS )                                  \
  { printf("Error: %d %s\n", __LINE__, hiptensorGetErrorString(__err)); exit(-1); } \
};

#define HANDLE_CUDA_ERROR(x)                                          \
{ auto const __err = x;                                               \
  if( __err != hipSuccess )                                          \
  { printf("Error: %d %s\n", __LINE__, hipGetErrorString(__err)); exit(-1); } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, nullptr);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, nullptr);
    }

    float seconds() 
    {
        hipEventRecord(stop_, nullptr);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return static_cast<float>(time * 1e-3);
    }
    private:
    hipEvent_t start_, stop_;
};


int main()
{
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeCompute;

    hiptensorDataType_t          const typeA       = HIPTENSOR_R_32F;
    hiptensorDataType_t          const typeB       = HIPTENSOR_R_32F;
    hiptensorDataType_t          const typeC       = HIPTENSOR_R_32F;
    hiptensorComputeDescriptor_t const descCompute = HIPTENSOR_COMPUTE_DESC_32F;

    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta  = (floatTypeCompute)1.3f;;
    floatTypeCompute gamma = (floatTypeCompute)1.2f;

    /**********************
     * Computing: D_{a,b,c} = alpha * A_{b,a,c} + beta * B_{c,a,b} + gamma * C_{a,b,c}
     **********************/

    std::vector<int> modeC{'a','b','c'};
    std::vector<int> modeA{'c','b','a'};
    std::vector<int> modeB{'c','a','b'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    std::unordered_map<int, int64_t> extent;
    extent['a'] = 400;
    extent['b'] = 200;
    extent['c'] = 300;
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);
    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);

    /**********************
     * Allocating data
     **********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeB = sizeof(floatTypeB) * elementsB;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC)/1024./1024./1024);

    void *A_d, *B_d, *C_d, *D_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &B_d, sizeB));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &D_d, sizeC));

    const uint32_t kAlignment = 256; // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(B_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);
    assert(uintptr_t(D_d) % kAlignment == 0);

    floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
    floatTypeB *B = (floatTypeB*) malloc(sizeof(floatTypeB) * elementsB);
    floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

    if (A == nullptr || B == nullptr || C == nullptr)
    {
        printf("Error: Host allocation of A, B, or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (size_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX)*100;
    for (size_t i = 0; i < elementsB; i++)
        B[i] = (((float) rand())/RAND_MAX)*100;
    for (size_t i = 0; i < elementsC; i++)
        C[i] = (((float) rand())/RAND_MAX)*100;

    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(C_d, sizeC, C, sizeC, sizeC, 1, hipMemcpyDefault, nullptr));
    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(D_d, sizeC, C, sizeC, sizeC, 1, hipMemcpyDefault, nullptr));
    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(A_d, sizeA, A, sizeA, sizeA, 1, hipMemcpyDefault, nullptr));
    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(B_d, sizeB, B, sizeB, sizeB, 1, hipMemcpyDefault, nullptr));

    /*************************
     * Memcpy perf 
     *************************/

    double minTimeMEMCPY = 1e100;
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());
    GPUTimer timer;
    timer.start();
    HANDLE_CUDA_ERROR(hipMemcpy2DAsync(D_d, sizeC, C_d, sizeC, sizeC, 1, hipMemcpyDefault, nullptr));
    HANDLE_CUDA_ERROR(hipDeviceSynchronize());
    minTimeMEMCPY = timer.seconds();

    /*************************
     * cuTENSOR
     *************************/

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t  descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descA, nmodeA, extentA.data(),
                                                nullptr /* stride */,
                                                typeA,
                                                kAlignment));

    hiptensorTensorDescriptor_t  descB;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descB, nmodeB, extentB.data(),
                                                nullptr /* stride */,
                                                typeB,
                                                kAlignment));

    hiptensorTensorDescriptor_t  descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                                                &descC, nmodeC, extentC.data(),
                                                nullptr /* stride */,
                                                typeC,
                                                kAlignment));

    /*******************************
     * Create Elementwise Trinary Descriptor
     *******************************/

    hiptensorOperationDescriptor_t desc;
    HANDLE_ERROR(hiptensorCreateElementwiseTrinary(handle, 
                                                  &desc,
                                                  descA, modeA.data(), /* unary operator A */ HIPTENSOR_OP_IDENTITY,
                                                  descB, modeB.data(), /* unary operator B */ HIPTENSOR_OP_IDENTITY,
                                                  descC, modeC.data(), /* unary operator C */ HIPTENSOR_OP_IDENTITY,
                                                  descC, modeC.data(),
                                                  /* binary operator AC  */ HIPTENSOR_OP_ADD,
                                                  /* binary operator ABC */ HIPTENSOR_OP_ADD,
                                                  descCompute));

    /**************************
    * Set the algorithm to use
    ***************************/

    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    hiptensorPlanPreference_t  planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(handle,
                                              &planPref,
                                              algo,
                                              HIPTENSOR_JIT_MODE_NONE));

    /**************************
     * Create Plan
     **************************/

    hiptensorPlan_t  plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                                    &plan,
                                    desc,
                                    planPref,
                                    0 /*workspaceSizeEstimate*/));

    /**********************
     * Run
     **********************/

    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    double minTimeCUTENSOR = 1e100;
    for (int i = 0; i < 3; i++)
    {
        HANDLE_CUDA_ERROR(hipMemcpy2DAsync(D_d, sizeC, C, sizeC, sizeC, 1, hipMemcpyDefault, nullptr));
        timer.start();
        HANDLE_ERROR(hiptensorElementwiseTrinaryExecute(handle, plan,
                                                (void*)&alpha, A_d,
                                                (void*)&beta , B_d,
                                                (void*)&gamma, C_d,
                                                               D_d, stream));
        auto time = timer.seconds();
        minTimeCUTENSOR = (minTimeCUTENSOR < time)? minTimeCUTENSOR : time;
    }

    /*************************/

    double transferedBytes = sizeC;
    transferedBytes += ((float) alpha != 0.f) ? sizeA : 0;
    transferedBytes += ((float) beta != 0.f) ? sizeB : 0;
    transferedBytes += ((float) gamma != 0.f) ? sizeC : 0;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GB/s\n", transferedBytes/ minTimeCUTENSOR);
    printf("memcpy: %.2f GB/s\n", 2 * sizeC / minTimeMEMCPY / 1e9 );

    HANDLE_ERROR(hiptensorDestroy(handle));
    HANDLE_ERROR(hiptensorDestroyPlan(plan));
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descB));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
    HANDLE_CUDA_ERROR(hipStreamDestroy(stream));

    if (A) free(A);
    if (B) free(B);
    if (C) free(C);
    if (A_d) hipFree(A_d);
    if (B_d) hipFree(B_d);
    if (C_d) hipFree(C_d);
    if (D_d) hipFree(D_d);

    return 0;
}
