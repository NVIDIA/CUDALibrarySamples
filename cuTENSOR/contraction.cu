/*  
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); exit(-1); } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); exit(-1); } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;

    hiptensorDataType_t typeA = HIPTENSOR_R_32F;
    hiptensorDataType_t typeB = HIPTENSOR_R_32F;
    hiptensorDataType_t typeC = HIPTENSOR_R_32F;
    const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;

    /**********************
     * Computing: C_{m,u,n,v} = alpha * A_{m,h,k,n} B_{u,k,v,h} + beta * C_{m,u,n,v}
     **********************/

    std::vector<int> modeC{'m','u','n','v'};
    std::vector<int> modeA{'m','h','k','n'};
    std::vector<int> modeB{'u','k','v','h'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    std::unordered_map<int, int64_t> extent;
    extent['m'] = 96;
    extent['n'] = 96;
    extent['u'] = 96;
    extent['v'] = 64;
    extent['h'] = 64;
    extent['k'] = 64;

    double gflops = (2.0 * extent['m'] * extent['n'] * extent['u'] * extent['v'] * extent['k'] * extent['h']) /1e9;

    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);

    /**********************
     * Allocating data
     **********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeB = sizeof(floatTypeB) * elementsB;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC)/1024./1024./1024);

    void *A_d, *B_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &B_d, sizeB));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));

    floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
    floatTypeB *B = (floatTypeB*) malloc(sizeof(floatTypeB) * elementsB);
    floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

    if (A == NULL || B == NULL || C == NULL)
    {
        printf("Error: Host allocation of A or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsB; i++)
        B[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = (((float) rand())/RAND_MAX - 0.5)*100;

    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

    const uint32_t kAlignment = 128; // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(B_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);

    /*************************
     * cuTENSOR
     *************************/ 

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL,/*stride*/
                 typeA, kAlignment));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descB,
                 nmodeB,
                 extentB.data(),
                 NULL,/*stride*/
                 typeB, kAlignment));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL,/*stride*/
                 typeC, kAlignment));

    /*******************************
     * Create Contraction Descriptor
     *******************************/

    hiptensorOperationDescriptor_t desc;
    HANDLE_ERROR(hiptensorCreateContraction(handle, 
                 &desc,
                 descA, modeA.data(), /* unary operator A*/HIPTENSOR_OP_IDENTITY,
                 descB, modeB.data(), /* unary operator B*/HIPTENSOR_OP_IDENTITY,
                 descC, modeC.data(), /* unary operator C*/HIPTENSOR_OP_IDENTITY,
                 descC, modeC.data(),
                 descCompute));

    /*****************************
     * Optional (but recommended): ensure that the scalar type is correct.
     *****************************/

    hiptensorDataType_t scalarType;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle,
        desc,
        HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
        (void*)&scalarType,
        sizeof(scalarType)));

    assert(scalarType == HIPTENSOR_R_32F);
    typedef float floatTypeCompute;
    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta  = (floatTypeCompute)0.f;

    /**************************
    * Set the algorithm to use
    ***************************/

    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    hiptensorPlanPreference_t planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(
                               handle,
                               &planPref,
                               algo,
                               HIPTENSOR_JIT_MODE_NONE));

    /**********************
     * Query workspace estimate
     **********************/

    uint64_t workspaceSizeEstimate = 0;
    const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
    HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle,
                                          desc,
                                          planPref,
                                          workspacePref,
                                          &workspaceSizeEstimate));

    /**************************
     * Create Contraction Plan
     **************************/

    hiptensorPlan_t plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                 &plan,
                 desc,
                 planPref,
                 workspaceSizeEstimate));

    /**************************
     * Optional: Query information about the created plan
     **************************/

    // query actually used workspace
    uint64_t actualWorkspaceSize = 0;
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
        plan,
        HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
        &actualWorkspaceSize,
        sizeof(actualWorkspaceSize)));

    // At this point the user knows exactly how much memory is need by the operation and
    // only the smaller actual workspace needs to be allocated
    assert(actualWorkspaceSize <= workspaceSizeEstimate);

    void *work = nullptr;
    if (actualWorkspaceSize > 0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
    }

    /**********************
     * Run
     **********************/

    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    double minTimeCUTENSOR = 1e100;
    for (int i=0; i < 3; ++i)
    {
        hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        // Set up timing
        GPUTimer timer;
        timer.start();

        HANDLE_ERROR(hiptensorContract(handle,
                               plan,
                               (void*) &alpha, A_d, B_d,
                               (void*) &beta,  C_d, C_d, 
                               work, actualWorkspaceSize, stream));

        // Synchronize and measure timing
        auto time = timer.seconds();
        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }

    /*************************/

    double transferedBytes = sizeC + sizeA + sizeB;
    transferedBytes += ((float) beta != 0.f) ? sizeC : 0;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GFLOPs/s %.2f GB/s\n", gflops / minTimeCUTENSOR, transferedBytes/ minTimeCUTENSOR);

    HANDLE_ERROR(hiptensorDestroy(handle));
    HANDLE_ERROR(hiptensorDestroyPlan(plan));
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descB));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
    HANDLE_CUDA_ERROR(hipStreamDestroy(stream));

    if (A) free(A);
    if (B) free(B);
    if (C) free(C);
    if (A_d) hipFree(A_d);
    if (B_d) hipFree(B_d);
    if (C_d) hipFree(C_d);
    if (work) hipFree(work);

    return 0;
}
