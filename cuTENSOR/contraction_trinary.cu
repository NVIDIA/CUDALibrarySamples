/*
 * SPDX-FileCopyrightText: Copyright (c) 2019-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); exit(-1); } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); exit(-1); } \
};

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef float floatTypeA;
    typedef float floatTypeB;
    typedef float floatTypeC;
    typedef float floatTypeD;

    hiptensorDataType_t typeA = HIPTENSOR_R_32F;
    hiptensorDataType_t typeB = HIPTENSOR_R_32F;
    hiptensorDataType_t typeC = HIPTENSOR_R_32F;
    hiptensorDataType_t typeD = HIPTENSOR_R_32F;
    const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;

    /**********************
     * Computing: D_{m,n,b,r,a} = alpha * A_{m,k,a,j,b,i} B_{k,n,i} C_{r,j} + beta * D_{m,n,b,r,a}
     **********************/

    std::vector<int> modeD{'m','n','b','r','a'};
    std::vector<int> modeA{'m','k','a','j','b','i'};
    std::vector<int> modeB{'k','n','i'};
    std::vector<int> modeC{'r','j'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();
    int nmodeD = modeD.size();

    std::unordered_map<int, int64_t> extent;
    extent['m'] = 256;
    extent['a'] = 32;
    extent['b'] = 32;
    extent['n'] = 64;
    extent['r'] = 64;
    extent['k'] = 8;
    extent['i'] = 8;
    extent['j'] = 64;

    double gflopsFirstContraction  = 2.0 * extent['m'] * extent['a'] * extent['b'] * extent['j'] * extent['n'] * extent['k'] * extent['i'] / 1e9;
    double gflopsSecondContraction = 2.0 * extent['m'] * extent['a'] * extent['b'] * extent['n'] * extent['r'] * extent['j'] / 1e9;
    double gflops = gflopsFirstContraction + gflopsSecondContraction;

    std::vector<int64_t> extentD;
    for (auto mode : modeD)
        extentD.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);
    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);

    /**********************
     * Allocating data
     **********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];
    size_t elementsD = 1;
    for (auto mode : modeD)
        elementsD *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeB = sizeof(floatTypeB) * elementsB;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    size_t sizeD = sizeof(floatTypeD) * elementsD;
    printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC + sizeD)/1024./1024./1024);

    void *A_d, *B_d, *C_d, *D_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &B_d, sizeB));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &D_d, sizeD));

    floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
    floatTypeB *B = (floatTypeB*) malloc(sizeof(floatTypeB) * elementsB);
    floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);
    floatTypeD *D = (floatTypeC*) malloc(sizeof(floatTypeD) * elementsD);

    if (A == NULL || B == NULL || C == NULL || D == NULL)
    {
        printf("Error: Host allocation of A or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsB; i++)
        B[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsC; i++)
        C[i] = (((float) rand())/RAND_MAX - 0.5)*100;
    for (int64_t i = 0; i < elementsD; i++)
        D[i] = (((float) rand())/RAND_MAX - 0.5)*100;

    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(D_d, D, sizeD, hipMemcpyHostToDevice));

    const uint32_t kAlignment = 128; // Alignment of the global-memory device pointers (bytes)
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(B_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);
    assert(uintptr_t(D_d) % kAlignment == 0);

    /*************************
     * cuTENSOR
     *************************/ 

    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL,/*stride*/
                 typeA, kAlignment));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descB,
                 nmodeB,
                 extentB.data(),
                 NULL,/*stride*/
                 typeB, kAlignment));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL,/*stride*/
                 typeC, kAlignment));

    hiptensorTensorDescriptor_t descD;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle,
                 &descD,
                 nmodeD,
                 extentD.data(),
                 NULL,/*stride*/
                 typeD, kAlignment));

    /*******************************
     * Create Contraction Descriptor
     *******************************/

    hiptensorOperationDescriptor_t desc;
    HANDLE_ERROR(cutensorCreateContractionTrinary(handle, 
                 &desc,
                 descA, modeA.data(), /* unary operator A*/HIPTENSOR_OP_IDENTITY,
                 descB, modeB.data(), /* unary operator B*/HIPTENSOR_OP_IDENTITY,
                 descC, modeC.data(), /* unary operator C*/HIPTENSOR_OP_IDENTITY,
                 descD, modeD.data(), /* unary operator D*/HIPTENSOR_OP_IDENTITY,
                 descD, modeD.data(),
                 descCompute));

    /*****************************
     * Optional (but recommended): ensure that the scalar type is correct.
     *****************************/

    hiptensorDataType_t scalarType;
    HANDLE_ERROR(hiptensorOperationDescriptorGetAttribute(handle,
        desc,
        HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE,
        (void*)&scalarType,
        sizeof(scalarType)));

    assert(scalarType == HIPTENSOR_R_32F);
    typedef float floatTypeCompute;
    floatTypeCompute alpha = (floatTypeCompute)1.1f;
    floatTypeCompute beta  = (floatTypeCompute)0.f;

    /**************************
    * Set the algorithm to use
    ***************************/

    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    hiptensorPlanPreference_t planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(
                               handle,
                               &planPref,
                               algo,
                               HIPTENSOR_JIT_MODE_NONE));

    /**********************
     * Query workspace estimate
     **********************/

    uint64_t workspaceSizeEstimate = 0;
    const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
    HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle,
                                          desc,
                                          planPref,
                                          workspacePref,
                                          &workspaceSizeEstimate));

    /**************************
     * Create Contraction Plan
     **************************/

    hiptensorPlan_t plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle,
                 &plan,
                 desc,
                 planPref,
                 workspaceSizeEstimate));

    /**************************
     * Optional: Query information about the created plan
     **************************/

    // query actually used workspace
    uint64_t actualWorkspaceSize = 0;
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle,
        plan,
        HIPTENSOR_PLAN_REQUIRED_WORKSPACE,
        &actualWorkspaceSize,
        sizeof(actualWorkspaceSize)));

    // At this point the user knows exactly how much memory is need by the operation and
    // only the smaller actual workspace needs to be allocated
    assert(actualWorkspaceSize <= workspaceSizeEstimate);

    void *work = nullptr;
    if (actualWorkspaceSize > 0)
    {
        HANDLE_CUDA_ERROR(hipMalloc(&work, actualWorkspaceSize));
        assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
    }

    /**********************
     * Run
     **********************/

    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    double minTimeCUTENSOR = 1e100;
    for (int i=0; i < 3; ++i)
    {
        hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        // Set up timing
        GPUTimer timer;
        timer.start();

        HANDLE_ERROR(cutensorContractTrinary(handle,
                               plan,
                               (void*) &alpha, A_d, B_d, C_d,
                               (void*) &beta,  D_d, D_d,
                               work, actualWorkspaceSize, stream));

        // Synchronize and measure timing
        auto time = timer.seconds();
        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }

    /*************************/

    double transferedBytes = sizeA + sizeB + sizeC + sizeD;
    transferedBytes += ((float) beta != 0.f) ? sizeD : 0;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GFLOPs/s %.2f GB/s\n", gflops / minTimeCUTENSOR, transferedBytes/ minTimeCUTENSOR);

    HANDLE_ERROR(hiptensorDestroy(handle));
    HANDLE_ERROR(hiptensorDestroyPlan(plan));
    HANDLE_ERROR(hiptensorDestroyOperationDescriptor(desc));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descA));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descB));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descC));
    HANDLE_ERROR(hiptensorDestroyTensorDescriptor(descD));
    HANDLE_CUDA_ERROR(hipStreamDestroy(stream));

    if (A) free(A);
    if (B) free(B);
    if (C) free(C);
    if (D) free(D);
    if (A_d) hipFree(A_d);
    if (B_d) hipFree(B_d);
    if (C_d) hipFree(C_d);
    if (D_d) hipFree(D_d);
    if (work) hipFree(work);

    return 0;
}