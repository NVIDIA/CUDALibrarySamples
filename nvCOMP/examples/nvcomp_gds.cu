#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2024 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
*/

// Simple example of how to use GDS with nvcomp.
// GDS (GPU Direct Storage) allows to read and write from/to NVMe drives
// directly from the GPU, bypassing the CPU.
//
// For best performance, the I/O buffer should be registered but it is not
// mandatory. Registration can be expensive but done only once, and allows the
// I/O to be performed directly from the registered buffer. Otherwise, GDS will
// use it's own intermediate buffer, at the expense of extra memory copies.
// Similarly, I/Os with a base address or size which is not aligned on 4KB will
// go through GDS's internal buffer and will be less efficient.
//
// For more details on GDS, included the supported GPUs, please see the
// documentation. https://docs.nvidia.com/gpudirect-storage/
//
// To compile this GDS example, GDS must be installed, and the following
// option must be passed when configuring cmake:
// cmake -DBUILD_GDS_EXAMPLE=on <...>

#include <fcntl.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <cufile.h>
#include <nvtx3/nvToolsExt.h>

#include "nvcomp.hpp"
#include "nvcomp/lz4.hpp"

using namespace nvcomp;

#define CUDA_CHECK(func)                                                       \
  do {                                                                         \
    hipError_t rt = (func);                                                   \
    if (rt != hipSuccess) {                                                   \
      std::cout << "API call failure \"" #func "\" with " << rt << " at "      \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      throw;                                                                   \
    }                                                                          \
  } while (0);

// Kernel to initialize the input data with sequential bytes
__global__ void initialize(uint8_t* data, size_t n)
{
  size_t i = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    data[i] = i & 0xff;
}

// Kernel to compare 2 buffers. Invalid flag must be set to zero before.
__global__ void
compare(const uint8_t* ref, const uint8_t* val, int* invalid, size_t n)
{
  size_t i = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  while (i < n) {
    if (ref[i] != val[i])
      *invalid = 1;
    i += stride;
  }
}

void usage(const char* str)
{
  printf("Argument: %s <filename>\n", str);
  exit(-1);
}

int main(int argc, char** argv)
{
  if (argc != 2)
    usage(argv[0]);

  // Open the file. Note: GDS requires O_DIRECT.
  const char* filename = argv[1];
  int fd = open(filename, O_RDWR | O_TRUNC | O_CREAT | O_DIRECT, 0666);
  if (fd == -1) {
    printf("Error, cannot create the file: %s\n", filename);
    return -1;
  }
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));
  printf("Using device: %s\n", deviceProp.name);
  int smcount = deviceProp.multiProcessorCount;

  // Uncompressed data = 100 MB
  const size_t n = 100000000;

  // Device pointers for the data to be compressed / decompressed
  uint8_t *d_input, *d_output;
  hipStream_t stream;
  CUDA_CHECK(hipMalloc(&d_input, n));
  CUDA_CHECK(hipMalloc(&d_output, n));
  CUDA_CHECK(hipStreamCreate(&stream));

  // Initialize the input data (sequential bytes)
  initialize<<<(n - 1) / 512 + 1, 512, 0, stream>>>(d_input, n);

  // Using NVTX to highlight the different phases of the program in the Nsight
  // Systems profiler
  nvtxRangePushA("Compressor setup");

  // Create an LZ4 compressor, get the max output size, and temp storage size
  LZ4Manager compressor(1 << 16, nvcompBatchedLZ4Opts_t{NVCOMP_TYPE_CHAR}, stream);
  const CompressionConfig comp_config = compressor.configure_compression(n);
  size_t lcompbuf = comp_config.max_compressed_buffer_size;

  // The compressed output buffer is padded to the next multiple of 4KB
  // for best I/O performance. Unaligned I/Os go through an extra
  // memory copy (GDS's internal aligned registered buffer)
  lcompbuf = ((lcompbuf - 1) / 4096 + 1) * 4096;
  uint8_t* d_compressed;
  CUDA_CHECK(hipMalloc(&d_compressed, lcompbuf));

  nvtxRangePop();
  nvtxRangePushA("GDS setup");

  // Initialize the cufile driver
  CUfileError_t status = cuFileDriverOpen();
  if (status.err != CU_FILE_SUCCESS) {
    printf("Error: cuFileDriverOpen failed (%d)\n", status.err);
    return -1;
  }

  // Register the file with GDS
  CUfileDescr_t cf_descr;
  memset(&cf_descr, 0, sizeof(CUfileDescr_t));
  cf_descr.handle.fd = fd;
  cf_descr.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
  CUfileHandle_t cf_handle;
  status = cuFileHandleRegister(&cf_handle, &cf_descr);
  if (status.err != CU_FILE_SUCCESS) {
    printf("Error: cuFileHandleRegister failed (%d)\n", status.err);
    return -1;
  }

  // Buffer registration is not mandatory but recommended for best performance.
  // I/Os from/to unregistered buffers will go through GDS's internal registered
  // buffer (extra copy).
  // Let's ignore if it fails (e.g. not enough BAR memory on this GPU)
  bool registered = true;
  status = cuFileBufRegister(d_compressed, lcompbuf, 0);
  if (status.err != CU_FILE_SUCCESS) {
    printf("Warning: GDS buffer registration failed\n");
    registered = false;
  }

  nvtxRangePop();
  nvtxRangePushA("Compression");

  // The compressed size must be device-accessible, using pinned memory.

  // Compress the data (asynchronous)
  compressor.compress(d_input, d_compressed, comp_config);
  const size_t compressed_size = compressor.get_compressed_output_size(d_compressed);

  // Align the compressed size to the next multiple of 4KB.
  size_t aligned_compressed_size = ((compressed_size - 1) / 4096 + 1) * 4096;
  printf(
      "Data compressed from %lu Bytes to %lu Bytes, aligned to %lu Bytes\n",
      n,
      compressed_size,
      aligned_compressed_size);

  nvtxRangePop();
  nvtxRangePushA("GDS Write");

  // Write the data (padded to next 4KB), directly from the device, with GDS
  ssize_t nb;
  if ((nb = cuFileWrite(cf_handle, d_compressed, aligned_compressed_size, 0, 0)) != aligned_compressed_size) {
    printf("Error, write returned %ld instead of %lu \n", nb, aligned_compressed_size);
    return -1;
  } else
    printf("Wrote %ld bytes to file %s using GDS\n", nb, filename);

  nvtxRangePop();
  nvtxRangePushA("Cleaning up compressor");

  // Erase the content of the compressed buffer
  CUDA_CHECK(hipMemsetAsync(d_compressed, 0xff, compressed_size, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  nvtxRangePop();
  nvtxRangePushA("GDS Read");

  // Read the compressed data from the GDS file into the device buffer
  if ((nb = cuFileRead(cf_handle, d_compressed, aligned_compressed_size, 0, 0)) != aligned_compressed_size) {
    nvtxRangePop();
    printf("Error, GDS read returned %ld instead of %lu \n", nb, aligned_compressed_size);
    return -1;
  } else {
    nvtxRangePop();
    printf("Read %ld bytes from file %s using GDS\n", nb, filename);
  }

  nvtxRangePushA("Decompressor setup");

  CUDA_CHECK(hipStreamSynchronize(stream));
  // Decompressor, configured with the compressed data
  const DecompressionConfig decomp_config
      = compressor.configure_decompression(comp_config);
  size_t ldecomp = decomp_config.decomp_data_size;
  if (ldecomp != n) {
    printf("Error: Uncompressed size does not match the original size\n");
    return -1;
  }

  // Device-accessible flag to compare the data
  int* dh_invalid;
  CUDA_CHECK(hipHostMalloc(&dh_invalid, sizeof(int)));
  *dh_invalid = 0;

  nvtxRangePop();
  nvtxRangePushA("Decompression and comparison");
  printf("Decompressing\n");

  // Decompress the data (asynchronous)
  compressor.decompress(d_output, d_compressed, decomp_config);
  // Compare the uncompressed data with the original, in the same stream
  compare<<<2 * smcount, 1024, 0, stream>>>(d_input, d_output, dh_invalid, n);

  // Sync the stream before we check the result
  CUDA_CHECK(hipStreamSynchronize(stream));
  if (*dh_invalid)
    printf("FAILED: Uncompressed data does not match the original\n");
  else
    printf("PASSED: Uncompressed data is identical to the input\n");

  nvtxRangePop();
  nvtxRangePushA("Final cleanup");

  // Cleanup
  if (registered) {
    status = cuFileBufDeregister(d_compressed);
    if (status.err != CU_FILE_SUCCESS) {
      printf("Error: cuFileBufDeregister failed(%d)\n", status.err);
      return -1;
    }
  }
  close(fd);
  status = cuFileDriverClose();
  if (status.err != CU_FILE_SUCCESS) {
    printf("Error: cuFileDriverClose failed(%d)\n", status.err);
    return -1;
  }

  // Deallocate internal memory structures of LZ4Manager
  compressor.deallocate_gpu_mem();

  CUDA_CHECK(hipHostFree(dh_invalid));
  CUDA_CHECK(hipFree(d_input));
  CUDA_CHECK(hipFree(d_output));
  CUDA_CHECK(hipFree(d_compressed));
  CUDA_CHECK(hipStreamDestroy(stream));

  printf("All done, exiting...\n");
  nvtxRangePop();

  return 0;
}
