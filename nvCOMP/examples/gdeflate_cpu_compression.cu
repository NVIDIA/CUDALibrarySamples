#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020-2024 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
*/

#include "BatchData.h"

#include <nvcomp/native/gdeflate_cpu.h>
#include <nvcomp/gdeflate.h>

// Benchmark performance from the binary data file fname
static void run_example(const std::vector<std::vector<char>>& data)
{
  size_t total_bytes = 0;
  for (const std::vector<char>& part : data) {
    total_bytes += part.size();
  }

  std::cout << "----------" << std::endl;
  std::cout << "files: " << data.size() << std::endl;
  std::cout << "uncompressed (B): " << total_bytes << std::endl;

  const size_t chunk_size = 1 << 16;

  // build up input batch on CPU
  BatchDataCPU input_data_cpu(data, chunk_size);
  std::cout << "chunks: " << input_data_cpu.size() << std::endl;

  // compression

  // Get max output size per chunk
  nvcompStatus_t status;
  size_t max_out_bytes;
  status = nvcompBatchedGdeflateCompressGetMaxOutputChunkSize(
      chunk_size, nvcompBatchedGdeflateDefaultOpts, &max_out_bytes);
  if( status != nvcompSuccess){
    throw std::runtime_error("ERROR: nvcompBatchedGdeflateCompressGetMaxOutputChunkSize() not successful");
  }

  // Allocate and prepare output/compressed batch
  BatchDataCPU compress_data_cpu(max_out_bytes, input_data_cpu.size());

  // Compress on the CPU using gdeflate CPU batched API
  gdeflate::compressCPU(
      input_data_cpu.ptrs(),
      input_data_cpu.sizes(),
      chunk_size,
      input_data_cpu.size(),
      compress_data_cpu.ptrs(),
      compress_data_cpu.sizes());

  // compute compression ratio
  size_t* compressed_sizes_host = compress_data_cpu.sizes();
  size_t comp_bytes = 0;
  for (size_t i = 0; i < compress_data_cpu.size(); ++i)
    comp_bytes += compressed_sizes_host[i];

  std::cout << "comp_size: " << comp_bytes
            << ", compressed ratio: " << std::fixed << std::setprecision(2)
            << (double)total_bytes / comp_bytes << std::endl;

  // Copy compressed data to GPU
  BatchData compress_data(compress_data_cpu, true);

  // Allocate and build up decompression batch on GPU
  BatchData decomp_data(input_data_cpu, false);

  // Create CUDA stream
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // CUDA events to measure decompression time
  hipEvent_t start, end;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));

  // gdeflate GPU decompression
  size_t decomp_temp_bytes;
  status = nvcompBatchedGdeflateDecompressGetTempSize(
      compress_data.size(), chunk_size, &decomp_temp_bytes);
  if( status != nvcompSuccess){
    throw std::runtime_error("ERROR: nvcompBatchedGdeflateDecompressGetTempSize() not successful");
  }

  void* d_decomp_temp;
  CUDA_CHECK(hipMalloc(&d_decomp_temp, decomp_temp_bytes));

  size_t* d_decomp_sizes;
  CUDA_CHECK(hipMalloc(&d_decomp_sizes, decomp_data.size() * sizeof(size_t)));

  nvcompStatus_t* d_statuses;
  CUDA_CHECK(hipMalloc(&d_statuses, decomp_data.size() * sizeof(nvcompStatus_t)));

  CUDA_CHECK(hipStreamSynchronize(stream));

  // Run decompression
  status = nvcompBatchedGdeflateDecompressAsync(
      compress_data.ptrs(),
      compress_data.sizes(),
      decomp_data.sizes(),
      d_decomp_sizes,
      compress_data.size(),
      d_decomp_temp,
      decomp_temp_bytes,
      decomp_data.ptrs(),
      d_statuses,
      stream);
  if( status != nvcompSuccess){
    throw std::runtime_error("ERROR: nvcompBatchedGdeflateDecompressAsync() not successful");
  }

  // Validate decompressed data against input
  if (!(input_data_cpu == decomp_data))
    throw std::runtime_error("Failed to validate decompressed data");
  else
    std::cout << "decompression validated :)" << std::endl;

  // Re-run decompression to get throughput
  CUDA_CHECK(hipEventRecord(start, stream));
  status = nvcompBatchedGdeflateDecompressAsync(
      compress_data.ptrs(),
      compress_data.sizes(),
      decomp_data.sizes(),
      d_decomp_sizes,
      compress_data.size(),
      d_decomp_temp,
      decomp_temp_bytes,
      decomp_data.ptrs(),
      d_statuses,
      stream);
  CUDA_CHECK(hipEventRecord(end, stream));
  if( status != nvcompSuccess){
    throw std::runtime_error("ERROR: nvcompBatchedGdeflateDecompressAsync() not successful");
  }

  CUDA_CHECK(hipStreamSynchronize(stream));

  float ms;
  CUDA_CHECK(hipEventElapsedTime(&ms, start, end));

  double decompression_throughput = ((double)total_bytes / ms) * 1e-6;
  std::cout << "decompression throughput (GB/s): " << decompression_throughput
            << std::endl;

  CUDA_CHECK(hipFree(d_decomp_temp));
  CUDA_CHECK(hipFree(d_decomp_sizes));
  CUDA_CHECK(hipFree(d_statuses));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(end));
  CUDA_CHECK(hipStreamDestroy(stream));
}

std::vector<char> readFile(const std::string& filename)
{
  std::vector<char> buffer(4096);
  std::vector<char> host_data;

  std::ifstream fin(filename, std::ifstream::binary);
  fin.exceptions(std::ifstream::failbit | std::ifstream::badbit);

  size_t num;
  do {
    num = fin.readsome(buffer.data(), buffer.size());
    host_data.insert(host_data.end(), buffer.begin(), buffer.begin() + num);
  } while (num > 0);

  return host_data;
}

std::vector<std::vector<char>>
multi_file(const std::vector<std::string>& filenames)
{
  std::vector<std::vector<char>> split_data;

  for (auto const& filename : filenames) {
    split_data.emplace_back(readFile(filename));
  }

  return split_data;
}

int main(int argc, char* argv[])
{
  std::vector<std::string> file_names(argc - 1);

  if (argc == 1) {
    std::cerr << "Must specify at least one file." << std::endl;
    return 1;
  }

  // if `-f` is specified, assume single file mode
  if (strcmp(argv[1], "-f") == 0) {
    if (argc == 2) {
      std::cerr << "Missing file name following '-f'" << std::endl;
      return 1;
    } else if (argc > 3) {
      std::cerr << "Unknown extra arguments with '-f'." << std::endl;
      return 1;
    }

    file_names = {argv[2]};
  } else {
    // multi-file mode
    for (int i = 1; i < argc; ++i) {
      file_names[i - 1] = argv[i];
    }
  }

  auto data = multi_file(file_names);

  run_example(data);

  return 0;
}
