/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
*/

#include "snappy.h"
#include "nvcomp/snappy.h"
#include "BatchData.h"


static void run_example(const std::vector<std::vector<char>>& data,
                        size_t warmup_iteration_count, size_t total_iteration_count)
{
  assert(!data.empty());
  if(warmup_iteration_count >= total_iteration_count) {
    throw std::runtime_error("ERROR: the total iteration count must be greater than the warmup iteration count");
  }

  size_t total_bytes =
    std::accumulate(data.begin(), data.end(), size_t(0), [](const size_t& a, const std::vector<char>& part) {
        return a + part.size();
  });

  std::cout << "----------" << std::endl;
  std::cout << "files: " << data.size() << std::endl;
  std::cout << "uncompressed (B): " << total_bytes << std::endl;

  const size_t chunk_size = 1 << 16;
  static_assert(chunk_size <= snappy::kBlockSize, "Chunk size must be less than the constant specified in the Snappy library");

  // Build up input batch on CPU
  BatchDataCPU input_data_cpu(data, chunk_size);
  const size_t chunk_count = input_data_cpu.size();
  std::cout << "chunks: " << chunk_count << std::endl;

  // compression

  // Allocate and prepare output/compressed batch
  BatchDataCPU compressed_data_cpu(
      snappy::MaxCompressedLength(chunk_size), chunk_count);

  // loop over chunks on the CPU, compressing each one
  for (size_t i = 0; i < chunk_count; ++i) {
    snappy::RawCompress(static_cast<const char*>(input_data_cpu.ptrs()[i]),
                   input_data_cpu.sizes()[i],
                   static_cast<char*>(compressed_data_cpu.ptrs()[i]),
                   &compressed_data_cpu.sizes()[i]);
    if (compressed_data_cpu.sizes()[i] == 0) {
      throw std::runtime_error(
          "Snappy CPU failed to compress chunk " + std::to_string(i) + ".");
    }
  }

  // compute compression ratio
  size_t comp_bytes = std::accumulate(compressed_data_cpu.sizes(), compressed_data_cpu.sizes() + chunk_count, size_t(0));

  std::cout << "comp_size: " << comp_bytes
            << ", compressed ratio: " << std::fixed << std::setprecision(2)
            << (double)total_bytes / comp_bytes << std::endl;

  // Copy compressed data to GPU
  BatchData compressed_data(compressed_data_cpu, true, nvcompBatchedSnappyDecompressRequiredAlignments.input);

  // Allocate and build up decompression batch on GPU
  BatchData decomp_data(input_data_cpu, false, nvcompBatchedSnappyDecompressRequiredAlignments.output);

  // Create CUDA stream
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // CUDA events to measure decompression time
  hipEvent_t start, end;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));

  // Snappy GPU decompression
  size_t decomp_temp_bytes;
  nvcompStatus_t status = nvcompBatchedSnappyDecompressGetTempSize(
      chunk_count, chunk_size, &decomp_temp_bytes);
  if (status != nvcompSuccess) {
    throw std::runtime_error("nvcompBatchedSnappyDecompressGetTempSize() failed.");
  }

  void* d_decomp_temp;
  CUDA_CHECK(hipMalloc(&d_decomp_temp, decomp_temp_bytes));

  size_t* d_decomp_sizes;
  CUDA_CHECK(hipMalloc(&d_decomp_sizes, chunk_count * sizeof(size_t)));

  nvcompStatus_t* d_status_ptrs;
  CUDA_CHECK(hipMalloc(&d_status_ptrs, chunk_count * sizeof(nvcompStatus_t)));

  CUDA_CHECK(hipStreamSynchronize(stream));

  auto perform_decompression = [&]() {
    if (nvcompBatchedSnappyDecompressAsync(
          compressed_data.ptrs(),
          compressed_data.sizes(),
          decomp_data.sizes(),
          d_decomp_sizes,
          chunk_count,
          d_decomp_temp,
          decomp_temp_bytes,
          decomp_data.ptrs(),
          d_status_ptrs,
          stream) != nvcompSuccess) {
      throw std::runtime_error("ERROR: nvcompBatchedSnappyDecompressAsync() not successful");
    }
  };

  // Run warm-up decompression
  for (size_t iter = 0; iter < warmup_iteration_count; ++iter) {
    perform_decompression();
  }

  // Re-run decompression to get throughput
  CUDA_CHECK(hipEventRecord(start, stream));
  for (size_t iter = warmup_iteration_count; iter < total_iteration_count; ++iter) {
    perform_decompression();
  }
  CUDA_CHECK(hipEventRecord(end, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  // Validate decompressed data against input
  if (!(input_data_cpu == decomp_data)) {
    throw std::runtime_error("Failed to validate decompressed data");
  } else {
    std::cout << "decompression validated :)" << std::endl;
  }

  float ms;
  CUDA_CHECK(hipEventElapsedTime(&ms, start, end));
  ms /= total_iteration_count - warmup_iteration_count;

  double decompression_throughput = ((double)total_bytes / ms) * 1e-6;
  std::cout << "decompression throughput (GB/s): " << decompression_throughput
            << std::endl;

  CUDA_CHECK(hipFree(d_decomp_temp));
  CUDA_CHECK(hipFree(d_decomp_sizes));
  CUDA_CHECK(hipFree(d_status_ptrs));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(end));
  CUDA_CHECK(hipStreamDestroy(stream));
}

int main(int argc, char* argv[])
{
  std::vector<std::string> file_names;

  size_t warmup_iteration_count = 2;
  size_t total_iteration_count = 5;

  do {
    if (argc < 3) {
      break;
    }

    int i = 1;
    while (i < argc) {
      const char* current_argv = argv[i++];
      if (strcmp(current_argv, "-f") == 0) {
        // parse until next `-` argument
        while (i < argc && argv[i][0] != '-') {
          file_names.emplace_back(argv[i++]);
        }
      } else {
        std::cerr << "Unknown argument: " << current_argv << std::endl;
        return 1;
      }
    }
  } while (0);

  if (file_names.empty()) {
    std::cerr << "Must specify at least one file via '-f <file>'." << std::endl;
    return 1;
  }

  auto data = multi_file(file_names);

  run_example(data, warmup_iteration_count, total_iteration_count);

  return 0;
}
