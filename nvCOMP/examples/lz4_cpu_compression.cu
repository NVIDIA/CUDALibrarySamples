/*
 * Copyright (c) 2020-2025 NVIDIA CORPORATION AND AFFILIATES. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *  * Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 *  * Neither the name of the NVIDIA CORPORATION nor the names of its contributors
 *    may be used to endorse or promote products derived from this software without
 *    specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY DIRECT,
 * INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "BatchData.h"

#include "lz4.h"
#include "lz4hc.h"
#include "nvcomp/lz4.h"

// Benchmark performance from the binary data file fname
static void run_example(const std::vector<std::vector<char>>& data,
                        size_t warmup_iteration_count, size_t total_iteration_count)
{
  assert(!data.empty());
  if(warmup_iteration_count >= total_iteration_count) {
    throw std::runtime_error("ERROR: the total iteration count must be greater than the warmup iteration count");
  }

  size_t total_bytes = 0;
  for (const std::vector<char>& part : data) {
    total_bytes += part.size();
  }

  std::cout << "----------" << std::endl;
  std::cout << "files: " << data.size() << std::endl;
  std::cout << "uncompressed (B): " << total_bytes << std::endl;

  const size_t chunk_size = 1 << 16;

  // Build up input batch on CPU
  BatchDataCPU input_data_cpu(data, chunk_size);
  const size_t chunk_count = input_data_cpu.size();
  std::cout << "chunks: " << chunk_count << std::endl;

  // compression

  // Allocate and prepare output/compressed batch
  BatchDataCPU compressed_data_cpu(
      LZ4_compressBound(chunk_size), chunk_count);

  // loop over chunks on the CPU, compressing each one
  for (size_t i = 0; i < chunk_count; ++i) {
    // could use LZ4_compress_default or LZ4_compress_fast instead
    const int size = LZ4_compress_HC(
        static_cast<const char*>(input_data_cpu.ptrs()[i]),
        static_cast<char*>(compressed_data_cpu.ptrs()[i]),
        static_cast<int>(input_data_cpu.sizes()[i]),
        static_cast<int>(compressed_data_cpu.sizes()[i]),
        12);
    if (size == 0) {
      throw std::runtime_error(
          "LZ4 CPU failed to compress chunk " + std::to_string(i) + ".");
    }

    // set the actual compressed size
    compressed_data_cpu.sizes()[i] = static_cast<size_t>(size);
  }

  // compute compression ratio
  size_t* compressed_sizes_host = compressed_data_cpu.sizes();
  size_t comp_bytes = 0;
  for (size_t i = 0; i < chunk_count; ++i)
    comp_bytes += compressed_sizes_host[i];

  std::cout << "comp_size: " << comp_bytes
            << ", compressed ratio: " << std::fixed << std::setprecision(2)
            << (double)total_bytes / comp_bytes << std::endl;

  // Decompression options
  nvcompBatchedLZ4DecompressOpts_t decompress_opts = nvcompBatchedLZ4DecompressDefaultOpts;

  // Query decompression alignment requirements
  nvcompAlignmentRequirements_t decompression_alignment_reqs;
  nvcompStatus_t status = nvcompBatchedLZ4DecompressGetRequiredAlignments(
    decompress_opts,
    &decompression_alignment_reqs);
  if (status != nvcompSuccess) {
    throw std::runtime_error("ERROR: nvcompBatchedLZ4DecompressGetRequiredAlignments() not successful");
  }

  // Copy compressed data to GPU
  BatchData compressed_data(compressed_data_cpu, true, decompression_alignment_reqs.input);

  // Allocate and build up decompression batch on GPU
  BatchData decomp_data(input_data_cpu, false, decompression_alignment_reqs.output);

  // Create CUDA stream
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // CUDA events to measure decompression time
  hipEvent_t start, end;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));

  // LZ4 GPU decompression
  size_t decomp_temp_bytes;
  status = nvcompBatchedLZ4DecompressGetTempSizeAsync(
      chunk_count,
      chunk_size,
      decompress_opts,
      &decomp_temp_bytes,
      chunk_count * chunk_size);
  if (status != nvcompSuccess) {
    throw std::runtime_error("nvcompBatchedLZ4DecompressGetTempSizeAsync() failed.");
  }

  void* d_decomp_temp;
  CUDA_CHECK(hipMalloc(&d_decomp_temp, decomp_temp_bytes));

  size_t* d_decomp_sizes;
  CUDA_CHECK(hipMalloc(&d_decomp_sizes, chunk_count * sizeof(size_t)));

  nvcompStatus_t* d_status_ptrs;
  CUDA_CHECK(hipMalloc(&d_status_ptrs, chunk_count * sizeof(nvcompStatus_t)));

  CUDA_CHECK(hipStreamSynchronize(stream));

  auto perform_decompression = [&]() {
    if (nvcompBatchedLZ4DecompressAsync(
          compressed_data.ptrs(),
          compressed_data.sizes(),
          decomp_data.sizes(),
          d_decomp_sizes,
          chunk_count,
          d_decomp_temp,
          decomp_temp_bytes,
          decomp_data.ptrs(),
          decompress_opts,
          d_status_ptrs,
          stream) != nvcompSuccess) {
      throw std::runtime_error("ERROR: nvcompBatchedLZ4DecompressAsync() not successful");
    }
  };

  // Run warm-up decompression
  for (size_t iter = 0; iter < warmup_iteration_count; ++iter) {
    perform_decompression();
  }

  // Re-run decompression to get throughput
  CUDA_CHECK(hipEventRecord(start, stream));
  for (size_t iter = warmup_iteration_count; iter < total_iteration_count; ++iter) {
    perform_decompression();
  }
  CUDA_CHECK(hipEventRecord(end, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  // Validate decompressed data against input
  if (!(input_data_cpu == decomp_data)) {
    throw std::runtime_error("Failed to validate decompressed data");
  } else {
    std::cout << "decompression validated :)" << std::endl;
  }

  float ms;
  CUDA_CHECK(hipEventElapsedTime(&ms, start, end));
  ms /= total_iteration_count - warmup_iteration_count;

  double decompression_throughput = ((double)total_bytes / ms) * 1e-6;
  std::cout << "decompression throughput (GB/s): " << decompression_throughput
            << std::endl;

  CUDA_CHECK(hipFree(d_decomp_temp));
  CUDA_CHECK(hipFree(d_decomp_sizes));
  CUDA_CHECK(hipFree(d_status_ptrs));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(end));
  CUDA_CHECK(hipStreamDestroy(stream));
}

int main(int argc, char* argv[])
{
  std::vector<std::string> file_names;

  size_t warmup_iteration_count = 2;
  size_t total_iteration_count = 5;

  do {
    if (argc < 3) {
      break;
    }

    int i = 1;
    while (i < argc) {
      const char* current_argv = argv[i++];
      if (strcmp(current_argv, "-f") == 0) {
        // parse until next `-` argument
        while (i < argc && argv[i][0] != '-') {
          file_names.emplace_back(argv[i++]);
        }
      } else {
        std::cerr << "Unknown argument: " << current_argv << std::endl;
        return 1;
      }
    }
  } while (0);

  if (file_names.empty()) {
    std::cerr << "Must specify at least one file via '-f <file>'." << std::endl;
    return 1;
  }

  auto data = multi_file(file_names);

  run_example(data, warmup_iteration_count, total_iteration_count);

  return 0;
}
