/*
 * SPDX-FileCopyrightText: Copyright (c) 2020-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include "BatchData.h"

#include "lz4.h"
#include "lz4hc.h"
#include "nvcomp/lz4.h"

BatchDataCPU GetBatchDataCPU(const BatchData& batch_data, bool copy_data)
{
  BatchDataCPU batch_data_cpu(
      batch_data.ptrs(),
      batch_data.sizes(),
      batch_data.data(),
      batch_data.size(),
      copy_data);
  return batch_data_cpu;
}

// Benchmark performance from the binary data file fname
static void run_example(const std::vector<std::vector<char>>& data,
                        size_t warmup_iteration_count, size_t total_iteration_count)
{
  assert(!data.empty());
  if(warmup_iteration_count >= total_iteration_count) {
    throw std::runtime_error("ERROR: the total iteration count must be greater than the warmup iteration count");
  }

  size_t total_bytes = 0;
  for (const std::vector<char>& part : data) {
    total_bytes += part.size();
  }

  std::cout << "----------" << std::endl;
  std::cout << "files: " << data.size() << std::endl;
  std::cout << "uncompressed (B): " << total_bytes << std::endl;

  constexpr size_t chunk_size = 1 << 16;
  static_assert(chunk_size <= nvcompLZ4CompressionMaxAllowedChunkSize, "Chunk size must be less than the constant specified in the nvCOMP library");

  auto nvcompBatchedLZ4Opts = nvcompBatchedLZ4CompressDefaultOpts;

  // Query compression alignment requirements
  nvcompAlignmentRequirements_t compression_alignment_reqs;
  nvcompStatus_t status = nvcompBatchedLZ4CompressGetRequiredAlignments(
    nvcompBatchedLZ4Opts,
    &compression_alignment_reqs);
  if (status != nvcompSuccess) {
    throw std::runtime_error("ERROR: nvcompBatchedLZ4CompressGetRequiredAlignments() not successful");
  }

  // Build up GPU data
  BatchData input_data(data, chunk_size, compression_alignment_reqs.input);
  const size_t chunk_count = input_data.size();

  // Compress on the GPU using batched API
  size_t comp_temp_bytes;
  status = nvcompBatchedLZ4CompressGetTempSizeAsync(
      chunk_count,
      chunk_size,
      nvcompBatchedLZ4Opts,
      &comp_temp_bytes,
      chunk_count * chunk_size);
  if (status != nvcompSuccess) {
    throw std::runtime_error("ERROR: nvcompBatchedLZ4CompressGetTempSizeAsync() not successful");
  }

  void* d_comp_temp;
  CUDA_CHECK(hipMalloc(&d_comp_temp, comp_temp_bytes));

  size_t max_out_bytes;
  status = nvcompBatchedLZ4CompressGetMaxOutputChunkSize(
      chunk_size, nvcompBatchedLZ4Opts, &max_out_bytes);
  if (status != nvcompSuccess) {
    throw std::runtime_error("ERROR: nvcompBatchedLZ4CompressGetMaxOutputChunkSize() not successful");
  }

  BatchData compressed_data(max_out_bytes, chunk_count, compression_alignment_reqs.output);

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  hipEvent_t start, end;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));

  auto perform_compression = [&]() {
    if (nvcompBatchedLZ4CompressAsync(
          input_data.ptrs(),
          input_data.sizes(),
          chunk_size,
          chunk_count,
          d_comp_temp,
          comp_temp_bytes,
          compressed_data.ptrs(),
          compressed_data.sizes(),
          nvcompBatchedLZ4Opts,
          nullptr,
          stream) != nvcompSuccess) {
      throw std::runtime_error("nvcompBatchedLZ4CompressAsync() failed.");
    }
  };

  // Run warm-up compression
  for (size_t iter = 0; iter < warmup_iteration_count; ++iter) {
    perform_compression();
  }

  // Re-run compression to get throughput
  CUDA_CHECK(hipEventRecord(start, stream));
  for (size_t iter = warmup_iteration_count; iter < total_iteration_count; ++iter) {
    perform_compression();
  }
  CUDA_CHECK(hipEventRecord(end, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  float ms;
  CUDA_CHECK(hipEventElapsedTime(&ms, start, end));
  ms /= total_iteration_count - warmup_iteration_count;

  // compute compression ratio
  std::vector<size_t> compressed_sizes_host(chunk_count);
  CUDA_CHECK(hipMemcpy(
      compressed_sizes_host.data(),
      compressed_data.sizes(),
      chunk_count * sizeof(*compressed_data.sizes()),
      hipMemcpyDeviceToHost));

  size_t comp_bytes = 0;
  for (const size_t s : compressed_sizes_host) {
    comp_bytes += s;
  }

  std::cout << "comp_size: " << comp_bytes
            << ", compressed ratio: " << std::fixed << std::setprecision(2)
            << (double)total_bytes / comp_bytes << std::endl;
  std::cout << "compression throughput (GB/s): "
            << (double)total_bytes / (1.0e6 * ms) << std::endl;

  // Allocate and prepare output/compressed batch
  BatchDataCPU compressed_data_cpu = GetBatchDataCPU(compressed_data, true);
  BatchDataCPU decompressed_data_cpu = GetBatchDataCPU(input_data, false);

  // loop over chunks on the CPU, decompressing each one
  for (size_t i = 0; i < chunk_count; ++i) {
    const int size = LZ4_decompress_safe(
        static_cast<const char*>(compressed_data_cpu.ptrs()[i]),
        static_cast<char*>(decompressed_data_cpu.ptrs()[i]),
        static_cast<int>(compressed_data_cpu.sizes()[i]),
        static_cast<int>(decompressed_data_cpu.sizes()[i]));
    if (size == 0) {
      throw std::runtime_error(
          "LZ4 CPU failed to decompress chunk " + std::to_string(i) + ".");
    }
  }
  // Validate decompressed data against input
  if (!(decompressed_data_cpu == input_data)) {
    throw std::runtime_error("Failed to validate CPU decompressed data");
  } else {
    std::cout << "CPU decompression validated :)" << std::endl;
  }

  CUDA_CHECK(hipFree(d_comp_temp));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(end));
  CUDA_CHECK(hipStreamDestroy(stream));
}

int main(int argc, char* argv[])
{
  std::vector<std::string> file_names;

  size_t warmup_iteration_count = 2;
  size_t total_iteration_count = 5;

  do {
    if (argc < 3) {
      break;
    }

    int i = 1;
    while (i < argc) {
      const char* current_argv = argv[i++];
      if (strcmp(current_argv, "-f") == 0) {
        // parse until next `-` argument
        while (i < argc && argv[i][0] != '-') {
          file_names.emplace_back(argv[i++]);
        }
      } else {
        std::cerr << "Unknown argument: " << current_argv << std::endl;
        return 1;
      }
    }
  } while (0);

  if (file_names.empty()) {
    std::cerr << "Must specify at least one file via '-f <file>'." << std::endl;
    return 1;
  }

  auto data = multi_file(file_names);

  run_example(data, warmup_iteration_count, total_iteration_count);

  return 0;
}