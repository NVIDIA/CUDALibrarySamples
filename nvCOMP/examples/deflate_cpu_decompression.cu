/*
 * SPDX-FileCopyrightText: Copyright (c) 2020-2024 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
*/

#include "BatchData.h"
#include "zlib.h"
#include "libdeflate.h"
#include "nvcomp/deflate.h"

BatchDataCPU GetBatchDataCPU(const BatchData& batch_data, bool copy_data)
{
  BatchDataCPU compress_data_cpu(
      batch_data.ptrs(),
      batch_data.sizes(),
      batch_data.data(),
      batch_data.size(),
      copy_data);
  return compress_data_cpu;
}

// Benchmark performance from the binary data file fname
static void run_example(const std::vector<std::vector<char>>& data, int algo)
{
  size_t total_bytes = 0;
  for (const std::vector<char>& part : data) {
    total_bytes += part.size();
  }

  std::cout << "----------" << std::endl;
  std::cout << "files: " << data.size() << std::endl;
  std::cout << "uncompressed (B): " << total_bytes << std::endl;

  const size_t chunk_size = 1 << 16;

  // build up metadata
  BatchData input_data(data, chunk_size);
  static nvcompBatchedDeflateOpts_t nvcompBatchedDeflateOpts = nvcompBatchedDeflateDefaultOpts;
  // Compress on the GPU using batched API
  size_t comp_temp_bytes;
  nvcompStatus_t status = nvcompBatchedDeflateCompressGetTempSize(
      input_data.size(),
      chunk_size,
      nvcompBatchedDeflateOpts,
      &comp_temp_bytes);
  if( status != nvcompSuccess){
    throw std::runtime_error("ERROR: nvcompBatchedDeflateCompressGetTempSize() not successful");
  }

  void* d_comp_temp;
  CUDA_CHECK(hipMalloc(&d_comp_temp, comp_temp_bytes));

  size_t max_out_bytes;
  status = nvcompBatchedDeflateCompressGetMaxOutputChunkSize(
      chunk_size, nvcompBatchedDeflateOpts, &max_out_bytes);
  if( status != nvcompSuccess){
    throw std::runtime_error("ERROR: nvcompBatchedDeflateCompressGetMaxOutputChunkSize() not successful");
  }

  BatchData compress_data(max_out_bytes, input_data.size());

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  hipEvent_t start, end;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));
  CUDA_CHECK(hipEventRecord(start, stream));

  status = nvcompBatchedDeflateCompressAsync(
      input_data.ptrs(),
      input_data.sizes(),
      chunk_size,
      input_data.size(),
      d_comp_temp,
      comp_temp_bytes,
      compress_data.ptrs(),
      compress_data.sizes(),
      nvcompBatchedDeflateOpts,
      stream);
  if (status != nvcompSuccess) {
    throw std::runtime_error("nvcompBatchedDeflateCompressAsync() failed.");
  }
  
  CUDA_CHECK(hipEventRecord(end, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  // free compression memory
  CUDA_CHECK(hipFree(d_comp_temp));

  float ms;
  CUDA_CHECK(hipEventElapsedTime(&ms, start, end));

  // compute compression ratio
  std::vector<size_t> compressed_sizes_host(compress_data.size());
  CUDA_CHECK(hipMemcpy(
      compressed_sizes_host.data(),
      compress_data.sizes(),
      compress_data.size() * sizeof(*compress_data.sizes()),
      hipMemcpyDeviceToHost));

  size_t comp_bytes = 0;
  for (const size_t s : compressed_sizes_host) {
    comp_bytes += s;
  }

  std::cout << "comp_size: " << comp_bytes
            << ", compressed ratio: " << std::fixed << std::setprecision(2)
            << (double)total_bytes / comp_bytes << std::endl;
  std::cout << "compression throughput (GB/s): "
            << (double)total_bytes / (1.0e6 * ms) << std::endl;

  // Allocate and prepare output/compressed batch
  BatchDataCPU compress_data_cpu = GetBatchDataCPU(compress_data, true);
  BatchDataCPU decompress_data_cpu = GetBatchDataCPU(input_data, false);

  // loop over chunks on the CPU, decompressing each one
  for (size_t i = 0; i < input_data.size(); ++i) {
    if(algo==0){
        struct libdeflate_decompressor  *decompressor;
        decompressor = libdeflate_alloc_decompressor();
        enum libdeflate_result res = libdeflate_deflate_decompress(decompressor, compress_data_cpu.ptrs()[i], compress_data_cpu.sizes()[i], 
                                                   decompress_data_cpu.ptrs()[i], decompress_data_cpu.sizes()[i], NULL);
    
       if (res != LIBDEFLATE_SUCCESS) {
       throw std::runtime_error(
           "libdeflate CPU failed to decompress chunk " + std::to_string(i) + ".");
       }
    }else if (algo==1){
        z_stream zs1;
        zs1.zalloc = NULL;
        zs1.zfree = NULL;
        zs1.msg = NULL;
        zs1.next_in = (Bytef*)compress_data_cpu.ptrs()[i];
        zs1.avail_in = static_cast<uInt>(compress_data_cpu.sizes()[i]);
        zs1.next_out = (Bytef*)decompress_data_cpu.ptrs()[i];
        zs1.avail_out = static_cast<uInt>(decompress_data_cpu.sizes()[i]);

        int ret = inflateInit2(&zs1, -15);
        if (ret != Z_OK) {
           throw std::runtime_error("inflateInit2 error " + std::to_string(ret));
        }
        if ((ret = inflate(&zs1, Z_FINISH)) != Z_STREAM_END) {
           throw std::runtime_error("zlib::inflate operation fail " + std::to_string(ret));;
            if ((ret = inflateEnd(&zs1)) != Z_OK) {
               throw std::runtime_error("Call to inflateEnd failed: " + std::to_string(ret));
            }
        }
        if ((ret = inflateEnd(&zs1)) != Z_OK) {
           throw std::runtime_error("Call to inflateEnd failed: " + std::to_string(ret));
        }
    }
  }
  // Validate decompressed data against input
  if (!(decompress_data_cpu == input_data))
    throw std::runtime_error("Failed to validate CPU decompressed data");
  else
    std::cout << "CPU decompression validated :)" << std::endl;

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(end));
  CUDA_CHECK(hipStreamDestroy(stream));
}

std::vector<char> readFile(const std::string& filename)
{
  std::vector<char> buffer(4096);
  std::vector<char> host_data;

  std::ifstream fin(filename, std::ifstream::binary);
  fin.exceptions(std::ifstream::failbit | std::ifstream::badbit);

  size_t num;
  do {
    num = fin.readsome(buffer.data(), buffer.size());
    host_data.insert(host_data.end(), buffer.begin(), buffer.begin() + num);
  } while (num > 0);

  return host_data;
}

std::vector<std::vector<char>>
multi_file(const std::vector<std::string>& filenames)
{
  std::vector<std::vector<char>> split_data;

  for (auto const& filename : filenames) {
    split_data.emplace_back(readFile(filename));
  }

  return split_data;
}

int main(int argc, char* argv[])
{
 std::vector<std::string> file_names;

 if (argc < 5) {
   std::cerr << "Must choose the algorithm (-a <0>) and specify at least one file (-f <inputfile>)." << std::endl;
   return 1;
 }
 int algo = 0;
 int i = 1; bool choose_algo = false; bool input_file = false;
 do{
  if(strcmp(argv[i], "-a") !=0 && strcmp(argv[i], "-f") != 0){
    std::cerr << "The config only could be -a (choose algorithm: 0 libdeflate, 1 zlib_inflate) or -f (add input files)." << std::endl;
    return 1;
  }else if(strcmp(argv[i], "-a") ==0){
    choose_algo = true;
    i++;
    if( (i < argc) && (atoi(argv[i]) == 0 ||  atoi(argv[i]) == 1)){
      algo = atoi(argv[2]);
      i++;
    }else{
      std::cerr<<"`-a` could only be 0, 1. (0 libdeflate, 1 zlib_inflate)"<<std::endl;
      return 1;
    }
  }else if (strcmp(argv[i], "-f") == 0){
    i++;
    if(i >= argc){
      std::cerr<<"Specify at least one input file." <<std::endl;
      return 1;
    }
    do{
      input_file = true;
      file_names.push_back(argv[i]);
      i++;
    }while(i < argc && strcmp(argv[i], "-a") !=0);
  }
 }while(i < argc);

 if(!choose_algo){
  std::cerr<<"Have to choose an algorithm use `-a`. `-a` could be 0, 1. (0 libdeflate, 1 zlib_inflate)"<<std::endl;
  return 1;
 }

 if(!input_file){
   std::cerr<<"Specify at least one input file by using `-f`"<<std::endl;
   return 1;
 }

  auto data = multi_file(file_names);
  run_example(data, algo);

  return 0;
}

