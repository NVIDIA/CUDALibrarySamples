/*
 * SPDX-FileCopyrightText: Copyright (c) 2020-2025 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
*/

#include "zlib.h"
#include "libdeflate.h"
#include "nvcomp/deflate.h"
#include "BatchData.h"


BatchDataCPU GetBatchDataCPU(const BatchData& batch_data, bool copy_data)
{
  BatchDataCPU batch_data_cpu(
      batch_data.ptrs(),
      batch_data.sizes(),
      batch_data.data(),
      batch_data.size(),
      copy_data);
  return batch_data_cpu;
}

static void run_example(const std::vector<std::vector<char>>& data,
                        int algo,
                        size_t warmup_iteration_count, size_t total_iteration_count)
{
  assert(!data.empty());
  assert(algo >= 0 && algo <= 1);
    if(warmup_iteration_count >= total_iteration_count) {
    throw std::runtime_error("ERROR: the total iteration count must be greater than the warmup iteration count");
  }

  size_t total_bytes = 0;
  for (const std::vector<char>& part : data) {
    total_bytes += part.size();
  }

  std::cout << "----------" << std::endl;
  std::cout << "files: " << data.size() << std::endl;
  std::cout << "uncompressed (B): " << total_bytes << std::endl;

  const size_t chunk_size = 1 << 16;
  static_assert(chunk_size <= nvcompDeflateCompressionMaxAllowedChunkSize, "Chunk size must be less than the constant specified in the nvCOMP library");

  auto nvcompBatchedDeflateOpts = nvcompBatchedDeflateDefaultOpts;

  // Query compression alignment requirements
  nvcompAlignmentRequirements_t compression_alignment_reqs;
  nvcompStatus_t status = nvcompBatchedDeflateCompressGetRequiredAlignments(
      nvcompBatchedDeflateOpts,
      &compression_alignment_reqs);
  if (status != nvcompSuccess) {
    throw std::runtime_error("ERROR: nvcompBatchedDeflateCompressGetRequiredAlignments() not successful");
  }

  // Build up GPU data
  BatchData input_data(data, chunk_size, compression_alignment_reqs.input);
  const size_t chunk_count = input_data.size();

  // Compress on the GPU using batched API
  size_t comp_temp_bytes;
  status = nvcompBatchedDeflateCompressGetTempSize(
      chunk_count,
      chunk_size,
      nvcompBatchedDeflateOpts,
      &comp_temp_bytes);
  if (status != nvcompSuccess) {
    throw std::runtime_error("ERROR: nvcompBatchedDeflateCompressGetTempSize() not successful");
  }

  void* d_comp_temp;
  CUDA_CHECK(hipMalloc(&d_comp_temp, comp_temp_bytes));

  size_t max_out_bytes;
  status = nvcompBatchedDeflateCompressGetMaxOutputChunkSize(
      chunk_size, nvcompBatchedDeflateOpts, &max_out_bytes);
  if (status != nvcompSuccess) {
    throw std::runtime_error("ERROR: nvcompBatchedDeflateCompressGetMaxOutputChunkSize() not successful");
  }

  BatchData compressed_data(max_out_bytes, chunk_count, compression_alignment_reqs.output);

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  hipEvent_t start, end;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&end));

  auto perform_compression = [&]() {
    if (nvcompBatchedDeflateCompressAsync(
          input_data.ptrs(),
          input_data.sizes(),
          chunk_size,
          chunk_count,
          d_comp_temp,
          comp_temp_bytes,
          compressed_data.ptrs(),
          compressed_data.sizes(),
          nvcompBatchedDeflateOpts,
          stream) != nvcompSuccess) {
      throw std::runtime_error("nvcompBatchedDeflateCompressAsync() failed.");
    }
  };

  // Warm-up compression iterations
  for (size_t iter = 0; iter < warmup_iteration_count; ++iter) {
    perform_compression();
  }

  CUDA_CHECK(hipEventRecord(start, stream));
  for (size_t iter = warmup_iteration_count; iter < total_iteration_count; ++iter) {
    perform_compression();
  }
  CUDA_CHECK(hipEventRecord(end, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  float ms;
  CUDA_CHECK(hipEventElapsedTime(&ms, start, end));
  ms /= total_iteration_count - warmup_iteration_count;

  // compute compression ratio
  std::vector<size_t> compressed_sizes_host(chunk_count);
  CUDA_CHECK(hipMemcpy(
      compressed_sizes_host.data(),
      compressed_data.sizes(),
      chunk_count * sizeof(*compressed_data.sizes()),
      hipMemcpyDeviceToHost));

  size_t comp_bytes = 0;
  for (const size_t s : compressed_sizes_host) {
    comp_bytes += s;
  }

  std::cout << "comp_size: " << comp_bytes
            << ", compressed ratio: " << std::fixed << std::setprecision(2)
            << (double)total_bytes / comp_bytes << std::endl;
  std::cout << "compression throughput (GB/s): "
            << (double)total_bytes / (1.0e6 * ms) << std::endl;

  // Allocate and prepare output/compressed batch
  BatchDataCPU compressed_data_cpu = GetBatchDataCPU(compressed_data, true);
  BatchDataCPU decompressed_data_cpu = GetBatchDataCPU(input_data, false);

  // loop over chunks on the CPU, decompressing each one
  for (size_t i = 0; i < chunk_count; ++i) {
    if (algo == 0) {
        struct libdeflate_decompressor  *decompressor;
        decompressor = libdeflate_alloc_decompressor();
        enum libdeflate_result res = libdeflate_deflate_decompress(decompressor, compressed_data_cpu.ptrs()[i], compressed_data_cpu.sizes()[i], 
                                                   decompressed_data_cpu.ptrs()[i], decompressed_data_cpu.sizes()[i], NULL);

       if (res != LIBDEFLATE_SUCCESS) {
       throw std::runtime_error(
           "libdeflate CPU failed to decompress chunk " + std::to_string(i) + ".");
       }
    } else if (algo == 1) {
        z_stream zs1;
        zs1.zalloc = NULL;
        zs1.zfree = NULL;
        zs1.msg = NULL;
        zs1.next_in = (Bytef*)compressed_data_cpu.ptrs()[i];
        zs1.avail_in = static_cast<uInt>(compressed_data_cpu.sizes()[i]);
        zs1.next_out = (Bytef*)decompressed_data_cpu.ptrs()[i];
        zs1.avail_out = static_cast<uInt>(decompressed_data_cpu.sizes()[i]);
        // -15 to disable zlib header/footer (raw deflate)
        int ret = inflateInit2(&zs1, -15);
        if (ret != Z_OK) {
           throw std::runtime_error("inflateInit2 error " + std::to_string(ret));
        }
        if ((ret = inflate(&zs1, Z_FINISH)) != Z_STREAM_END) {
           throw std::runtime_error("zlib::inflate operation fail " + std::to_string(ret));;
            if ((ret = inflateEnd(&zs1)) != Z_OK) {
               throw std::runtime_error("Call to inflateEnd failed: " + std::to_string(ret));
            }
        }
        if ((ret = inflateEnd(&zs1)) != Z_OK) {
           throw std::runtime_error("Call to inflateEnd failed: " + std::to_string(ret));
        }
    }
  }
  // Validate decompressed data against input
  if (!(decompressed_data_cpu == input_data)) {
    throw std::runtime_error("Failed to validate CPU decompressed data");
  } else {
    std::cout << "CPU decompression validated :)" << std::endl;
  }

  CUDA_CHECK(hipFree(d_comp_temp));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(end));
  CUDA_CHECK(hipStreamDestroy(stream));
}

int main(int argc, char* argv[])
{
  std::vector<std::string> file_names;

  int algo = -1;
  size_t warmup_iteration_count = 2;
  size_t total_iteration_count = 5;

  do {
    if (argc < 5) {
      break;
    }

    int i = 1;
    while (i < argc) {
      const char* current_argv = argv[i++];
      if (strcmp(current_argv, "-a") == 0) {
        if(i >= argc) {
          std::cerr << "Missing value for argument '-a <algorithm>'" << std::endl;
          return 1;
        }
        algo = atoi(argv[i++]);
      } else if (strcmp(current_argv, "-f") == 0) {
          // parse until next `-` argument
          while (i < argc && argv[i][0] != '-') {
            file_names.emplace_back(argv[i++]);
          }
      } else {
        std::cerr << "Unknown argument: " << current_argv << std::endl;
        return 1;
      }
    }
  } while (0);

  if (argc < 5) {
    std::cerr << "Must choose an algorithm via '-a <algo>', and must specify at least one file via '-f <file>'." << std::endl;
    return 1;
  } else if (algo < 0 || algo > 1) {
    std::cerr << "Must choose an algorithm via '-a <algo>'. '<algo>' can be 0 or 1. (0 libdeflate, 1 zlib_inflate)" << std::endl;
    return 1;
  } else if (file_names.empty()) {
   std::cerr << "Must specify at least one file via '-f <file>'" << std::endl;
   return 1;
  }

  auto data = multi_file(file_names);

  run_example(data, algo, warmup_iteration_count, total_iteration_count);

  return 0;
}
