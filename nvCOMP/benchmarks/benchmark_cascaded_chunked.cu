/*
 * SPDX-FileCopyrightText: Copyright (c) 2020-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include "benchmark_template_chunked.cuh"
#include "nvcomp/cascaded.h"

#include <iostream>
#include <vector>

static nvcompBatchedCascadedCompressOpts_t nvcompBatchedCascadedCompressOpts =
  {4096, NVCOMP_TYPE_UINT, 2, 1, 1, {0}};
static nvcompBatchedCascadedDecompressOpts_t nvcompBatchedCascadedDecompressOpts =
  nvcompBatchedCascadedDecompressDefaultOpts;

static bool handleCommandLineArgument(
    const std::string& arg,
    const char* const* additionalArgs,
    size_t& additionalArgsUsed)
{
  if (arg == "--type" || arg == "-t") {
    const char* const typeArg = *additionalArgs;
    additionalArgsUsed = 1;
    bool valid;
    nvcompBatchedCascadedCompressOpts.type = string_to_data_type(typeArg, valid);
    return valid;
  }
  if (arg == "--num_rles" || arg == "-r") {
    int n = atoi(*additionalArgs);
    additionalArgsUsed = 1;
    if (n < 0) {
      std::cerr << "ERROR: num_rles can't be negative, but it is " << n
                << std::endl;
      return false;
    }
    nvcompBatchedCascadedCompressOpts.num_RLEs = n;
    return true;
  }
  if (arg == "--num_deltas" || arg == "-d") {
    int n = atoi(*additionalArgs);
    additionalArgsUsed = 1;
    if (n < 0) {
      std::cerr << "ERROR: num_deltas can't be negative, but it is " << n
                << std::endl;
      return false;
    }
    nvcompBatchedCascadedCompressOpts.num_deltas = n;
    return true;
  }
  if (arg == "--num_bps" || arg == "-b") {
    int n = atoi(*additionalArgs);
    additionalArgsUsed = 1;
    if (n < 0 || n > 1) {
      std::cerr << "ERROR: num_bps can only be 0 or 1, but it is " << n
                << std::endl;
      return false;
    }
    nvcompBatchedCascadedCompressOpts.use_bp = n;
    return true;
  }
  return false;
}

static bool isCascadedInputValid(const std::vector<std::vector<char>>& data,
                                 bool compressed_inputs)
{
  // Find the type size, to check that all chunk sizes are a multiple of it.
  size_t typeSize = 1;
  auto type = nvcompBatchedCascadedCompressOpts.type;
  switch (type) {
  case NVCOMP_TYPE_CHAR:
  case NVCOMP_TYPE_UCHAR:
    // Type size is 1 byte, so chunk sizes are always a multiple of it.
    return true;
  case NVCOMP_TYPE_SHORT:
  case NVCOMP_TYPE_USHORT:
    typeSize = sizeof(uint16_t);
    break;
  case NVCOMP_TYPE_INT:
  case NVCOMP_TYPE_UINT:
    typeSize = sizeof(uint32_t);
    break;
  case NVCOMP_TYPE_LONGLONG:
  case NVCOMP_TYPE_ULONGLONG:
    typeSize = sizeof(uint64_t);
    break;
  default:
    std::cerr << "ERROR: Cascaded data type must be 0-7 (CHAR, UCHAR, SHORT, "
                 "USHORT, INT, UINT, LONGLONG, or ULONGLONG), "
                 "but it is "
              << int(type) << std::endl;
    return false;
  }

  if(!compressed_inputs) {
    for (const auto& chunk : data) {
      if ((chunk.size() % typeSize) != 0) {
        std::cerr << "ERROR: Input data must have a length and chunk size that "
                    "are a multiple of "
                  << typeSize << ", the size of the specified data type."
                  << std::endl;
        return false;
      }
    }
  }
  return true;
}

void run_benchmark(
    const std::vector<std::vector<char>>& data,
    const bool warmup,
    const size_t count,
    const bool csv_output,
    const nvcompDecompressBackend_t decompress_backend,
    const bool tab_separator,
    const size_t duplicate_count,
    const size_t num_files,
    const bool compressed_inputs,
    const bool single_output_buffer,
    const std::string& output_compressed_filename,
    const std::string& output_decompressed_filename)
{
  run_benchmark_template(
      nvcompBatchedCascadedCompressGetTempSizeAsync,
      nvcompBatchedCascadedCompressGetMaxOutputChunkSize,
      nvcompBatchedCascadedCompressAsync,
      nvcompBatchedCascadedCompressGetRequiredAlignments,
      nvcompBatchedCascadedDecompressGetTempSizeAsync,
      nvcompBatchedCascadedDecompressGetTempSizeSync,
      nvcompBatchedCascadedDecompressAsync,
      nvcompBatchedCascadedGetDecompressSizeAsync,
      nvcompBatchedCascadedDecompressGetRequiredAlignments,
      isCascadedInputValid,
      nvcompBatchedCascadedCompressOpts,
      nvcompBatchedCascadedDecompressOpts,
      data,
      warmup,
      count,
      csv_output,
      decompress_backend,
      tab_separator,
      duplicate_count,
      num_files,
      compressed_inputs,
      single_output_buffer,
      output_compressed_filename,
      output_decompressed_filename);
}