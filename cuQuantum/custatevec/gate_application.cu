/*
 * Copyright (c) 2021, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
 *
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nIndexBits = 3;
    const int nSvSize    = (1 << nIndexBits);
    const int nTargets   = 1;
    const int nControls  = 2;
    const int adjoint    = 0;

    const int targets[]  = {2};
    const int controls[] = {0, 1};

    hipDoubleComplex h_sv[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2}, 
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.4, 0.5}};
    hipDoubleComplex h_sv_result[] = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.4, 0.5}, 
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.1, 0.2}};
    hipDoubleComplex matrix[] = {{0.0, 0.0}, {1.0, 0.0},
                                {1.0, 0.0}, {0.0, 0.0}};

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex)) );

    HANDLE_CUDA_ERROR( hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex), 
                       hipMemcpyHostToDevice) );

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // check the size of external workspace
    HANDLE_ERROR( custatevecApplyMatrix_bufferSize(
                  handle, HIP_C_64F, nIndexBits, matrix, HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW,
                  adjoint, nTargets, nControls, CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes) );

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

    // apply gate
    HANDLE_ERROR( custatevecApplyMatrix(
                  handle, d_sv, HIP_C_64F, nIndexBits, matrix, HIP_C_64F,
                  CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, controls, nControls, 
                  nullptr, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    for (int i = 0; i < nSvSize; i++) {
        if (!almost_equal(h_sv[i], h_sv_result[i])) {
            correct = false;
            break;
        }
    }

    HANDLE_CUDA_ERROR( hipFree(d_sv) );
    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );

    if (correct) {
        printf("example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}
