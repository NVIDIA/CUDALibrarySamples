/*
 * Copyright (c) 2021, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
 *
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

int main(void) {

    const int nIndexBits = 3;
    const int nSvSize    = (1 << nIndexBits);
    const int nBasisBits = 1;

    const int basisBits[] = {1};

    hipDoubleComplex expect;
    hipDoubleComplex expect_result = {4.1, 0.0};

    hipDoubleComplex h_sv[]        = {{ 0.0, 0.0}, { 0.0, 0.1}, { 0.1, 0.1}, { 0.1, 0.2}, 
                                     { 0.2, 0.2}, { 0.3, 0.3}, { 0.3, 0.4}, { 0.4, 0.5}};
    hipDoubleComplex matrix[] = {{1.0, 0.0}, {2.0, 1.0},
                                {2.0,-1.0}, {3.0, 0.0}};

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex)) );

    HANDLE_CUDA_ERROR( hipMemcpy(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex), 
                       hipMemcpyHostToDevice) );

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // check the size of external workspace
    HANDLE_ERROR( custatevecExpectation_bufferSize(
                  handle, HIP_C_64F, nIndexBits, matrix, HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW,
                  nBasisBits, CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes) );

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

    // compute expectation
    HANDLE_ERROR( custatevecExpectation(
                  handle, d_sv, HIP_C_64F, nIndexBits, &expect, HIP_C_64F, nullptr, matrix, 
                  HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW, basisBits, nBasisBits, 
                  CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    HANDLE_CUDA_ERROR( hipMemcpy(h_sv, d_sv, nSvSize * sizeof(hipDoubleComplex),
                       hipMemcpyDeviceToHost) );

    bool correct = true;
    if ( !almost_equal(expect, expect_result) ) {
        correct = false;
    }

    HANDLE_CUDA_ERROR( hipFree(d_sv) );
    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );

    if (correct) {
        printf("example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }

}
