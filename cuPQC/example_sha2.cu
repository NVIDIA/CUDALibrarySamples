#include "hip/hip_runtime.h"
#include <vector>
#include <iomanip>
#include <cuhash.hpp>
#include <stdio.h>

using namespace cupqc;

using SHA2_256_THREAD = decltype(SHA2_256() + Thread());

__global__ void hash_sha2_kernel(uint8_t* digest, const uint8_t* msg, size_t inbuf_len)
{
    SHA2_256_THREAD hash {};
    if (threadIdx.x == 0) {
        hash.reset();
        hash.update(msg, inbuf_len);
        hash.finalize();
        hash.digest(digest, SHA2_256_THREAD::digest_size);
    }
}


void hash_sha2(std::vector<uint8_t>& digest, std::vector<uint8_t>& msg)
{
    uint8_t* d_msg;
    uint8_t* d_digest;
    hipMalloc(reinterpret_cast<void**>(&d_msg), msg.size());
    hipMalloc(reinterpret_cast<void**>(&d_digest), digest.size());

    hipMemcpy(d_msg, msg.data(), msg.size(), hipMemcpyHostToDevice);

    hash_sha2_kernel<<<1, 32>>>(d_digest, d_msg, msg.size());

    hipMemcpy(digest.data(), d_digest, digest.size(), hipMemcpyDeviceToHost);

    hipFree(d_msg);
    hipFree(d_digest);
}

int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[]) {
    const char * msg_str = "The quick brown fox jumps over the lazy dog";
    std::vector<uint8_t> msg(reinterpret_cast<const uint8_t*>(msg_str), reinterpret_cast<const uint8_t*>(msg_str) + strlen(msg_str));
    std::vector<uint8_t> digest(SHA2_256::digest_size, 0);
    hash_sha2(digest, msg);
    printf("SHA2-256: ");
    for (uint8_t num : digest) {
        printf("%02x", num);
    }
    printf("\n");
}
