#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblaslt.h>
#include <hip/hip_runtime.h>

#include "sample_cublasLt_LtPlanarComplex.h"
#include "helpers.h"

/// Use hipblasLtMatmul to perform tensor-op Cgemm using planar complex memory layout and half-precision inputs.
///
/// For better performance data order transforms should be offline as much as possible.
///
/// transa, transb assumed N; alpha, beta are host pointers, tensor ops allowed, alpha assumed 1, beta assumed 0,
/// stream assumed 0
/// outputs can be either single or half precision, half precision is used in this example
void LtPlanarCgemm(hipblasLtHandle_t ltHandle,
                   hipblasOperation_t transa,
                   hipblasOperation_t transb,
                   int m,
                   int n,
                   int k,
                   const __half *A_real,
                   const __half *A_imag,
                   int lda,
                   const __half *B_real,
                   const __half *B_imag,
                   int ldb,
                   __half *C_real,
                   __half *C_imag,
                   int ldc) {
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipComplex alpha = {1, 0}, beta = {0, 0};

    // cublasLt expects offests in bytes
    int64_t AplaneOffset = (A_imag - A_real) * sizeof(A_real[0]);
    int64_t BplaneOffset = (B_imag - B_real) * sizeof(B_real[0]);
    int64_t CplaneOffset = (C_imag - C_real) * sizeof(C_real[0]);

    checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F, HIP_C_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for planar complex matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_C_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, CUBLASLT_MATRIX_LAYOUT_PLANE_OFFSET, &AplaneOffset, sizeof(AplaneOffset)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_C_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, CUBLASLT_MATRIX_LAYOUT_PLANE_OFFSET, &BplaneOffset, sizeof(BplaneOffset)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_C_16F, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, CUBLASLT_MATRIX_LAYOUT_PLANE_OFFSET, &CplaneOffset, sizeof(CplaneOffset)));

    // ---------------------------------------------------------------------------------------------
    // Launch computation

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     A_real,
                                     Adesc,
                                     B_real,
                                     Bdesc,
                                     &beta,
                                     C_real,
                                     Cdesc,
                                     C_real,
                                     Cdesc,
                                     NULL,
                                     NULL,
                                     0,
                                     0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
}
