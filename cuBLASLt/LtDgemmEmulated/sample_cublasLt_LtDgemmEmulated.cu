/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hipblaslt.h>

#include "sample_cublasLt_LtDgemmEmulated.h"
#include "helpers.h"

/// Sample wrapper executing double precision GEMM with a predefined algorithm using hipblasLtMatmul, nearly a drop-in
/// replacement for hipblasDgemm, with addition of the workspace to support fp64 emulation.
///
/// Pointer mode is always host. To change it, configure the appropriate matmul descriptor attribute.
/// Matmul is not using cuBLAS handle's configuration of math mode (where tensor ops are implicitly allowed).
/// To change this, configure the appropriate attribute in the preference handle.
///
/// NOTE: This sample may not work on all architectures or all problem sizes.
void LtDgemmEmulated(hipblasLtHandle_t ltHandle,
        hipblasOperation_t transa,
        hipblasOperation_t transb,
        int m,
        int n,
        int k,
        const double *alpha, /* host pointer */
        const double *A,
        int lda,
        const double *B,
        int ldb,
        const double *beta, /* host pointer */
        double *C,
        int ldc,
        void *workspace,
        size_t workspaceSize,
        hipblasEmulationStrategy_t emulationStrategy,
        cudaEmulationSpecialValuesSupport specialValuesSupport,
        cudaEmulationMantissaControl mantissaControl,
        int maxMantissaBitCount,
        int mantissaBitOffset,
        hipStream_t stream) {
    cublasLtEmulationDescOpaque_t emulationDescOpaque = {};
    hipblasLtMatmulDescOpaque_t operationDesc = {};
    hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
    hipblasLtMatmulPreferenceOpaque_t preference = {};

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // Create emulation descriptor; see cublasLtEmulationDescAttributes_t for details about defaults.
    cublasLtEmulationDesc_t emulationDesc = &emulationDescOpaque;
    checkCublasStatus(cublasLtEmulationDescInit(emulationDesc));
    checkCublasStatus(cublasLtEmulationDescSetAttribute(emulationDesc, CUBLASLT_EMULATION_DESC_STRATEGY, &emulationStrategy, sizeof(emulationStrategy)));
    checkCublasStatus(cublasLtEmulationDescSetAttribute(emulationDesc, CUBLASLT_EMULATION_DESC_SPECIAL_VALUES_SUPPORT,
                &specialValuesSupport, sizeof(specialValuesSupport)));
    checkCublasStatus(cublasLtEmulationDescSetAttribute(emulationDesc, CUBLASLT_EMULATION_DESC_FIXEDPOINT_MANTISSA_CONTROL,
                &mantissaControl, sizeof(mantissaControl)));
    checkCublasStatus(cublasLtEmulationDescSetAttribute(emulationDesc, CUBLASLT_EMULATION_DESC_FIXEDPOINT_MAX_MANTISSA_BIT_COUNT,
                &maxMantissaBitCount, sizeof(maxMantissaBitCount)));
    checkCublasStatus(cublasLtEmulationDescSetAttribute(emulationDesc, CUBLASLT_EMULATION_DESC_FIXEDPOINT_MANTISSA_BIT_OFFSET,
                &mantissaBitOffset, sizeof(mantissaBitOffset)));

    // Create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults.
    // Here we just need to set the transforms for A and B.
    checkCublasStatus(cublasLtMatmulDescInit(&operationDesc, CUBLAS_COMPUTE_64F_EMULATED_FIXEDPOINT, HIP_R_64F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(&operationDesc, CUBLASLT_MATMUL_DESC_EMULATION_DESCRIPTOR, &emulationDesc, sizeof(emulationDesc)));

    // Create matrix descriptors. 
    // We are good with the details here so no need to set any extra attributes.
    checkCublasStatus(cublasLtMatrixLayoutInit(&Adesc, HIP_R_64F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(cublasLtMatrixLayoutInit(&Bdesc, HIP_R_64F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(cublasLtMatrixLayoutInit(&Cdesc, HIP_R_64F, m, n, ldc));

    // Create preference handle.
    // Here we could use extra attributes to disable tensor ops, or we could make sure that the algorithm that is selected will work with badly aligned A, B, C.
    // For simplicity, we just assume A, B, C are always well aligned (e.g. directly come from hipMalloc).
    checkCublasStatus(cublasLtMatmulPreferenceInit(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(&preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // We just need the best available heuristic to try and run matmul. 
    // There is no guarantee this will work (e.g. if A is badly aligned). 
    // You can request more (e.g. 32) algorithms and try to run them one by one until something works.
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     &operationDesc,
                                     alpha,
                                     A,
                                     &Adesc,
                                     B,
                                     &Bdesc,
                                     beta,
                                     C,
                                     &Cdesc,
                                     C,
                                     &Cdesc,
                                     &heuristicResult.algo,
                                     workspace,
                                     workspaceSize,
                                     stream));
}