#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblaslt.h>

#include "helpers.h"
#include "sample_cublasLt_LtNvfp4Matmul.h"

/// Sample wrapper executing nvfp4 matmul with hipblasLtMatmul, with addition of per-tensor block scaling, and
/// the workspace to support split-K algorithms.
///
/// pointer mode is for alpha and beta is always host, to change it configure the appropriate matmul descriptor
/// attribute matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to
/// change this configure appropriate attribute in the preference handle
void LtNvfp4Matmul(hipblasLtHandle_t ltHandle,
                 int m,
                 int n,
                 int k,
                 const float *alpha, /* host pointer */
                 const __hip_fp8_e4m3_fnuz *a_scale, /* device pointer */
                 const typename StorageType<__hip_fp4_e2m1>::type *A,
                 int lda,
                 const __hip_fp8_e4m3_fnuz *b_scale, /* device pointer */
                 const typename StorageType<__hip_fp4_e2m1>::type *B,
                 int ldb,
                 const float *beta,
                 const __hip_fp8_e4m3_fnuz *c_scale, /* device pointer */
                 __hip_bfloat16 *C,
                 int ldc,
                 const float *d_scale, /* device pointer */
                 typename StorageType<__hip_fp4_e2m1>::type *D,
                 int ldd,
                 __hip_fp8_e4m3_fnuz *d_out_scale, /* device pointer */
                 void *workspace,
                 size_t workspaceSize,
                 hipblasLtMatmulMatrixScale_t AScaleMode,
                 hipblasLtMatmulMatrixScale_t BScaleMode,
                 hipblasLtMatmulMatrixScale_t CScaleMode,
                 hipblasLtMatmulMatrixScale_t DScaleMode,
                 hipblasLtMatmulMatrixScale_t DOutScaleMode) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL, Ddesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));


    // set block scaling mode
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_MODE, &AScaleMode, sizeof(AScaleMode)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_MODE, &BScaleMode, sizeof(BScaleMode)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_D_SCALE_MODE, &DScaleMode, sizeof(DScaleMode)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_D_OUT_SCALE_MODE, &DOutScaleMode, sizeof(DOutScaleMode)));

    // set scaling factors
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &a_scale, sizeof(a_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &b_scale, sizeof(b_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &d_scale, sizeof(d_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_D_OUT_SCALE_POINTER, &d_out_scale, sizeof(d_out_scale)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    // table of supported type combinations can be found in the documentation: https://docs.nvidia.com/cuda/cublas/index.html#cublasltmatmul
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_4F_E2M1, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_4F_E2M1, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16BF, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_4F_E2M1, m, n, ldd));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     &beta,
                                     C,
                                     Cdesc,
                                     D,
                                     Ddesc,
                                     &heuristicResult.algo,
                                     workspace,
                                     workspaceSize,
                                     0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Ddesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}
