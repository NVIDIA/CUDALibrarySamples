#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hipblaslt.h>

#include "helpers.h"
#include "sample_cublasLt_LtNvfp4Matmul.h"

/// Sample wrapper executing nvfp4 matmul with hipblasLtMatmul, with addition of per-tensor block scaling, and
/// the workspace to support split-K algorithms.
///
/// pointer mode is for alpha and beta is always host, to change it configure the appropriate matmul descriptor
/// attribute matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to
/// change this configure appropriate attribute in the preference handle
void LtNvfp4Matmul(hipblasLtHandle_t ltHandle,
                 hipblasOperation_t transa,
                 hipblasOperation_t transb,
                 int m,
                 int n,
                 int k,
                 const float *alpha, /* host pointer */
                 const __hip_fp8_e4m3_fnuz *a_scale, /* device pointer */
                 const typename StorageType<__hip_fp4_e2m1>::type *A,
                 int lda,
                 const __hip_fp8_e4m3_fnuz *b_scale, /* device pointer */
                 const typename StorageType<__hip_fp4_e2m1>::type *B,
                 int ldb,
                 const float *beta,
                 const __hip_fp8_e4m3_fnuz *c_scale, /* device pointer */
                 __hip_bfloat16 *C,
                 int ldc,
                 const float *d_scale, /* device pointer */
                 typename StorageType<__hip_fp4_e2m1>::type *D,
                 int ldd,
                 __hip_fp8_e4m3_fnuz *d_out_scale, /* device pointer */
                 void *workspace,
                 size_t workspaceSize,
                 hipblasLtMatmulMatrixScale_t AScaleMode,
                 hipblasLtMatmulMatrixScale_t BScaleMode,
                 hipblasLtMatmulMatrixScale_t CScaleMode,
                 hipblasLtMatmulMatrixScale_t DScaleMode,
                 hipblasLtMatmulMatrixScale_t DOutScaleMode) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL, Ddesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));


    // set block scaling mode
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_MODE, &AScaleMode, sizeof(AScaleMode)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_MODE, &BScaleMode, sizeof(BScaleMode)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_D_SCALE_MODE, &DScaleMode, sizeof(DScaleMode)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_D_OUT_SCALE_MODE, &DOutScaleMode, sizeof(DOutScaleMode)));

    // set scaling factors
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &a_scale, sizeof(a_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &b_scale, sizeof(b_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &d_scale, sizeof(d_scale)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_D_OUT_SCALE_POINTER, &d_out_scale, sizeof(d_out_scale)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    // table of supported type combinations can be found in the documentation: https://docs.nvidia.com/cuda/cublas/index.html#cublasltmatmul
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_4F_E2M1, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_4F_E2M1, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16BF, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_4F_E2M1, m, n, ldd));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     &beta,
                                     C,
                                     Cdesc,
                                     D,
                                     Ddesc,
                                     &heuristicResult.algo,
                                     workspace,
                                     workspaceSize,
                                     0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Ddesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}