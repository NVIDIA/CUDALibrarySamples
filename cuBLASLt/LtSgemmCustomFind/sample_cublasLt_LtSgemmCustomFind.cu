/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipblaslt.h>

#include "sample_cublasLt_LtSgemmCustomFind.h"
#include "helpers.h"

/* Structure to store information about different run trials */
typedef struct {
    hipblasLtMatmulAlgo_t algo;
    hipblasStatus_t status;
    float time;
    size_t workspaceSize;  // actual memory workspace needed
    hipblasMath_t mathMode;
    cublasLtReductionScheme_t reductionScheme;
    int customOption;
    float wavesCount;
} customMatmulPerf_t;

/* CAUTION : must match cublasLtMatmulTile_t */
const char * const matmulTileName[] = {
    "UNDEF",
    "8x8",
    "8x16",
    "16x8"   ,
    "8x32"   ,
    "16x16"  ,
    "32x8"   ,
    "8x64"   ,
    "16x32"  ,
    "32x16"  ,
    "64x8"   ,
    "32x32"  ,
    "32x64"  ,
    "64x32"  ,
    "32x128" ,
    "64x64"  ,
    "128x32" ,
    "64x128" ,
    "128x64" ,
    "64x256" ,
    "128x128",
    "256x64" ,
    "64x512" ,
    "128x256",
    "256x128",
    "512x64" ,
};

// Utility function to print customMatmulPerf_t structure
static void printPerfStructure(const customMatmulPerf_t &perf) {
    int algoId, tile, swizzle, customOption, numSplitsK, reductionScheme, stages;
    
    const hipblasLtMatmulAlgo_t *matmulAlgo = &perf.algo;
    cublasLtMatmulAlgoConfigGetAttribute( matmulAlgo,  CUBLASLT_ALGO_CONFIG_ID, &algoId, sizeof(algoId), NULL);
    cublasLtMatmulAlgoConfigGetAttribute( matmulAlgo,  CUBLASLT_ALGO_CONFIG_TILE_ID, &tile, sizeof(tile), NULL);
    cublasLtMatmulAlgoConfigGetAttribute( matmulAlgo,  CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &numSplitsK, sizeof(numSplitsK), NULL);
    cublasLtMatmulAlgoConfigGetAttribute( matmulAlgo,  CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &reductionScheme, sizeof(reductionScheme), NULL);
    cublasLtMatmulAlgoConfigGetAttribute( matmulAlgo,  CUBLASLT_ALGO_CONFIG_CTA_SWIZZLING, &swizzle, sizeof(swizzle), NULL);
    cublasLtMatmulAlgoConfigGetAttribute( matmulAlgo,  CUBLASLT_ALGO_CONFIG_CUSTOM_OPTION, &customOption, sizeof(customOption), NULL);
    cublasLtMatmulAlgoConfigGetAttribute( matmulAlgo,  CUBLASLT_ALGO_CONFIG_STAGES_ID, &stages, sizeof(stages), NULL);

    printf("algo={ Id=%d, tileIdx=%d (%s) splitK=%d reduc=%d swizzle=%d custom=%d stages=%d} status %d "
        "time %f workspace=%d mathMode=%d waves=%f\n",       
        algoId, tile, matmulTileName[tile],
        numSplitsK, reductionScheme,
        swizzle, customOption, stages,
        perf.status,
        perf.time,
        (int)perf.workspaceSize,
        (int)perf.mathMode,
        perf.wavesCount);
}

static inline bool time_compare(const customMatmulPerf_t &perf_a, const customMatmulPerf_t &perf_b) {
    return ((perf_a.status == HIPBLAS_STATUS_SUCCESS) && (perf_a.time < perf_b.time));
}

static hipblasStatus_t customMatmulRun(hipblasLtHandle_t ltHandle,  // to get the capabilities (required a GPU)
                 hipblasLtMatmulDesc_t operationDesc,
                 const void *alpha, /* host or device pointer */
                 const void *A,
                 hipblasLtMatrixLayout_t Adesc,
                 const void *B,
                 hipblasLtMatrixLayout_t Bdesc,
                 const void *beta, /* host or device pointer */
                 const void *C,
                 hipblasLtMatrixLayout_t Cdesc,
                 void *D,
                 hipblasLtMatrixLayout_t Ddesc,
                 const hipblasLtMatmulAlgo_t &algo,
                 int kernelRepeats,  
                 void *workSpace,
                 size_t workSpaceSizeInBytes,                 
                 customMatmulPerf_t &perfResults,                 
                 hipStream_t stream,
                 hipEvent_t &startEvent,
                 hipEvent_t &stopEvent) {
    hipblasLtMatmulHeuristicResult_t heurResult;
    /* Looping over the Algo */
    int repeats = kernelRepeats;    

    hipblasStatus_t algoStatus = cublasLtMatmulAlgoCheck( ltHandle,
                                                         operationDesc,
                                                         Adesc,
                                                         Bdesc,
                                                         Cdesc,
                                                         Ddesc,
                                                         &algo, 
                                                         &heurResult);     
                                                                                 
    if (algoStatus == HIPBLAS_STATUS_SUCCESS) {
        if (heurResult.workspaceSize <= workSpaceSizeInBytes) {
            hipError_t err, err1, err2, err3;
            err  = hipEventRecord(startEvent, stream);
            for (int loop = 0; loop < repeats; loop++) {
                hipblasStatus_t oneRunStatus = hipblasLtMatmul( ltHandle,
                                                              operationDesc,
                                                              alpha,
                                                              A, Adesc,
                                                              B, Bdesc,
                                                              beta,
                                                              C, Cdesc,
                                                              D, Ddesc,
                                                              &algo,
                                                              workSpace,
                                                              workSpaceSizeInBytes,
                                                              stream);
                if (oneRunStatus != HIPBLAS_STATUS_SUCCESS) {
                    algoStatus = oneRunStatus;
                    break;
                }
            }
            err1 = hipEventRecord(stopEvent, stream);
            err2 = hipEventSynchronize(stopEvent);
            float time;
            err3 = hipEventElapsedTime(&time, startEvent, stopEvent);
            if ((err != hipSuccess) || (err1 != hipSuccess) || (err2 != hipSuccess) || (err3 != hipSuccess)) {
                algoStatus = HIPBLAS_STATUS_INTERNAL_ERROR;
            }                                     
            // For the moment only add successful findings
            if (algoStatus == HIPBLAS_STATUS_SUCCESS) {
                perfResults.algo = algo;  
                perfResults.time = time;  
                perfResults.workspaceSize = heurResult.workspaceSize; 
                perfResults.wavesCount = heurResult.wavesCount;                                                                       
            }
        }
        else {
            algoStatus = HIPBLAS_STATUS_NOT_SUPPORTED; //Not enough workspace
        }        
    }
    
    return algoStatus;
}

/// Sample wrapper running through multiple algo and config attributes combination for single precision gemm using cublasLt low-level API
void LtSgemmCustomFind(hipblasLtHandle_t ltHandle,
                      hipblasOperation_t transa,
                      hipblasOperation_t transb,
                      int m,
                      int n,
                      int k,
                      const float *alpha, /* host pointer */
                      const float *A,
                      int lda,
                      const float *B,
                      int ldb,
                      const float *beta, /* host pointer */
                      float *C,
                      int ldc,
                      void *workSpace,
                      size_t workSpaceSize) {
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;
    hipEvent_t startEvent = NULL, stopEvent = NULL;
    hipStream_t stream = NULL;
    // SplitK value that we are going to try when SplitK is supported for a given algo
    const int splitKSequenceA[] = {2, 3, 4, 5, 6, 8, 12, 16, 32};
     // Let try a fixed number of combinations
    #define ALGO_COMBINATIONS 100
    int AlgoCombinations = ALGO_COMBINATIONS;
    int AlgoCount = 0;
    int kernelRepeats = 10; //number of time the CUDA kernels will be run back to back
    customMatmulPerf_t perfResults[ALGO_COMBINATIONS];
    int nbAlgoIds = 0;
    #define ALGO_IDS 4
    int algoIdA[ALGO_IDS];
    hipDataType scaleType = HIP_R_32F, Atype = HIP_R_32F, Btype = HIP_R_32F, Ctype = HIP_R_32F;
    hipblasComputeType_t computeType = HIPBLAS_COMPUTE_32F;
    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));
    
    // Request the 4 first AlgoId available for SGEMM ( computeType = scaleType = Atype = Btype = Ctype = Dtype = HIP_R_32F)
    checkCublasStatus(cublasLtMatmulAlgoGetIds(ltHandle, computeType, scaleType, Atype, Btype, Ctype, Ctype, ALGO_IDS, algoIdA, &nbAlgoIds));
    
    // Create CUDA event to time the execution time of each algo    
    checkCudaStatus(hipEventCreate(&startEvent, hipEventBlockingSync));
    checkCudaStatus(hipEventCreate(&stopEvent, hipEventBlockingSync));

    // Loop over the Algo IDs
    for (int idx = 0; (idx < nbAlgoIds) && (AlgoCount < AlgoCombinations); idx++) {   
        hipblasLtMatmulAlgo_t algo;
        size_t sizeWritten = 0;
        /* Initialize algo structure with given Algp ID */
        status = cublasLtMatmulAlgoInit(ltHandle, computeType, scaleType, Atype, Btype, Ctype, Ctype, algoIdA[idx], &algo);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            continue;
        }
        // Query the tiles enums supported by that algo
        checkCublasStatus(cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_TILE_IDS, NULL, 0, &sizeWritten));
        int nbTiles = int(sizeWritten/sizeof(int));
        int *tileA = new int[ nbTiles == 0 ? 1:nbTiles];
        if(nbTiles == 0){
            tileA[0] = CUBLASLT_MATMUL_TILE_UNDEFINED;
            nbTiles = 1;
        }
        
        checkCublasStatus(cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_STAGES_IDS, NULL, 0, &sizeWritten));
        int nbStages = int(sizeWritten/sizeof(int));
        std::vector<int> stagesA(nbStages == 0 ? 1 : nbStages);
        if (nbStages == 0) {
            stagesA[0] = CUBLASLT_MATMUL_STAGES_UNDEFINED;
            nbStages = 1;
        } else {
            checkCublasStatus(cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_STAGES_IDS, stagesA.data(), sizeof(int)*nbStages, &sizeWritten));
        }

        int splitkSupport, redMask, swizzlingMax, customOptionMax;
        // Retrieve Algo Capabilities attributes to be able to setup loop over the different combinations
        cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_TILE_IDS, tileA, sizeof(int)*nbTiles, &sizeWritten);
        cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_SPLITK_SUPPORT, &splitkSupport, sizeof(splitkSupport), &sizeWritten);
        cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_REDUCTION_SCHEME_MASK, &redMask, sizeof(redMask), &sizeWritten);
        cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_CTA_SWIZZLING_SUPPORT, &swizzlingMax, sizeof(swizzlingMax), &sizeWritten);        
        cublasLtMatmulAlgoCapGetAttribute(&algo, CUBLASLT_ALGO_CAP_CUSTOM_OPTION_MAX, &customOptionMax, sizeof(customOptionMax), &sizeWritten);
        
        /* Loop over the different tiles */
        for (int tileIdx = 0; tileIdx < nbTiles; tileIdx++) {
            /* Loop over different stages count */
            for (int stagesIdx = 0; stagesIdx < nbStages; stagesIdx++) {
                checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_STAGES_ID, &stagesA[stagesIdx], sizeof(stagesA[stagesIdx])));
                /* loop over the different custom option if any */
                for (int customOption = 0; customOption <= customOptionMax; customOption++) {
                    checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_CUSTOM_OPTION, &customOption, sizeof(customOption)));
                    /* loop over the CTAs swizzling support */
                    for (int k = 0; k <= swizzlingMax; k++) {
                        int splitK_trial = 0;
                        if (splitkSupport) {
                            splitK_trial += sizeof(splitKSequenceA) / sizeof(splitKSequenceA[0]);
                        }
                        // Loop over the splitK value over a fixed sequence splitKSequenceA in addtion to the case where splitK is not enabled
                        for (int l = 0; (l < (1 + splitK_trial)) && (AlgoCount < AlgoCombinations); l++) {
                            /* Setup attribute of the algo to run */
                            checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &tileA[tileIdx], sizeof(tileA[tileIdx])));
                            int splitK_val = 0;
                            int redScheme = CUBLASLT_REDUCTION_SCHEME_NONE;
                            checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &splitK_val, sizeof(splitK_val)));
                            checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_CTA_SWIZZLING, &k, sizeof(k)));
                            checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &redScheme, sizeof(int)));
                                                                            
                            if (l > 0) { // Split-K case
                                splitK_val = splitKSequenceA[l - 1];
                                checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &splitKSequenceA[l - 1], sizeof(splitKSequenceA[l - 1])));
                                /* Going over all the reduction scheme  */
                                for (redScheme = 1 ; redScheme < (int)CUBLASLT_REDUCTION_SCHEME_MASK && (AlgoCount < AlgoCombinations); redScheme = redScheme << 1) {
                                    if (redScheme & redMask) {
                                        checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &redScheme, sizeof(redScheme)));
                                        
                                        status = customMatmulRun( ltHandle,
                                                                operationDesc,
                                                                alpha, /* host or device pointer */
                                                                A, Adesc,
                                                                B, Bdesc,
                                                                beta, /* host or device pointer */
                                                                C, Cdesc,
                                                                C, Cdesc,
                                                                algo,
                                                                kernelRepeats,  
                                                                workSpace,
                                                                workSpaceSize,                 
                                                                perfResults[AlgoCount],
                                                                stream,
                                                                startEvent, stopEvent);
                                        perfResults[AlgoCount].status = status;
                                        if (status == HIPBLAS_STATUS_SUCCESS) AlgoCount++;
                                                                    
                                    } // end if
                                } // end for
                            } else { // Non-splitK case
                                /* if user preference is ok with workspace */
                                if (AlgoCount < AlgoCombinations) {       
                                    status = customMatmulRun( ltHandle,
                                                            operationDesc,
                                                            alpha, /* host or device pointer */
                                                            A, Adesc,
                                                            B, Bdesc,
                                                            beta, /* host or device pointer */
                                                            C, Cdesc,
                                                            C, Cdesc,
                                                            algo,
                                                            kernelRepeats,  
                                                            workSpace,
                                                            workSpaceSize,                 
                                                            perfResults[AlgoCount],
                                                            stream,
                                                            startEvent, stopEvent);
                                    perfResults[AlgoCount].status = status;
                                    if (status == HIPBLAS_STATUS_SUCCESS) AlgoCount++;
                                }
                            }
                        }  // end l
                    }  // end k
                } //end customOption
            } // end stagesIdx
        } // end tileIdx
        delete [] tileA;
    } // end idx
    
    // Sort the results per run duration 
    std::sort(perfResults, perfResults + AlgoCount, time_compare);
    // Print timing and perf details 
    for (int i = 0; i < AlgoCount; i++) {                
        printf( "result %03d : ", i);
        printPerfStructure(perfResults[i]);                          
    }

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
    if (startEvent) checkCudaStatus(hipEventDestroy(startEvent));
    if (stopEvent) checkCudaStatus(hipEventDestroy(stopEvent));
}
