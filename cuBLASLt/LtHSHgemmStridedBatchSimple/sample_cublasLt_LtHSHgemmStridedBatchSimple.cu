#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hipblaslt.h>

#include "sample_cublasLt_LtHSHgemmStridedBatchSimple.h"
#include "helpers.h"

/// Sample wrapper executing mixed precision gemm with hipblasLtMatmul, nearly a drop-in replacement for hipblasGemmEx,
/// with addition of the workspace to support split-K algorithms
///
/// pointer mode is always host, to change it configure the appropriate matmul descriptor attribute
/// matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed
void LtHSHgemmStridedBatchSimple(hipblasLtHandle_t ltHandle,
                                 hipblasOperation_t transa,
                                 hipblasOperation_t transb,
                                 int m,
                                 int n,
                                 int k,
                                 const float *alpha, /* host pointer */
                                 const __half *A,
                                 int lda,
                                 int64_t stridea,
                                 const __half *B,
                                 int ldb,
                                 int64_t strideb,
                                 const float *beta, /* host pointer */
                                 __half *C,
                                 int ldc,
                                 int64_t stridec,
                                 int batchCount,
                                 void *workspace,
                                 size_t workspaceSize) {

    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create matrix descriptors, we need to configure batch size and counts in this case
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridea, sizeof(stridea)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideb, sizeof(strideb)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16F, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridec, sizeof(stridec)));

    // in this simplified example we take advantage of hipblasLtMatmul shortcut notation with algo=NULL which will force
    // matmul to get the basic heuristic result internally. Downsides of this approach are that there is no way to
    // configure search preferences (e.g. disallow tensor operations or some reduction schemes) and no way to store the
    // algo for later use
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     beta,
                                     C,
                                     Cdesc,
                                      C,
                                     Cdesc,
                                     NULL,
                                     workspace,
                                     workspaceSize,
                                     0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}