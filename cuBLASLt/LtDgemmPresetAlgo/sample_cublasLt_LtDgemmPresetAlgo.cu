/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hipblaslt.h>

#include "sample_cublasLt_LtDgemmPresetAlgo.h"
#include "helpers.h"

/// Sample wrapper executing double precision gemm with a predefined algorithm using hipblasLtMatmul, nearly a drop-in
/// replacement for hipblasDgemm, with addition of the workspace to support split-K algorithms
///
/// pointer mode is always host, to change it configure the appropriate matmul descriptor attribute
/// matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to change
/// this configure appropriate attribute in the preference handle
///
/// NOTE: this sample may not work on all architectures or all problem sizes
void LtDgemmPresetAlgo(hipblasLtHandle_t ltHandle,
        hipblasOperation_t transa,
        hipblasOperation_t transb,
        int m,
        int n,
        int k,
        const double *alpha, /* host pointer */
        const double *A,
        int lda,
        const double *B,
        int ldb,
        const double *beta, /* host pointer */
        double *C,
        int ldc,
        void *workspace,
        size_t workspaceSize,
        hipStream_t stream) {
    hipblasLtMatmulDescOpaque_t operationDesc = {};
    hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
    hipblasLtMatmulAlgo_t algo = {};

    const int32_t algoId = 10;
    const cublasLtMatmulTile_t tileId = CUBLASLT_MATMUL_TILE_16x16; // 5
    const cublasLtReductionScheme_t reductionMode = CUBLASLT_REDUCTION_SCHEME_INPLACE; // 1
    const int32_t splitKFactor = 256;

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_64F, HIP_R_64F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(cublasLtMatrixLayoutInit(&Adesc, HIP_R_64F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(cublasLtMatrixLayoutInit(&Bdesc, HIP_R_64F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(cublasLtMatrixLayoutInit(&Cdesc, HIP_R_64F, m, n, ldc));

    checkCublasStatus(cublasLtMatmulAlgoInit(ltHandle,  //
                                             HIPBLAS_COMPUTE_64F,   // compute
                                             HIP_R_64F,   // scale
                                             HIP_R_64F,   // A
                                             HIP_R_64F,   // B
                                             HIP_R_64F,   // C
                                             HIP_R_64F,   // D
                                             algoId,
                                             &algo));

    checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &tileId, sizeof(tileId)));
    checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &reductionMode, sizeof(reductionMode)));
    checkCublasStatus(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &splitKFactor, sizeof(splitKFactor)));

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     &operationDesc,
                                     alpha,
                                     A,
                                     &Adesc,
                                     B,
                                     &Bdesc,
                                     beta,
                                     C,
                                     &Cdesc,
                                     C,
                                     &Cdesc,
                                     &algo,
                                     workspace,
                                     workspaceSize,
                                     stream));
}