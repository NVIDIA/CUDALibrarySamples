/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdio>
#include <vector>
#include <algorithm>

#include <hipblaslt.h>
#include <hip/hip_runtime_api.h>

#include "sample_cublasLt_LtSgemmSimpleAutoTuning.h"
#include "helpers.h"

float median(std::vector<float>& times) {
    const size_t size = times.size();
    if (size == 0) {
        return 0;
    }

    std::sort(times.begin(), times.end());

    const size_t mid = size / 2;
    if (size % 2 == 0) {
        return (times[mid] + times[mid - 1]) / 2;
    }
    else {
        return times[mid];
    }
}

/// Sample wrapper executing single precision gemm algorithm auto tuning by querying cublasLt heuristics for best algorithms,
/// iterate over the results and pick the algorithm that have the best performance for the given problem
///
/// pointer mode is always host, to change it configure the appropriate matmul descriptor attribute
/// matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to change
/// this configure appropriate attribute in the preference handle
void LtSgemmSimpleAutoTuning(hipblasLtHandle_t ltHandle,
                             hipblasOperation_t transa,
                             hipblasOperation_t transb,
                             int m,
                             int n,
                             int k,
                             const float *alpha, /* host pointer */
                             const float *A,
                             int lda,
                             const float *B,
                             int ldb,
                             const float *beta, /* host pointer */
                             float *C,
                             int ldc,
                             void *workspace,
                             size_t workspaceSize,
                             hipblasLtMatmulAlgo_t& algo) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    const int requestedAlgoCount = 8;
    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = { 0 };
    int bestAlgoIdx = 0;
    float time = 0;
    float bestAlgoTime = 0;
    hipStream_t stream;
    hipEvent_t startEvent, stopEvent;

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
        &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference,
        requestedAlgoCount, heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    checkCudaStatus(hipStreamCreate(&stream));
    checkCudaStatus(hipEventCreate(&startEvent));
    checkCudaStatus(hipEventCreate(&stopEvent));

    constexpr int repeatAlgoCheck = 5;
    std::vector<float> algoTimes(repeatAlgoCheck);

    for (int algoIdx = 0; algoIdx < returnedResults; algoIdx++) {
        for (int checkIdx = 0; checkIdx < repeatAlgoCheck; checkIdx++) {
            checkCudaStatus(hipEventRecord(startEvent, stream));

            checkCublasStatus(hipblasLtMatmul(ltHandle,
                                            operationDesc,
                                            alpha,
                                            A,
                                            Adesc,
                                            B,
                                            Bdesc,
                                            beta,
                                            C,
                                            Cdesc,
                                            C,
                                            Cdesc,
                                            &heuristicResult[algoIdx].algo,
                                            workspace,
                                            workspaceSize,
                                            stream));

            checkCudaStatus(hipEventRecord(stopEvent, stream));
            checkCudaStatus(hipEventSynchronize(stopEvent));
            checkCudaStatus(hipEventElapsedTime(&time, startEvent, stopEvent));
            algoTimes[checkIdx] = time;
        }

        time = median(algoTimes);

        if (algoIdx == 0 || time < bestAlgoTime) {
            bestAlgoTime = time;
            bestAlgoIdx = algoIdx;
        }
    }

    memcpy(&algo, &heuristicResult[bestAlgoIdx].algo, sizeof(algo));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) hipblasLtMatmulDescDestroy(operationDesc);
    if (stream) checkCudaStatus(hipStreamDestroy(stream));
    if (startEvent) checkCudaStatus(hipEventDestroy(startEvent));
    if (stopEvent) checkCudaStatus(hipEventDestroy(stopEvent));
}
