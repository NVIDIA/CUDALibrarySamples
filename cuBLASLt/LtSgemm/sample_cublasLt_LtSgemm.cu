/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblaslt.h>

#include "sample_cublasLt_LtSgemm.h"
#include "helpers.h"

/// Sample wrapper executing single precision gemm with hipblasLtMatmul, nearly a drop-in replacement for hipblasSgemm,
/// with addition of the workspace to support split-K algorithms
///
/// pointer mode is always host, to change it configure the appropriate matmul descriptor attribute
/// matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to change
/// this configure appropriate attribute in the preference handle
void LtSgemm(hipblasLtHandle_t ltHandle,
             hipblasOperation_t transa,
             hipblasOperation_t transb,
             int m,
             int n,
             int k,
             const float *alpha, /* host pointer */
             const float *A,
             int lda,
             const float *B,
             int ldb,
             const float *beta, /* host pointer */
             float *C,
             int ldc,
             void *workspace,
             size_t workspaceSize) {
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     beta,
                                     C,
                                     Cdesc,
                                     C,
                                     Cdesc,
                                     &heuristicResult.algo,
                                     workspace,
                                     workspaceSize,
                                     0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}
