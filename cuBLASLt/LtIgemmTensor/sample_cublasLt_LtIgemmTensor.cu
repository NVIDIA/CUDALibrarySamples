/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <cstdint>

#include "sample_cublasLt_LtIgemmTensor.h"
#include "helpers.h"

int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}

/// Use hipblasLtMatmul to perform tensor-op Igemm with memory order transforms on all buffers
///
/// For better performance data order transforms should be offline as much as possible.
///
/// transa, transb assumed N; alpha, beta are host pointers, tensor ops allowed, alpha assumed 1, beta assumed 0,
/// stream assumed 0
void LtIgemmTensor(hipblasLtHandle_t ltHandle,
                   hipblasOperation_t transa,
                   hipblasOperation_t transb,
                   int m,
                   int n,
                   int k,
                   const int8_t *A,
                   int lda,
                   const int8_t *B,
                   int ldb,
                   int32_t *C,
                   int ldc) {
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    int32_t alpha = 1, beta = 0;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;

    // tensor op igemm kernels require specialized memory order of data
    hipblasLtMatrixTransformDesc_t transformDesc = NULL;
    int8_t *Atransform = NULL, *Btransform = NULL;
    int32_t *Ctransform                   = NULL;
    hipblasLtMatrixLayout_t AtransformDesc = NULL, BtransformDesc = NULL, CtransformDesc = NULL;
    float transformAlpha = 1.0f, transformBeta = 0.0f;
    hipblasLtOrder_t order_COL32       = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t order_COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;

    int ldatransform = 32 * m;
    int ldbtransform = 32 * roundoff(n, 8);
    int ldctransform = 32 * m;

    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Atransform), sizeof(int8_t) * roundoff(k, 32) / 32 * ldatransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Btransform), sizeof(int8_t) * roundoff(k, 32) / 32 * ldbtransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Ctransform), sizeof(int32_t) * roundoff(n, 32) / 32 * ldctransform));

    checkCublasStatus(hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_32F));

    checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I));
    // tensor op igemm kernels only support NT gemm
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for original matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, k, n, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, ldc));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for transformed matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&AtransformDesc, HIP_R_8I, m, k, ldatransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // data memory order is set to CUBLASLT_ORDER_COL4_4R2_8C in order to achieve best performance on Turing devices.
    // for best performance on Ampere, consider setting the memory order to CUBLASLT_ORDER_COL32_2R_4R4.
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&BtransformDesc, HIP_R_8I, n, k, ldbtransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL4_4R2_8C, sizeof(order_COL4_4R2_8C)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&CtransformDesc, HIP_R_32I, m, n, ldctransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // ---------------------------------------------------------------------------------------------
    // transforms and computation

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, A, Adesc, &transformBeta, NULL, NULL, Atransform, AtransformDesc, 0));

    // B matrix is non-transposed, but transposed matrix is needed - add transpose operation in matrix transform.
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, B, Bdesc, &transformBeta, NULL, NULL, Btransform, BtransformDesc, 0));

    // no need to transform C matrix as beta is assumed to be 0
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     Atransform,
                                     AtransformDesc,
                                     Btransform,
                                     BtransformDesc,
                                     &beta,
                                     Ctransform,
                                     CtransformDesc,
                                     Ctransform,
                                     CtransformDesc,
                                     NULL,
                                     NULL,
                                     0,
                                     0));

    opTranspose = HIPBLAS_OP_N;
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    // transform outputs to COL order
    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, Ctransform, CtransformDesc, &transformBeta, NULL, NULL, C, Cdesc, 0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (CtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(CtransformDesc));
    if (BtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(BtransformDesc));
    if (AtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(AtransformDesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
    if (transformDesc) checkCublasStatus(hipblasLtMatrixTransformDescDestroy(transformDesc));

    // wait until device is done before freeing transformed buffers
    checkCudaStatus(hipDeviceSynchronize());
    if (Ctransform) checkCudaStatus(hipFree(Ctransform));
    if (Btransform) checkCudaStatus(hipFree(Btransform));
    if (Atransform) checkCudaStatus(hipFree(Atransform));
}
