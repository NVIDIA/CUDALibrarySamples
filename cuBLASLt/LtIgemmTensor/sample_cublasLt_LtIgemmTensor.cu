/*
 * SPDX-FileCopyrightText: Copyright (c) 2020 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <cstdint>

#include "sample_cublasLt_LtIgemmTensor.h"
#include "helpers.h"

int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}

/// Use hipblasLtMatmul to perform tensor-op Igemm with memory order transforms on all buffers
///
/// For better performance data order transforms should be offline as much as possible.
///
/// transa, transb assumed N; alpha, beta are host pointers, tensor ops allowed, alpha assumed 1, beta assumed 0,
/// stream assumed 0
void LtIgemmTensor(hipblasLtHandle_t ltHandle,
                   hipblasOperation_t transa,
                   hipblasOperation_t transb,
                   int m,
                   int n,
                   int k,
                   const int8_t *A,
                   int lda,
                   const int8_t *B,
                   int ldb,
                   int32_t *C,
                   int ldc) {
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    int32_t alpha = 1, beta = 0;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;

    // tensor op igemm kernels require specialized memory order of data
    hipblasLtMatrixTransformDesc_t transformDesc = NULL;
    int8_t *Atransform = NULL, *Btransform = NULL;
    int32_t *Ctransform                   = NULL;
    hipblasLtMatrixLayout_t AtransformDesc = NULL, BtransformDesc = NULL, CtransformDesc = NULL;
    float transformAlpha = 1.0f, transformBeta = 0.0f;
    hipblasLtOrder_t order_COL32       = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t order_COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;

    int ldatransform = 32 * m;
    int ldbtransform = 32 * roundoff(n, 8);
    int ldctransform = 32 * m;

    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Atransform), sizeof(int8_t) * roundoff(k, 32) / 32 * ldatransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Btransform), sizeof(int8_t) * roundoff(k, 32) / 32 * ldbtransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Ctransform), sizeof(int32_t) * roundoff(n, 32) / 32 * ldctransform));

    checkCublasStatus(hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_32F));

    checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I));
    // tensor op igemm kernels only support NT gemm
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for original matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, k, n, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, ldc));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for transformed matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&AtransformDesc, HIP_R_8I, m, k, ldatransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // data memory order is set to CUBLASLT_ORDER_COL4_4R2_8C in order to achieve best performance on Turing devices.
    // for best performance on Ampere, consider setting the memory order to CUBLASLT_ORDER_COL32_2R_4R4.
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&BtransformDesc, HIP_R_8I, n, k, ldbtransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL4_4R2_8C, sizeof(order_COL4_4R2_8C)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&CtransformDesc, HIP_R_32I, m, n, ldctransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // ---------------------------------------------------------------------------------------------
    // transforms and computation

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, A, Adesc, &transformBeta, NULL, NULL, Atransform, AtransformDesc, 0));

    // B matrix is non-transposed, but transposed matrix is needed - add transpose operation in matrix transform.
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, B, Bdesc, &transformBeta, NULL, NULL, Btransform, BtransformDesc, 0));

    // no need to transform C matrix as beta is assumed to be 0
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     Atransform,
                                     AtransformDesc,
                                     Btransform,
                                     BtransformDesc,
                                     &beta,
                                     Ctransform,
                                     CtransformDesc,
                                     Ctransform,
                                     CtransformDesc,
                                     NULL,
                                     NULL,
                                     0,
                                     0));

    opTranspose = HIPBLAS_OP_N;
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    // transform outputs to COL order
    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, Ctransform, CtransformDesc, &transformBeta, NULL, NULL, C, Cdesc, 0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (CtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(CtransformDesc));
    if (BtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(BtransformDesc));
    if (AtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(AtransformDesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
    if (transformDesc) checkCublasStatus(hipblasLtMatrixTransformDescDestroy(transformDesc));

    // wait until device is done before freeing transformed buffers
    checkCudaStatus(hipDeviceSynchronize());
    if (Ctransform) checkCudaStatus(hipFree(Ctransform));
    if (Btransform) checkCudaStatus(hipFree(Btransform));
    if (Atransform) checkCudaStatus(hipFree(Atransform));
}